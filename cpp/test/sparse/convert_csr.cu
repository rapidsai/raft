#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/random/rng.hpp>
#include "../test_utils.h"

#include <raft/sparse/convert/csr.hpp>
#include <raft/sparse/coo.hpp>

#include <iostream>

namespace raft {
namespace sparse {

/**************************** sorted COO to CSR ****************************/

template <typename T>
struct SparseConvertCSRInputs {
  int m, n, nnz;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const SparseConvertCSRInputs<T>& dims)
{
  return os;
}

template <typename T>
class SparseConvertCSRTest : public ::testing::TestWithParam<SparseConvertCSRInputs<T>> {
 protected:
  void SetUp() override {}

  void TearDown() override {}

 protected:
  SparseConvertCSRInputs<T> params;
};

const std::vector<SparseConvertCSRInputs<float>> inputsf = {{5, 10, 5, 1234ULL}};

typedef SparseConvertCSRTest<float> SortedCOOToCSR;
TEST_P(SortedCOOToCSR, Result)
{
  hipStream_t stream;
  hipStreamCreate(&stream);

  int nnz = 8;

  int* in_h  = new int[nnz]{0, 0, 1, 1, 2, 2, 3, 3};
  int* exp_h = new int[4]{0, 2, 4, 6};

  rmm::device_uvector<int> in(nnz, stream);
  rmm::device_uvector<int> exp(4, stream);
  rmm::device_uvector<int> out(4, stream);
  CUDA_CHECK(hipMemsetAsync(in.data(), 0, in.size() * sizeof(int), stream));
  CUDA_CHECK(hipMemsetAsync(exp.data(), 0, exp.size() * sizeof(int), stream));
  CUDA_CHECK(hipMemsetAsync(out.data(), 0, out.size() * sizeof(int), stream));

  raft::update_device(in.data(), in_h, nnz, stream);
  raft::update_device(exp.data(), exp_h, 4, stream);

  convert::sorted_coo_to_csr<int>(in.data(), nnz, out.data(), 4, stream);

  ASSERT_TRUE(raft::devArrMatch<int>(out.data(), exp.data(), 4, raft::Compare<int>()));

  hipStreamDestroy(stream);

  delete[] in_h;
  delete[] exp_h;
}

INSTANTIATE_TEST_CASE_P(SparseConvertCSRTest, SortedCOOToCSR, ::testing::ValuesIn(inputsf));

/******************************** adj graph ********************************/

template <typename Index_>
struct CSRAdjGraphInputs {
  Index_ n_rows;
  Index_ n_cols;
  std::vector<Index_> row_ind;
  std::vector<uint8_t> adj;  // To avoid vector<bool> optimization
  std::vector<Index_> verify;
};

template <typename Index_>
class CSRAdjGraphTest : public ::testing::TestWithParam<CSRAdjGraphInputs<Index_>> {
 public:
  CSRAdjGraphTest()
    : params(::testing::TestWithParam<CSRAdjGraphInputs<Index_>>::GetParam()),
      stream(handle.get_stream()),
      row_ind(params.n_rows, stream),
      adj(params.n_rows * params.n_cols, stream),
      result(params.verify.size(), stream),
      verify(params.verify.size(), stream)
  {
  }

 protected:
  void SetUp() override { nnz = params.verify.size(); }

  void Run()
  {
    raft::update_device(row_ind.data(), params.row_ind.data(), params.n_rows, stream);
    raft::update_device(adj.data(),
                        reinterpret_cast<bool*>(params.adj.data()),
                        params.n_rows * params.n_cols,
                        stream);
    raft::update_device(verify.data(), params.verify.data(), nnz, stream);

    convert::csr_adj_graph_batched<Index_>(
      row_ind.data(), params.n_cols, nnz, params.n_rows, adj.data(), result.data(), stream);

    ASSERT_TRUE(
      raft::devArrMatch<Index_>(verify.data(), result.data(), nnz, raft::Compare<Index_>()));
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  CSRAdjGraphInputs<Index_> params;
  Index_ nnz;
  rmm::device_uvector<Index_> row_ind, result, verify;
  rmm::device_uvector<bool> adj;
};

using CSRAdjGraphTestI = CSRAdjGraphTest<int>;
TEST_P(CSRAdjGraphTestI, Result) { Run(); }

using CSRAdjGraphTestL = CSRAdjGraphTest<int64_t>;
TEST_P(CSRAdjGraphTestL, Result) { Run(); }

const std::vector<CSRAdjGraphInputs<int>> csradjgraph_inputs_i = {
  {3,
   6,
   {0, 3, 6},
   {1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
   {0, 1, 2, 0, 1, 2, 0, 1, 2}},
};
const std::vector<CSRAdjGraphInputs<int64_t>> csradjgraph_inputs_l = {
  {3,
   6,
   {0, 3, 6},
   {1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
   {0, 1, 2, 0, 1, 2, 0, 1, 2}},
};

INSTANTIATE_TEST_CASE_P(SparseConvertCSRTest,
                        CSRAdjGraphTestI,
                        ::testing::ValuesIn(csradjgraph_inputs_i));
INSTANTIATE_TEST_CASE_P(SparseConvertCSRTest,
                        CSRAdjGraphTestL,
                        ::testing::ValuesIn(csradjgraph_inputs_l));

}  // namespace sparse
}  // namespace raft
