/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>

#include <raft/sparse/csr.hpp>
#include <raft/sparse/op/row_op.cuh>

#include "../test_utils.cuh"
#include <raft/util/cudart_utils.hpp>

#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

template <typename Type_f, typename Index_>
struct CSRRowOpInputs {
  std::vector<Index_> ex_scan;
  std::vector<Type_f> verify;
};

/** Wrapper to call csr_row_op because the enclosing function of a __device__
 *  lambda cannot have private ot protected access within the class. */
template <typename Type_f, typename Index_>
void csr_row_op_wrapper(
  const Index_* row_ind, Index_ n_rows, Index_ nnz, Type_f* result, hipStream_t stream)
{
  op::csr_row_op<Index_>(
    row_ind,
    n_rows,
    nnz,
    [result] __device__(Index_ row, Index_ start_idx, Index_ stop_idx) {
      for (Index_ i = start_idx; i < stop_idx; i++)
        result[i] = row;
    },
    stream);
}

template <typename Type_f, typename Index_>
class CSRRowOpTest : public ::testing::TestWithParam<CSRRowOpInputs<Type_f, Index_>> {
 public:
  CSRRowOpTest()
    : params(::testing::TestWithParam<CSRRowOpInputs<Type_f, Index_>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      verify(params.verify.size(), stream),
      ex_scan(params.ex_scan.size(), stream),
      result(params.verify.size(), stream)
  {
  }

 protected:
  void SetUp() override
  {
    n_rows = params.ex_scan.size();
    nnz    = params.verify.size();
  }

  void Run()
  {
    raft::update_device(ex_scan.data(), params.ex_scan.data(), n_rows, stream);
    raft::update_device(verify.data(), params.verify.data(), nnz, stream);

    csr_row_op_wrapper<Type_f, Index_>(ex_scan.data(), n_rows, nnz, result.data(), stream);

    ASSERT_TRUE(raft::devArrMatch<Type_f>(
      verify.data(), result.data(), nnz, raft::Compare<Type_f>(), stream));
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  CSRRowOpInputs<Type_f, Index_> params;
  Index_ n_rows, nnz;
  rmm::device_uvector<Index_> ex_scan;
  rmm::device_uvector<Type_f> result, verify;
};

using CSRRowOpTestF = CSRRowOpTest<float, int>;
TEST_P(CSRRowOpTestF, Result) { Run(); }

using CSRRowOpTestD = CSRRowOpTest<double, int>;
TEST_P(CSRRowOpTestD, Result) { Run(); }

const std::vector<CSRRowOpInputs<float, int>> csrrowop_inputs_f = {
  {{0, 4, 8, 9}, {0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 2.0, 3.0}},
};
const std::vector<CSRRowOpInputs<double, int>> csrrowop_inputs_d = {
  {{0, 4, 8, 9}, {0.0, 0.0, 0.0, 0.0, 1.0, 1.0, 1.0, 1.0, 2.0, 3.0}},
};

INSTANTIATE_TEST_CASE_P(SparseRowOpTest, CSRRowOpTestF, ::testing::ValuesIn(csrrowop_inputs_f));
INSTANTIATE_TEST_CASE_P(SparseRowOpTest, CSRRowOpTestD, ::testing::ValuesIn(csrrowop_inputs_d));

}  // namespace sparse
}  // namespace raft
