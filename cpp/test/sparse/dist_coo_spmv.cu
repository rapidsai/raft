#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include <hipsparse.h>

#include <raft/cudart_utils.h>
#include <raft/linalg/distance_type.h>
#include <raft/sparse/cusparse_wrappers.h>
#include <raft/linalg/unary_op.cuh>
#include <raft/mr/device/allocator.hpp>

#include <raft/sparse/convert/coo.cuh>
#include <raft/sparse/distance/coo_spmv.cuh>
#include <raft/sparse/distance/operators.cuh>

#include "../test_utils.h"

#include <type_traits>

namespace raft {
namespace sparse {
namespace distance {

using namespace raft;
using namespace raft::sparse;

template <typename value_idx, typename value_t>
struct InputConfiguration {
  value_idx n_cols;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_t> out_dists_ref_h;

  raft::distance::DistanceType metric;

  float metric_arg = 0.0;
};

using dense_smem_strategy_t = dense_smem_strategy<int, float, 1024>;
using hash_strategy_t = hash_strategy<int, float, 1024>;

template <typename value_idx, typename value_t, typename strategy_t>
struct SparseDistanceCOOSPMVInputs {
  InputConfiguration<value_idx, value_t> input_configuration;

  float capacity_threshold = 0.5;
  int map_size = hash_strategy<value_idx, value_t, 1024>::get_map_size();
};

template <typename value_idx, typename value_t, typename strategy_t>
::std::ostream &operator<<(
  ::std::ostream &os,
  const SparseDistanceCOOSPMVInputs<value_idx, value_t, strategy_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t, typename strategy_t>
class SparseDistanceCOOSPMVTest
  : public ::testing::TestWithParam<
      SparseDistanceCOOSPMVInputs<value_idx, value_t, strategy_t>> {
 public:
  SparseDistanceCOOSPMVTest() : dist_config(handle) {}

  template <typename U,
            std::enable_if_t<std::is_same_v<U, hash_strategy_t>> * = nullptr>
  U make_strategy() {
    return strategy_t(dist_config, params.capacity_threshold, params.map_size);
  }

  template <typename U, std::enable_if_t<
                          std::is_same_v<U, dense_smem_strategy_t>> * = nullptr>
  U make_strategy() {
    return strategy_t(dist_config);
  }

  template <typename reduce_f, typename accum_f, typename write_f>
  void compute_dist(reduce_f reduce_func, accum_f accum_func,
                    write_f write_func, bool rev = true) {
    raft::mr::device::buffer<value_idx> coo_rows(
      dist_config.handle.get_device_allocator(),
      dist_config.handle.get_stream(),
      max(dist_config.b_nnz, dist_config.a_nnz));

    raft::sparse::convert::csr_to_coo(dist_config.b_indptr, dist_config.b_nrows,
                                      coo_rows.data(), dist_config.b_nnz,
                                      dist_config.handle.get_stream());

    strategy_t selected_strategy = make_strategy<strategy_t>();
    balanced_coo_pairwise_generalized_spmv<value_idx, value_t>(
      out_dists, dist_config, coo_rows.data(), reduce_func, accum_func,
      write_func, selected_strategy);

    if (rev) {
      raft::sparse::convert::csr_to_coo(
        dist_config.a_indptr, dist_config.a_nrows, coo_rows.data(),
        dist_config.a_nnz, dist_config.handle.get_stream());

      balanced_coo_pairwise_generalized_spmv_rev<value_idx, value_t>(
        out_dists, dist_config, coo_rows.data(), reduce_func, accum_func,
        write_func, selected_strategy);
    }
  }

  void run_spmv() {
    switch (params.input_configuration.metric) {
      case raft::distance::DistanceType::InnerProduct:
        compute_dist(Product(), Sum(), AtomicAdd(), true);
        break;
      case raft::distance::DistanceType::L2Unexpanded:
        compute_dist(SqDiff(), Sum(), AtomicAdd());
        break;
      case raft::distance::DistanceType::Canberra:
        compute_dist(
          [] __device__(value_t a, value_t b) {
            return fabsf(a - b) / (fabsf(a) + fabsf(b));
          },
          Sum(), AtomicAdd());
        break;
      case raft::distance::DistanceType::L1:
        compute_dist(AbsDiff(), Sum(), AtomicAdd());
        break;
      case raft::distance::DistanceType::Linf:
        compute_dist(AbsDiff(), Max(), AtomicMax());
        break;
      case raft::distance::DistanceType::LpUnexpanded: {
        compute_dist(PDiff(params.input_configuration.metric_arg), Sum(),
                     AtomicAdd());
        float p = 1.0f / params.input_configuration.metric_arg;
        raft::linalg::unaryOp<value_t>(
          out_dists, out_dists, dist_config.a_nrows * dist_config.b_nrows,
          [=] __device__(value_t input) { return powf(input, p); },
          dist_config.handle.get_stream());

      } break;
      default:
        throw raft::exception("Unknown distance");
    }
  }

 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.input_configuration.indptr_h;
    std::vector<value_idx> indices_h = params.input_configuration.indices_h;
    std::vector<value_t> data_h = params.input_configuration.data_h;

    allocate(indptr, indptr_h.size());
    allocate(indices, indices_h.size());
    allocate(data, data_h.size());

    update_device(indptr, indptr_h.data(), indptr_h.size(),
                  handle.get_stream());
    update_device(indices, indices_h.data(), indices_h.size(),
                  handle.get_stream());
    update_device(data, data_h.data(), data_h.size(), handle.get_stream());

    std::vector<value_t> out_dists_ref_h =
      params.input_configuration.out_dists_ref_h;

    allocate(out_dists_ref, (indptr_h.size() - 1) * (indptr_h.size() - 1));

    update_device(out_dists_ref, out_dists_ref_h.data(), out_dists_ref_h.size(),
                  handle.get_stream());
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      SparseDistanceCOOSPMVInputs<value_idx, value_t, strategy_t>>::GetParam();

    make_data();

    dist_config.b_nrows = params.input_configuration.indptr_h.size() - 1;
    dist_config.b_ncols = params.input_configuration.n_cols;
    dist_config.b_nnz = params.input_configuration.indices_h.size();
    dist_config.b_indptr = indptr;
    dist_config.b_indices = indices;
    dist_config.b_data = data;
    dist_config.a_nrows = params.input_configuration.indptr_h.size() - 1;
    dist_config.a_ncols = params.input_configuration.n_cols;
    dist_config.a_nnz = params.input_configuration.indices_h.size();
    dist_config.a_indptr = indptr;
    dist_config.a_indices = indices;
    dist_config.a_data = data;

    int out_size = dist_config.a_nrows * dist_config.b_nrows;

    allocate(out_dists, out_size);

    run_spmv();

    CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
  }

  void TearDown() override {
    CUDA_CHECK(hipStreamSynchronize(handle.get_stream()));
    CUDA_CHECK(hipFree(indptr));
    CUDA_CHECK(hipFree(indices));
    CUDA_CHECK(hipFree(data));
    CUDA_CHECK(hipFree(out_dists));
    CUDA_CHECK(hipFree(out_dists_ref));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_dists_ref, out_dists,
                            params.input_configuration.out_dists_ref_h.size(),
                            CompareApprox<value_t>(1e-3)));
  }

 protected:
  raft::handle_t handle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_t *out_dists, *out_dists_ref;

  raft::sparse::distance::distances_config_t<value_idx, value_t> dist_config;

  SparseDistanceCOOSPMVInputs<value_idx, value_t, strategy_t> params;
};

const InputConfiguration<int, float> input_inner_product = {
  2,
  {0, 2, 4, 6, 8},
  {0, 1, 0, 1, 0, 1, 0, 1},
  {1.0f, 2.0f, 1.0f, 2.0f, 1.0f, 2.0f, 1.0f, 2.0f},
  {5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0, 5.0,
   5.0},
  raft::distance::DistanceType::InnerProduct,
  0.0};

const InputConfiguration<int, float> input_l2_unexpanded = {
  2,
  {0, 2, 4, 6, 8},
  {0, 1, 0, 1, 0, 1, 0, 1},  // indices
  {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
  {
    // dense output
    0.0,
    4.0,
    3026.0,
    226.0,
    4.0,
    0.0,
    2930.0,
    234.0,
    3026.0,
    2930.0,
    0.0,
    1832.0,
    226.0,
    234.0,
    1832.0,
    0.0,
  },
  raft::distance::DistanceType::L2Unexpanded,
  0.0};

const InputConfiguration<int, float> input_canberra =
  {10,
   {0, 5, 11, 15, 20, 27, 32, 36, 43, 47, 50},
   {0, 1, 3, 6, 8, 0, 1, 2, 3, 5, 6, 1, 2, 4, 8, 0, 2,
    3, 4, 7, 0, 1, 2, 3, 4, 6, 8, 0, 1, 2, 5, 7, 1, 5,
    8, 9, 0, 1, 2, 5, 6, 8, 9, 2, 4, 5, 7, 0, 3, 9},  // indices
   {0.5438, 0.2695, 0.4377, 0.7174, 0.9251, 0.7648, 0.3322, 0.7279, 0.4131,
    0.5167, 0.8655, 0.0730, 0.0291, 0.9036, 0.7988, 0.5019, 0.7663, 0.2190,
    0.8206, 0.3625, 0.0411, 0.3995, 0.5688, 0.7028, 0.8706, 0.3199, 0.4431,
    0.0535, 0.2225, 0.8853, 0.1932, 0.3761, 0.3379, 0.1771, 0.2107, 0.228,
    0.5279, 0.4885, 0.3495, 0.5079, 0.2325, 0.2331, 0.3018, 0.6231, 0.2645,
    0.8429, 0.6625, 0.0797, 0.2724, 0.4218},
   {0.0,
    3.3954660629919076,
    5.6469232737388815,
    6.373112846266441,
    4.0212880272531715,
    6.916281504639404,
    5.741508386786526,
    5.411470999663036,
    9.0,
    4.977014354725805,
    3.3954660629919076,
    0.0,
    7.56256082439209,
    5.540261147481582,
    4.832322929216881,
    4.62003193872216,
    6.498056792320361,
    4.309846252268695,
    6.317531174829905,
    6.016362684141827,
    5.6469232737388815,
    7.56256082439209,
    0.0,
    5.974878731322299,
    4.898357301336036,
    6.442097410320605,
    5.227077347287883,
    7.134101195584642,
    5.457753923371659,
    7.0,
    6.373112846266441,
    5.540261147481582,
    5.974878731322299,
    0.0,
    5.5507273748583,
    4.897749658726415,
    9.0,
    8.398776718824767,
    3.908281400328807,
    4.83431066343688,
    4.0212880272531715,
    4.832322929216881,
    4.898357301336036,
    5.5507273748583,
    0.0,
    6.632989819428174,
    7.438852294822894,
    5.6631570310967465,
    7.579428202635459,
    6.760811985364303,
    6.916281504639404,
    4.62003193872216,
    6.442097410320605,
    4.897749658726415,
    6.632989819428174,
    0.0,
    5.249404187382862,
    6.072559523278559,
    4.07661278488929,
    6.19678948003145,
    5.741508386786526,
    6.498056792320361,
    5.227077347287883,
    9.0,
    7.438852294822894,
    5.249404187382862,
    0.0,
    3.854811639654704,
    6.652724827169063,
    5.298236851430971,
    5.411470999663036,
    4.309846252268695,
    7.134101195584642,
    8.398776718824767,
    5.6631570310967465,
    6.072559523278559,
    3.854811639654704,
    0.0,
    7.529184598969917,
    6.903282911791188,
    9.0,
    6.317531174829905,
    5.457753923371659,
    3.908281400328807,
    7.579428202635459,
    4.07661278488929,
    6.652724827169063,
    7.529184598969917,
    0.0,
    7.0,
    4.977014354725805,
    6.016362684141827,
    7.0,
    4.83431066343688,
    6.760811985364303,
    6.19678948003145,
    5.298236851430971,
    6.903282911791188,
    7.0,
    0.0},
   raft::distance::DistanceType::Canberra,
   0.0};

const InputConfiguration<int, float> input_lp_unexpanded =
  {10,
   {0, 5, 11, 15, 20, 27, 32, 36, 43, 47, 50},
   {0, 1, 3, 6, 8, 0, 1, 2, 3, 5, 6, 1, 2, 4, 8, 0, 2,
    3, 4, 7, 0, 1, 2, 3, 4, 6, 8, 0, 1, 2, 5, 7, 1, 5,
    8, 9, 0, 1, 2, 5, 6, 8, 9, 2, 4, 5, 7, 0, 3, 9},  // indices
   {0.5438, 0.2695, 0.4377, 0.7174, 0.9251, 0.7648, 0.3322, 0.7279, 0.4131,
    0.5167, 0.8655, 0.0730, 0.0291, 0.9036, 0.7988, 0.5019, 0.7663, 0.2190,
    0.8206, 0.3625, 0.0411, 0.3995, 0.5688, 0.7028, 0.8706, 0.3199, 0.4431,
    0.0535, 0.2225, 0.8853, 0.1932, 0.3761, 0.3379, 0.1771, 0.2107, 0.228,
    0.5279, 0.4885, 0.3495, 0.5079, 0.2325, 0.2331, 0.3018, 0.6231, 0.2645,
    0.8429, 0.6625, 0.0797, 0.2724, 0.4218},
   {0.0,
    1.31462855332296,
    1.3690307816129905,
    1.698603990921237,
    1.3460470789553531,
    1.6636670712582544,
    1.2651744044972217,
    1.1938329352055201,
    1.8811409082590185,
    1.3653115050624267,
    1.31462855332296,
    0.0,
    1.9447722703291133,
    1.42818777206562,
    1.4685491458946494,
    1.3071999866010466,
    1.4988622861692171,
    0.9698559287406783,
    1.4972023224597841,
    1.5243383567266802,
    1.3690307816129905,
    1.9447722703291133,
    0.0,
    1.2748400840107568,
    1.0599569946448246,
    1.546591282841402,
    1.147526531928459,
    1.447002179128145,
    1.5982242387673176,
    1.3112533607072414,
    1.698603990921237,
    1.42818777206562,
    1.2748400840107568,
    0.0,
    1.038121552545461,
    1.011788365364402,
    1.3907391109256988,
    1.3128200942311496,
    1.19595706584447,
    1.3233328139624725,
    1.3460470789553531,
    1.4685491458946494,
    1.0599569946448246,
    1.038121552545461,
    0.0,
    1.3642741698145529,
    1.3493868683808095,
    1.394942694628328,
    1.572881849642552,
    1.380122665319464,
    1.6636670712582544,
    1.3071999866010466,
    1.546591282841402,
    1.011788365364402,
    1.3642741698145529,
    0.0,
    1.018961640373018,
    1.0114394258945634,
    0.8338711034820684,
    1.1247823842299223,
    1.2651744044972217,
    1.4988622861692171,
    1.147526531928459,
    1.3907391109256988,
    1.3493868683808095,
    1.018961640373018,
    0.0,
    0.7701238110357329,
    1.245486437864406,
    0.5551259549534626,
    1.1938329352055201,
    0.9698559287406783,
    1.447002179128145,
    1.3128200942311496,
    1.394942694628328,
    1.0114394258945634,
    0.7701238110357329,
    0.0,
    1.1886800117391216,
    1.0083692448135637,
    1.8811409082590185,
    1.4972023224597841,
    1.5982242387673176,
    1.19595706584447,
    1.572881849642552,
    0.8338711034820684,
    1.245486437864406,
    1.1886800117391216,
    0.0,
    1.3661374102525012,
    1.3653115050624267,
    1.5243383567266802,
    1.3112533607072414,
    1.3233328139624725,
    1.380122665319464,
    1.1247823842299223,
    0.5551259549534626,
    1.0083692448135637,
    1.3661374102525012,
    0.0},
   raft::distance::DistanceType::LpUnexpanded,
   2.0};

const InputConfiguration<int, float> input_linf =
  {10,
   {0, 5, 11, 15, 20, 27, 32, 36, 43, 47, 50},
   {0, 1, 3, 6, 8, 0, 1, 2, 3, 5, 6, 1, 2, 4, 8, 0, 2,
    3, 4, 7, 0, 1, 2, 3, 4, 6, 8, 0, 1, 2, 5, 7, 1, 5,
    8, 9, 0, 1, 2, 5, 6, 8, 9, 2, 4, 5, 7, 0, 3, 9},  // indices
   {0.5438, 0.2695, 0.4377, 0.7174, 0.9251, 0.7648, 0.3322, 0.7279, 0.4131,
    0.5167, 0.8655, 0.0730, 0.0291, 0.9036, 0.7988, 0.5019, 0.7663, 0.2190,
    0.8206, 0.3625, 0.0411, 0.3995, 0.5688, 0.7028, 0.8706, 0.3199, 0.4431,
    0.0535, 0.2225, 0.8853, 0.1932, 0.3761, 0.3379, 0.1771, 0.2107, 0.228,
    0.5279, 0.4885, 0.3495, 0.5079, 0.2325, 0.2331, 0.3018, 0.6231, 0.2645,
    0.8429, 0.6625, 0.0797, 0.2724, 0.4218},
   {0.0,
    0.9251771844789913,
    0.9036452083899731,
    0.9251771844789913,
    0.8706483735804971,
    0.9251771844789913,
    0.717493881903289,
    0.6920214832303888,
    0.9251771844789913,
    0.9251771844789913,
    0.9251771844789913,
    0.0,
    0.9036452083899731,
    0.8655339692155823,
    0.8706483735804971,
    0.8655339692155823,
    0.8655339692155823,
    0.6329837991017668,
    0.8655339692155823,
    0.8655339692155823,
    0.9036452083899731,
    0.9036452083899731,
    0.0,
    0.7988276152181608,
    0.7028075145996631,
    0.9036452083899731,
    0.9036452083899731,
    0.9036452083899731,
    0.8429599432532096,
    0.9036452083899731,
    0.9251771844789913,
    0.8655339692155823,
    0.7988276152181608,
    0.0,
    0.48376552205293305,
    0.8206394616536681,
    0.8206394616536681,
    0.8206394616536681,
    0.8429599432532096,
    0.8206394616536681,
    0.8706483735804971,
    0.8706483735804971,
    0.7028075145996631,
    0.48376552205293305,
    0.0,
    0.8706483735804971,
    0.8706483735804971,
    0.8706483735804971,
    0.8429599432532096,
    0.8706483735804971,
    0.9251771844789913,
    0.8655339692155823,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.0,
    0.8853924473642432,
    0.535821510936138,
    0.6497196601457607,
    0.8853924473642432,
    0.717493881903289,
    0.8655339692155823,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.8853924473642432,
    0.0,
    0.5279604218147174,
    0.6658348373853169,
    0.33799874888632914,
    0.6920214832303888,
    0.6329837991017668,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.535821510936138,
    0.5279604218147174,
    0.0,
    0.662579808115858,
    0.5079750812968089,
    0.9251771844789913,
    0.8655339692155823,
    0.8429599432532096,
    0.8429599432532096,
    0.8429599432532096,
    0.6497196601457607,
    0.6658348373853169,
    0.662579808115858,
    0.0,
    0.8429599432532096,
    0.9251771844789913,
    0.8655339692155823,
    0.9036452083899731,
    0.8206394616536681,
    0.8706483735804971,
    0.8853924473642432,
    0.33799874888632914,
    0.5079750812968089,
    0.8429599432532096,
    0.0},
   raft::distance::DistanceType::Linf,
   0.0};

const InputConfiguration<int, float> input_l1 = {
  4,
  {0, 1, 1, 2, 4},
  {3, 2, 0, 1},  // indices
  {0.99296, 0.42180, 0.11687, 0.305869},
  {
    // dense output
    0.0,
    0.99296,
    1.41476,
    1.415707,
    0.99296,
    0.0,
    0.42180,
    0.42274,
    1.41476,
    0.42180,
    0.0,
    0.84454,
    1.41570,
    0.42274,
    0.84454,
    0.0,
  },
  raft::distance::DistanceType::L1,
  0.0};

// test dense smem strategy
const std::vector<
  SparseDistanceCOOSPMVInputs<int, float, dense_smem_strategy_t>>
  inputs_dense_strategy = {{input_inner_product}, {input_l2_unexpanded},
                           {input_canberra},      {input_lp_unexpanded},
                           {input_linf},          {input_l1}};

typedef SparseDistanceCOOSPMVTest<int, float, dense_smem_strategy_t>
  SparseDistanceCOOSPMVTestDenseStrategyF;
TEST_P(SparseDistanceCOOSPMVTestDenseStrategyF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(SparseDistanceCOOSPMVTests,
                        SparseDistanceCOOSPMVTestDenseStrategyF,
                        ::testing::ValuesIn(inputs_dense_strategy));

// test hash and chunk strategy
const std::vector<SparseDistanceCOOSPMVInputs<int, float, hash_strategy_t>>
  inputs_hash_strategy = {{input_inner_product},
                          {input_inner_product, 0.5, 2},
                          {input_l2_unexpanded},
                          {input_l2_unexpanded, 0.5, 2},
                          {input_canberra},
                          {input_canberra, 0.5, 2},
                          {input_canberra, 0.5, 6},
                          {input_lp_unexpanded},
                          {input_lp_unexpanded, 0.5, 2},
                          {input_lp_unexpanded, 0.5, 6},
                          {input_linf},
                          {input_linf, 0.5, 2},
                          {input_linf, 0.5, 6},
                          {input_l1},
                          {input_l1, 0.5, 2}};

typedef SparseDistanceCOOSPMVTest<int, float, hash_strategy_t>
  SparseDistanceCOOSPMVTestHashStrategyF;
TEST_P(SparseDistanceCOOSPMVTestHashStrategyF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(SparseDistanceCOOSPMVTests,
                        SparseDistanceCOOSPMVTestHashStrategyF,
                        ::testing::ValuesIn(inputs_hash_strategy));

};  // namespace distance
};  // end namespace sparse
};  // end namespace raft
