#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../preprocess_utils.cu"
#include "../test_utils.cuh"

#include <raft/core/device_coo_matrix.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/sparse/matrix/preprocessing.cuh>
#include <raft/util/cudart_utils.hpp>

#include <gtest/gtest.h>

#include <cmath>
#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

template <typename Type_f, typename Index_>
struct SparsePreprocessInputs {
  int n_rows;
  int n_cols;
  int nnz_edges;
};

template <typename Type_f, typename Index_>
class SparsePreprocessCoo
  : public ::testing::TestWithParam<SparsePreprocessInputs<Type_f, Index_>> {
 public:
  SparsePreprocessCoo()
    : params(::testing::TestWithParam<SparsePreprocessInputs<Type_f, Index_>>::GetParam()),
      stream(resource::get_cuda_stream(handle))
  {
  }

 protected:
  void SetUp() override {}

  void Run(bool bm25_on)
  {
    hipStream_t stream = raft::resource::get_cuda_stream(handle);

    int num_rows = pow(2, params.n_rows);
    int num_cols = pow(2, params.n_cols);

    auto rows    = raft::make_device_vector<Index_, int64_t>(handle, params.nnz_edges);
    auto columns = raft::make_device_vector<Index_, int64_t>(handle, params.nnz_edges);
    auto values  = raft::make_device_vector<Type_f, int64_t>(handle, params.nnz_edges);

    rmm::device_uvector<Index_> rows_uvec(rows.size(), stream);
    rmm::device_uvector<Index_> cols_uvec(rows.size(), stream);
    rmm::device_uvector<Type_f> vals_uvec(rows.size(), stream);

    raft::util::create_dataset<Index_, Type_f>(
      handle, rows.view(), columns.view(), values.view(), 5, params.n_rows, params.n_cols);

    raft::sparse::op::coo_sort(int(rows.size()),
                               int(columns.size()),
                               int(values.size()),
                               rows.data_handle(),
                               columns.data_handle(),
                               values.data_handle(),
                               stream);

    raft::copy(rows_uvec.data(), rows.data_handle(), rows.size(), stream);
    raft::copy(cols_uvec.data(), columns.data_handle(), columns.size(), stream);
    raft::copy(vals_uvec.data(), values.data_handle(), values.size(), stream);

    raft::sparse::COO<Type_f, Index_> coo(stream);
    raft::sparse::op::max_duplicates(handle,
                                     coo,
                                     rows_uvec.data(),
                                     cols_uvec.data(),
                                     vals_uvec.data(),
                                     params.nnz_edges,
                                     num_rows,
                                     num_cols);

    auto rows_csr = raft::make_device_vector<Index_, int64_t>(handle, num_rows + 1);

    raft::sparse::convert::sorted_coo_to_csr(
      coo.rows(), coo.nnz, rows_csr.data_handle(), num_rows + 1, stream);

    auto csr_struct_view = raft::make_device_compressed_structure_view(
      rows_csr.data_handle(), coo.cols(), num_rows, num_cols, coo.nnz);

    auto csr_matrix =
      raft::make_device_csr_matrix<Type_f, Index_, Index_, Index_>(handle, csr_struct_view);
    raft::update_device<Type_f>(
      csr_matrix.view().get_elements().data(), coo.vals(), coo.nnz, stream);

    auto coo_struct_view = raft::make_device_coordinate_structure_view(
      coo.rows(), coo.cols(), num_rows, num_cols, int(coo.nnz));
    auto c_matrix =
      raft::make_device_coo_matrix<Type_f, Index_, Index_, Index_>(handle, coo_struct_view);
    raft::update_device<Type_f>(c_matrix.view().get_elements().data(), coo.vals(), coo.nnz, stream);

    auto result = raft::make_device_vector<Type_f, int64_t>(handle, coo.nnz);

    if (bm25_on) {
      auto bm25_vals = raft::make_device_vector<Type_f, int64_t>(handle, coo.nnz);
      sparse::matrix::encode_bm25<Index_, Type_f>(handle, c_matrix.view(), result.view());
      raft::util::calc_tfidf_bm25<Index_, Type_f>(handle, csr_matrix.view(), bm25_vals.view());
      ASSERT_TRUE(raft::devArrMatch<Type_f>(bm25_vals.data_handle(),
                                            result.data_handle(),
                                            result.size(),
                                            raft::CompareApprox<Type_f>(2e-5),
                                            stream));
    } else {
      auto tfidf_vals = raft::make_device_vector<Type_f, int64_t>(handle, coo.nnz);
      sparse::matrix::encode_tfidf<Index_, Type_f>(handle, c_matrix.view(), result.view());
      raft::util::calc_tfidf_bm25<Index_, Type_f>(
        handle, csr_matrix.view(), tfidf_vals.view(), true);
      ASSERT_TRUE(raft::devArrMatch<Type_f>(tfidf_vals.data_handle(),
                                            result.data_handle(),
                                            result.size(),
                                            raft::CompareApprox<Type_f>(2e-5),
                                            stream));
    }

    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  SparsePreprocessInputs<Type_f, Index_> params;
};

using SparsePreprocessTfidfCoo = SparsePreprocessCoo<float, int>;
TEST_P(SparsePreprocessTfidfCoo, Result) { Run(false); }

using SparsePreprocessBm25Coo = SparsePreprocessCoo<float, int>;
TEST_P(SparsePreprocessBm25Coo, Result) { Run(true); }

using SparsePreprocessTfidfCooBig = SparsePreprocessCoo<float, int>;
TEST_P(SparsePreprocessTfidfCooBig, Result) { Run(false); }

using SparsePreprocessBm25CooBig = SparsePreprocessCoo<float, int>;
TEST_P(SparsePreprocessBm25CooBig, Result) { Run(true); }

const std::vector<SparsePreprocessInputs<float, int>> sparse_preprocess_inputs = {
  {
    10,   // n_rows_factor
    10,   // n_cols_factor
    1000  // nnz_edges
  },
};

const std::vector<SparsePreprocessInputs<float, int>> sparse_preprocess_inputs_big = {
  {
    15,      // n_rows_factor
    15,      // n_cols_factor
    1000000  // nnz_edges
  },
};

INSTANTIATE_TEST_CASE_P(SparsePreprocessCoo,
                        SparsePreprocessTfidfCoo,
                        ::testing::ValuesIn(sparse_preprocess_inputs));
INSTANTIATE_TEST_CASE_P(SparsePreprocessCoo,
                        SparsePreprocessBm25Coo,
                        ::testing::ValuesIn(sparse_preprocess_inputs));

INSTANTIATE_TEST_CASE_P(SparsePreprocessCoo,
                        SparsePreprocessTfidfCooBig,
                        ::testing::ValuesIn(sparse_preprocess_inputs_big));
INSTANTIATE_TEST_CASE_P(SparsePreprocessCoo,
                        SparsePreprocessBm25CooBig,
                        ::testing::ValuesIn(sparse_preprocess_inputs_big));

}  // namespace sparse
}  // namespace raft