/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>

#include <raft/core/resources.hpp>
#include <raft/sparse/csr.hpp>
#include <raft/sparse/linalg/add.cuh>

#include "../test_utils.cuh"
#include <raft/util/cudart_utils.hpp>

#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

template <typename Type_f, typename Index_>
struct CSRMatrixVal {
  std::vector<Index_> row_ind;
  std::vector<Index_> row_ind_ptr;
  std::vector<Type_f> values;
};

template <typename Type_f, typename Index_>
struct CSRAddInputs {
  CSRMatrixVal<Type_f, Index_> matrix_a;
  CSRMatrixVal<Type_f, Index_> matrix_b;
  CSRMatrixVal<Type_f, Index_> matrix_verify;
};

template <typename Type_f, typename Index_>
class CSRAddTest : public ::testing::TestWithParam<CSRAddInputs<Type_f, Index_>> {
 public:
  CSRAddTest()
    : params(::testing::TestWithParam<CSRAddInputs<Type_f, Index_>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      ind_a(params.matrix_a.row_ind.size(), stream),
      ind_ptr_a(params.matrix_a.row_ind_ptr.size(), stream),
      values_a(params.matrix_a.row_ind_ptr.size(), stream),
      ind_b(params.matrix_a.row_ind.size(), stream),
      ind_ptr_b(params.matrix_b.row_ind_ptr.size(), stream),
      values_b(params.matrix_b.row_ind_ptr.size(), stream),
      ind_verify(params.matrix_a.row_ind.size(), stream),
      ind_ptr_verify(params.matrix_verify.row_ind_ptr.size(), stream),
      values_verify(params.matrix_verify.row_ind_ptr.size(), stream),
      ind_result(params.matrix_a.row_ind.size(), stream),
      ind_ptr_result(params.matrix_verify.row_ind_ptr.size(), stream),
      values_result(params.matrix_verify.row_ind_ptr.size(), stream)
  {
  }

 protected:
  void SetUp() override
  {
    n_rows     = params.matrix_a.row_ind.size();
    nnz_a      = params.matrix_a.row_ind_ptr.size();
    nnz_b      = params.matrix_b.row_ind_ptr.size();
    nnz_result = params.matrix_verify.row_ind_ptr.size();
  }

  void Run()
  {
    raft::update_device(ind_a.data(), params.matrix_a.row_ind.data(), n_rows, stream);
    raft::update_device(ind_ptr_a.data(), params.matrix_a.row_ind_ptr.data(), nnz_a, stream);
    raft::update_device(values_a.data(), params.matrix_a.values.data(), nnz_a, stream);

    raft::update_device(ind_b.data(), params.matrix_b.row_ind.data(), n_rows, stream);
    raft::update_device(ind_ptr_b.data(), params.matrix_b.row_ind_ptr.data(), nnz_b, stream);
    raft::update_device(values_b.data(), params.matrix_b.values.data(), nnz_b, stream);

    raft::update_device(ind_verify.data(), params.matrix_verify.row_ind.data(), n_rows, stream);
    raft::update_device(
      ind_ptr_verify.data(), params.matrix_verify.row_ind_ptr.data(), nnz_result, stream);
    raft::update_device(
      values_verify.data(), params.matrix_verify.values.data(), nnz_result, stream);

    Index_ nnz = linalg::csr_add_calc_inds<Type_f>(ind_a.data(),
                                                   ind_ptr_a.data(),
                                                   values_a.data(),
                                                   nnz_a,
                                                   ind_b.data(),
                                                   ind_ptr_b.data(),
                                                   values_b.data(),
                                                   nnz_b,
                                                   n_rows,
                                                   ind_result.data(),
                                                   stream);

    ASSERT_TRUE(nnz == nnz_result);
    ASSERT_TRUE(raft::devArrMatch<Index_>(
      ind_verify.data(), ind_result.data(), n_rows, raft::Compare<Index_>(), stream));

    linalg::csr_add_finalize<Type_f>(ind_a.data(),
                                     ind_ptr_a.data(),
                                     values_a.data(),
                                     nnz_a,
                                     ind_b.data(),
                                     ind_ptr_b.data(),
                                     values_b.data(),
                                     nnz_b,
                                     n_rows,
                                     ind_result.data(),
                                     ind_ptr_result.data(),
                                     values_result.data(),
                                     stream);

    ASSERT_TRUE(raft::devArrMatch<Index_>(
      ind_ptr_verify.data(), ind_ptr_result.data(), nnz, raft::Compare<Index_>(), stream));
    ASSERT_TRUE(raft::devArrMatch<Type_f>(
      values_verify.data(), values_result.data(), nnz, raft::Compare<Type_f>(), stream));
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  CSRAddInputs<Type_f, Index_> params;
  Index_ n_rows, nnz_a, nnz_b, nnz_result;
  rmm::device_uvector<Index_> ind_a, ind_b, ind_verify, ind_result, ind_ptr_a, ind_ptr_b,
    ind_ptr_verify, ind_ptr_result;
  rmm::device_uvector<Type_f> values_a, values_b, values_verify, values_result;
};

using CSRAddTestF = CSRAddTest<float, int>;
TEST_P(CSRAddTestF, Result) { Run(); }

using CSRAddTestD = CSRAddTest<double, int>;
TEST_P(CSRAddTestD, Result) { Run(); }

const std::vector<CSRAddInputs<float, int>> csradd_inputs_f = {
  {{{0, 4, 8, 9},
    {1, 2, 3, 4, 1, 2, 3, 5, 0, 1},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 4, 8, 9},
    {1, 2, 5, 4, 0, 2, 3, 5, 1, 0},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 5, 10, 12},
    {1, 2, 3, 4, 5, 1, 2, 3, 5, 0, 0, 1, 1, 0},
    {2.0, 2.0, 0.5, 1.0, 0.5, 1.0, 2.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0}}},
};
const std::vector<CSRAddInputs<double, int>> csradd_inputs_d = {
  {{{0, 4, 8, 9},
    {1, 2, 3, 4, 1, 2, 3, 5, 0, 1},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 4, 8, 9},
    {1, 2, 5, 4, 0, 2, 3, 5, 1, 0},
    {1.0, 1.0, 0.5, 0.5, 1.0, 1.0, 0.5, 0.5, 1.0, 1.0}},
   {{0, 5, 10, 12},
    {1, 2, 3, 4, 5, 1, 2, 3, 5, 0, 0, 1, 1, 0},
    {2.0, 2.0, 0.5, 1.0, 0.5, 1.0, 2.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0}}},
};

INSTANTIATE_TEST_CASE_P(SparseAddTest, CSRAddTestF, ::testing::ValuesIn(csradd_inputs_f));
INSTANTIATE_TEST_CASE_P(SparseAddTest, CSRAddTestD, ::testing::ValuesIn(csradd_inputs_d));

}  // namespace sparse
}  // namespace raft
