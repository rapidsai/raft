/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resource/cusparse_handle.hpp>

#include "../test_utils.cuh"

#include <raft/core/resources.hpp>
#include <raft/linalg/transpose.cuh>
#include <raft/sparse/detail/cusparse_wrappers.h>
#include <raft/util/cudart_utils.hpp>
#include <rmm/device_uvector.hpp>

#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

struct SPGemmiInputs {
  int n_rows, n_cols;
};

template <typename data_t>
class SPGemmiTest : public ::testing::TestWithParam<SPGemmiInputs> {
 public:
  SPGemmiTest()
    : params(::testing::TestWithParam<SPGemmiInputs>::GetParam()),
      stream(resource::get_cuda_stream(handle))
  {
  }

 protected:
  void SetUp() override {}

  void Run()
  {
    // Host problem definition
    float alpha    = 1.0f;
    float beta     = 0.0f;
    int A_num_rows = 5;
    int A_num_cols = 3;
    // int   B_num_rows      = A_num_cols;
    int B_num_cols      = 4;
    int B_nnz           = 9;
    int lda             = A_num_rows;
    int ldc             = A_num_rows;
    int A_size          = lda * A_num_cols;
    int C_size          = ldc * B_num_cols;
    int hB_cscOffsets[] = {0, 3, 4, 7, 9};
    int hB_rows[]       = {0, 2, 3, 1, 0, 2, 3, 1, 3};
    float hB_values[]   = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};
    float hA[]          = {1.0f,
                           2.0f,
                           3.0f,
                           4.0f,
                           5.0f,
                           6.0f,
                           7.0f,
                           8.0f,
                           9.0f,
                           10.0f,
                           11.0f,
                           12.0f,
                           13.0f,
                           14.0f,
                           15.0f};
    std::vector<float> hC(C_size);
    std::vector<float> hC_expected{23, 26, 29, 32,  35,  24, 28, 32, 36, 40,
                                   71, 82, 93, 104, 115, 48, 56, 64, 72, 80};
    //--------------------------------------------------------------------------
    // Device memory management
    rmm::device_uvector<int> dB_cscOffsets(B_num_cols + 1, stream);
    rmm::device_uvector<int> dB_rows(B_nnz, stream);
    rmm::device_uvector<float> dB_values(B_nnz, stream);
    rmm::device_uvector<float> dA(A_size, stream);
    rmm::device_uvector<float> dC(C_size, stream);
    rmm::device_uvector<float> dCT(C_size, stream);

    raft::update_device(dB_cscOffsets.data(), hB_cscOffsets, B_num_cols + 1, stream);
    raft::update_device(dB_rows.data(), hB_rows, B_nnz, stream);
    raft::update_device(dB_values.data(), hB_values, B_nnz, stream);
    raft::update_device(dA.data(), hA, A_size, stream);
    raft::update_device(dC.data(), hC.data(), C_size, stream);

    //--------------------------------------------------------------------------
    // execute gemmi
    RAFT_CUSPARSE_TRY(raft::sparse::detail::cusparsegemmi(resource::get_cusparse_handle(handle),
                                                          A_num_rows,
                                                          B_num_cols,
                                                          A_num_cols,
                                                          B_nnz,
                                                          &alpha,
                                                          dA.data(),
                                                          lda,
                                                          dB_values.data(),
                                                          dB_cscOffsets.data(),
                                                          dB_rows.data(),
                                                          &beta,
                                                          dC.data(),
                                                          ldc,
                                                          resource::get_cuda_stream(handle)));

    //--------------------------------------------------------------------------
    // result check
    raft::update_host(hC.data(), dC.data(), C_size, stream);
    ASSERT_TRUE(hostVecMatch(hC_expected, hC, raft::Compare<float>()));
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  SPGemmiInputs params;
};

using SPGemmiTestF = SPGemmiTest<float>;
TEST_P(SPGemmiTestF, Result) { Run(); }

using SPGemmiTestD = SPGemmiTest<double>;
TEST_P(SPGemmiTestD, Result) { Run(); }

const std::vector<SPGemmiInputs> csc_inputs_f = {{5, 4}};
const std::vector<SPGemmiInputs> csc_inputs_d = {{5, 4}};

INSTANTIATE_TEST_CASE_P(SparseGemmi, SPGemmiTestF, ::testing::ValuesIn(csc_inputs_f));
INSTANTIATE_TEST_CASE_P(SparseGemmi, SPGemmiTestD, ::testing::ValuesIn(csc_inputs_d));

}  // namespace sparse
}  // namespace raft
