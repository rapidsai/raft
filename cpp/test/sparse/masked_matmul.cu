#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"

#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/random/make_blobs.cuh>
#include <raft/sparse/linalg/masked_matmul.hpp>
#include <raft/util/cudart_utils.hpp>

#include <thrust/reduce.h>

#include <hipsparse.h>
#include <gtest/gtest.h>

#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

template <typename value_t, typename output_t, typename index_t>
struct MaskedMatmulInputs {
  output_t tolerance;

  index_t m;
  index_t k;
  index_t n;

  float sparsity;

  unsigned long long int seed;
};

template <typename value_t>
struct sum_abs_op {
  __host__ __device__ value_t operator()(const value_t& x, const value_t& y) const
  {
    return y >= value_t(0.0) ? (x + y) : (x - y);
  }
};

struct float_to_half {
  __host__ __device__ __half operator()(const float x) const { return __float2half(x); }
};

template <typename value_t, typename output_t, typename index_t>
::std::ostream& operator<<(::std::ostream& os,
                           const MaskedMatmulInputs<value_t, output_t, index_t>& params)
{
  os << " m: " << params.m << "\tk: " << params.k << "\tn: " << params.n
     << "\tsparsity: " << params.sparsity;

  return os;
}

bool isCuSparseVersionGreaterThan_12_0_1()
{
  int version;
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  hipsparseGetVersion(handle, &version);

  int major = version / 1000;
  int minor = (version % 1000) / 100;
  int patch = version % 100;

  hipsparseDestroy(handle);

  return (major > 12) || (major == 12 && minor > 0) || (major == 12 && minor == 0 && patch >= 2);
}

template <typename value_t,
          typename output_t,
          typename index_t,
          typename bitmap_t      = uint32_t,
          typename LayoutPolicyA = raft::row_major,
          typename LayoutPolicyB = raft::row_major>
class MaskedMatmulTest
  : public ::testing::TestWithParam<MaskedMatmulInputs<value_t, output_t, index_t>> {
 public:
  MaskedMatmulTest()
    : params(::testing::TestWithParam<MaskedMatmulInputs<value_t, output_t, index_t>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      a_data_d(0, resource::get_cuda_stream(handle)),
      b_data_d(0, resource::get_cuda_stream(handle)),
      bitmap_d(0, resource::get_cuda_stream(handle)),
      c_indptr_d(0, resource::get_cuda_stream(handle)),
      c_indices_d(0, resource::get_cuda_stream(handle)),
      c_data_d(0, resource::get_cuda_stream(handle)),
      c_expected_data_d(0, resource::get_cuda_stream(handle))
  {
  }

 protected:
  index_t create_sparse_matrix(index_t m, index_t n, float sparsity, std::vector<bitmap_t>& bitmap)
  {
    index_t total    = static_cast<index_t>(m * n);
    index_t num_ones = static_cast<index_t>((total * 1.0f) * sparsity);
    index_t res      = num_ones;

    for (auto& item : bitmap) {
      item = static_cast<bitmap_t>(0);
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<index_t> dis(0, total - 1);

    while (num_ones > 0) {
      index_t index = dis(gen);

      bitmap_t& element    = bitmap[index / (8 * sizeof(bitmap_t))];
      index_t bit_position = index % (8 * sizeof(bitmap_t));

      if (((element >> bit_position) & 1) == 0) {
        element |= (static_cast<index_t>(1) << bit_position);
        num_ones--;
      }
    }
    return res;
  }

  void cpu_convert_to_csr(std::vector<bitmap_t>& bitmap,
                          index_t rows,
                          index_t cols,
                          std::vector<index_t>& indices,
                          std::vector<index_t>& indptr)
  {
    index_t offset_indptr   = 0;
    index_t offset_values   = 0;
    indptr[offset_indptr++] = 0;

    index_t index        = 0;
    bitmap_t element     = 0;
    index_t bit_position = 0;

    for (index_t i = 0; i < rows; ++i) {
      for (index_t j = 0; j < cols; ++j) {
        index        = i * cols + j;
        element      = bitmap[index / (8 * sizeof(bitmap_t))];
        bit_position = index % (8 * sizeof(bitmap_t));

        if (((element >> bit_position) & 1)) {
          indices[offset_values] = static_cast<index_t>(j);
          offset_values++;
        }
      }
      indptr[offset_indptr++] = static_cast<index_t>(offset_values);
    }
  }

  void cpu_sddmm(const std::vector<value_t>& A,
                 const std::vector<value_t>& B,
                 std::vector<output_t>& vals,
                 const std::vector<index_t>& cols,
                 const std::vector<index_t>& row_ptrs,
                 bool is_row_major_A,
                 bool is_row_major_B)
  {
    if (params.m * params.k != static_cast<index_t>(A.size()) ||
        params.k * params.n != static_cast<index_t>(B.size())) {
      std::cerr << "Matrix dimensions and vector size do not match!" << std::endl;
      return;
    }

    for (index_t i = 0; i < params.m; ++i) {
      for (index_t j = row_ptrs[i]; j < row_ptrs[i + 1]; ++j) {
        output_t sum = 0;
        for (index_t l = 0; l < params.k; ++l) {
          index_t a_index = i * params.k + l;
          index_t b_index = cols[j] * params.k + l;
          if constexpr ((std::is_same_v<output_t, float> && std::is_same_v<value_t, half>)) {
            sum += __half2float(A[a_index]) * __half2float(B[b_index]);
          } else {
            sum += A[a_index] * B[b_index];
          }
        }
        vals[j] = sum;
      }
    }
  }

  void make_data()
  {
    index_t a_size = params.m * params.k;
    index_t b_size = params.k * params.n;
    index_t c_size = params.m * params.n;

    index_t element = raft::ceildiv(params.m * params.n, index_t(sizeof(bitmap_t) * 8));
    std::vector<bitmap_t> bitmap_h(element);

    std::vector<value_t> a_data_h(a_size);
    std::vector<value_t> b_data_h(b_size);

    a_data_d.resize(a_size, stream);
    b_data_d.resize(b_size, stream);
    bitmap_d.resize(bitmap_h.size(), stream);

    auto blobs_a_b = raft::make_device_matrix<output_t, index_t>(handle, 1, a_size + b_size);
    auto labels    = raft::make_device_vector<index_t, index_t>(handle, 1);

    raft::random::make_blobs<output_t, index_t>(blobs_a_b.data_handle(),
                                                labels.data_handle(),
                                                1,
                                                a_size + b_size,
                                                1,
                                                stream,
                                                false,
                                                nullptr,
                                                nullptr,
                                                output_t(1.0),
                                                false,
                                                output_t(-1.0f),
                                                output_t(1.0f),
                                                uint64_t(2024));

    if constexpr ((std::is_same_v<output_t, float> && std::is_same_v<value_t, half>)) {
      {
        thrust::device_ptr<output_t> d_output_ptr =
          thrust::device_pointer_cast(blobs_a_b.data_handle());
        thrust::device_ptr<value_t> d_value_ptr = thrust::device_pointer_cast(a_data_d.data());
        thrust::transform(thrust::cuda::par.on(stream),
                          d_output_ptr,
                          d_output_ptr + a_size,
                          d_value_ptr,
                          float_to_half());
      }
      {
        thrust::device_ptr<output_t> d_output_ptr =
          thrust::device_pointer_cast(blobs_a_b.data_handle() + a_size);
        thrust::device_ptr<value_t> d_value_ptr = thrust::device_pointer_cast(b_data_d.data());
        thrust::transform(thrust::cuda::par.on(stream),
                          d_output_ptr,
                          d_output_ptr + b_size,
                          d_value_ptr,
                          float_to_half());
      }
      raft::copy(a_data_h.data(), a_data_d.data(), a_size, stream);
      raft::copy(b_data_h.data(), b_data_d.data(), b_size, stream);
    } else {
      raft::copy(a_data_h.data(), blobs_a_b.data_handle(), a_size, stream);
      raft::copy(b_data_h.data(), blobs_a_b.data_handle() + a_size, b_size, stream);

      raft::copy(a_data_d.data(), blobs_a_b.data_handle(), a_size, stream);
      raft::copy(b_data_d.data(), blobs_a_b.data_handle() + a_size, b_size, stream);
    }

    resource::sync_stream(handle);

    index_t c_true_nnz = create_sparse_matrix(params.m, params.n, params.sparsity, bitmap_h);

    std::vector<index_t> c_indptr_h(params.m + 1);
    std::vector<index_t> c_indices_h(c_true_nnz);
    std::vector<output_t> c_data_h(c_true_nnz);

    cpu_convert_to_csr(bitmap_h, params.m, params.n, c_indices_h, c_indptr_h);

    c_data_d.resize(c_data_h.size(), stream);

    update_device(c_data_d.data(), c_data_h.data(), c_data_h.size(), stream);
    update_device(bitmap_d.data(), bitmap_h.data(), bitmap_h.size(), stream);
    resource::sync_stream(handle);

    cpu_sddmm(a_data_h, b_data_h, c_data_h, c_indices_h, c_indptr_h, true, true);

    c_indptr_d.resize(c_indptr_h.size(), stream);
    c_indices_d.resize(c_indices_h.size(), stream);
    c_expected_data_d.resize(c_data_h.size(), stream);

    update_device(c_indptr_d.data(), c_indptr_h.data(), c_indptr_h.size(), stream);
    update_device(c_indices_d.data(), c_indices_h.data(), c_indices_h.size(), stream);
    update_device(c_expected_data_d.data(), c_data_h.data(), c_data_h.size(), stream);

    resource::sync_stream(handle);
  }

  void SetUp() override
  {
    if (std::is_same_v<value_t, half> && !isCuSparseVersionGreaterThan_12_0_1()) {
      GTEST_SKIP() << "Skipping all tests for half-float as cuSparse doesn't support it.";
    }
    make_data();
  }

  void Run()
  {
    auto A =
      raft::make_device_matrix_view<const value_t, index_t>(a_data_d.data(), params.m, params.k);
    auto B =
      raft::make_device_matrix_view<const value_t, index_t>(b_data_d.data(), params.n, params.k);

    auto mask =
      raft::core::bitmap_view<const bitmap_t, index_t>(bitmap_d.data(), params.m, params.n);

    auto c_structure = raft::make_device_compressed_structure_view<index_t, index_t, index_t>(
      c_indptr_d.data(),
      c_indices_d.data(),
      params.m,
      params.n,
      static_cast<index_t>(c_indices_d.size()));

    auto C = raft::make_device_csr_matrix_view<output_t>(c_data_d.data(), c_structure);

    raft::sparse::linalg::masked_matmul(handle, A, B, mask, C);

    resource::sync_stream(handle);

    ASSERT_TRUE(raft::devArrMatch<output_t>(c_expected_data_d.data(),
                                            C.get_elements().data(),
                                            c_expected_data_d.size(),
                                            raft::CompareApprox<output_t>(params.tolerance),
                                            stream));

    thrust::device_ptr<output_t> expected_data_ptr =
      thrust::device_pointer_cast(c_expected_data_d.data());
    output_t sum_abs = thrust::reduce(thrust::cuda::par.on(stream),
                                      expected_data_ptr,
                                      expected_data_ptr + c_expected_data_d.size(),
                                      output_t(0.0f),
                                      sum_abs_op<output_t>());
    output_t avg     = sum_abs / (1.0f * c_expected_data_d.size());

    ASSERT_GE(avg, (params.tolerance * static_cast<output_t>(0.001f)));
  }

  raft::resources handle;
  hipStream_t stream;
  MaskedMatmulInputs<value_t, output_t, index_t> params;

  rmm::device_uvector<value_t> a_data_d;
  rmm::device_uvector<value_t> b_data_d;
  rmm::device_uvector<bitmap_t> bitmap_d;

  rmm::device_uvector<index_t> c_indptr_d;
  rmm::device_uvector<index_t> c_indices_d;
  rmm::device_uvector<output_t> c_data_d;

  rmm::device_uvector<output_t> c_expected_data_d;
};

using MaskedMatmulTestF = MaskedMatmulTest<float, float, int>;
TEST_P(MaskedMatmulTestF, Result) { Run(); }

using MaskedMatmulTestD = MaskedMatmulTest<double, double, int>;
TEST_P(MaskedMatmulTestD, Result) { Run(); }

using MaskedMatmulTestH = MaskedMatmulTest<half, float, int>;
TEST_P(MaskedMatmulTestH, Result) { Run(); }

const std::vector<MaskedMatmulInputs<float, float, int>> sddmm_inputs_f = {
  {0.001f, 2, 255, 1023, 0.19, 1234ULL},
  {0.001f, 2, 255, 1023 * 2, 0.19, 1234ULL},
  {0.001f, 2, 255, 1023 * 3, 0.38, 1234ULL},
  {0.0001f, 10, 255, 13000, 0.01, 1234ULL},
  {0.0001f, 10, 5, 32, 0.1, 1234ULL},
  {0.001f, 11, 255, 1023, 0.19, 1234ULL},
  {0.001f, 11, 255, 1023 * 2, 0.19, 1234ULL},
  {0.001f, 11, 255, 1023 * 3, 0.38, 1234ULL},
  {0.0003f, 32, 1024, 1024, 0.2, 1234ULL},
  {0.0001f, 1024, 32, 1024, 0.1, 1234ULL},
  {0.001f, 1024, 1024, 1024, 0.19, 1234ULL},
  {0.001f, 1023, 1023, 1023 * 3, 0.38, 1234ULL},
  {0.001f, 1025, 1025, 1025 * 3, 0.31, 1234ULL},
  {0.0001f, 1024, 1024, 32, 0.3, 1234ULL},
  {0.0001f, 1024, 32, 1024, 0.4, 1234ULL},
  {0.0003f, 31, 1025, 1025, 0.19, 1234ULL},
  {0.001f, 1024, 1024, 1024, 0.1, 1234ULL}};

const std::vector<MaskedMatmulInputs<double, double, int>> sddmm_inputs_d = {
  {0.0001f, 2, 255, 1023, 0.19, 1234ULL},
  {0.0001f, 2, 255, 1023 * 2, 0.19, 1234ULL},
  {0.0001f, 2, 255, 1023 * 3, 0.38, 1234ULL},
  {0.0001f, 10, 255, 13000, 0.01, 1234ULL},
  {0.0001f, 10, 5, 32, 0.1, 1234ULL},
  {0.0001f, 11, 255, 1023, 0.19, 1234ULL},
  {0.0001f, 11, 255, 1023 * 2, 0.19, 1234ULL},
  {0.0001f, 11, 255, 1023 * 3, 0.38, 1234ULL},
  {0.0001f, 32, 1024, 1024, 0.2, 1234ULL},
  {0.0001f, 1024, 32, 1024, 0.1, 1234ULL},
  {0.0001f, 1024, 1024, 1024, 0.19, 1234ULL},
  {0.0001f, 1023, 1023, 1023 * 3, 0.38, 1234ULL},
  {0.0001f, 1025, 1025, 1025 * 3, 0.31, 1234ULL},
  {0.0001f, 1024, 1024, 32, 0.3, 1234ULL},
  {0.0001f, 1024, 32, 1024, 0.4, 1234ULL},
  {0.0001f, 31, 1025, 1025, 0.19, 1234ULL},
  {0.0001f, 1024, 1024, 1024, 0.1, 1234ULL}};

const std::vector<MaskedMatmulInputs<half, float, int>> sddmm_inputs_h = {
  {0.001f, 2, 255, 1023, 0.19, 1234ULL},
  {0.001f, 2, 255, 1023 * 2, 0.19, 1234ULL},
  {0.001f, 2, 255, 1023 * 3, 0.38, 1234ULL},
  {0.0001f, 10, 255, 13000, 0.01, 1234ULL},
  {0.0001f, 10, 5, 32, 0.1, 1234ULL},
  {0.001f, 11, 255, 1023, 0.19, 1234ULL},
  {0.001f, 11, 255, 1023 * 2, 0.19, 1234ULL},
  {0.001f, 11, 255, 1023 * 3, 0.38, 1234ULL},
  {0.0003f, 32, 1024, 1024, 0.2, 1234ULL},
  {0.0001f, 1024, 32, 1024, 0.1, 1234ULL},
  {0.001f, 1024, 1024, 1024, 0.19, 1234ULL},
  {0.001f, 1023, 1023, 1023 * 3, 0.38, 1234ULL},
  {0.001f, 1025, 1025, 1025 * 3, 0.31, 1234ULL},
  {0.0001f, 1024, 1024, 32, 0.3, 1234ULL},
  {0.0001f, 1024, 32, 1024, 0.4, 1234ULL},
  {0.0003f, 31, 1025, 1025, 0.19, 1234ULL},
  {0.001f, 1024, 1024, 1024, 0.1, 1234ULL}};

INSTANTIATE_TEST_CASE_P(MaskedMatmulTest, MaskedMatmulTestF, ::testing::ValuesIn(sddmm_inputs_f));

INSTANTIATE_TEST_CASE_P(MaskedMatmulTest, MaskedMatmulTestD, ::testing::ValuesIn(sddmm_inputs_d));

INSTANTIATE_TEST_CASE_P(MaskedMatmulTest, MaskedMatmulTestH, ::testing::ValuesIn(sddmm_inputs_h));

}  // namespace sparse
}  // namespace raft
