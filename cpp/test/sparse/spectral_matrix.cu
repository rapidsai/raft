/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <iostream>
#include <memory>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resource/device_id.hpp>
#include <raft/core/resources.hpp>

#include <raft/spectral/matrix_wrappers.hpp>

namespace raft {
namespace spectral {
namespace matrix {
namespace {
template <typename index_type, typename value_type>
struct csr_view_t {
  index_type* offsets;
  index_type* indices;
  value_type* edge_data;
  index_type number_of_vertices;
  index_type number_of_edges;
};
}  // namespace
TEST(Raft, SpectralMatrices)
{
  using index_type = int;
  using value_type = double;

  raft::resources h;
  ASSERT_EQ(0, raft::resource::get_device_id(h));

  csr_view_t<index_type, value_type> csr_v{nullptr, nullptr, nullptr, 0, 0};

  int const sz = 10;
  vector_t<index_type> d_v{h, sz};

  index_type* ro{nullptr};
  index_type* ci{nullptr};
  value_type* vs{nullptr};
  index_type nnz   = 0;
  index_type nrows = 0;
  sparse_matrix_t<index_type, value_type> sm1{h, ro, ci, vs, nrows, nnz};
  sparse_matrix_t<index_type, value_type> sm2{h, csr_v};
  ASSERT_EQ(nullptr, sm1.row_offsets_);
  ASSERT_EQ(nullptr, sm2.row_offsets_);

  auto stream = resource::get_cuda_stream(h);

  auto cnstr_lm1 = [&h, ro, ci, vs, nrows, nnz](void) {
    laplacian_matrix_t<index_type, value_type> lm1{h, ro, ci, vs, nrows, nnz};
  };
  EXPECT_ANY_THROW(cnstr_lm1());  // because of nullptr ptr args

  auto cnstr_lm2 = [&h, &sm2](void) { laplacian_matrix_t<index_type, value_type> lm2{h, sm2}; };
  EXPECT_ANY_THROW(cnstr_lm2());  // because of nullptr ptr args

  auto cnstr_mm1 = [&h, ro, ci, vs, nrows, nnz](void) {
    modularity_matrix_t<index_type, value_type> mm1{h, ro, ci, vs, nrows, nnz};
  };
  EXPECT_ANY_THROW(cnstr_mm1());  // because of nullptr ptr args

  auto cnstr_mm2 = [&h, &sm2](void) { modularity_matrix_t<index_type, value_type> mm2{h, sm2}; };
  EXPECT_ANY_THROW(cnstr_mm2());  // because of nullptr ptr args
}

}  // namespace matrix
}  // namespace spectral
}  // namespace raft
