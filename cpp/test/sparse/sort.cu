/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/random/rng.hpp>

#include <raft/sparse/op/sort.hpp>

#include <iostream>

namespace raft {
namespace sparse {

template <typename T>
struct SparseSortInput {
  int m, n, nnz;
  unsigned long long int seed;
};

template <typename T>
class SparseSortTest : public ::testing::TestWithParam<SparseSortInput<T>> {
 protected:
  void SetUp() override {}

  void TearDown() override {}

 protected:
  SparseSortInput<T> params;
};

const std::vector<SparseSortInput<float>> inputsf = {{5, 10, 5, 1234ULL}};

typedef SparseSortTest<float> COOSort;
TEST_P(COOSort, Result)
{
  params = ::testing::TestWithParam<SparseSortInput<float>>::GetParam();
  raft::random::Rng r(params.seed);
  hipStream_t stream;
  RAFT_CUDA_TRY(hipStreamCreate(&stream));

  rmm::device_uvector<int> in_rows(params.nnz, stream);
  rmm::device_uvector<int> in_cols(params.nnz, stream);
  rmm::device_uvector<int> verify(params.nnz, stream);
  rmm::device_uvector<float> in_vals(params.nnz, stream);

  r.uniform(in_vals.data(), params.nnz, float(-1.0), float(1.0), stream);

  int* in_rows_h = (int*)malloc(params.nnz * sizeof(int));
  int* in_cols_h = (int*)malloc(params.nnz * sizeof(int));
  int* verify_h  = (int*)malloc(params.nnz * sizeof(int));

  for (int i = 0; i < params.nnz; i++) {
    in_rows_h[i] = params.nnz - i - 1;
    verify_h[i]  = i;
    in_cols_h[i] = i;
  }

  raft::update_device(in_rows.data(), in_rows_h, params.nnz, stream);

  raft::update_device(in_cols.data(), in_cols_h, params.nnz, stream);
  raft::update_device(verify.data(), verify_h, params.nnz, stream);

  op::coo_sort(
    params.m, params.n, params.nnz, in_rows.data(), in_cols.data(), in_vals.data(), stream);

  ASSERT_TRUE(raft::devArrMatch<int>(
    verify.data(), in_rows.data(), params.nnz, raft::Compare<int>(), stream));

  delete[] in_rows_h;
  delete[] in_cols_h;
  delete[] verify_h;

  RAFT_CUDA_TRY(hipStreamDestroy(stream));
}

INSTANTIATE_TEST_CASE_P(SparseSortTest, COOSort, ::testing::ValuesIn(inputsf));

}  // namespace sparse
}  // namespace raft
