/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

#include <raft/sparse/op/sort.cuh>

#include <iostream>
#include <memory>

namespace raft {
namespace sparse {

template <typename T>
struct SparseSortInput {
  int m, n, nnz;
  unsigned long long int seed;
};

template <typename T>
class SparseSortTest : public ::testing::TestWithParam<SparseSortInput<T>> {
 protected:
  void SetUp() override {}

  void TearDown() override {}

 protected:
  SparseSortInput<T> params;
};

const std::vector<SparseSortInput<float>> inputsf = {{5, 10, 5, 1234ULL}};

typedef SparseSortTest<float> COOSort;
TEST_P(COOSort, Result)
{
  params = ::testing::TestWithParam<SparseSortInput<float>>::GetParam();
  raft::random::RngState r(params.seed);
  raft::resources h;
  auto stream = resource::get_cuda_stream(h);

  rmm::device_uvector<int> in_rows(params.nnz, stream);
  rmm::device_uvector<int> in_cols(params.nnz, stream);
  rmm::device_uvector<int> verify(params.nnz, stream);
  rmm::device_uvector<float> in_vals(params.nnz, stream);

  uniform(h, r, in_vals.data(), params.nnz, float(-1.0), float(1.0));

  auto in_rows_h = std::make_unique<int[]>(params.nnz);
  auto in_cols_h = std::make_unique<int[]>(params.nnz);
  auto verify_h  = std::make_unique<int[]>(params.nnz);

  for (int i = 0; i < params.nnz; i++) {
    in_rows_h[i] = params.nnz - i - 1;
    verify_h[i]  = i;
    in_cols_h[i] = i;
  }

  raft::update_device(in_rows.data(), in_rows_h.get(), params.nnz, stream);

  raft::update_device(in_cols.data(), in_cols_h.get(), params.nnz, stream);
  raft::update_device(verify.data(), verify_h.get(), params.nnz, stream);

  op::coo_sort(
    params.m, params.n, params.nnz, in_rows.data(), in_cols.data(), in_vals.data(), stream);

  ASSERT_TRUE(raft::devArrMatch<int>(
    verify.data(), in_rows.data(), params.nnz, raft::Compare<int>(), stream));
}

INSTANTIATE_TEST_CASE_P(SparseSortTest, COOSort, ::testing::ValuesIn(inputsf));

}  // namespace sparse
}  // namespace raft
