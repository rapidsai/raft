/*
 * Copyright (c) 2018-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipsparse.h>

#include <gtest/gtest.h>

#include <raft/cudart_utils.h>
#include <raft/sparse/cusparse_wrappers.h>

#include <raft/sparse/linalg/transpose.h>

#include "../test_utils.h"

namespace raft {
namespace sparse {

using namespace raft;
using namespace raft::sparse;

template <typename value_idx, typename value_t>
struct CSRTransposeInputs {
  value_idx nrows;
  value_idx ncols;
  value_idx nnz;

  std::vector<value_idx> indptr_h;
  std::vector<value_idx> indices_h;
  std::vector<value_t> data_h;

  std::vector<value_idx> out_indptr_ref_h;
  std::vector<value_idx> out_indices_ref_h;
  std::vector<value_t> out_data_ref_h;
};

template <typename value_idx, typename value_t>
::std::ostream &operator<<(::std::ostream &os,
                           const CSRTransposeInputs<value_idx, value_t> &dims) {
  return os;
}

template <typename value_idx, typename value_t>
class CSRTransposeTest
  : public ::testing::TestWithParam<CSRTransposeInputs<value_idx, value_t>> {
 protected:
  void make_data() {
    std::vector<value_idx> indptr_h = params.indptr_h;
    std::vector<value_idx> indices_h = params.indices_h;
    std::vector<value_t> data_h = params.data_h;

    raft::allocate(indptr, indptr_h.size(), stream);
    raft::allocate(indices, indices_h.size(), stream);
    raft::allocate(data, data_h.size(), stream);

    update_device(indptr, indptr_h.data(), indptr_h.size(), stream);
    update_device(indices, indices_h.data(), indices_h.size(), stream);
    update_device(data, data_h.data(), data_h.size(), stream);

    std::vector<value_idx> out_indptr_ref_h = params.out_indptr_ref_h;
    std::vector<value_idx> out_indices_ref_h = params.out_indices_ref_h;
    std::vector<value_t> out_data_ref_h = params.out_data_ref_h;

    raft::allocate(out_indptr_ref, out_indptr_ref_h.size(), stream);
    raft::allocate(out_indices_ref, out_indices_ref_h.size(), stream);
    raft::allocate(out_data_ref, out_data_ref_h.size(), stream);

    update_device(out_indptr_ref, out_indptr_ref_h.data(),
                  out_indptr_ref_h.size(), stream);
    update_device(out_indices_ref, out_indices_ref_h.data(),
                  out_indices_ref_h.size(), stream);
    update_device(out_data_ref, out_data_ref_h.data(), out_data_ref_h.size(),
                  stream);

    raft::allocate(out_indptr, out_indptr_ref_h.size(), stream);
    raft::allocate(out_indices, out_indices_ref_h.size(), stream);
    raft::allocate(out_data, out_data_ref_h.size(), stream);
  }

  void SetUp() override {
    params = ::testing::TestWithParam<
      CSRTransposeInputs<value_idx, value_t>>::GetParam();
    CUDA_CHECK(hipStreamCreate(&stream));
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    make_data();

    raft::sparse::linalg::csr_transpose(
      handle, indptr, indices, data, out_indptr, out_indices, out_data,
      params.nrows, params.ncols, params.nnz, stream);

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUSPARSE_CHECK(hipsparseDestroy(handle));
  }

  void TearDown() override {
    raft::deallocate_all(stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void compare() {
    ASSERT_TRUE(devArrMatch(out_indptr, out_indptr_ref,
                            params.out_indptr_ref_h.size(),
                            Compare<value_t>()));
    ASSERT_TRUE(devArrMatch(out_indices, out_indices_ref,
                            params.out_indices_ref_h.size(),
                            Compare<value_t>()));
    ASSERT_TRUE(devArrMatch(out_data, out_data_ref,
                            params.out_data_ref_h.size(), Compare<value_t>()));
  }

 protected:
  hipStream_t stream;
  hipsparseHandle_t handle;

  // input data
  value_idx *indptr, *indices;
  value_t *data;

  // output data
  value_idx *out_indptr, *out_indices;
  value_t *out_data;

  // expected output data
  value_idx *out_indptr_ref, *out_indices_ref;
  value_t *out_data_ref;

  CSRTransposeInputs<value_idx, value_t> params;
};

const std::vector<CSRTransposeInputs<int, float>> inputs_i32_f = {
  {
    4,
    2,
    8,
    {0, 2, 4, 6, 8},
    {0, 1, 0, 1, 0, 1, 0, 1},  // indices
    {1.0f, 3.0f, 1.0f, 5.0f, 50.0f, 28.0f, 16.0f, 2.0f},
    {0, 4, 8},
    {0, 1, 2, 3, 0, 1, 2, 3},  // indices
    {1.0f, 1.0f, 50.0f, 16.0f, 3.0f, 5.0f, 28.0f, 2.0f},
  },
};
typedef CSRTransposeTest<int, float> CSRTransposeTestF;
TEST_P(CSRTransposeTestF, Result) { compare(); }
INSTANTIATE_TEST_CASE_P(CSRTransposeTest, CSRTransposeTestF,
                        ::testing::ValuesIn(inputs_i32_f));

};  // end namespace sparse
};  // end namespace raft
