/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"

#include <raft/core/handle.hpp>
#include <raft/spectral/modularity_maximization.cuh>
#include <raft/spectral/partition.cuh>

#include <gtest/gtest.h>

#include <iostream>
#include <memory>

namespace raft {
namespace cluster {

/**
 * Warning: There appears to be a CUDA 12.2 bug in cusparse that causes an
 * alignment issue. We've fixed the bug in our code through a workaround
 * (see raft/sparse/linalg/spmm.hpp for fix). This test is meant to fail
 * in the case where the fix is accidentally reverted, so that it doesn't
 * break any downstream libraries that depend on RAFT
 */
TEST(Raft, Spectral)
{
  raft::handle_t handle;

  std::vector<int32_t> h_offsets({0, 2, 4, 7, 10, 12, 14});
  std::vector<int32_t> h_indices({1, 2, 0, 2, 0, 1, 3, 2, 4, 5, 3, 5, 3, 4});
  std::vector<float> h_values(
    {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0});
  std::vector<int32_t> expected_clustering({1, 1, 1, 0, 0, 0});

  int32_t n_clusters{2};
  int32_t n_eigenvectors{2};
  int32_t evs_max_it{100};
  int32_t kmean_max_it{100};
  int32_t restartIter_lanczos = 15 + n_eigenvectors;
  float evs_tol{0.001};
  float kmean_tol{0.001};
  unsigned long long seed1{1234567};
  unsigned long long seed2{12345678};
  bool reorthog{false};

  rmm::device_uvector<int32_t> offsets(h_offsets.size(), handle.get_stream());
  rmm::device_uvector<int32_t> indices(h_indices.size(), handle.get_stream());
  rmm::device_uvector<float> values(h_indices.size(), handle.get_stream());
  rmm::device_uvector<int32_t> clustering(expected_clustering.size(), handle.get_stream());
  rmm::device_uvector<float> eigenvalues(n_eigenvectors, handle.get_stream());
  rmm::device_uvector<float> eigenvectors(n_eigenvectors * expected_clustering.size(),
                                          handle.get_stream());

  rmm::device_uvector<int32_t> exp_dev(expected_clustering.size(), handle.get_stream());

  raft::update_device(
    exp_dev.data(), expected_clustering.data(), expected_clustering.size(), handle.get_stream());

  raft::update_device(offsets.data(), h_offsets.data(), h_offsets.size(), handle.get_stream());
  raft::update_device(indices.data(), h_indices.data(), h_indices.size(), handle.get_stream());
  raft::update_device(values.data(), h_values.data(), h_values.size(), handle.get_stream());

  raft::spectral::matrix::sparse_matrix_t<int32_t, float> const matrix{
    handle,
    offsets.data(),
    indices.data(),
    values.data(),
    static_cast<int32_t>(offsets.size() - 1),
    static_cast<int32_t>(indices.size())};

  raft::spectral::eigen_solver_config_t<int32_t, float> eig_cfg{
    n_eigenvectors, evs_max_it, restartIter_lanczos, evs_tol, reorthog, seed1};
  raft::spectral::lanczos_solver_t<int32_t, float> eig_solver{eig_cfg};

  raft::spectral::cluster_solver_config_t<int32_t, float> clust_cfg{
    n_clusters, kmean_max_it, kmean_tol, seed2};
  raft::spectral::kmeans_solver_t<int32_t, float> cluster_solver{clust_cfg};

  raft::spectral::partition(handle,
                            matrix,
                            eig_solver,
                            cluster_solver,
                            clustering.data(),
                            eigenvalues.data(),
                            eigenvectors.data());

  ASSERT_TRUE(devArrMatch(expected_clustering.data(),
                          exp_dev.data(),
                          exp_dev.size(),
                          1,
                          raft::Compare<int32_t>(),
                          handle.get_stream()));
}

}  // namespace cluster
}  // namespace raft