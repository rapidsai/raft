/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <iostream>
#include <memory>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>

#include <raft/spectral/cluster_solvers_deprecated.cuh>

namespace raft {
namespace spectral {

TEST(Raft, ClusterSolvers)
{
  using namespace matrix;
  using index_type = int;
  using value_type = double;

  raft::resources h;

  index_type maxiter{100};
  value_type tol{1.0e-10};
  unsigned long long seed{100110021003};

  auto stream = resource::get_cuda_stream(h);

  index_type n{100};
  index_type d{10};
  index_type k{5};

  // nullptr expected to trigger exceptions:
  //
  value_type* eigvecs{nullptr};
  index_type* codes{nullptr};

  cluster_solver_config_deprecated_t<index_type, value_type> cfg{k, maxiter, tol, seed};
  kmeans_solver_deprecated_t<index_type, value_type> cluster_solver{cfg};

  EXPECT_ANY_THROW(cluster_solver.solve(h, n, d, eigvecs, codes));
}

}  // namespace spectral
}  // namespace raft
