#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include <gtest/gtest.h>
#include <raft/matrix/math.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace matrix {

template <typename Type>
__global__ void nativePowerKernel(Type* in, Type* out, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) { out[idx] = in[idx] * in[idx]; }
}

template <typename Type>
void naivePower(Type* in, Type* out, int len, hipStream_t stream)
{
  static const int TPB = 64;
  int nblks            = raft::ceildiv(len, TPB);
  nativePowerKernel<Type><<<nblks, TPB, 0, stream>>>(in, out, len);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename Type>
__global__ void nativeSqrtKernel(Type* in, Type* out, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) { out[idx] = sqrt(in[idx]); }
}

template <typename Type>
void naiveSqrt(Type* in, Type* out, int len, hipStream_t stream)
{
  static const int TPB = 64;
  int nblks            = raft::ceildiv(len, TPB);
  nativeSqrtKernel<Type><<<nblks, TPB, 0, stream>>>(in, out, len);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename Type>
__global__ void naiveSignFlipKernel(Type* in, Type* out, int rowCount, int colCount)
{
  int d_i = blockIdx.x * rowCount;
  int end = d_i + rowCount;

  if (blockIdx.x < colCount) {
    Type max      = 0.0;
    int max_index = 0;
    for (int i = d_i; i < end; i++) {
      Type val = in[i];
      if (val < 0.0) { val = -val; }
      if (val > max) {
        max       = val;
        max_index = i;
      }
    }

    for (int i = d_i; i < end; i++) {
      if (in[max_index] < 0.0) {
        out[i] = -in[i];
      } else {
        out[i] = in[i];
      }
    }
  }

  __syncthreads();
}

template <typename Type>
void naiveSignFlip(Type* in, Type* out, int rowCount, int colCount, hipStream_t stream)
{
  naiveSignFlipKernel<Type><<<colCount, 1, 0, stream>>>(in, out, rowCount, colCount);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename T>
struct MathInputs {
  T tolerance;
  int n_row;
  int n_col;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const MathInputs<T>& dims)
{
  return os;
}

template <typename T>
class MathTest : public ::testing::TestWithParam<MathInputs<T>> {
 public:
  MathTest()
    : params(::testing::TestWithParam<MathInputs<T>>::GetParam()),
      stream(handle.get_stream()),
      in_power(params.len, stream),
      out_power_ref(params.len, stream),
      in_sqrt(params.len, stream),
      out_sqrt_ref(params.len, stream),
      in_sign_flip(params.len, stream),
      out_sign_flip_ref(params.len, stream),
      in_ratio(4, stream),
      out_ratio_ref(4, stream),
      in_recip(4, stream),
      in_recip_ref(4, stream),
      out_recip(4, stream),
      in_smallzero(4, stream),
      out_smallzero(4, stream),
      out_smallzero_ref(4, stream)
  {
  }

 protected:
  void SetUp() override
  {
    random::RngState r(params.seed);
    int len         = params.len;
    T in_ratio_h[4] = {1.0, 2.0, 2.0, 3.0};
    update_device(in_ratio.data(), in_ratio_h, 4, stream);

    T out_ratio_ref_h[4] = {0.125, 0.25, 0.25, 0.375};
    update_device(out_ratio_ref.data(), out_ratio_ref_h, 4, stream);

    uniform(handle, r, in_power.data(), len, T(-1.0), T(1.0));
    uniform(handle, r, in_sqrt.data(), len, T(0.0), T(1.0));
    // uniform(r, in_ratio, len, T(0.0), T(1.0));
    uniform(handle, r, in_sign_flip.data(), len, T(-100.0), T(100.0));

    naivePower(in_power.data(), out_power_ref.data(), len, stream);
    power(in_power.data(), len, stream);

    naiveSqrt(in_sqrt.data(), out_sqrt_ref.data(), len, stream);
    seqRoot(in_sqrt.data(), len, stream);

    ratio(handle, in_ratio.data(), in_ratio.data(), 4, stream);

    naiveSignFlip(
      in_sign_flip.data(), out_sign_flip_ref.data(), params.n_row, params.n_col, stream);
    signFlip(in_sign_flip.data(), params.n_row, params.n_col, stream);

    // default threshold is 1e-15
    std::vector<T> in_recip_h     = {0.1, 0.01, -0.01, 0.1e-16};
    std::vector<T> in_recip_ref_h = {10.0, 100.0, -100.0, 0.0};
    update_device(in_recip.data(), in_recip_h.data(), 4, stream);
    update_device(in_recip_ref.data(), in_recip_ref_h.data(), 4, stream);
    T recip_scalar = T(1.0);

    // this `reciprocal()` has to go first bc next one modifies its input
    reciprocal(in_recip.data(), out_recip.data(), recip_scalar, 4, stream);

    reciprocal(in_recip.data(), recip_scalar, 4, stream, true);

    std::vector<T> in_small_val_zero_h     = {0.1, 1e-16, -1e-16, -0.1};
    std::vector<T> in_small_val_zero_ref_h = {0.1, 0.0, 0.0, -0.1};

    update_device(in_smallzero.data(), in_small_val_zero_h.data(), 4, stream);
    update_device(out_smallzero_ref.data(), in_small_val_zero_ref_h.data(), 4, stream);
    setSmallValuesZero(out_smallzero.data(), in_smallzero.data(), 4, stream);
    setSmallValuesZero(in_smallzero.data(), 4, stream);
    handle.sync_stream(stream);
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  MathInputs<T> params;
  rmm::device_uvector<T> in_power, out_power_ref, in_sqrt, out_sqrt_ref, in_ratio, out_ratio_ref,
    in_sign_flip, out_sign_flip_ref, in_recip, in_recip_ref, out_recip, in_smallzero, out_smallzero,
    out_smallzero_ref;
};

const std::vector<MathInputs<float>> inputsf = {{0.00001f, 1024, 1024, 1024 * 1024, 1234ULL}};

const std::vector<MathInputs<double>> inputsd = {{0.00001, 1024, 1024, 1024 * 1024, 1234ULL}};

typedef MathTest<float> MathPowerTestF;
TEST_P(MathPowerTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_power.data(),
                          out_power_ref.data(),
                          params.len,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathPowerTestD;
TEST_P(MathPowerTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_power.data(),
                          out_power_ref.data(),
                          params.len,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

typedef MathTest<float> MathSqrtTestF;
TEST_P(MathSqrtTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_sqrt.data(),
                          out_sqrt_ref.data(),
                          params.len,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathSqrtTestD;
TEST_P(MathSqrtTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_sqrt.data(),
                          out_sqrt_ref.data(),
                          params.len,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

typedef MathTest<float> MathRatioTestF;
TEST_P(MathRatioTestF, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_ratio.data(), out_ratio_ref.data(), 4, CompareApprox<float>(params.tolerance), stream));
}

typedef MathTest<double> MathRatioTestD;
TEST_P(MathRatioTestD, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_ratio.data(), out_ratio_ref.data(), 4, CompareApprox<double>(params.tolerance), stream));
}

typedef MathTest<float> MathSignFlipTestF;
TEST_P(MathSignFlipTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_sign_flip.data(),
                          out_sign_flip_ref.data(),
                          params.len,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathSignFlipTestD;
TEST_P(MathSignFlipTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_sign_flip.data(),
                          out_sign_flip_ref.data(),
                          params.len,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

typedef MathTest<float> MathReciprocalTestF;
TEST_P(MathReciprocalTestF, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_recip.data(), in_recip_ref.data(), 4, CompareApprox<float>(params.tolerance), stream));

  // 4-th term tests `setzero=true` functionality, not present in this version of `reciprocal`.
  ASSERT_TRUE(devArrMatch(
    out_recip.data(), in_recip_ref.data(), 3, CompareApprox<float>(params.tolerance), stream));
}

typedef MathTest<double> MathReciprocalTestD;
TEST_P(MathReciprocalTestD, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_recip.data(), in_recip_ref.data(), 4, CompareApprox<double>(params.tolerance), stream));

  // 4-th term tests `setzero=true` functionality, not present in this version of `reciprocal`.
  ASSERT_TRUE(devArrMatch(
    out_recip.data(), in_recip_ref.data(), 3, CompareApprox<double>(params.tolerance), stream));
}

typedef MathTest<float> MathSetSmallZeroTestF;
TEST_P(MathSetSmallZeroTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<float>(params.tolerance),
                          stream));

  ASSERT_TRUE(devArrMatch(out_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathSetSmallZeroTestD;
TEST_P(MathSetSmallZeroTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<double>(params.tolerance),
                          stream));

  ASSERT_TRUE(devArrMatch(out_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

INSTANTIATE_TEST_SUITE_P(MathTests, MathPowerTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathPowerTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSqrtTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSqrtTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathRatioTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathRatioTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSignFlipTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSignFlipTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathReciprocalTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathReciprocalTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSetSmallZeroTestF, ::testing::ValuesIn(inputsf));
INSTANTIATE_TEST_SUITE_P(MathTests, MathSetSmallZeroTestD, ::testing::ValuesIn(inputsd));

}  // namespace matrix
}  // namespace raft
