#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>

#include <raft/core/device_mdspan.hpp>
#include <raft/matrix/power.cuh>
#include <raft/matrix/ratio.cuh>
#include <raft/matrix/reciprocal.cuh>
#include <raft/matrix/sign_flip.cuh>
#include <raft/matrix/sqrt.cuh>
#include <raft/matrix/threshold.cuh>

#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace matrix {

template <typename Type>
RAFT_KERNEL naivePowerKernel(Type* in, Type* out, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) { out[idx] = in[idx] * in[idx]; }
}

template <typename Type>
void naivePower(Type* in, Type* out, int len, hipStream_t stream)
{
  static const int TPB = 64;
  int nblks            = raft::ceildiv(len, TPB);
  naivePowerKernel<Type><<<nblks, TPB, 0, stream>>>(in, out, len);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename Type>
RAFT_KERNEL naiveSqrtKernel(Type* in, Type* out, int len)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) { out[idx] = raft::sqrt(in[idx]); }
}

template <typename Type>
void naiveSqrt(Type* in, Type* out, int len, hipStream_t stream)
{
  static const int TPB = 64;
  int nblks            = raft::ceildiv(len, TPB);
  naiveSqrtKernel<Type><<<nblks, TPB, 0, stream>>>(in, out, len);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename Type>
RAFT_KERNEL naiveSignFlipKernel(Type* in, Type* out, int rowCount, int colCount)
{
  int d_i = blockIdx.x * rowCount;
  int end = d_i + rowCount;

  if (blockIdx.x < colCount) {
    Type max      = 0.0;
    int max_index = 0;
    for (int i = d_i; i < end; i++) {
      Type val = in[i];
      if (val < 0.0) { val = -val; }
      if (val > max) {
        max       = val;
        max_index = i;
      }
    }

    for (int i = d_i; i < end; i++) {
      if (in[max_index] < 0.0) {
        out[i] = -in[i];
      } else {
        out[i] = in[i];
      }
    }
  }

  __syncthreads();
}

template <typename Type>
void naiveSignFlip(Type* in, Type* out, int rowCount, int colCount, hipStream_t stream)
{
  naiveSignFlipKernel<Type><<<colCount, 1, 0, stream>>>(in, out, rowCount, colCount);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename T>
struct MathInputs {
  T tolerance;
  int n_row;
  int n_col;
  int len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const MathInputs<T>& dims)
{
  return os;
}

template <typename T>
class MathTest : public ::testing::TestWithParam<MathInputs<T>> {
 public:
  MathTest()
    : params(::testing::TestWithParam<MathInputs<T>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      in_power(params.len, stream),
      out_power_ref(params.len, stream),
      in_sqrt(params.len, stream),
      out_sqrt_ref(params.len, stream),
      in_sign_flip(params.len, stream),
      out_sign_flip_ref(params.len, stream),
      in_ratio(4, stream),
      out_ratio_ref(4, stream),
      in_recip(4, stream),
      in_recip_ref(4, stream),
      out_recip(4, stream),
      in_smallzero(4, stream),
      out_smallzero(4, stream),
      out_smallzero_ref(4, stream)
  {
  }

 protected:
  void SetUp() override
  {
    random::RngState r(params.seed);
    int len         = params.len;
    T in_ratio_h[4] = {1.0, 2.0, 2.0, 3.0};
    update_device(in_ratio.data(), in_ratio_h, 4, stream);

    T out_ratio_ref_h[4] = {0.125, 0.25, 0.25, 0.375};
    update_device(out_ratio_ref.data(), out_ratio_ref_h, 4, stream);

    uniform(handle, r, in_power.data(), len, T(-1.0), T(1.0));
    uniform(handle, r, in_sqrt.data(), len, T(0.0), T(1.0));
    // uniform(r, in_ratio, len, T(0.0), T(1.0));
    uniform(handle, r, in_sign_flip.data(), len, T(-100.0), T(100.0));

    naivePower(in_power.data(), out_power_ref.data(), len, stream);

    auto in_power_view = raft::make_device_matrix_view<T>(in_power.data(), len, 1);
    power<T>(handle, in_power_view);

    naiveSqrt(in_sqrt.data(), out_sqrt_ref.data(), len, stream);

    auto in_sqrt_view = raft::make_device_matrix_view(in_sqrt.data(), len, 1);
    sqrt<T>(handle, in_sqrt_view);

    auto in_ratio_view = raft::make_device_matrix_view<T>(in_ratio.data(), 4, 1);
    ratio<T>(handle, in_ratio_view);

    naiveSignFlip(
      in_sign_flip.data(), out_sign_flip_ref.data(), params.n_row, params.n_col, stream);

    auto in_sign_flip_view = raft::make_device_matrix_view<T, int, col_major>(
      in_sign_flip.data(), params.n_row, params.n_col);
    sign_flip<T>(handle, in_sign_flip_view);

    // default threshold is 1e-15
    std::vector<T> in_recip_h     = {0.1, 0.01, -0.01, 0.1e-16};
    std::vector<T> in_recip_ref_h = {10.0, 100.0, -100.0, 0.0};
    update_device(in_recip.data(), in_recip_h.data(), 4, stream);
    update_device(in_recip_ref.data(), in_recip_ref_h.data(), 4, stream);
    T recip_scalar = T(1.0);

    auto in_recip_view  = raft::make_device_matrix_view<const T>(in_recip.data(), 4, 1);
    auto out_recip_view = raft::make_device_matrix_view<T>(out_recip.data(), 4, 1);

    // this `reciprocal()` has to go first bc next one modifies its input
    reciprocal<T>(
      handle, in_recip_view, out_recip_view, raft::make_host_scalar_view(&recip_scalar));

    auto inout_recip_view = raft::make_device_matrix_view<T>(in_recip.data(), 4, 1);

    reciprocal<T>(handle, inout_recip_view, raft::make_host_scalar_view(&recip_scalar), true);

    std::vector<T> in_small_val_zero_h     = {0.1, 1e-16, -1e-16, -0.1};
    std::vector<T> in_small_val_zero_ref_h = {0.1, 0.0, 0.0, -0.1};

    auto in_smallzero_view    = raft::make_device_matrix_view<const T>(in_smallzero.data(), 4, 1);
    auto inout_smallzero_view = raft::make_device_matrix_view<T>(in_smallzero.data(), 4, 1);
    auto out_smallzero_view   = raft::make_device_matrix_view<T>(out_smallzero.data(), 4, 1);

    update_device(in_smallzero.data(), in_small_val_zero_h.data(), 4, stream);
    update_device(out_smallzero_ref.data(), in_small_val_zero_ref_h.data(), 4, stream);
    zero_small_values<T>(handle, in_smallzero_view, out_smallzero_view);
    zero_small_values<T>(handle, inout_smallzero_view);
    resource::sync_stream(handle, stream);
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  MathInputs<T> params;
  rmm::device_uvector<T> in_power, out_power_ref, in_sqrt, out_sqrt_ref, in_ratio, out_ratio_ref,
    in_sign_flip, out_sign_flip_ref, in_recip, in_recip_ref, out_recip, in_smallzero, out_smallzero,
    out_smallzero_ref;
};

const std::vector<MathInputs<float>> inputsf = {{0.00001f, 1024, 1024, 1024 * 1024, 1234ULL}};

const std::vector<MathInputs<double>> inputsd = {{0.00001, 1024, 1024, 1024 * 1024, 1234ULL}};

typedef MathTest<float> MathPowerTestF;
TEST_P(MathPowerTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_power.data(),
                          out_power_ref.data(),
                          params.len,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathPowerTestD;
TEST_P(MathPowerTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_power.data(),
                          out_power_ref.data(),
                          params.len,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

typedef MathTest<float> MathSqrtTestF;
TEST_P(MathSqrtTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_sqrt.data(),
                          out_sqrt_ref.data(),
                          params.len,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathSqrtTestD;
TEST_P(MathSqrtTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_sqrt.data(),
                          out_sqrt_ref.data(),
                          params.len,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

typedef MathTest<float> MathRatioTestF;
TEST_P(MathRatioTestF, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_ratio.data(), out_ratio_ref.data(), 4, CompareApprox<float>(params.tolerance), stream));
}

typedef MathTest<double> MathRatioTestD;
TEST_P(MathRatioTestD, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_ratio.data(), out_ratio_ref.data(), 4, CompareApprox<double>(params.tolerance), stream));
}

typedef MathTest<float> MathSignFlipTestF;
TEST_P(MathSignFlipTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_sign_flip.data(),
                          out_sign_flip_ref.data(),
                          params.len,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathSignFlipTestD;
TEST_P(MathSignFlipTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_sign_flip.data(),
                          out_sign_flip_ref.data(),
                          params.len,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

typedef MathTest<float> MathReciprocalTestF;
TEST_P(MathReciprocalTestF, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_recip.data(), in_recip_ref.data(), 4, CompareApprox<float>(params.tolerance), stream));

  // 4-th term tests `setzero=true` functionality, not present in this version of `reciprocal`.
  ASSERT_TRUE(devArrMatch(
    out_recip.data(), in_recip_ref.data(), 3, CompareApprox<float>(params.tolerance), stream));
}

typedef MathTest<double> MathReciprocalTestD;
TEST_P(MathReciprocalTestD, Result)
{
  ASSERT_TRUE(devArrMatch(
    in_recip.data(), in_recip_ref.data(), 4, CompareApprox<double>(params.tolerance), stream));

  // 4-th term tests `setzero=true` functionality, not present in this version of `reciprocal`.
  ASSERT_TRUE(devArrMatch(
    out_recip.data(), in_recip_ref.data(), 3, CompareApprox<double>(params.tolerance), stream));
}

typedef MathTest<float> MathSetSmallZeroTestF;
TEST_P(MathSetSmallZeroTestF, Result)
{
  ASSERT_TRUE(devArrMatch(in_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<float>(params.tolerance),
                          stream));

  ASSERT_TRUE(devArrMatch(out_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<float>(params.tolerance),
                          stream));
}

typedef MathTest<double> MathSetSmallZeroTestD;
TEST_P(MathSetSmallZeroTestD, Result)
{
  ASSERT_TRUE(devArrMatch(in_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<double>(params.tolerance),
                          stream));

  ASSERT_TRUE(devArrMatch(out_smallzero.data(),
                          out_smallzero_ref.data(),
                          4,
                          CompareApprox<double>(params.tolerance),
                          stream));
}

INSTANTIATE_TEST_SUITE_P(MathTests, MathPowerTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathPowerTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSqrtTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSqrtTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathRatioTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathRatioTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSignFlipTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSignFlipTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathReciprocalTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(MathTests, MathReciprocalTestD, ::testing::ValuesIn(inputsd));

INSTANTIATE_TEST_SUITE_P(MathTests, MathSetSmallZeroTestF, ::testing::ValuesIn(inputsf));
INSTANTIATE_TEST_SUITE_P(MathTests, MathSetSmallZeroTestD, ::testing::ValuesIn(inputsd));

}  // namespace matrix
}  // namespace raft
