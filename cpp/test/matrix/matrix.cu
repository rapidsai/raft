/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/matrix/matrix.cuh>
#include <raft/random/rng.cuh>
#include "../test_utils.h"

namespace raft {
namespace matrix {

template <typename T>
struct MatrixInputs {
  T tolerance;
  int n_row;
  int n_col;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const MatrixInputs<T> &dims) {
  return os;
}

template <typename T>
class MatrixTest : public ::testing::TestWithParam<MatrixInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MatrixInputs<T>>::GetParam();
    raft::random::Rng r(params.seed);
    int len = params.n_row * params.n_col;
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    raft::allocate(in1, len);
    raft::allocate(in2, len);
    raft::allocate(in1_revr, len);
    r.uniform(in1, len, T(-1.0), T(1.0), stream);

    copy(in1, in2, params.n_row, params.n_col, stream);
    // copy(in1, in1_revr, params.n_row, params.n_col);
    // colReverse(in1_revr, params.n_row, params.n_col);

    T *outTrunc;
    raft::allocate(outTrunc, 6);
    truncZeroOrigin(in1, params.n_row, outTrunc, 3, 2, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in1));
    CUDA_CHECK(hipFree(in2));
    // CUDA_CHECK(hipFree(in1_revr));
  }

 protected:
  MatrixInputs<T> params;
  T *in1, *in2, *in1_revr;
};

const std::vector<MatrixInputs<float>> inputsf2 = {{0.000001f, 4, 4, 1234ULL}};

const std::vector<MatrixInputs<double>> inputsd2 = {
  {0.00000001, 4, 4, 1234ULL}};

typedef MatrixTest<float> MatrixTestF;
TEST_P(MatrixTestF, Result) {
  ASSERT_TRUE(raft::devArrMatch(in1, in2, params.n_row * params.n_col,
                                raft::CompareApprox<float>(params.tolerance)));
}

typedef MatrixTest<double> MatrixTestD;
TEST_P(MatrixTestD, Result) {
  ASSERT_TRUE(raft::devArrMatch(in1, in2, params.n_row * params.n_col,
                                raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_SUITE_P(MatrixTests, MatrixTestF,
                         ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_SUITE_P(MatrixTests, MatrixTestD,
                         ::testing::ValuesIn(inputsd2));

template <typename T>
class MatrixCopyRowsTest : public ::testing::Test {
  using math_t = typename std::tuple_element<0, T>::type;
  using idx_t = typename std::tuple_element<1, T>::type;
  using idx_array_t = typename std::tuple_element<2, T>::type;

 protected:
  MatrixCopyRowsTest()
    : allocator(handle.get_device_allocator()),
      input(allocator, handle.get_stream(), n_cols * n_rows),
      indices(allocator, handle.get_stream(), n_selected),
      output(allocator, handle.get_stream(), n_cols * n_selected) {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);
    raft::update_device(indices.data(), indices_host, n_selected, stream);
    // Init input array
    thrust::counting_iterator<idx_t> first(0);
    thrust::device_ptr<math_t> ptr(input.data());
    thrust::copy(thrust::cuda::par.on(stream), first, first + n_cols * n_rows,
                 ptr);
  }

  void TearDown() override { CUDA_CHECK(hipStreamDestroy(stream)); }

  void testCopyRows() {
    copyRows(input.data(), n_rows, n_cols, output.data(), indices.data(),
             n_selected, stream, false);
    EXPECT_TRUE(raft::devArrMatchHost(output_exp_colmajor, output.data(),
                                      n_selected * n_cols,
                                      raft::Compare<math_t>()));
    copyRows(input.data(), n_rows, n_cols, output.data(), indices.data(),
             n_selected, stream, true);
    EXPECT_TRUE(raft::devArrMatchHost(output_exp_rowmajor, output.data(),
                                      n_selected * n_cols,
                                      raft::Compare<math_t>()));
  }

 protected:
  int n_rows = 10;
  int n_cols = 3;
  int n_selected = 5;

  idx_array_t indices_host[5] = {0, 3, 4, 7, 9};
  math_t output_exp_colmajor[15] = {0,  3,  4,  7,  9,  10, 13, 14,
                                    17, 19, 20, 23, 24, 27, 29};
  math_t output_exp_rowmajor[15] = {0,  1,  2,  9,  10, 11, 12, 13,
                                    14, 21, 22, 23, 27, 28, 29};
  raft::handle_t handle;
  hipStream_t stream;
  std::shared_ptr<raft::mr::device::allocator> allocator;
  raft::mr::device::buffer<math_t> input;
  raft::mr::device::buffer<math_t> output;
  raft::mr::device::buffer<idx_array_t> indices;
};

using TypeTuple =
  ::testing::Types<std::tuple<float, int, int>, std::tuple<float, int64_t, int>,
                   std::tuple<double, int, int>,
                   std::tuple<double, int64_t, int>>;

TYPED_TEST_CASE(MatrixCopyRowsTest, TypeTuple);
TYPED_TEST(MatrixCopyRowsTest, CopyRows) { this->testCopyRows(); }
}  // namespace matrix
}  // namespace raft
