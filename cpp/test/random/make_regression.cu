/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/core/resource/cublas_handle.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <thrust/count.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include "../test_utils.cuh"
#include <raft/core/resources.hpp>
#include <raft/linalg/detail/cublas_wrappers.hpp>
#include <raft/linalg/subtract.cuh>

#include <raft/linalg/transpose.cuh>
#include <raft/random/make_regression.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft::random {

template <typename T>
struct MakeRegressionInputs {
  T tolerance;
  int n_samples, n_features, n_informative, n_targets, effective_rank;
  T bias;
  bool shuffle;
  raft::random::GeneratorType gtype;
  uint64_t seed;
};

template <typename T>
class MakeRegressionTest : public ::testing::TestWithParam<MakeRegressionInputs<T>> {
 public:
  MakeRegressionTest()
    : params(::testing::TestWithParam<MakeRegressionInputs<T>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      values_ret(params.n_samples * params.n_targets, stream),
      values_prod(params.n_samples * params.n_targets, stream)
  {
  }

 protected:
  void SetUp() override
  {
    // Noise must be zero to compare the actual and expected values
    T noise = (T)0.0, tail_strength = (T)0.5;

    rmm::device_uvector<T> data(params.n_samples * params.n_features, stream);
    rmm::device_uvector<T> values_cm(params.n_samples * params.n_targets, stream);
    rmm::device_uvector<T> coef(params.n_features * params.n_targets, stream);

    // Create the regression problem
    make_regression(handle,
                    data.data(),
                    values_ret.data(),
                    params.n_samples,
                    params.n_features,
                    params.n_informative,
                    stream,
                    coef.data(),
                    params.n_targets,
                    params.bias,
                    params.effective_rank,
                    tail_strength,
                    noise,
                    params.shuffle,
                    params.seed,
                    params.gtype);

    // FIXME (mfh 2022/09/07) This test passes even if it doesn't call
    // make_regression.  Please see
    // https://github.com/rapidsai/raft/issues/814.

    // Calculate the values from the data and coefficients (column-major)
    T alpha = (T)1.0, beta = (T)0.0;
    RAFT_CUBLAS_TRY(raft::linalg::detail::cublasgemm(resource::get_cublas_handle(handle),
                                                     HIPBLAS_OP_T,
                                                     HIPBLAS_OP_T,
                                                     params.n_samples,
                                                     params.n_targets,
                                                     params.n_features,
                                                     &alpha,
                                                     data.data(),
                                                     params.n_features,
                                                     coef.data(),
                                                     params.n_targets,
                                                     &beta,
                                                     values_cm.data(),
                                                     params.n_samples,
                                                     stream));

    // Transpose the values to row-major
    raft::linalg::transpose(
      handle, values_cm.data(), values_prod.data(), params.n_samples, params.n_targets, stream);

    // Add the bias
    raft::linalg::addScalar(values_prod.data(),
                            values_prod.data(),
                            params.bias,
                            params.n_samples * params.n_targets,
                            stream);

    // Count the number of zeroes in the coefficients
    thrust::device_ptr<T> __coef = thrust::device_pointer_cast(coef.data());
    zero_count = thrust::count(__coef, __coef + params.n_features * params.n_targets, (T)0.0);
  }

 protected:
  raft::resources handle;
  hipStream_t stream = 0;

  MakeRegressionInputs<T> params;
  rmm::device_uvector<T> values_ret, values_prod;
  int zero_count;
};

typedef MakeRegressionTest<float> MakeRegressionTestF;
const std::vector<MakeRegressionInputs<float>> inputsf_t = {
  {0.01f, 256, 32, 16, 1, -1, 0.f, true, raft::random::GenPC, 1234ULL},
  {0.01f, 1000, 100, 47, 4, 65, 4.2f, true, raft::random::GenPC, 1234ULL},
  {0.01f, 20000, 500, 450, 13, -1, -3.f, false, raft::random::GenPC, 1234ULL},
  {0.01f, 256, 32, 16, 1, -1, 0.f, true, raft::random::GenPhilox, 1234ULL},
  {0.01f, 1000, 100, 47, 4, 65, 4.2f, true, raft::random::GenPhilox, 1234ULL},
  {0.01f, 20000, 500, 450, 13, -1, -3.f, false, raft::random::GenPhilox, 1234ULL}};

TEST_P(MakeRegressionTestF, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<float>(params.tolerance),
                          stream));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionTests, MakeRegressionTestF, ::testing::ValuesIn(inputsf_t));

typedef MakeRegressionTest<double> MakeRegressionTestD;
const std::vector<MakeRegressionInputs<double>> inputsd_t = {
  {0.01, 256, 32, 16, 1, -1, 0.0, true, raft::random::GenPC, 1234ULL},
  {0.01, 1000, 100, 47, 4, 65, 4.2, true, raft::random::GenPC, 1234ULL},
  {0.01, 20000, 500, 450, 13, -1, -3.0, false, raft::random::GenPC, 1234ULL},
  {0.01, 256, 32, 16, 1, -1, 0.0, true, raft::random::GenPhilox, 1234ULL},
  {0.01, 1000, 100, 47, 4, 65, 4.2, true, raft::random::GenPhilox, 1234ULL},
  {0.01, 20000, 500, 450, 13, -1, -3.0, false, raft::random::GenPhilox, 1234ULL}};

TEST_P(MakeRegressionTestD, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<double>(params.tolerance),
                          stream));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionTests, MakeRegressionTestD, ::testing::ValuesIn(inputsd_t));

template <typename T>
class MakeRegressionMdspanTest : public ::testing::TestWithParam<MakeRegressionInputs<T>> {
 public:
  MakeRegressionMdspanTest() = default;

 protected:
  void SetUp() override
  {
    auto stream = resource::get_cuda_stream(handle);

    // Noise must be zero to compare the actual and expected values
    T noise = (T)0.0, tail_strength = (T)0.5;

    rmm::device_uvector<T> data(params.n_samples * params.n_features, stream);
    rmm::device_uvector<T> values_cm(params.n_samples * params.n_targets, stream);
    rmm::device_uvector<T> coef(params.n_features * params.n_targets, stream);

    using index_type = typename rmm::device_uvector<T>::index_type;
    using matrix_view =
      raft::device_matrix_view<T, raft::matrix_extent<index_type>, raft::row_major>;
    matrix_view out_mat(data.data(), params.n_samples, params.n_features);
    matrix_view values_mat(values_ret.data(), params.n_samples, params.n_targets);
    matrix_view coef_mat(coef.data(), params.n_features, params.n_targets);

    // Create the regression problem
    make_regression(handle,
                    out_mat,
                    values_mat,
                    params.n_informative,
                    coef_mat,
                    params.bias,
                    params.effective_rank,
                    tail_strength,
                    noise,
                    params.shuffle,
                    params.seed,
                    params.gtype);

    // FIXME (mfh 2022/09/07) This test passes even if it doesn't call
    // make_regression.  Please see
    // https://github.com/rapidsai/raft/issues/814.

    // Calculate the values from the data and coefficients (column-major)
    T alpha{};
    T beta{};
    RAFT_CUBLAS_TRY(raft::linalg::detail::cublasgemm(resource::get_cublas_handle(handle),
                                                     HIPBLAS_OP_T,
                                                     HIPBLAS_OP_T,
                                                     params.n_samples,
                                                     params.n_targets,
                                                     params.n_features,
                                                     &alpha,
                                                     data.data(),
                                                     params.n_features,
                                                     coef.data(),
                                                     params.n_targets,
                                                     &beta,
                                                     values_cm.data(),
                                                     params.n_samples,
                                                     stream));

    // Transpose the values to row-major
    raft::linalg::transpose(
      handle, values_cm.data(), values_prod.data(), params.n_samples, params.n_targets, stream);

    // Add the bias
    raft::linalg::addScalar(values_prod.data(),
                            values_prod.data(),
                            params.bias,
                            params.n_samples * params.n_targets,
                            stream);

    // Count the number of zeroes in the coefficients
    thrust::device_ptr<T> __coef = thrust::device_pointer_cast(coef.data());
    constexpr T ZERO{};
    zero_count = thrust::count(__coef, __coef + params.n_features * params.n_targets, ZERO);
  }

 private:
  MakeRegressionInputs<T> params{::testing::TestWithParam<MakeRegressionInputs<T>>::GetParam()};
  raft::resources handle;
  rmm::device_uvector<T> values_ret{params.n_samples * params.n_targets,
                                    resource::get_cuda_stream(handle)};
  rmm::device_uvector<T> values_prod{params.n_samples * params.n_targets,
                                     resource::get_cuda_stream(handle)};
  int zero_count = -1;
};

using MakeRegressionMdspanTestF = MakeRegressionTest<float>;

TEST_P(MakeRegressionMdspanTestF, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<float>(params.tolerance),
                          resource::get_cuda_stream(handle)));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionMdspanTests,
                        MakeRegressionMdspanTestF,
                        ::testing::ValuesIn(inputsf_t));

using MakeRegressionMdspanTestD = MakeRegressionTest<double>;

TEST_P(MakeRegressionMdspanTestD, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<double>(params.tolerance),
                          resource::get_cuda_stream(handle)));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionMdspanTests,
                        MakeRegressionMdspanTestD,
                        ::testing::ValuesIn(inputsd_t));

}  // end namespace raft::random
