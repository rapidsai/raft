#include "hip/hip_runtime.h"
#include "../test_utils.cuh"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>
#include <raft/random/rng.cuh>

namespace raft {
namespace random {

// CPT - Calls Per Thread, How many calls to custom_next is made by a single thread
// IPC - Items Per Call, How many items are returned by a single call to custom_next (usually IPC = 1 or 2)
template<typename DType, typename ParamType, int CPT, int IPC>
__host__ __device__ void single_thread_fill(DType* buffer, DeviceState<PCGenerator> r, ParamType params, const size_t total_threads, const size_t len, const size_t tid) {

  PCGenerator gen(r, tid);
 
  for (size_t i = 0; i < CPT; i++) {
    DType val[IPC];
    size_t index = (tid * CPT * IPC) + i * IPC;
    custom_next(gen, val, params, index, total_threads);
    for (int j = 0; j < IPC; j++) {
      if (index + j < len) {
        buffer[index + j] = val[j];
      }
    }
  }
}

template<typename DType, typename ParamType, int CPT, int IPC>
__global__ void pcg_device_kernel(DType* buffer, DeviceState<PCGenerator> r, ParamType params, const size_t total_threads, const size_t len) {
  int tid = int(blockIdx.x) * blockDim.x + threadIdx.x;

  single_thread_fill<DType, ParamType, CPT, IPC>(buffer, r, params, total_threads, len, tid);

}

/*void trying_func(){
  constexpr int IPC = 1;
  constexpr size_t len = total_threads * CPT * IPC;
  printf("len = %lu\n", len);
  UniformDistParams<T> uniform_params = { .start = params.start, .end = params.end};

  d_buffer.resize(len, stream);
  h_buffer.resize(len);

  pcg_device_kernel<T, UniformDistParams<T>, CPT, IPC><<<n_blocks, n_threads>>>(d_buffer.data(), d_state, uniform_params, total_threads, len);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));

  for(size_t tid = 0; tid < total_threads; tid++) {
    single_thread_fill<T, UniformDistParams<T>, CPT, IPC>(h_buffer.data(), d_state, uniform_params, total_threads, len, tid);
  }
}*/

TEST(RNG, demo)
{
  hipStream_t stream;

/*
  constexpr UniformDistParams<double> uniform_params = { .start = (0.0), .end = double(1.0)};

  constexpr UniformIntDistParams<int, uint32_t> uniform_int_params {.start = int(0), .end = int(1000), .diff = uint32_t(1000) };

constexpr NormalDistParams<double> normal_params = { .mu = double(0.5), .sigma = double(0.1)};

constexpr NormalIntDistParams<int> normal_params = {.mu = int(10), .sigma = int(5)};

constexpr NormalTableDistParams<int, double> . = { 
  LenType n_rows;
  LenType n_cols;
  const OutType* mu_vec;
  OutType sigma;
  const OutType* sigma_vec;
};

constexpr BernoulliDistParams<double> bernoulli_params = { .prob = double(0.75)};

constexpr ScaledBernoulliDistParams<double> scaled_bernoulli_params = {.prob = double(0.4), .scale = double(0.75) };

constexpr GumbelDistParams<double> _params = {
  .mu = double(),
  .beta = double()
};

constexpr LogNormalDistParams <double> log_normal_params = {
  .mu = double(),
  .sigma = double()
};

constexpr LogisticDistParams<double> logistic_params = {
  .mu = double(),
  .scale = double()
};

constexpr ExponentialDistParams<double> exponential_params = {
  .lambda = double()
};

constexpr RayleighDistParams<double> rayleigh_params = {
  .sigma = double()
};

constexpr LaplaceDistParams<double> laplace_params = {
  .mu = double(),
  .scale = double()
};
*/

  raft::resources handle;
  stream = resource::get_cuda_stream(handle);

  using dtype          = double;
  constexpr size_t len = size_t(128) * 1024 * 1024;

  rmm::device_uvector<dtype> d_buffer(len, stream);
  std::vector<dtype> h_buffer(len);
  std::vector<dtype> buffer(len);

  
  RngState r(uint64_t(0x4d5cd6fc9a66e8c2), GenPC);

  int n_blocks = 128;
  int n_threads = 64;
  constexpr int TPB = 16;
  int total_threads = n_blocks * n_threads;
  constexpr InvariantDistParams<double> invariant_params = {.const_val = double(42.0)};
  //demo_rng_kernel<double, InvariantDistParams<double>, TPB><<<n_blocks, n_threads>>>(r, d_buffer.data(), len, invariant_params);

  update_host(h_buffer.data(), d_buffer.data(), len, stream);

  for(int tid = 0; tid < total_threads; tid++) {
    PCGenerator gen0(r.seed, r.base_subsequence + tid, tid);
    dtype val = 0.0; 
    for (int i = 0; i < TPB; i++) {
      custom_next(gen0, &val, invariant_params, i, 1);
      if (tid*TPB + i < int(len)) {
        buffer[tid*TPB + i] = val;
      }
    }
  }
  /*for (int i = 0; i < 2*TPB; i++) {
    printf("%f - %f\n", buffer[i], h_buffer[i]);
  }*/
  
  ASSERT_TRUE(devArrMatchHost(buffer.data(), d_buffer.data(), total_threads, raft::Compare<double>(), stream));

}

enum RandomType {
  RNG_Normal,
  RNG_LogNormal,
  RNG_Uniform,
  RNG_Gumbel,
  RNG_Logistic,
  RNG_Exp,
  RNG_Rayleigh,
  RNG_Laplace
};

template <typename ParamType>
struct RngInputs {
  // Meaning of 'start' and 'end' parameter for various distributions
  //
  //         Uniform   Normal/Log-Normal   Gumbel   Logistic   Laplace   Exponential   Rayleigh
  // start    start          mean           mean     mean       mean       lambda       sigma
  // end       end           sigma          beta     scale      scale      Unused       Unused
  ParamType distro_param;
  //T start, end;
  RandomType type;
  uint64_t seed;
};

template <typename ParamType, typename DataType, int CPT, int IPC>
class RngPcgHostTest : public ::testing::TestWithParam<RngInputs<ParamType>> {
 public:
  RngPcgHostTest()
    : params(::testing::TestWithParam<RngInputs<ParamType>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      d_buffer(0, stream)
  {
    printf("Constructor called\n");
    len = total_threads * CPT * IPC;
    d_buffer.resize(len, stream);
    h_buffer.resize(len);
  }

 protected:
  void SetUp() override
  {
    RngState r(params.seed, GenPC);
    DeviceState<PCGenerator> d_state(r);
    
    pcg_device_kernel<DataType, ParamType, CPT, IPC><<<n_blocks, n_threads>>>(d_buffer.data(), d_state, params.distro_param, total_threads, len);

    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
    for(size_t tid = 0; tid < len; tid++) {
      single_thread_fill<DataType, ParamType, CPT, IPC>(h_buffer.data(), d_state, params.distro_param, total_threads, len, tid);
    }
    ASSERT_TRUE(false);
    /*switch (params.type) {
      case RNG_Normal:
      {
        constexpr int IPC = 2;
        constexpr size_t len = total_threads * CPT * IPC;
        NormalDistParams<T> normal_params = { .mu = params.start, .sigma = params.end};

        d_buffer.resize(len, stream);
        h_buffer.resize(len);

        pcg_device_kernel<T, NormalDistParams<T>, CPT, IPC><<<n_blocks, n_threads>>>(d_buffer.data(), d_state, normal_params, total_threads, len);
        RAFT_CUDA_TRY(hipStreamSynchronize(stream));

        for(size_t tid = 0; tid < total_threads; tid++) {
          single_thread_fill<T, NormalDistParams<T>, CPT, IPC>(h_buffer.data(), d_state, normal_params, total_threads, len, tid);
        }
        break;
      }
      case RNG_LogNormal:
        printf("running for lognormal\n");
        break;
      case RNG_Uniform:
      {
        constexpr int IPC = 1;
        constexpr size_t len = total_threads * CPT * IPC;
        printf("len = %lu\n", len);
        UniformDistParams<T> uniform_params = { .start = params.start, .end = params.end};

        d_buffer.resize(len, stream);
        h_buffer.resize(len);

        pcg_device_kernel<T, UniformDistParams<T>, CPT, IPC><<<n_blocks, n_threads>>>(d_buffer.data(), d_state, uniform_params, total_threads, len);
        RAFT_CUDA_TRY(hipStreamSynchronize(stream));

        for(size_t tid = 0; tid < total_threads; tid++) {
          single_thread_fill<T, UniformDistParams<T>, CPT, IPC>(h_buffer.data(), d_state, uniform_params, total_threads, len, tid);
        }
        break;
      }
      case RNG_Gumbel: printf("Running for gumbel\n"); break;
      case RNG_Logistic:
        printf("running for logistic\n");
        break;
      case RNG_Exp: printf("running for exponential\n"); break;
      case RNG_Rayleigh: printf("running for rayleigh\n"); break;
      case RNG_Laplace:
        printf("running for laplace\n");
        break;
    };*/
    
  }


 protected:

  static const int n_blocks = 128;
  static const int n_threads = 64;
  static const size_t total_threads = size_t(n_blocks) * n_threads;
  size_t len;
  raft::resources handle;
  hipStream_t stream;

  RngInputs<ParamType> params;
  rmm::device_uvector<DataType> d_buffer;
  std::vector<DataType> h_buffer;
};

const std::vector<RngInputs<NormalDistParams<double>>> inputsf = {
  {NormalDistParams<double>({0.5, 0.5}), RNG_Laplace, 1234ULL}};

/*const std::vector<RngInputs<double>> inputsf = {
  {3.0f, 1.3f, RNG_Normal, 1234ULL},
  {1.2f, 0.1f, RNG_LogNormal, 1234ULL},
  {1.2f, 5.5f, RNG_Uniform, 1234ULL},
  {0.1f, 1.3f, RNG_Gumbel, 1234ULL},
  {1.6f, 0.0f, RNG_Exp, 1234ULL},
  {1.6f, 0.0f, RNG_Rayleigh, 1234ULL},
  {2.6f, 1.3f, RNG_Laplace, 1234ULL}};*/


/*TEST_F(RngPcgHostTest, Result) {
  ASSERT_TRUE(devArrMatchHost(h_buffer.data(), d_buffer.data(), len, raft::CompareApprox<double>(1e-5), stream));
}*/

//constexpr NormalDistParams<double> normal_params = { .mu = double(0.5), .sigma = double(0.1)};
using RngPcgHostTestD = RngPcgHostTest<NormalDistParams<double>, double, 16, 2>;
RngInputs<NormalDistParams<double>> value = {{.mu=3.0, .sigma=1.3}, RNG_Laplace, 1234ULL};
INSTANTIATE_TEST_SUITE_P(RngPcgHostTest, RngPcgHostTestD, testing::Values(value));

using RngPcgHostTestD1 = RngPcgHostTest<LogNormalDistParams<double>, double, 16, 2>;
RngInputs<LogNormalDistParams<double>> value1 = {{.mu=1.2, .sigma=0.1}, RNG_Laplace, 1234ULL};
INSTANTIATE_TEST_SUITE_P(RngPcgHostTest, RngPcgHostTestD1, testing::Values(value1));

template <typename ParamType, typename DataType, int CPT, int IPC>
class HostApiTest {
  public:
    HostApiTest()
      : stream(resource::get_cuda_stream(handle)),
      d_buffer(0, stream) {

      len = total_threads * CPT * IPC;
      d_buffer.resize(len, stream);
      h_buffer.resize(len);
      printf("### len = %lu\n", len);
    }
    void FillBuffers(uint64_t seed) {
      printf("seed = %lu\n", seed);
      RngState r(seed, GenPC);
      DeviceState<PCGenerator> d_state(r);
      
      pcg_device_kernel<DataType, ParamType, CPT, IPC><<<n_blocks, n_threads, 0, stream>>>(d_buffer.data(), d_state, dist_params, total_threads, len);

      RAFT_CUDA_TRY(hipStreamSynchronize(stream));
      for(size_t tid = 0; tid < len; tid++) {
        single_thread_fill<DataType, ParamType, CPT, IPC>(h_buffer.data(), d_state, dist_params, total_threads, len, tid);
      }
    }    
    void SetParams(ParamType _dist_params) {
      dist_params = _dist_params;
    }

    void test() {
      ASSERT_TRUE(devArrMatchHost(h_buffer.data(), d_buffer.data(), len, raft::CompareApprox<double>(1e-5), stream));
    }
    ParamType dist_params;
    raft::resources handle;
    hipStream_t stream;

    static const int n_blocks = 128;
    static const int n_threads = 64;
    static const size_t total_threads = size_t(n_blocks) * n_threads;

    size_t len;
    rmm::device_uvector<DataType> d_buffer;
    std::vector<DataType> h_buffer;
};


template <typename T>
class TestWrapper : public testing::Test {
  protected:
    void SetUp() override {
      test_obj.SetParams(p);  
      test_obj.FillBuffers(seed);
    }
  public:
    void print_foo() {
      test_obj.test();
    }
  T test_obj;
  using ParamType = decltype(T::dist_params);
  static ParamType p;
  static uint64_t seed;
};

TYPED_TEST_SUITE_P(TestWrapper);

TYPED_TEST_P(TestWrapper, print) {
  this->print_foo();
}

REGISTER_TYPED_TEST_SUITE_P(TestWrapper, print);

using InvariantDistType = HostApiTest<InvariantDistParams<int>, int, 16, 1>;
template<> InvariantDistParams<int> TestWrapper<InvariantDistType>::p = {.const_val = 431601};
template<> uint64_t TestWrapper<InvariantDistType>::seed = 23242423;

using UniformDistType = HostApiTest<UniformDistParams<double>, double, 16, 1>;
template<> UniformDistParams<double> TestWrapper<UniformDistType>::p = {.start = 0.0, .end = 1.0};
template<> uint64_t TestWrapper<UniformDistType>::seed = 23242423;

using UniformInt32DistType = HostApiTest<UniformIntDistParams<uint32_t, uint32_t>, uint32_t, 16, 1>;
template<> UniformIntDistParams<uint32_t, uint32_t> TestWrapper<UniformInt32DistType>::p = {.start = 0, .end = 100000, .diff = 100000};
template<> uint64_t TestWrapper<UniformInt32DistType>::seed = 23242423;

using UniformInt64DistType = HostApiTest<UniformIntDistParams<uint64_t, uint64_t>, uint64_t, 16, 1>;
template<> UniformIntDistParams<uint64_t, uint64_t> TestWrapper<UniformInt64DistType>::p = {.start = 0, .end = 100000, .diff = 100000};
template<> uint64_t TestWrapper<UniformInt64DistType>::seed = 23242423;

using NormalDistType = HostApiTest<NormalDistParams<double>, double, 16, 2>;
template<> NormalDistParams<double> TestWrapper<NormalDistType>::p = {.mu = 0.5, .sigma = 0.5};
template<> uint64_t TestWrapper<NormalDistType>::seed = 892357182;

using NormalIntDistType = HostApiTest<NormalIntDistParams<uint32_t>, uint32_t, 16, 2>;
template<> NormalIntDistParams<uint32_t> TestWrapper<NormalIntDistType>::p = {.mu = 1, .sigma = 1};
template<> uint64_t TestWrapper<NormalIntDistType>::seed = 892357182;

using BernoulliDistType = HostApiTest<BernoulliDistParams<double>, double, 16, 1>;
template<> BernoulliDistParams<double> TestWrapper<BernoulliDistType>::p = {.prob = 0.7};
template<> uint64_t TestWrapper<BernoulliDistType>::seed = 892357182;

using ScaledBernoulliDistType = HostApiTest<ScaledBernoulliDistParams<double>, double, 16, 1>;
template<> ScaledBernoulliDistParams<double> TestWrapper<ScaledBernoulliDistType>::p = {.prob = 0.7, .scale = 0.5};
template<> uint64_t TestWrapper<ScaledBernoulliDistType>::seed = 892357182;

using GumbelDistType = HostApiTest<GumbelDistParams<double>, double, 16, 1>;
template<> GumbelDistParams<double> TestWrapper<GumbelDistType>::p = {.mu = 0.7, .beta = 0.5};
template<> uint64_t TestWrapper<GumbelDistType>::seed = 892357182;

using LogNormalDistType = HostApiTest<LogNormalDistParams<double>, double, 16, 2>;
template<> LogNormalDistParams<double> TestWrapper<LogNormalDistType>::p = {.mu = 0.5, .sigma = 0.5};
template<> uint64_t TestWrapper<LogNormalDistType>::seed = 892357182;

using LogisticDistType = HostApiTest<LogisticDistParams<double>, double, 16, 1>;
template<> LogisticDistParams<double> TestWrapper<LogisticDistType>::p = {.mu = 0.2, .scale = 0.3};
template<> uint64_t TestWrapper<LogisticDistType>::seed = 892357182;

using ExponentialDistType = HostApiTest<ExponentialDistParams<double>, double, 16, 1>;
template<> ExponentialDistParams<double> TestWrapper<ExponentialDistType>::p = {.lambda = 1.6};
template<> uint64_t TestWrapper<ExponentialDistType>::seed = 892357182;

using RayleighDistType = HostApiTest<RayleighDistParams<double>, double, 16, 1>;
template<> RayleighDistParams<double> TestWrapper<RayleighDistType>::p = {.sigma = 1.6};
template<> uint64_t TestWrapper<RayleighDistType>::seed = 892357182;

using LaplaceDistType = HostApiTest<LaplaceDistParams<double>, double, 16, 1>;
template<> LaplaceDistParams<double> TestWrapper<LaplaceDistType>::p = {.mu = 0.2, .scale = 0.3};
template<> uint64_t TestWrapper<LaplaceDistType>::seed = 892357182;

using TestingTypes1 = testing::Types<InvariantDistType, UniformDistType, UniformInt32DistType, /*UniformInt64DistType, */NormalDistType,
/*NormalIntDistType, */BernoulliDistType, ScaledBernoulliDistType, GumbelDistType, LogisticDistType, ExponentialDistType, RayleighDistType, LaplaceDistType>;

INSTANTIATE_TYPED_TEST_SUITE_P(My1, TestWrapper, TestingTypes1);

template <typename T>
class TypedTestExample : public testing::Test {
  public:
    void calculate_size() {
      printf("calculate_size called %lu\n", sizeof(params));
    }
    using ParamType = typename T::first_type;
    using DataType = typename T::second_type;
    static ParamType params;
    static DataType d;
    static std::string distro_name;
};

//using TestTypes = testing::Types<char, int, unsigned long, std::pair<NormalDistParams<float>, float>>;
using TestTypes = testing::Types<std::pair<NormalDistParams<float>, float>, std::pair<NormalDistParams<double>, double>>;

TYPED_TEST_SUITE_P(TypedTestExample);

TYPED_TEST_P(TypedTestExample, printSize) {
  this->calculate_size(); 
  printf("Test passed for %s\n", this->distro_name.c_str());
}
REGISTER_TYPED_TEST_SUITE_P(TypedTestExample,
                            printSize);

//template<> float TypedTestExample<char>::start = 1.0;
//template<> float TypedTestExample<int>::start = 2.0;
//template<> float TypedTestExample<unsigned long>::start = 2.0;
template<> NormalDistParams<float> TypedTestExample<std::pair<NormalDistParams<float>, float>>::params = { .mu = 100, .sigma = double(0.1)};
template<> float TypedTestExample<std::pair<NormalDistParams<float>, float>>::d = float(1.2);
template<> std::string TypedTestExample<std::pair<NormalDistParams<float>, float>>::distro_name = std::string("Normal distribution"); 
template<> NormalDistParams<double> TypedTestExample<std::pair<NormalDistParams<double>, double>>::params = { .mu = double(0.5), .sigma = double(0.1)};
template<> double TypedTestExample<std::pair<NormalDistParams<double>, double>>::d = double(1.2);
template<> std::string TypedTestExample<std::pair<NormalDistParams<double>, double>>::distro_name = std::string("Normal double distribution"); 
INSTANTIATE_TYPED_TEST_SUITE_P(My, TypedTestExample, TestTypes);


} // namespace random
} // namespace raft
