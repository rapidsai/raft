/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <algorithm>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/random/permute.cuh>
#include <raft/random/rng.cuh>

#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>
#include <vector>

namespace raft {
namespace random {

template <typename T>
struct PermInputs {
  int N, D;
  bool needPerms, needShuffle, rowMajor;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const PermInputs<T>& dims)
{
  return os;
}

template <typename T>
class PermTest : public ::testing::TestWithParam<PermInputs<T>> {
 public:
  using test_data_type = T;

 protected:
  PermTest()
    : in(0, resource::get_cuda_stream(handle)),
      out(0, resource::get_cuda_stream(handle)),
      outPerms(0, resource::get_cuda_stream(handle))
  {
  }

  void SetUp() override
  {
    auto stream = resource::get_cuda_stream(handle);
    params      = ::testing::TestWithParam<PermInputs<T>>::GetParam();
    // forcefully set needPerms, since we need it for unit-testing!
    if (params.needShuffle) { params.needPerms = true; }
    raft::random::RngState r(params.seed);
    int N   = params.N;
    int D   = params.D;
    int len = N * D;
    if (params.needPerms) {
      outPerms.resize(N, stream);
      outPerms_ptr = outPerms.data();
    }
    if (params.needShuffle) {
      in.resize(len, stream);
      out.resize(len, stream);
      in_ptr  = in.data();
      out_ptr = out.data();
      uniform(handle, r, in_ptr, len, T(-1.0), T(1.0));
    }
    permute(outPerms_ptr, out_ptr, in_ptr, D, N, params.rowMajor, stream);
    resource::sync_stream(handle);
  }

 protected:
  raft::resources handle;
  PermInputs<T> params;
  rmm::device_uvector<T> in, out;
  T* in_ptr  = nullptr;
  T* out_ptr = nullptr;
  rmm::device_uvector<int> outPerms;
  int* outPerms_ptr = nullptr;
};

template <typename T>
class PermMdspanTest : public ::testing::TestWithParam<PermInputs<T>> {
 public:
  using test_data_type = T;

 protected:
  PermMdspanTest()
    : in(0, resource::get_cuda_stream(handle)),
      out(0, resource::get_cuda_stream(handle)),
      outPerms(0, resource::get_cuda_stream(handle))
  {
  }

 private:
  using index_type = int;

  template <class ElementType, class Layout>
  using matrix_view_t = raft::device_matrix_view<ElementType, index_type, Layout>;

  template <class ElementType>
  using vector_view_t = raft::device_vector_view<ElementType, index_type>;

 protected:
  void SetUp() override
  {
    auto stream = resource::get_cuda_stream(handle);
    params      = ::testing::TestWithParam<PermInputs<T>>::GetParam();
    // forcefully set needPerms, since we need it for unit-testing!
    if (params.needShuffle) { params.needPerms = true; }
    raft::random::RngState r(params.seed);
    int N   = params.N;
    int D   = params.D;
    int len = N * D;
    if (params.needPerms) {
      outPerms.resize(N, stream);
      outPerms_ptr = outPerms.data();
    }
    if (params.needShuffle) {
      in.resize(len, stream);
      out.resize(len, stream);
      in_ptr  = in.data();
      out_ptr = out.data();
      uniform(handle, r, in_ptr, len, T(-1.0), T(1.0));
    }

    auto set_up_views_and_test = [&](auto layout) {
      using layout_type = std::decay_t<decltype(layout)>;

      matrix_view_t<const T, layout_type> in_view(in_ptr, N, D);
      std::optional<matrix_view_t<T, layout_type>> out_view;
      if (out_ptr != nullptr) { out_view.emplace(out_ptr, N, D); }
      std::optional<vector_view_t<index_type>> outPerms_view;
      if (outPerms_ptr != nullptr) { outPerms_view.emplace(outPerms_ptr, N); }

      permute(handle, in_view, outPerms_view, out_view);

      // None of these three permute calls should have an effect.
      // The point is to test whether the function can deduce the
      // element type of outPerms if given nullopt.
      std::optional<matrix_view_t<T, layout_type>> out_view_empty;
      std::optional<vector_view_t<index_type>> outPerms_view_empty;
      permute(handle, in_view, std::nullopt, out_view_empty);
      permute(handle, in_view, outPerms_view_empty, std::nullopt);
      permute(handle, in_view, std::nullopt, std::nullopt);
    };

    if (params.rowMajor) {
      set_up_views_and_test(raft::row_major{});
    } else {
      set_up_views_and_test(raft::col_major{});
    }

    resource::sync_stream(handle);
  }

 protected:
  raft::resources handle;
  PermInputs<T> params;
  rmm::device_uvector<T> in, out;
  T* in_ptr  = nullptr;
  T* out_ptr = nullptr;
  rmm::device_uvector<int> outPerms;
  int* outPerms_ptr = nullptr;
};

template <typename T, typename L>
::testing::AssertionResult devArrMatchRange(
  const T* actual, size_t size, T start, L eq_compare, bool doSort = true, hipStream_t stream = 0)
{
  std::vector<T> act_h(size);
  raft::update_host<T>(&(act_h[0]), actual, size, stream);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  if (doSort) std::sort(act_h.begin(), act_h.end());
  for (size_t i(0); i < size; ++i) {
    auto act      = act_h[i];
    auto expected = start + i;
    if (!eq_compare(expected, act)) {
      return ::testing::AssertionFailure()
             << "actual=" << act << " != expected=" << expected << " @" << i;
    }
  }
  return ::testing::AssertionSuccess();
}

template <typename T, typename L>
::testing::AssertionResult devArrMatchShuffle(const int* perms,
                                              const T* out,
                                              const T* in,
                                              int D,
                                              int N,
                                              bool rowMajor,
                                              L eq_compare,
                                              hipStream_t stream = 0)
{
  std::vector<int> h_perms(N);
  raft::update_host<int>(&(h_perms[0]), perms, N, stream);
  std::vector<T> h_out(N * D), h_in(N * D);
  raft::update_host<T>(&(h_out[0]), out, N * D, stream);
  raft::update_host<T>(&(h_in[0]), in, N * D, stream);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < D; ++j) {
      int outPos    = rowMajor ? i * D + j : j * N + i;
      int inPos     = rowMajor ? h_perms[i] * D + j : j * N + h_perms[i];
      auto act      = h_out[outPos];
      auto expected = h_in[inPos];
      if (!eq_compare(expected, act)) {
        return ::testing::AssertionFailure()
               << "actual=" << act << " != expected=" << expected << " @" << i << ", " << j;
      }
    }
  }
  return ::testing::AssertionSuccess();
}

const std::vector<PermInputs<float>> inputsf = {
  // only generate permutations
  {32, 8, true, false, true, 1234ULL},
  {32, 8, true, false, true, 1234567890ULL},
  {1024, 32, true, false, true, 1234ULL},
  {1024, 32, true, false, true, 1234567890ULL},
  {2 * 1024, 32, true, false, true, 1234ULL},
  {2 * 1024, 32, true, false, true, 1234567890ULL},
  {2 * 1024 + 500, 32, true, false, true, 1234ULL},
  {2 * 1024 + 500, 32, true, false, true, 1234567890ULL},
  {100000, 32, true, false, true, 1234ULL},
  {100000, 32, true, false, true, 1234567890ULL},
  {100001, 33, true, false, true, 1234567890ULL},
  // permute and shuffle the data row major
  {32, 8, true, true, true, 1234ULL},
  {32, 8, true, true, true, 1234567890ULL},
  {1024, 32, true, true, true, 1234ULL},
  {1024, 32, true, true, true, 1234567890ULL},
  {2 * 1024, 32, true, true, true, 1234ULL},
  {2 * 1024, 32, true, true, true, 1234567890ULL},
  {2 * 1024 + 500, 32, true, true, true, 1234ULL},
  {2 * 1024 + 500, 32, true, true, true, 1234567890ULL},
  {100000, 32, true, true, true, 1234ULL},
  {100000, 32, true, true, true, 1234567890ULL},
  {100001, 31, true, true, true, 1234567890ULL},
  // permute and shuffle the data column major
  {32, 8, true, true, false, 1234ULL},
  {32, 8, true, true, false, 1234567890ULL},
  {1024, 32, true, true, false, 1234ULL},
  {1024, 32, true, true, false, 1234567890ULL},
  {2 * 1024, 32, true, true, false, 1234ULL},
  {2 * 1024, 32, true, true, false, 1234567890ULL},
  {2 * 1024 + 500, 32, true, true, false, 1234ULL},
  {2 * 1024 + 500, 32, true, true, false, 1234567890ULL},
  {100000, 32, true, true, false, 1234ULL},
  {100000, 32, true, true, false, 1234567890ULL},
  {100001, 33, true, true, false, 1234567890ULL}};

#define _PERMTEST_BODY(DATA_TYPE)                                                     \
  do {                                                                                \
    if (params.needPerms) {                                                           \
      ASSERT_TRUE(devArrMatchRange(outPerms_ptr, params.N, 0, raft::Compare<int>())); \
    }                                                                                 \
    if (params.needShuffle) {                                                         \
      ASSERT_TRUE(devArrMatchShuffle(outPerms_ptr,                                    \
                                     out_ptr,                                         \
                                     in_ptr,                                          \
                                     params.D,                                        \
                                     params.N,                                        \
                                     params.rowMajor,                                 \
                                     raft::Compare<DATA_TYPE>()));                    \
    }                                                                                 \
  } while (false)

using PermTestF = PermTest<float>;
TEST_P(PermTestF, Result)
{
  using test_data_type = PermTestF::test_data_type;
  _PERMTEST_BODY(test_data_type);
}
INSTANTIATE_TEST_CASE_P(PermTests, PermTestF, ::testing::ValuesIn(inputsf));

using PermMdspanTestF = PermMdspanTest<float>;
TEST_P(PermMdspanTestF, Result)
{
  using test_data_type = PermTestF::test_data_type;
  _PERMTEST_BODY(test_data_type);
}
INSTANTIATE_TEST_CASE_P(PermMdspanTests, PermMdspanTestF, ::testing::ValuesIn(inputsf));

const std::vector<PermInputs<double>> inputsd = {
  // only generate permutations
  {32, 8, true, false, true, 1234ULL},
  {32, 8, true, false, true, 1234567890ULL},
  {1024, 32, true, false, true, 1234ULL},
  {1024, 32, true, false, true, 1234567890ULL},
  {2 * 1024, 32, true, false, true, 1234ULL},
  {2 * 1024, 32, true, false, true, 1234567890ULL},
  {2 * 1024 + 500, 32, true, false, true, 1234ULL},
  {2 * 1024 + 500, 32, true, false, true, 1234567890ULL},
  {100000, 32, true, false, true, 1234ULL},
  {100000, 32, true, false, true, 1234567890ULL},
  {100001, 33, true, false, true, 1234567890ULL},
  // permute and shuffle the data row major
  {32, 8, true, true, true, 1234ULL},
  {32, 8, true, true, true, 1234567890ULL},
  {1024, 32, true, true, true, 1234ULL},
  {1024, 32, true, true, true, 1234567890ULL},
  {2 * 1024, 32, true, true, true, 1234ULL},
  {2 * 1024, 32, true, true, true, 1234567890ULL},
  {2 * 1024 + 500, 32, true, true, true, 1234ULL},
  {2 * 1024 + 500, 32, true, true, true, 1234567890ULL},
  {100000, 32, true, true, true, 1234ULL},
  {100000, 32, true, true, true, 1234567890ULL},
  {100001, 31, true, true, true, 1234567890ULL},
  // permute and shuffle the data column major
  {32, 8, true, true, false, 1234ULL},
  {32, 8, true, true, false, 1234567890ULL},
  {1024, 32, true, true, false, 1234ULL},
  {1024, 32, true, true, false, 1234567890ULL},
  {2 * 1024, 32, true, true, false, 1234ULL},
  {2 * 1024, 32, true, true, false, 1234567890ULL},
  {2 * 1024 + 500, 32, true, true, false, 1234ULL},
  {2 * 1024 + 500, 32, true, true, false, 1234567890ULL},
  {100000, 32, true, true, false, 1234ULL},
  {100000, 32, true, true, false, 1234567890ULL},
  {100001, 33, true, true, false, 1234567890ULL}};

using PermTestD = PermTest<double>;
TEST_P(PermTestD, Result)
{
  using test_data_type = PermTestF::test_data_type;
  _PERMTEST_BODY(test_data_type);
}
INSTANTIATE_TEST_CASE_P(PermTests, PermTestD, ::testing::ValuesIn(inputsd));

using PermMdspanTestD = PermMdspanTest<double>;
TEST_P(PermMdspanTestD, Result)
{
  using test_data_type = PermTestF::test_data_type;
  _PERMTEST_BODY(test_data_type);
}
INSTANTIATE_TEST_CASE_P(PermMdspanTests, PermMdspanTestD, ::testing::ValuesIn(inputsd));

}  // end namespace random
}  // end namespace raft
