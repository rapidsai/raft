#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include "../test_utils.h"
#include <raft/core/math.hpp>
#include <rmm/hip_stream.hpp>
#include <rmm/device_scalar.hpp>

template <typename OutT, typename OpT, typename... Args>
__global__ void math_eval_kernel(OutT* out, OpT op, Args... args)
{
  out[0] = op(std::forward<Args>(args)...);
}

template <typename OpT, typename... Args>
auto math_eval(OpT op, Args&&... args)
{
  typedef decltype(op(args...)) OutT;
  auto stream = rmm::cuda_stream_default;
  rmm::device_scalar<OutT> result(stream);
  math_eval_kernel<<<1, 1, 0, stream>>>(result.data(), op, std::forward<Args>(args)...);
  return result.value(stream);
}

struct abs_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::abs(in);
  }
};

TEST(MathDevice, Abs)
{
  // Integer abs
  ASSERT_TRUE(
    raft::match(int8_t{123}, math_eval(abs_test_op{}, int8_t{-123}), raft::Compare<int8_t>()));
  ASSERT_TRUE(raft::match(12345, math_eval(abs_test_op{}, -12345), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(12345l, math_eval(abs_test_op{}, -12345l), raft::Compare<long int>()));
  ASSERT_TRUE(raft::match(123451234512345ll,
                          math_eval(abs_test_op{}, -123451234512345ll),
                          raft::Compare<long long int>()));
  // Floating-point abs
  ASSERT_TRUE(
    raft::match(12.34f, math_eval(abs_test_op{}, -12.34f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(
    raft::match(12.34, math_eval(abs_test_op{}, -12.34), raft::CompareApprox<double>(0.000001)));
}

struct acos_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::acos(in);
  }
};

TEST(MathDevice, Acos)
{
  ASSERT_TRUE(raft::match(
    std::acos(0.123f), math_eval(acos_test_op{}, 0.123f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::acos(0.123), math_eval(acos_test_op{}, 0.123), raft::CompareApprox<double>(0.000001)));
}

struct asin_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::asin(in);
  }
};

TEST(MathDevice, Asin)
{
  ASSERT_TRUE(raft::match(
    std::asin(0.123f), math_eval(asin_test_op{}, 0.123f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::asin(0.123), math_eval(asin_test_op{}, 0.123), raft::CompareApprox<double>(0.000001)));
}

struct atanh_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::atanh(in);
  }
};

TEST(MathDevice, Atanh)
{
  ASSERT_TRUE(raft::match(
    std::atanh(0.123f), math_eval(atanh_test_op{}, 0.123f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::atanh(0.123), math_eval(atanh_test_op{}, 0.123), raft::CompareApprox<double>(0.000001)));
}

struct cos_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::cos(in);
  }
};

TEST(MathDevice, Cos)
{
  ASSERT_TRUE(raft::match(
    std::cos(12.34f), math_eval(cos_test_op{}, 12.34f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::cos(12.34), math_eval(cos_test_op{}, 12.34), raft::CompareApprox<double>(0.000001)));
}

struct exp_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::exp(in);
  }
};

TEST(MathDevice, Exp)
{
  ASSERT_TRUE(raft::match(
    std::exp(12.34f), math_eval(exp_test_op{}, 12.34f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::exp(12.34), math_eval(exp_test_op{}, 12.34), raft::CompareApprox<double>(0.000001)));
}

struct log_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::log(in);
  }
};

TEST(MathDevice, Log)
{
  ASSERT_TRUE(raft::match(
    std::log(12.34f), math_eval(log_test_op{}, 12.34f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::log(12.34), math_eval(log_test_op{}, 12.34), raft::CompareApprox<double>(0.000001)));
}

struct max_test_op {
  template <typename... Args>
  constexpr RAFT_INLINE_FUNCTION auto operator()(Args&&... args) const
  {
    return raft::max(std::forward<Args>(args)...);
  }
};

TEST(MathDevice, Max2)
{
  ASSERT_TRUE(raft::match(1234, math_eval(max_test_op{}, -1234, 1234), raft::Compare<int>()));
  ASSERT_TRUE(
    raft::match(1234u, math_eval(max_test_op{}, 1234u, 123u), raft::Compare<unsigned int>()));
  ASSERT_TRUE(
    raft::match(1234ll, math_eval(max_test_op{}, -1234ll, 1234ll), raft::Compare<long long int>()));
  ASSERT_TRUE(raft::match(
    1234ull, math_eval(max_test_op{}, 1234ull, 123ull), raft::Compare<unsigned long long int>()));

  ASSERT_TRUE(
    raft::match(12.34f, math_eval(max_test_op{}, -12.34f, 12.34f), raft::Compare<float>()));
  ASSERT_TRUE(raft::match(12.34, math_eval(max_test_op{}, -12.34, 12.34), raft::Compare<double>()));
  ASSERT_TRUE(raft::match(
    12.34, math_eval(max_test_op{}, -12.34f, 12.34), raft::CompareApprox<double>(0.000001)));
  ASSERT_TRUE(raft::match(
    12.34, math_eval(max_test_op{}, -12.34, 12.34f), raft::CompareApprox<double>(0.000001)));
}

TEST(MathDevice, Max3)
{
  ASSERT_TRUE(raft::match(1234, math_eval(max_test_op{}, 1234, 0, -1234), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(1234, math_eval(max_test_op{}, -1234, 1234, 0), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(1234, math_eval(max_test_op{}, 0, -1234, 1234), raft::Compare<int>()));

  ASSERT_TRUE(raft::match(
    12.34, math_eval(max_test_op{}, 12.34f, 0., -12.34), raft::CompareApprox<double>(0.000001)));
  ASSERT_TRUE(raft::match(
    12.34, math_eval(max_test_op{}, -12.34, 12.34f, 0.), raft::CompareApprox<double>(0.000001)));
  ASSERT_TRUE(raft::match(
    12.34, math_eval(max_test_op{}, 0., -12.34, 12.34f), raft::CompareApprox<double>(0.000001)));
}

struct min_test_op {
  template <typename... Args>
  constexpr RAFT_INLINE_FUNCTION auto operator()(Args&&... args) const
  {
    return raft::min(std::forward<Args>(args)...);
  }
};

TEST(MathDevice, Min2)
{
  ASSERT_TRUE(raft::match(-1234, math_eval(min_test_op{}, -1234, 1234), raft::Compare<int>()));
  ASSERT_TRUE(
    raft::match(123u, math_eval(min_test_op{}, 1234u, 123u), raft::Compare<unsigned int>()));
  ASSERT_TRUE(raft::match(
    -1234ll, math_eval(min_test_op{}, -1234ll, 1234ll), raft::Compare<long long int>()));
  ASSERT_TRUE(raft::match(
    123ull, math_eval(min_test_op{}, 1234ull, 123ull), raft::Compare<unsigned long long int>()));

  ASSERT_TRUE(
    raft::match(-12.34f, math_eval(min_test_op{}, -12.34f, 12.34f), raft::Compare<float>()));
  ASSERT_TRUE(
    raft::match(-12.34, math_eval(min_test_op{}, -12.34, 12.34), raft::Compare<double>()));
  ASSERT_TRUE(raft::match(
    -12.34, math_eval(min_test_op{}, -12.34f, 12.34), raft::CompareApprox<double>(0.000001)));
  ASSERT_TRUE(raft::match(
    -12.34, math_eval(min_test_op{}, -12.34, 12.34f), raft::CompareApprox<double>(0.000001)));
}

TEST(MathDevice, Min3)
{
  ASSERT_TRUE(raft::match(-1234, math_eval(min_test_op{}, 1234, 0, -1234), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(-1234, math_eval(min_test_op{}, -1234, 1234, 0), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(-1234, math_eval(min_test_op{}, 0, -1234, 1234), raft::Compare<int>()));

  ASSERT_TRUE(raft::match(
    -12.34, math_eval(min_test_op{}, 12.34f, 0., -12.34), raft::CompareApprox<double>(0.000001)));
  ASSERT_TRUE(raft::match(
    -12.34, math_eval(min_test_op{}, -12.34, 12.34f, 0.), raft::CompareApprox<double>(0.000001)));
  ASSERT_TRUE(raft::match(
    -12.34, math_eval(min_test_op{}, 0., -12.34, 12.34f), raft::CompareApprox<double>(0.000001)));
}

struct pow_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& x, const Type& y) const
  {
    return raft::pow(x, y);
  }
};

TEST(MathDevice, Pow)
{
  ASSERT_TRUE(raft::match(std::pow(12.34f, 2.f),
                          math_eval(pow_test_op{}, 12.34f, 2.f),
                          raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(std::pow(12.34, 2.),
                          math_eval(pow_test_op{}, 12.34, 2.),
                          raft::CompareApprox<double>(0.000001)));
}

struct sgn_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::sgn(in);
  }
};

TEST(MathDevice, Sgn)
{
  ASSERT_TRUE(raft::match(-1, math_eval(sgn_test_op{}, -1234), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(0, math_eval(sgn_test_op{}, 0), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(1, math_eval(sgn_test_op{}, 1234), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(-1, math_eval(sgn_test_op{}, -12.34f), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(0, math_eval(sgn_test_op{}, 0.f), raft::Compare<int>()));
  ASSERT_TRUE(raft::match(1, math_eval(sgn_test_op{}, 12.34f), raft::Compare<int>()));
}

struct sin_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::sin(in);
  }
};

TEST(MathDevice, Sin)
{
  ASSERT_TRUE(raft::match(
    std::sin(12.34f), math_eval(sin_test_op{}, 12.34f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::sin(12.34), math_eval(sin_test_op{}, 12.34), raft::CompareApprox<double>(0.000001)));
}

struct sincos_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& x, Type* s, Type* c) const
  {
    raft::sincos(x, s, c);
    return x;  // unused, just to avoid creating another helper
  }
};

TEST(MathDevice, SinCos)
{
  auto stream = rmm::cuda_stream_default;
  float xf    = 12.34f;
  rmm::device_scalar<float> sf(stream);
  rmm::device_scalar<float> cf(stream);
  math_eval(sincos_test_op{}, xf, sf.data(), cf.data());
  ASSERT_TRUE(raft::match(std::sin(12.34f), sf.value(stream), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(std::cos(12.34f), cf.value(stream), raft::CompareApprox<float>(0.0001f)));
  double xd = 12.34f;
  rmm::device_scalar<double> sd(stream);
  rmm::device_scalar<double> cd(stream);
  math_eval(sincos_test_op{}, xd, sd.data(), cd.data());
  ASSERT_TRUE(raft::match(std::sin(12.34), sd.value(stream), raft::CompareApprox<double>(0.0001f)));
  ASSERT_TRUE(raft::match(std::cos(12.34), cd.value(stream), raft::CompareApprox<double>(0.0001f)));
}

struct sqrt_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::sqrt(in);
  }
};

TEST(MathDevice, Sqrt)
{
  ASSERT_TRUE(raft::match(
    std::sqrt(12.34f), math_eval(sqrt_test_op{}, 12.34f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::sqrt(12.34), math_eval(sqrt_test_op{}, 12.34), raft::CompareApprox<double>(0.000001)));
}

struct tanh_test_op {
  template <typename Type>
  constexpr RAFT_INLINE_FUNCTION auto operator()(const Type& in) const
  {
    return raft::tanh(in);
  }
};

TEST(MathDevice, Tanh)
{
  ASSERT_TRUE(raft::match(
    std::tanh(12.34f), math_eval(tanh_test_op{}, 12.34f), raft::CompareApprox<float>(0.0001f)));
  ASSERT_TRUE(raft::match(
    std::tanh(12.34), math_eval(tanh_test_op{}, 12.34), raft::CompareApprox<double>(0.000001)));
}
