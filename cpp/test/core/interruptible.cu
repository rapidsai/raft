#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstddef>
#include <gtest/gtest.h>
#include <iostream>
#include <memory>
#include <omp.h>
#include <raft/common/nvtx.hpp>
#include <raft/core/detail/macros.hpp>
#include <raft/core/interruptible.hpp>
#include <rmm/hip_stream.hpp>
#include <thread>
#include <vector>

namespace raft {

RAFT_KERNEL gpu_wait(int millis)
{
  for (auto i = millis; i > 0; i--) {
#if __CUDA_ARCH__ >= 700
    __nanosleep(1000000);
#else
    // For older CUDA devices:
    // just do some random work that takes more or less the same time from run to run.
    volatile double x = 0;
    for (int i = 0; i < 10000; i++) {
      x = x + double(i);
      x /= 2.0;
      __syncthreads();
    }
#endif
  }
}

TEST(Raft, InterruptibleBasic)
{
  ASSERT_TRUE(interruptible::yield_no_throw());

  // Cancel using the token
  interruptible::get_token()->cancel();
  ASSERT_FALSE(interruptible::yield_no_throw());
  ASSERT_TRUE(interruptible::yield_no_throw());

  // Cancel by thread id
  interruptible::cancel(std::this_thread::get_id());
  ASSERT_FALSE(interruptible::yield_no_throw());
  ASSERT_TRUE(interruptible::yield_no_throw());
}

TEST(Raft, InterruptibleRepeatedGetToken)
{
  auto i     = std::this_thread::get_id();
  auto a1    = interruptible::get_token();
  auto count = a1.use_count();
  auto a2    = interruptible::get_token();
  ASSERT_LT(count, a1.use_count());
  count   = a1.use_count();
  auto b1 = interruptible::get_token(i);
  ASSERT_LT(count, a1.use_count());
  count   = a1.use_count();
  auto b2 = interruptible::get_token(i);
  ASSERT_LT(count, a1.use_count());

  ASSERT_EQ(a1, a2);
  ASSERT_EQ(a1, b2);
  ASSERT_EQ(b1, b2);
}

TEST(Raft, InterruptibleDelayedInit)
{
  std::thread([&]() {
    auto a = interruptible::get_token(std::this_thread::get_id());
    ASSERT_EQ(a.use_count(), 1);  // the only pointer here is [a]
    auto b = interruptible::get_token();
    ASSERT_EQ(a.use_count(), 3);  // [a, b, thread_local]
    auto c = interruptible::get_token();
    ASSERT_EQ(a.use_count(), 4);  // [a, b, c, thread_local]
    ASSERT_EQ(a.get(), b.get());
    ASSERT_EQ(a.get(), c.get());
  }).join();
}

TEST(Raft, InterruptibleOpenMP)
{
  // number of threads must be smaller than max number of resident grids for GPU
  const int n_threads = 10;
  // 1 <= n_expected_succeed <= n_threads
  const int n_expected_succeed = 5;
  // How many milliseconds passes between a thread i and i+1 finishes.
  // i.e. thread i executes (C + i*n_expected_succeed) milliseconds in total.
  const int thread_delay_millis = 20;
  common::nvtx::range fun_scope("interruptible");

  std::vector<std::shared_ptr<interruptible>> thread_tokens(n_threads);
  int n_finished  = 0;
  int n_cancelled = 0;

  omp_set_dynamic(0);
  omp_set_num_threads(n_threads);
#pragma omp parallel reduction(+ : n_finished) reduction(+ : n_cancelled) num_threads(n_threads)
  {
    auto i = omp_get_thread_num();
    common::nvtx::range omp_scope("interruptible::thread-%d", i);
    rmm::hip_stream stream;
    gpu_wait<<<1, 1, 0, stream.value()>>>(1);
    interruptible::synchronize(stream);
    thread_tokens[i] = interruptible::get_token();

#pragma omp barrier
    try {
      common::nvtx::range wait_scope("interruptible::wait-%d", i);
      gpu_wait<<<1, 1, 0, stream.value()>>>((1 + i) * thread_delay_millis);
      interruptible::synchronize(stream);
      n_finished = 1;
    } catch (interrupted_exception&) {
      n_cancelled = 1;
    }
    if (i == n_expected_succeed - 1) {
      common::nvtx::range cancel_scope("interruptible::cancel-%d", i);
      for (auto token : thread_tokens)
        token->cancel();
    }

#pragma omp barrier
    // clear the cancellation state to not disrupt other tests
    interruptible::yield_no_throw();
  }
  ASSERT_EQ(n_finished, n_expected_succeed);
  ASSERT_EQ(n_cancelled, n_threads - n_expected_succeed);
}
}  // namespace raft
