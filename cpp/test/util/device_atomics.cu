/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <array>
#include <cstddef>
#include <gtest/gtest.h>
#include <iostream>
#include <memory>
#include <numeric>
#include <raft/util/cudart_utils.hpp>
#include <raft/util/device_atomics.cuh>
#include <rmm/cuda_stream_pool.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

namespace raft {

RAFT_KERNEL test_atomic_inc_warp_kernel(int* counter, int* out_array)
{
  int global_tid                    = blockDim.x * blockIdx.x + threadIdx.x;
  out_array[atomicIncWarp(counter)] = global_tid;
}

TEST(Raft, AtomicIncWarp)
{
  const int num_blocks        = 1024;
  const int threads_per_block = 1024;
  const int num_elts          = num_blocks * threads_per_block;

  rmm::cuda_stream_pool pool{1};
  auto s = pool.get_stream();

  rmm::device_scalar<int> counter{0, s};
  rmm::device_uvector<int> out_device{num_elts, s};
  std::array<int, num_elts> out_host{0};

  // Write all 1M thread indices to a unique location in `out_device`
  test_atomic_inc_warp_kernel<<<num_blocks, threads_per_block, 0, s>>>(counter.data(),
                                                                       out_device.data());
  // Copy data to host
  RAFT_CUDA_TRY(hipMemcpyAsync(out_host.data(),
                                (const void*)out_device.data(),
                                num_elts * sizeof(int),
                                hipMemcpyDeviceToHost,
                                s));

  // Check that count is correct and that each thread index is contained in the
  // array exactly once.
  ASSERT_EQ(num_elts, counter.value(s));  // NB: accessing the counter synchronizes `s`
  std::sort(out_host.begin(), out_host.end());
  for (int i = 0; i < num_elts; ++i) {
    ASSERT_EQ(i, out_host[i]);
  }
}

}  // namespace raft
