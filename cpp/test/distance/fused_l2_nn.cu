#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <gtest/gtest.h>
#include <raft/core/kvp.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/distance/detail/fused_l2_nn.cuh>
#include <raft/distance/fused_l2_nn.cuh>
#include <raft/linalg/norm.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace distance {

template <typename LabelT, typename DataT>
struct RaftKVPMinReduce {
  typedef raft::KeyValuePair<LabelT, DataT> KVP;

  DI KVP operator()(LabelT rit, const KVP& a, const KVP& b) { return b.value < a.value ? b : a; }

  DI KVP operator()(const KVP& a, const KVP& b) { return b.value < a.value ? b : a; }

};  // KVPMinReduce

template <typename DataT, bool Sqrt, typename ReduceOpT, int NWARPS>
RAFT_KERNEL naiveKernel(raft::KeyValuePair<int, DataT>* min,
                        DataT* x,
                        DataT* y,
                        int m,
                        int n,
                        int k,
                        int* workspace,
                        DataT maxVal)
{
  int midx  = threadIdx.y + blockIdx.y * blockDim.y;
  int nidx  = threadIdx.x + blockIdx.x * blockDim.x;
  DataT acc = DataT(0);
  for (int i = 0; i < k; ++i) {
    int xidx  = i + midx * k;
    int yidx  = i + nidx * k;
    auto diff = midx >= m || nidx >= n ? DataT(0) : x[xidx] - y[yidx];
    acc += diff * diff;
  }

  if (Sqrt) { acc = raft::sqrt(acc); }
  ReduceOpT redOp;
  typedef hipcub::WarpReduce<raft::KeyValuePair<int, DataT>> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp[NWARPS];
  int warpId = threadIdx.x / raft::WarpSize;
  raft::KeyValuePair<int, DataT> tmp;
  tmp.key   = nidx;
  tmp.value = midx >= m || nidx >= n ? maxVal : acc;
  tmp       = WarpReduce(temp[warpId]).Reduce(tmp, RaftKVPMinReduce<int, DataT>());
  if (threadIdx.x % raft::WarpSize == 0 && midx < m) {
    while (atomicCAS(workspace + midx, 0, 1) == 1)
      ;
    __threadfence();
    redOp(midx, min + midx, tmp);
    __threadfence();
    atomicCAS(workspace + midx, 1, 0);
  }
}

template <typename DataT, bool Sqrt>
void naive(raft::KeyValuePair<int, DataT>* min,
           DataT* x,
           DataT* y,
           int m,
           int n,
           int k,
           int* workspace,
           hipStream_t stream)
{
  static const dim3 TPB(32, 16, 1);
  dim3 nblks(raft::ceildiv(n, (int)TPB.x), raft::ceildiv(m, (int)TPB.y), 1);
  RAFT_CUDA_TRY(hipMemsetAsync(workspace, 0, sizeof(int) * m, stream));
  auto blks = raft::ceildiv(m, 256);
  MinAndDistanceReduceOp<int, DataT> op;
  detail::initKernel<DataT, raft::KeyValuePair<int, DataT>, int>
    <<<blks, 256, 0, stream>>>(min, m, std::numeric_limits<DataT>::max(), op);
  RAFT_CUDA_TRY(hipGetLastError());
  naiveKernel<DataT, Sqrt, MinAndDistanceReduceOp<int, DataT>, 16>
    <<<nblks, TPB, 0, stream>>>(min, x, y, m, n, k, workspace, std::numeric_limits<DataT>::max());
  RAFT_CUDA_TRY(hipGetLastError());
}

template <typename DataT>
struct Inputs {
  DataT tolerance;
  int m, n, k;
  unsigned long long int seed;

  friend std::ostream& operator<<(std::ostream& os, const Inputs& p)
  {
    return os << "m: " << p.m
              << ", "
                 "n: "
              << p.n
              << ", "
                 "k: "
              << p.k
              << ", "
                 "seed: "
              << p.seed
              << ", "
                 "tol: "
              << p.tolerance;
  }
};

template <typename DataT, bool Sqrt>
class FusedL2NNTest : public ::testing::TestWithParam<Inputs<DataT>> {
 public:
  FusedL2NNTest()
    : params(::testing::TestWithParam<Inputs<DataT>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      x(params.m * params.k, stream),
      y(params.n * params.k, stream),
      xn(params.m, stream),
      yn(params.n, stream),
      min(params.m, stream),
      min_ref(params.m, stream),
      workspace(params.m * sizeof(int), stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);
    int m = params.m;
    int n = params.n;
    int k = params.k;
    uniform(handle, r, x.data(), m * k, DataT(-1.0), DataT(1.0));
    uniform(handle, r, y.data(), n * k, DataT(-1.0), DataT(1.0));
    generateGoldenResult();
    raft::linalg::rowNorm(xn.data(), x.data(), k, m, raft::linalg::L2Norm, true, stream);
    raft::linalg::rowNorm(yn.data(), y.data(), k, n, raft::linalg::L2Norm, true, stream);
    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }

 protected:
  raft::resources handle;
  hipStream_t stream;
  Inputs<DataT> params;
  rmm::device_uvector<DataT> x;
  rmm::device_uvector<DataT> y;
  rmm::device_uvector<DataT> xn;
  rmm::device_uvector<DataT> yn;
  rmm::device_uvector<raft::KeyValuePair<int, DataT>> min;
  rmm::device_uvector<raft::KeyValuePair<int, DataT>> min_ref;
  rmm::device_uvector<char> workspace;

  virtual void generateGoldenResult()
  {
    int m = params.m;
    int n = params.n;
    int k = params.k;
    naive<DataT, Sqrt>(min_ref.data(), x.data(), y.data(), m, n, k, (int*)workspace.data(), stream);
  }

  void runTest(raft::KeyValuePair<int, DataT>* out)
  {
    int m = params.m;
    int n = params.n;
    int k = params.k;

    const bool init_out_buffer = true;
    fusedL2NNMinReduce<DataT, raft::KeyValuePair<int, DataT>, int>(out,
                                                                   x.data(),
                                                                   y.data(),
                                                                   xn.data(),
                                                                   yn.data(),
                                                                   m,
                                                                   n,
                                                                   k,
                                                                   (void*)workspace.data(),
                                                                   Sqrt,
                                                                   init_out_buffer,
                                                                   stream);
    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }
};

template <typename T>
struct CompareApproxAbsKVP {
  typedef typename raft::KeyValuePair<int, T> KVP;
  CompareApproxAbsKVP(T eps_) : eps(eps_) {}
  bool operator()(const KVP& a, const KVP& b) const
  {
    T diff  = std::abs(std::abs(a.value) - std::abs(b.value));
    T m     = std::max(std::abs(a.value), std::abs(b.value));
    T ratio = m >= eps ? diff / m : diff;
    return (ratio <= eps);
  }

 private:
  T eps;
};

template <typename T>
struct CompareExactKVP {
  typedef typename raft::KeyValuePair<int, T> KVP;
  bool operator()(const KVP& a, const KVP& b) const
  {
    if (a.value != b.value) return false;
    return true;
  }
};

template <typename K, typename V, typename L>
::testing::AssertionResult devArrMatch(const raft::KeyValuePair<K, V>* expected,
                                       const raft::KeyValuePair<K, V>* actual,
                                       size_t size,
                                       L eq_compare,
                                       hipStream_t stream = 0)
{
  typedef typename raft::KeyValuePair<K, V> KVP;
  std::shared_ptr<KVP> exp_h(new KVP[size]);
  std::shared_ptr<KVP> act_h(new KVP[size]);
  raft::update_host<KVP>(exp_h.get(), expected, size, stream);
  raft::update_host<KVP>(act_h.get(), actual, size, stream);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  for (size_t i(0); i < size; ++i) {
    auto exp = exp_h.get()[i];
    auto act = act_h.get()[i];
    if (!eq_compare(exp, act)) {
      return ::testing::AssertionFailure()
             << "actual=" << act.key << "," << act.value << " != expected=" << exp.key << ","
             << exp.value << " @" << i;
    }
  }
  return ::testing::AssertionSuccess();
}

const std::vector<Inputs<float>> inputsf = {
  {0.001f, 32, 32, 32, 1234ULL},
  {0.001f, 32, 64, 32, 1234ULL},
  {0.001f, 64, 32, 32, 1234ULL},
  {0.001f, 64, 64, 32, 1234ULL},
  {0.001f, 128, 32, 32, 1234ULL},
  {0.001f, 128, 64, 32, 1234ULL},
  {0.001f, 128, 128, 64, 1234ULL},
  {0.001f, 64, 128, 128, 1234ULL},

  {0.001f, 32, 32, 34, 1234ULL},
  {0.001f, 32, 64, 34, 1234ULL},
  {0.001f, 64, 32, 34, 1234ULL},
  {0.001f, 64, 64, 34, 1234ULL},
  {0.001f, 128, 32, 34, 1234ULL},
  {0.001f, 128, 64, 34, 1234ULL},
  {0.001f, 128, 128, 66, 1234ULL},
  {0.001f, 64, 128, 130, 1234ULL},

  {0.001f, 32, 32, 33, 1234ULL},
  {0.001f, 32, 64, 33, 1234ULL},
  {0.001f, 64, 32, 33, 1234ULL},
  {0.001f, 64, 64, 33, 1234ULL},
  {0.001f, 128, 32, 33, 1234ULL},
  {0.001f, 128, 64, 33, 1234ULL},
  {0.001f, 128, 128, 65, 1234ULL},
  {0.001f, 64, 128, 129, 1234ULL},
  {0.006f, 1805, 134, 2, 1234ULL},
  {0.006f, 8192, 1024, 64, 1234ULL},
  {0.006f, 8192, 1025, 64, 1234ULL},

  // Repeat with smaller values of k
  {0.006f, 32, 32, 1, 1234ULL},
  {0.001f, 32, 64, 2, 1234ULL},
  {0.001f, 64, 32, 3, 1234ULL},
  {0.001f, 64, 64, 4, 1234ULL},
  {0.001f, 128, 32, 5, 1234ULL},
  {0.001f, 128, 64, 6, 1234ULL},
  {0.001f, 128, 128, 7, 1234ULL},
  {0.001f, 64, 128, 8, 1234ULL},

  {0.001f, 32, 32, 9, 1234ULL},
  {0.001f, 32, 64, 10, 1234ULL},
  {0.001f, 64, 32, 11, 1234ULL},
  {0.001f, 64, 64, 12, 1234ULL},
  {0.001f, 128, 32, 13, 1234ULL},
  {0.001f, 128, 64, 14, 1234ULL},
  {0.001f, 128, 128, 15, 1234ULL},
  {0.001f, 64, 128, 16, 1234ULL},

  {0.001f, 32, 32, 17, 1234ULL},
  {0.001f, 32, 64, 18, 1234ULL},
  {0.001f, 64, 32, 19, 1234ULL},
  {0.001f, 64, 64, 20, 1234ULL},
  {0.001f, 128, 32, 21, 1234ULL},
  {0.001f, 128, 64, 22, 1234ULL},
  {0.001f, 128, 128, 23, 1234ULL},
  {0.00001, 64, 128, 24, 1234ULL},
  {0.001f, 1805, 134, 25, 1234ULL},
  {0.006f, 8192, 1024, 25, 1234ULL},
  {0.006f, 8192, 1024, 66, 1234ULL},
};
typedef FusedL2NNTest<float, false> FusedL2NNTestF_Sq;
TEST_P(FusedL2NNTestF_Sq, Result)
{
  runTest(min.data());
  ASSERT_TRUE(devArrMatch(
    min_ref.data(), min.data(), params.m, CompareApproxAbsKVP<float>(params.tolerance), stream));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestF_Sq, ::testing::ValuesIn(inputsf));
typedef FusedL2NNTest<float, true> FusedL2NNTestF_Sqrt;
TEST_P(FusedL2NNTestF_Sqrt, Result)
{
  runTest(min.data());
  ASSERT_TRUE(devArrMatch(
    min_ref.data(), min.data(), params.m, CompareApproxAbsKVP<float>(params.tolerance), stream));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestF_Sqrt, ::testing::ValuesIn(inputsf));

const std::vector<Inputs<double>> inputsd = {
  {0.00001, 32, 32, 32, 1234ULL},   {0.00001, 32, 64, 32, 1234ULL},
  {0.00001, 64, 32, 32, 1234ULL},   {0.00001, 64, 64, 32, 1234ULL},
  {0.00001, 128, 32, 32, 1234ULL},  {0.00001, 128, 64, 32, 1234ULL},
  {0.00001, 128, 128, 64, 1234ULL}, {0.00001, 64, 128, 128, 1234ULL},

  {0.00001, 32, 32, 34, 1234ULL},   {0.00001, 32, 64, 34, 1234ULL},
  {0.00001, 64, 32, 34, 1234ULL},   {0.00001, 64, 64, 34, 1234ULL},
  {0.00001, 128, 32, 34, 1234ULL},  {0.00001, 128, 64, 34, 1234ULL},
  {0.00001, 128, 128, 66, 1234ULL}, {0.00001, 64, 128, 130, 1234ULL},

  {0.00001, 32, 32, 33, 1234ULL},   {0.00001, 32, 64, 33, 1234ULL},
  {0.00001, 64, 32, 33, 1234ULL},   {0.00001, 64, 64, 33, 1234ULL},
  {0.00001, 128, 32, 33, 1234ULL},  {0.00001, 128, 64, 33, 1234ULL},
  {0.00001, 128, 128, 65, 1234ULL}, {0.00001, 64, 128, 129, 1234ULL},

  {0.00001, 1805, 134, 2, 1234ULL},  //{0.00001, 8192, 1024, 25, 1234ULL},
};
typedef FusedL2NNTest<double, false> FusedL2NNTestD_Sq;
TEST_P(FusedL2NNTestD_Sq, Result)
{
  runTest(min.data());
  ASSERT_TRUE(devArrMatch(
    min_ref.data(), min.data(), params.m, CompareApproxAbsKVP<double>(params.tolerance), stream));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestD_Sq, ::testing::ValuesIn(inputsd));
typedef FusedL2NNTest<double, true> FusedL2NNTestD_Sqrt;
TEST_P(FusedL2NNTestD_Sqrt, Result)
{
  runTest(min.data());
  ASSERT_TRUE(devArrMatch(
    min_ref.data(), min.data(), params.m, CompareApproxAbsKVP<double>(params.tolerance), stream));
}
INSTANTIATE_TEST_CASE_P(FusedL2NNTests, FusedL2NNTestD_Sqrt, ::testing::ValuesIn(inputsd));

/// This is to test output determinism of the prim
template <typename DataT, bool Sqrt>
class FusedL2NNDetTest : public FusedL2NNTest<DataT, Sqrt> {
 public:
  FusedL2NNDetTest() : stream(resource::get_cuda_stream(handle)), min1(0, stream) {}

  void SetUp() override
  {
    FusedL2NNTest<DataT, Sqrt>::SetUp();
    int m = this->params.m;
    min1.resize(m, stream);
    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }

  void TearDown() override { FusedL2NNTest<DataT, Sqrt>::TearDown(); }

 protected:
  raft::resources handle;
  hipStream_t stream;

  rmm::device_uvector<raft::KeyValuePair<int, DataT>> min1;

  static const int NumRepeats = 3;

  void generateGoldenResult() override {}
};

typedef FusedL2NNDetTest<float, false> FusedL2NNDetTestF_Sq;
TEST_P(FusedL2NNDetTestF_Sq, Result)
{
  runTest(min.data());  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1.data());
    ASSERT_TRUE(devArrMatch(min.data(), min1.data(), params.m, CompareExactKVP<float>(), stream));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestF_Sq, ::testing::ValuesIn(inputsf));
typedef FusedL2NNDetTest<float, true> FusedL2NNDetTestF_Sqrt;
TEST_P(FusedL2NNDetTestF_Sqrt, Result)
{
  runTest(min.data());  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1.data());
    ASSERT_TRUE(devArrMatch(min.data(), min1.data(), params.m, CompareExactKVP<float>(), stream));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestF_Sqrt, ::testing::ValuesIn(inputsf));

typedef FusedL2NNDetTest<double, false> FusedL2NNDetTestD_Sq;
TEST_P(FusedL2NNDetTestD_Sq, Result)
{
  runTest(min.data());  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1.data());
    ASSERT_TRUE(devArrMatch(min.data(), min1.data(), params.m, CompareExactKVP<double>(), stream));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestD_Sq, ::testing::ValuesIn(inputsd));
typedef FusedL2NNDetTest<double, true> FusedL2NNDetTestD_Sqrt;
TEST_P(FusedL2NNDetTestD_Sqrt, Result)
{
  runTest(min.data());  // assumed to be golden
  for (int i = 0; i < NumRepeats; ++i) {
    runTest(min1.data());
    ASSERT_TRUE(devArrMatch(min.data(), min1.data(), params.m, CompareExactKVP<double>(), stream));
  }
}
INSTANTIATE_TEST_CASE_P(FusedL2NNDetTests, FusedL2NNDetTestD_Sqrt, ::testing::ValuesIn(inputsd));

}  // end namespace distance
}  // end namespace raft
