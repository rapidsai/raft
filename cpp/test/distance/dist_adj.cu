#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include <gtest/gtest.h>
#include <raft/distance/distance.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>
#include <rmm/device_uvector.hpp>

namespace raft {
namespace distance {

template <typename DataType>
__global__ void naiveDistanceAdjKernel(bool* dist,
                                       const DataType* x,
                                       const DataType* y,
                                       int m,
                                       int n,
                                       int k,
                                       DataType eps,
                                       bool isRowMajor)
{
  int midx = threadIdx.x + blockIdx.x * blockDim.x;
  int nidx = threadIdx.y + blockIdx.y * blockDim.y;
  if (midx >= m || nidx >= n) return;
  DataType acc = DataType(0);
  for (int i = 0; i < k; ++i) {
    int xidx  = isRowMajor ? i + midx * k : i * m + midx;
    int yidx  = isRowMajor ? i + nidx * k : i * n + nidx;
    auto diff = x[xidx] - y[yidx];
    acc += diff * diff;
  }
  int outidx   = isRowMajor ? midx * n + nidx : midx + m * nidx;
  dist[outidx] = acc <= eps;
}

template <typename DataType>
void naiveDistanceAdj(bool* dist,
                      const DataType* x,
                      const DataType* y,
                      int m,
                      int n,
                      int k,
                      DataType eps,
                      bool isRowMajor,
                      hipStream_t stream)
{
  static const dim3 TPB(16, 32, 1);
  dim3 nblks(raft::ceildiv(m, (int)TPB.x), raft::ceildiv(n, (int)TPB.y), 1);
  naiveDistanceAdjKernel<DataType><<<nblks, TPB, 0, stream>>>(dist, x, y, m, n, k, eps, isRowMajor);
  RAFT_CUDA_TRY(hipPeekAtLastError());
}

template <typename DataType>
struct DistanceAdjInputs {
  DataType eps;
  int m, n, k;
  bool isRowMajor;
  unsigned long long int seed;
};

template <typename DataType>
::std::ostream& operator<<(::std::ostream& os, const DistanceAdjInputs<DataType>& dims)
{
  return os;
}

template <typename DataType>
class DistanceAdjTest : public ::testing::TestWithParam<DistanceAdjInputs<DataType>> {
 public:
  DistanceAdjTest()
    : params(::testing::TestWithParam<DistanceAdjInputs<DataType>>::GetParam()),
      stream(handle.get_stream()),
      dist(params.m * params.n, stream),
      dist_ref(params.m * params.n, stream)
  {
  }

  void SetUp() override
  {
    raft::random::RngState r(params.seed);
    int m           = params.m;
    int n           = params.n;
    int k           = params.k;
    bool isRowMajor = params.isRowMajor;

    rmm::device_uvector<DataType> x(m * k, stream);
    rmm::device_uvector<DataType> y(n * k, stream);

    uniform(handle, r, x.data(), m * k, DataType(-1.0), DataType(1.0));
    uniform(handle, r, y.data(), n * k, DataType(-1.0), DataType(1.0));

    DataType threshold = params.eps;

    naiveDistanceAdj(dist_ref.data(), x.data(), y.data(), m, n, k, threshold, isRowMajor, stream);
    size_t worksize = raft::distance::
      getWorkspaceSize<raft::distance::DistanceType::L2Expanded, DataType, DataType, bool>(
        x.data(), y.data(), m, n, k);
    rmm::device_uvector<char> workspace(worksize, stream);

    auto fin_op = [threshold] __device__(DataType d_val, int g_d_idx) {
      return d_val <= threshold;
    };
    raft::distance::distance<raft::distance::DistanceType::L2Expanded, DataType, DataType, bool>(
      x.data(),
      y.data(),
      dist.data(),
      m,
      n,
      k,
      workspace.data(),
      workspace.size(),
      fin_op,
      stream,
      isRowMajor);
    handle.sync_stream(stream);
  }

  void TearDown() override {}

 protected:
  DistanceAdjInputs<DataType> params;
  rmm::device_uvector<bool> dist_ref;
  rmm::device_uvector<bool> dist;
  raft::handle_t handle;
  hipStream_t stream;
};

const std::vector<DistanceAdjInputs<float>> inputsf = {
  {0.01f, 1024, 1024, 32, true, 1234ULL},
  {0.1f, 1024, 1024, 32, true, 1234ULL},
  {1.0f, 1024, 1024, 32, true, 1234ULL},
  {10.0f, 1024, 1024, 32, true, 1234ULL},
  {0.01f, 1024, 1024, 32, false, 1234ULL},
  {0.1f, 1024, 1024, 32, false, 1234ULL},
  {1.0f, 1024, 1024, 32, false, 1234ULL},
  {10.0f, 1024, 1024, 32, false, 1234ULL},
};
typedef DistanceAdjTest<float> DistanceAdjTestF;
TEST_P(DistanceAdjTestF, Result)
{
  int m = params.isRowMajor ? params.m : params.n;
  int n = params.isRowMajor ? params.n : params.m;
  ASSERT_TRUE(devArrMatch(dist_ref.data(), dist.data(), m, n, raft::Compare<bool>(), stream));
}
INSTANTIATE_TEST_CASE_P(DistanceAdjTests, DistanceAdjTestF, ::testing::ValuesIn(inputsf));

const std::vector<DistanceAdjInputs<double>> inputsd = {
  {0.01, 1024, 1024, 32, true, 1234ULL},
  {0.1, 1024, 1024, 32, true, 1234ULL},
  {1.0, 1024, 1024, 32, true, 1234ULL},
  {10.0, 1024, 1024, 32, true, 1234ULL},
  {0.01, 1024, 1024, 32, false, 1234ULL},
  {0.1, 1024, 1024, 32, false, 1234ULL},
  {1.0, 1024, 1024, 32, false, 1234ULL},
  {10.0, 1024, 1024, 32, false, 1234ULL},
};
typedef DistanceAdjTest<double> DistanceAdjTestD;
TEST_P(DistanceAdjTestD, Result)
{
  int m = params.isRowMajor ? params.m : params.n;
  int n = params.isRowMajor ? params.n : params.m;
  ASSERT_TRUE(devArrMatch(dist_ref.data(), dist.data(), m, n, raft::Compare<bool>(), stream));
}
INSTANTIATE_TEST_CASE_P(DistanceAdjTests, DistanceAdjTestD, ::testing::ValuesIn(inputsd));

}  // namespace distance
}  // end namespace raft
