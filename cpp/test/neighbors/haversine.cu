/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <gtest/gtest.h>
#include <iostream>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/distance/distance_types.hpp>
#include <raft/spatial/knn/detail/haversine_distance.cuh>
#include <rmm/device_uvector.hpp>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {

template <typename value_idx, typename value_t>
class HaversineKNNTest : public ::testing::Test {
 public:
  HaversineKNNTest()
    : stream(resource::get_cuda_stream(handle)),
      d_train_inputs(0, stream),
      d_ref_I(0, stream),
      d_ref_D(0, stream),
      d_pred_I(0, stream),
      d_pred_D(0, stream)
  {
  }

 protected:
  void basicTest()
  {
    // Allocate input
    d_train_inputs.resize(n * d, stream);

    // Allocate reference arrays
    d_ref_I.resize(n * n, stream);
    d_ref_D.resize(n * n, stream);

    // Allocate predicted arrays
    d_pred_I.resize(n * n, stream);
    d_pred_D.resize(n * n, stream);

    // make testdata on host
    std::vector<value_t> h_train_inputs = {0.71113885,
                                           -1.29215058,
                                           0.59613176,
                                           -2.08048115,
                                           0.74932804,
                                           -1.33634042,
                                           0.51486728,
                                           -1.65962873,
                                           0.53154002,
                                           -1.47049808,
                                           0.72891737,
                                           -1.54095137};

    h_train_inputs.resize(d_train_inputs.size());
    raft::update_device(
      d_train_inputs.data(), h_train_inputs.data(), d_train_inputs.size(), stream);

    std::vector<value_t> h_res_D = {0., 0.05041587, 0.18767063, 0.23048252, 0.35749438, 0.62925595,
                                    0., 0.36575755, 0.44288665, 0.5170737,  0.59501296, 0.62925595,
                                    0., 0.05041587, 0.152463,   0.2426416,  0.34925285, 0.59501296,
                                    0., 0.16461092, 0.2345792,  0.34925285, 0.35749438, 0.36575755,
                                    0., 0.16461092, 0.20535265, 0.23048252, 0.2426416,  0.5170737,
                                    0., 0.152463,   0.18767063, 0.20535265, 0.2345792,  0.44288665};
    h_res_D.resize(n * n);
    raft::update_device(d_ref_D.data(), h_res_D.data(), n * n, stream);

    std::vector<value_idx> h_res_I = {0, 2, 5, 4, 3, 1, 1, 3, 5, 4, 2, 0, 2, 0, 5, 4, 3, 1,
                                      3, 4, 5, 2, 0, 1, 4, 3, 5, 0, 2, 1, 5, 2, 0, 4, 3, 1};
    h_res_I.resize(n * n);
    raft::update_device<value_idx>(d_ref_I.data(), h_res_I.data(), n * n, stream);

    raft::spatial::knn::detail::haversine_knn(d_pred_I.data(),
                                              d_pred_D.data(),
                                              d_train_inputs.data(),
                                              d_train_inputs.data(),
                                              n,
                                              n,
                                              k,
                                              stream);

    resource::sync_stream(handle, stream);
  }

  void SetUp() override { basicTest(); }

 protected:
  raft::resources handle;
  hipStream_t stream;

  rmm::device_uvector<value_t> d_train_inputs;

  int n = 6;
  int d = 2;

  int k = 6;

  rmm::device_uvector<value_idx> d_pred_I;
  rmm::device_uvector<value_t> d_pred_D;

  rmm::device_uvector<value_idx> d_ref_I;
  rmm::device_uvector<value_t> d_ref_D;
};

typedef HaversineKNNTest<int, float> HaversineKNNTestF;

TEST_F(HaversineKNNTestF, Fit)
{
  ASSERT_TRUE(raft::devArrMatch(
    d_ref_D.data(), d_pred_D.data(), n * n, raft::CompareApprox<float>(1e-3), stream));
  ASSERT_TRUE(
    raft::devArrMatch(d_ref_I.data(), d_pred_I.data(), n * n, raft::Compare<int>(), stream));
}

}  // namespace knn
}  // namespace spatial
}  // namespace raft
