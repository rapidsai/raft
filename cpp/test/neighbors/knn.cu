/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <raft/core/resource/hip_stream.hpp>

#include <raft/core/device_mdspan.hpp>
#include <raft/core/logger.hpp>
#include <raft/distance/distance_types.hpp>
#include <raft/neighbors/brute_force.cuh>

#include <rmm/device_buffer.hpp>

#include <gtest/gtest.h>

#include <cstddef>
#include <iostream>
#include <vector>

namespace raft::neighbors::brute_force {
struct KNNInputs {
  std::vector<std::vector<float>> input;
  int k;
  std::vector<int> labels;
};

template <typename IdxT>
RAFT_KERNEL build_actual_output(
  int* output, int n_rows, int k, const int* idx_labels, const IdxT* indices)
{
  int element = threadIdx.x + blockDim.x * blockIdx.x;
  if (element >= n_rows * k) return;

  output[element] = idx_labels[indices[element]];
}

RAFT_KERNEL build_expected_output(int* output, int n_rows, int k, const int* labels)
{
  int row = threadIdx.x + blockDim.x * blockIdx.x;
  if (row >= n_rows) return;

  int cur_label = labels[row];
  for (int i = 0; i < k; i++) {
    output[row * k + i] = cur_label;
  }
}

template <typename T, typename IdxT>
class KNNTest : public ::testing::TestWithParam<KNNInputs> {
 public:
  KNNTest()
    : params_(::testing::TestWithParam<KNNInputs>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      actual_labels_(0, stream),
      expected_labels_(0, stream),
      input_(0, stream),
      search_data_(0, stream),
      indices_(0, stream),
      distances_(0, stream),
      search_labels_(0, stream)
  {
  }

 protected:
  void testBruteForce()
  {
    // #if (RAFT_ACTIVE_LEVEL >= RAFT_LEVEL_DEBUG)
    raft::print_device_vector("Input array: ", input_.data(), rows_ * cols_, std::cout);
    std::cout << "K: " << k_ << std::endl;
    raft::print_device_vector("Labels array: ", search_labels_.data(), rows_, std::cout);
    // #endif

    std::vector<device_matrix_view<const T, IdxT, row_major>> index = {
      make_device_matrix_view((const T*)(input_.data()), rows_, cols_)};
    auto search = raft::make_device_matrix_view<const T, IdxT, row_major>(
      (const T*)(search_data_.data()), rows_, cols_);

    auto indices = raft::make_device_matrix_view<IdxT, IdxT, row_major>(indices_.data(), rows_, k_);
    auto distances =
      raft::make_device_matrix_view<T, IdxT, row_major>(distances_.data(), rows_, k_);

    auto metric = raft::distance::DistanceType::L2Unexpanded;
    knn(handle, index, search, indices, distances, metric, std::make_optional<IdxT>(0));

    build_actual_output<<<raft::ceildiv(rows_ * k_, 32), 32, 0, stream>>>(
      actual_labels_.data(), rows_, k_, search_labels_.data(), indices_.data());

    build_expected_output<<<raft::ceildiv(rows_ * k_, 32), 32, 0, stream>>>(
      expected_labels_.data(), rows_, k_, search_labels_.data());

    ASSERT_TRUE(devArrMatch(
      expected_labels_.data(), actual_labels_.data(), rows_ * k_, raft::Compare<int>(), stream));
  }

  void SetUp() override
  {
    rows_ = params_.input.size();
    cols_ = params_.input[0].size();
    k_    = params_.k;

    actual_labels_.resize(rows_ * k_, stream);
    expected_labels_.resize(rows_ * k_, stream);
    input_.resize(rows_ * cols_, stream);
    search_data_.resize(rows_ * cols_, stream);
    indices_.resize(rows_ * k_, stream);
    distances_.resize(rows_ * k_, stream);
    search_labels_.resize(rows_, stream);

    RAFT_CUDA_TRY(
      hipMemsetAsync(actual_labels_.data(), 0, actual_labels_.size() * sizeof(int), stream));
    RAFT_CUDA_TRY(
      hipMemsetAsync(expected_labels_.data(), 0, expected_labels_.size() * sizeof(int), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(input_.data(), 0, input_.size() * sizeof(float), stream));
    RAFT_CUDA_TRY(
      hipMemsetAsync(search_data_.data(), 0, search_data_.size() * sizeof(float), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(indices_.data(), 0, indices_.size() * sizeof(IdxT), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(distances_.data(), 0, distances_.size() * sizeof(float), stream));
    RAFT_CUDA_TRY(
      hipMemsetAsync(search_labels_.data(), 0, search_labels_.size() * sizeof(int), stream));

    std::vector<float> row_major_input;
    for (std::size_t i = 0; i < params_.input.size(); ++i) {
      for (std::size_t j = 0; j < params_.input[i].size(); ++j) {
        row_major_input.push_back(params_.input[i][j]);
      }
    }
    rmm::device_buffer input_d =
      rmm::device_buffer(row_major_input.data(), row_major_input.size() * sizeof(float), stream);
    float* input_ptr = static_cast<float*>(input_d.data());

    rmm::device_buffer labels_d =
      rmm::device_buffer(params_.labels.data(), params_.labels.size() * sizeof(int), stream);
    int* labels_ptr = static_cast<int*>(labels_d.data());

    raft::copy(input_.data(), input_ptr, rows_ * cols_, stream);
    raft::copy(search_data_.data(), input_ptr, rows_ * cols_, stream);
    raft::copy(search_labels_.data(), labels_ptr, rows_, stream);
    resource::sync_stream(handle, stream);
  }

 private:
  raft::resources handle;
  hipStream_t stream;

  KNNInputs params_;
  int rows_;
  int cols_;
  rmm::device_uvector<float> input_;
  rmm::device_uvector<float> search_data_;
  rmm::device_uvector<IdxT> indices_;
  rmm::device_uvector<float> distances_;
  int k_;

  rmm::device_uvector<int> search_labels_;
  rmm::device_uvector<int> actual_labels_;
  rmm::device_uvector<int> expected_labels_;
};

const std::vector<KNNInputs> inputs = {
  // 2D
  {{
     {2.7810836, 2.550537003},
     {1.465489372, 2.362125076},
     {3.396561688, 4.400293529},
     {1.38807019, 1.850220317},
     {3.06407232, 3.005305973},
     {7.627531214, 2.759262235},
     {5.332441248, 2.088626775},
     {6.922596716, 1.77106367},
     {8.675418651, -0.242068655},
     {7.673756466, 3.508563011},
   },
   2,
   {0, 0, 0, 0, 0, 1, 1, 1, 1, 1}}};

typedef KNNTest<float, int> KNNTestFint32_t;
TEST_P(KNNTestFint32_t, BruteForce) { this->testBruteForce(); }
typedef KNNTest<float, uint32_t> KNNTestFuint32_t;
TEST_P(KNNTestFuint32_t, BruteForce) { this->testBruteForce(); }

INSTANTIATE_TEST_CASE_P(KNNTest, KNNTestFint32_t, ::testing::ValuesIn(inputs));
INSTANTIATE_TEST_CASE_P(KNNTest, KNNTestFuint32_t, ::testing::ValuesIn(inputs));

}  // namespace raft::neighbors::brute_force
