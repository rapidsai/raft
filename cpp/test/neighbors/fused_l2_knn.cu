/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"

#include <faiss/gpu/GpuDistance.h>
#include <faiss/gpu/StandardGpuResources.h>

#include <raft/core/device_mdspan.hpp>
#include <raft/distance/distance_types.hpp>
#include <raft/neighbors/brute_force.cuh>
#include <raft/random/rng.cuh>
#include <raft/spatial/knn/detail/common_faiss.h>
#include <raft/spatial/knn/knn.cuh>

#if defined RAFT_NN_COMPILED
#include <raft/neighbors/specializations.cuh>
#endif

#include <rmm/device_buffer.hpp>

#include <gtest/gtest.h>

#include <cstddef>
#include <iostream>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {
struct FusedL2KNNInputs {
  int num_queries;
  int num_db_vecs;
  int dim;
  int k;
  raft::distance::DistanceType metric_;
};

template <typename IdxT, typename DistT, typename compareDist>
struct idx_dist_pair {
  IdxT idx;
  DistT dist;
  compareDist eq_compare;
  bool operator==(const idx_dist_pair<IdxT, DistT, compareDist>& a) const
  {
    if (idx == a.idx) return true;
    if (eq_compare(dist, a.dist)) return true;
    return false;
  }
  idx_dist_pair(IdxT x, DistT y, compareDist op) : idx(x), dist(y), eq_compare(op) {}
};

template <typename T, typename DistT>
testing::AssertionResult devArrMatchKnnPair(const T* expected_idx,
                                            const T* actual_idx,
                                            const DistT* expected_dist,
                                            const DistT* actual_dist,
                                            size_t rows,
                                            size_t cols,
                                            const DistT eps,
                                            hipStream_t stream = 0)
{
  size_t size = rows * cols;
  std::unique_ptr<T[]> exp_idx_h(new T[size]);
  std::unique_ptr<T[]> act_idx_h(new T[size]);
  std::unique_ptr<DistT[]> exp_dist_h(new DistT[size]);
  std::unique_ptr<DistT[]> act_dist_h(new DistT[size]);
  raft::update_host<T>(exp_idx_h.get(), expected_idx, size, stream);
  raft::update_host<T>(act_idx_h.get(), actual_idx, size, stream);
  raft::update_host<DistT>(exp_dist_h.get(), expected_dist, size, stream);
  raft::update_host<DistT>(act_dist_h.get(), actual_dist, size, stream);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  for (size_t i(0); i < rows; ++i) {
    for (size_t j(0); j < cols; ++j) {
      auto idx      = i * cols + j;  // row major assumption!
      auto exp_idx  = exp_idx_h.get()[idx];
      auto act_idx  = act_idx_h.get()[idx];
      auto exp_dist = exp_dist_h.get()[idx];
      auto act_dist = act_dist_h.get()[idx];
      idx_dist_pair exp_kvp(exp_idx, exp_dist, raft::CompareApprox<DistT>(eps));
      idx_dist_pair act_kvp(act_idx, act_dist, raft::CompareApprox<DistT>(eps));
      if (!(exp_kvp == act_kvp)) {
        return testing::AssertionFailure()
               << "actual=" << act_kvp.idx << "," << act_kvp.dist << "!="
               << "expected" << exp_kvp.idx << "," << exp_kvp.dist << " @" << i << "," << j;
      }
    }
  }
  return testing::AssertionSuccess();
}

template <typename T>
class FusedL2KNNTest : public ::testing::TestWithParam<FusedL2KNNInputs> {
 public:
  FusedL2KNNTest()
    : stream_(handle_.get_stream()),
      params_(::testing::TestWithParam<FusedL2KNNInputs>::GetParam()),
      database(params_.num_db_vecs * params_.dim, stream_),
      search_queries(params_.num_queries * params_.dim, stream_),
      raft_indices_(params_.num_queries * params_.k, stream_),
      raft_distances_(params_.num_queries * params_.k, stream_),
      faiss_indices_(params_.num_queries * params_.k, stream_),
      faiss_distances_(params_.num_queries * params_.k, stream_)
  {
    RAFT_CUDA_TRY(hipMemsetAsync(database.data(), 0, database.size() * sizeof(T), stream_));
    RAFT_CUDA_TRY(
      hipMemsetAsync(search_queries.data(), 0, search_queries.size() * sizeof(T), stream_));
    RAFT_CUDA_TRY(
      hipMemsetAsync(raft_indices_.data(), 0, raft_indices_.size() * sizeof(int64_t), stream_));
    RAFT_CUDA_TRY(
      hipMemsetAsync(raft_distances_.data(), 0, raft_distances_.size() * sizeof(T), stream_));
    RAFT_CUDA_TRY(
      hipMemsetAsync(faiss_indices_.data(), 0, faiss_indices_.size() * sizeof(int64_t), stream_));
    RAFT_CUDA_TRY(
      hipMemsetAsync(faiss_distances_.data(), 0, faiss_distances_.size() * sizeof(T), stream_));
  }

 protected:
  void testBruteForce()
  {
    launchFaissBfknn();

    auto index_view =
      raft::make_device_matrix_view<const T, int64_t>(database.data(), num_db_vecs, dim);
    auto query_view =
      raft::make_device_matrix_view<const T, int64_t>(search_queries.data(), num_queries, dim);
    auto out_indices_view =
      raft::make_device_matrix_view<int64_t, int64_t>(raft_indices_.data(), num_queries, k_);
    auto out_dists_view =
      raft::make_device_matrix_view<T, int64_t>(raft_distances_.data(), num_queries, k_);
    raft::neighbors::brute_force::fused_l2_knn(
      handle_, index_view, query_view, out_indices_view, out_dists_view, metric);

    // verify.
    devArrMatchKnnPair(faiss_indices_.data(),
                       raft_indices_.data(),
                       faiss_distances_.data(),
                       raft_distances_.data(),
                       num_queries,
                       k_,
                       float(0.001),
                       stream_);
  }

  void SetUp() override
  {
    num_queries = params_.num_queries;
    num_db_vecs = params_.num_db_vecs;
    dim         = params_.dim;
    k_          = params_.k;
    metric      = params_.metric_;

    unsigned long long int seed = 1234ULL;
    raft::random::RngState r(seed);
    uniform(handle_, r, database.data(), num_db_vecs * dim, T(-1.0), T(1.0));
    uniform(handle_, r, search_queries.data(), num_queries * dim, T(-1.0), T(1.0));
  }

  void launchFaissBfknn()
  {
    faiss::MetricType m = detail::build_faiss_metric(metric);

    faiss::gpu::StandardGpuResources gpu_res;

    gpu_res.noTempMemory();
    int device;
    RAFT_CUDA_TRY(hipGetDevice(&device));
    gpu_res.setDefaultStream(device, stream_);

    faiss::gpu::GpuDistanceParams args;
    args.metric          = m;
    args.metricArg       = 0;
    args.k               = k_;
    args.dims            = dim;
    args.vectors         = database.data();
    args.vectorsRowMajor = true;
    args.numVectors      = num_db_vecs;
    args.queries         = search_queries.data();
    args.queriesRowMajor = true;
    args.numQueries      = num_queries;
    args.outDistances    = faiss_distances_.data();
    args.outIndices      = faiss_indices_.data();

    bfKnn(&gpu_res, args);
  }

 private:
  raft::handle_t handle_;
  hipStream_t stream_ = 0;
  FusedL2KNNInputs params_;
  int num_queries;
  int num_db_vecs;
  int dim;
  rmm::device_uvector<T> database;
  rmm::device_uvector<T> search_queries;
  rmm::device_uvector<int64_t> raft_indices_;
  rmm::device_uvector<T> raft_distances_;
  rmm::device_uvector<int64_t> faiss_indices_;
  rmm::device_uvector<T> faiss_distances_;
  int k_;
  raft::distance::DistanceType metric;
};

const std::vector<FusedL2KNNInputs> inputs = {
  {100, 1000, 16, 10, raft::distance::DistanceType::L2Expanded},
  {256, 256, 30, 10, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, raft::distance::DistanceType::L2Expanded},
  {100, 1000, 16, 50, raft::distance::DistanceType::L2Expanded},
  {20, 10000, 16, 10, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 50, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 32, 50, raft::distance::DistanceType::L2Expanded},
  {10000, 40000, 32, 30, raft::distance::DistanceType::L2Expanded},
  {131072, 131072, 8, 60, raft::distance::DistanceType::L2Expanded},
  // L2 unexpanded
  {100, 1000, 16, 10, raft::distance::DistanceType::L2Unexpanded},
  {1000, 10000, 16, 10, raft::distance::DistanceType::L2Unexpanded},
  {100, 1000, 16, 50, raft::distance::DistanceType::L2Unexpanded},
  {20, 10000, 16, 50, raft::distance::DistanceType::L2Unexpanded},
  {1000, 10000, 16, 50, raft::distance::DistanceType::L2Unexpanded},
  {1000, 10000, 32, 50, raft::distance::DistanceType::L2Unexpanded},
  {10000, 40000, 32, 30, raft::distance::DistanceType::L2Unexpanded},
  {131072, 131072, 8, 60, raft::distance::DistanceType::L2Unexpanded}};

typedef FusedL2KNNTest<float> FusedL2KNNTestF;
TEST_P(FusedL2KNNTestF, FusedBruteForce) { this->testBruteForce(); }

INSTANTIATE_TEST_CASE_P(FusedL2KNNTest, FusedL2KNNTestF, ::testing::ValuesIn(inputs));

}  // namespace knn
}  // namespace spatial
}  // namespace raft
