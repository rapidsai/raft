/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "unary_op.cuh"
#include <gtest/gtest.h>
#include <raft/linalg/multiply.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename T>
class MultiplyTest : public ::testing::TestWithParam<UnaryOpInputs<T>> {
 public:
  MultiplyTest()
    : params(::testing::TestWithParam<UnaryOpInputs<T>>::GetParam()),
      stream(handle.get_stream()),
      in(params.len, stream),
      out_ref(params.len, stream),
      out(params.len, stream)
  {
  }

 protected:
  void SetUp() override
  {
    params = ::testing::TestWithParam<UnaryOpInputs<T>>::GetParam();
    raft::random::RngState r(params.seed);
    int len = params.len;
    uniform(handle, r, in.data(), len, T(-1.0), T(1.0));
    naiveScale(out_ref.data(), in.data(), params.scalar, len, stream);
    multiplyScalar(out.data(), in.data(), params.scalar, len, stream);
    handle.sync_stream(stream);
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  UnaryOpInputs<T> params;
  rmm::device_uvector<T> in, out_ref, out;
};

const std::vector<UnaryOpInputs<float>> inputsf = {{0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef MultiplyTest<float> MultiplyTestF;
TEST_P(MultiplyTestF, Result)
{
  ASSERT_TRUE(devArrMatch(
    out_ref.data(), out.data(), params.len, raft::CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MultiplyTests, MultiplyTestF, ::testing::ValuesIn(inputsf));

typedef MultiplyTest<double> MultiplyTestD;
const std::vector<UnaryOpInputs<double>> inputsd = {{0.000001f, 1024 * 1024, 2.f, 1234ULL}};
TEST_P(MultiplyTestD, Result)
{
  ASSERT_TRUE(devArrMatch(
    out_ref.data(), out.data(), params.len, raft::CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MultiplyTests, MultiplyTestD, ::testing::ValuesIn(inputsd));

}  // end namespace linalg
}  // end namespace raft
