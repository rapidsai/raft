/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include "unary_op.cuh"
#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/linalg/multiply.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename T>
class MultiplyTest : public ::testing::TestWithParam<UnaryOpInputs<T>> {
 public:
  MultiplyTest()
    : params(::testing::TestWithParam<UnaryOpInputs<T>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      in(params.len, stream),
      out_ref(params.len, stream),
      out(params.len, stream)
  {
  }

 protected:
  void SetUp() override
  {
    params = ::testing::TestWithParam<UnaryOpInputs<T>>::GetParam();
    raft::random::RngState r(params.seed);
    int len = params.len;
    uniform(handle, r, in.data(), len, T(-1.0), T(1.0));
    naiveScale(out_ref.data(), in.data(), params.scalar, len, stream);
    auto out_view    = raft::make_device_vector_view(out.data(), len);
    auto in_view     = raft::make_device_vector_view<const T>(in.data(), len);
    auto scalar_view = raft::make_host_scalar_view<const T>(&params.scalar);
    multiply_scalar(handle, in_view, out_view, scalar_view);
    resource::sync_stream(handle, stream);
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  UnaryOpInputs<T> params;
  rmm::device_uvector<T> in, out_ref, out;
};

const std::vector<UnaryOpInputs<float>> inputsf = {{0.000001f, 1024 * 1024, 2.f, 1234ULL}};
typedef MultiplyTest<float> MultiplyTestF;
TEST_P(MultiplyTestF, Result)
{
  ASSERT_TRUE(devArrMatch(
    out_ref.data(), out.data(), params.len, raft::CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MultiplyTests, MultiplyTestF, ::testing::ValuesIn(inputsf));

typedef MultiplyTest<double> MultiplyTestD;
const std::vector<UnaryOpInputs<double>> inputsd = {{0.000001f, 1024 * 1024, 2.f, 1234ULL}};
TEST_P(MultiplyTestD, Result)
{
  ASSERT_TRUE(devArrMatch(
    out_ref.data(), out.data(), params.len, raft::CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MultiplyTests, MultiplyTestD, ::testing::ValuesIn(inputsd));

}  // end namespace linalg
}  // end namespace raft
