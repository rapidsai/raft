/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "reduce.cuh"
#include <gtest/gtest.h>
#include <raft/linalg/reduce.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename InType, typename OutType>
struct ReduceInputs {
  OutType tolerance;
  int rows, cols;
  bool rowMajor, alongRows;
  unsigned long long int seed;
};

template <typename InType, typename OutType>
::std::ostream& operator<<(::std::ostream& os, const ReduceInputs<InType, OutType>& dims)
{
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename InType, typename OutType>
void reduceLaunch(OutType* dots,
                  const InType* data,
                  int cols,
                  int rows,
                  bool rowMajor,
                  bool alongRows,
                  bool inplace,
                  hipStream_t stream)
{
  Apply apply     = alongRows ? Apply::ALONG_ROWS : Apply::ALONG_COLUMNS;
  int output_size = alongRows ? cols : rows;

  auto output_view_row_major = raft::make_device_vector_view(dots, output_size);
  auto input_view_row_major  = raft::make_device_matrix_view(data, rows, cols);

  auto output_view_col_major = raft::make_device_vector_view<OutType, uint32_t>(dots, output_size);
  auto input_view_col_major =
    raft::make_device_matrix_view<const InType, uint32_t, raft::col_major>(data, rows, cols);

  raft::handle_t handle{stream};

  if (rowMajor) {
    reduce(handle,
           input_view_row_major,
           output_view_row_major,
           (OutType)0,

           apply,
           inplace,
           [] __device__(InType in, int i) { return static_cast<OutType>(in * in); });
  } else {
    reduce(handle,
           input_view_col_major,
           output_view_col_major,
           (OutType)0,

           apply,
           inplace,
           [] __device__(InType in, int i) { return static_cast<OutType>(in * in); });
  }
}

template <typename InType, typename OutType>
class ReduceTest : public ::testing::TestWithParam<ReduceInputs<InType, OutType>> {
 public:
  ReduceTest()
    : params(::testing::TestWithParam<ReduceInputs<InType, OutType>>::GetParam()),
      stream(handle.get_stream()),
      data(params.rows * params.cols, stream),
      dots_exp(params.alongRows ? params.rows : params.cols, stream),
      dots_act(params.alongRows ? params.rows : params.cols, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;
    outlen  = params.alongRows ? rows : cols;
    uniform(handle, r, data.data(), len, InType(-1.0), InType(1.0));
    naiveReduction(
      dots_exp.data(), data.data(), cols, rows, params.rowMajor, params.alongRows, stream);

    // Perform reduction with default inplace = false first
    reduceLaunch(
      dots_act.data(), data.data(), cols, rows, params.rowMajor, params.alongRows, false, stream);
    // Add to result with inplace = true next, which shouldn't affect
    // in the case of coalescedReduction!
    if (!(params.rowMajor ^ params.alongRows)) {
      reduceLaunch(
        dots_act.data(), data.data(), cols, rows, params.rowMajor, params.alongRows, true, stream);
    }
    handle.sync_stream(stream);
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  ReduceInputs<InType, OutType> params;
  rmm::device_uvector<InType> data;
  rmm::device_uvector<OutType> dots_exp, dots_act;
  int outlen;
};

const std::vector<ReduceInputs<float, float>> inputsff = {
  {0.000002f, 1024, 32, true, true, 1234ULL},
  {0.000002f, 1024, 64, true, true, 1234ULL},
  {0.000002f, 1024, 128, true, true, 1234ULL},
  {0.000002f, 1024, 256, true, true, 1234ULL},
  {0.000002f, 1024, 32, true, false, 1234ULL},
  {0.000002f, 1024, 64, true, false, 1234ULL},
  {0.000002f, 1024, 128, true, false, 1234ULL},
  {0.000002f, 1024, 256, true, false, 1234ULL},
  {0.000002f, 1024, 32, false, true, 1234ULL},
  {0.000002f, 1024, 64, false, true, 1234ULL},
  {0.000002f, 1024, 128, false, true, 1234ULL},
  {0.000002f, 1024, 256, false, true, 1234ULL},
  {0.000002f, 1024, 32, false, false, 1234ULL},
  {0.000002f, 1024, 64, false, false, 1234ULL},
  {0.000002f, 1024, 128, false, false, 1234ULL},
  {0.000002f, 1024, 256, false, false, 1234ULL}};

const std::vector<ReduceInputs<double, double>> inputsdd = {
  {0.000000001, 1024, 32, true, true, 1234ULL},
  {0.000000001, 1024, 64, true, true, 1234ULL},
  {0.000000001, 1024, 128, true, true, 1234ULL},
  {0.000000001, 1024, 256, true, true, 1234ULL},
  {0.000000001, 1024, 32, true, false, 1234ULL},
  {0.000000001, 1024, 64, true, false, 1234ULL},
  {0.000000001, 1024, 128, true, false, 1234ULL},
  {0.000000001, 1024, 256, true, false, 1234ULL},
  {0.000000001, 1024, 32, false, true, 1234ULL},
  {0.000000001, 1024, 64, false, true, 1234ULL},
  {0.000000001, 1024, 128, false, true, 1234ULL},
  {0.000000001, 1024, 256, false, true, 1234ULL},
  {0.000000001, 1024, 32, false, false, 1234ULL},
  {0.000000001, 1024, 64, false, false, 1234ULL},
  {0.000000001, 1024, 128, false, false, 1234ULL},
  {0.000000001, 1024, 256, false, false, 1234ULL}};

const std::vector<ReduceInputs<float, double>> inputsfd = {
  {0.000002f, 1024, 32, true, true, 1234ULL},
  {0.000002f, 1024, 64, true, true, 1234ULL},
  {0.000002f, 1024, 128, true, true, 1234ULL},
  {0.000002f, 1024, 256, true, true, 1234ULL},
  {0.000002f, 1024, 32, true, false, 1234ULL},
  {0.000002f, 1024, 64, true, false, 1234ULL},
  {0.000002f, 1024, 128, true, false, 1234ULL},
  {0.000002f, 1024, 256, true, false, 1234ULL},
  {0.000002f, 1024, 32, false, true, 1234ULL},
  {0.000002f, 1024, 64, false, true, 1234ULL},
  {0.000002f, 1024, 128, false, true, 1234ULL},
  {0.000002f, 1024, 256, false, true, 1234ULL},
  {0.000002f, 1024, 32, false, false, 1234ULL},
  {0.000002f, 1024, 64, false, false, 1234ULL},
  {0.000002f, 1024, 128, false, false, 1234ULL},
  {0.000002f, 1024, 256, false, false, 1234ULL}};

typedef ReduceTest<float, float> ReduceTestFF;
TEST_P(ReduceTestFF, Result)
{
  ASSERT_TRUE(devArrMatch(
    dots_exp.data(), dots_act.data(), outlen, raft::CompareApprox<float>(params.tolerance)));
}

typedef ReduceTest<double, double> ReduceTestDD;
TEST_P(ReduceTestDD, Result)
{
  ASSERT_TRUE(devArrMatch(
    dots_exp.data(), dots_act.data(), outlen, raft::CompareApprox<double>(params.tolerance)));
}

typedef ReduceTest<float, double> ReduceTestFD;
TEST_P(ReduceTestFD, Result)
{
  ASSERT_TRUE(devArrMatch(
    dots_exp.data(), dots_act.data(), outlen, raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_CASE_P(ReduceTests, ReduceTestFF, ::testing::ValuesIn(inputsff));

INSTANTIATE_TEST_CASE_P(ReduceTests, ReduceTestDD, ::testing::ValuesIn(inputsdd));

INSTANTIATE_TEST_CASE_P(ReduceTests, ReduceTestFD, ::testing::ValuesIn(inputsfd));

}  // end namespace linalg
}  // end namespace raft
