/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include <gtest/gtest.h>
#include <raft/linalg/eltwise.cuh>
#include <raft/linalg/map.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename InType, typename IdxType, typename OutType>
void mapLaunch(OutType* out,
               const InType* in1,
               const InType* in2,
               const InType* in3,
               InType scalar,
               IdxType len,
               hipStream_t stream)
{
  map(
    out,
    len,
    [=] __device__(InType a, InType b, InType c) { return a + b + c + scalar; },
    stream,
    in1,
    in2,
    in3);
}

template <typename InType, typename IdxType = int, typename OutType = InType>
struct MapInputs {
  InType tolerance;
  IdxType len;
  unsigned long long int seed;
  InType scalar;
};

template <typename InType, typename IdxType, typename OutType = InType>
void create_ref(OutType* out_ref,
                const InType* in1,
                const InType* in2,
                const InType* in3,
                InType scalar,
                IdxType len,
                hipStream_t stream)
{
  rmm::device_uvector<InType> tmp(len, stream);
  eltwiseAdd(tmp.data(), in1, in2, len, stream);
  eltwiseAdd(out_ref, tmp.data(), in3, len, stream);
  scalarAdd(out_ref, out_ref, (OutType)scalar, len, stream);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));
}

template <typename InType, typename IdxType, typename OutType = InType>
class MapTest : public ::testing::TestWithParam<MapInputs<InType, IdxType, OutType>> {
 public:
  MapTest()
    : params(::testing::TestWithParam<MapInputs<InType, IdxType, OutType>>::GetParam()),
      stream(handle.get_stream()),
      in1(params.len, stream),
      in2(params.len, stream),
      in3(params.len, stream),
      out_ref(params.len, stream),
      out(params.len, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);

    IdxType len = params.len;
    uniform(handle, r, in1.data(), len, InType(-1.0), InType(1.0));
    uniform(handle, r, in2.data(), len, InType(-1.0), InType(1.0));
    uniform(handle, r, in3.data(), len, InType(-1.0), InType(1.0));

    create_ref(out_ref.data(), in1.data(), in2.data(), in3.data(), params.scalar, len, stream);
    mapLaunch(out.data(), in1.data(), in2.data(), in3.data(), params.scalar, len, stream);
    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  MapInputs<InType, IdxType, OutType> params;
  rmm::device_uvector<InType> in1, in2, in3;
  rmm::device_uvector<OutType> out_ref, out;
};

const std::vector<MapInputs<float, int>> inputsf_i32 = {{0.000001f, 1024 * 1024, 1234ULL, 3.2}};
typedef MapTest<float, int> MapTestF_i32;
TEST_P(MapTestF_i32, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapTests, MapTestF_i32, ::testing::ValuesIn(inputsf_i32));

const std::vector<MapInputs<float, size_t>> inputsf_i64 = {{0.000001f, 1024 * 1024, 1234ULL, 9.4}};
typedef MapTest<float, size_t> MapTestF_i64;
TEST_P(MapTestF_i64, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapTests, MapTestF_i64, ::testing::ValuesIn(inputsf_i64));

const std::vector<MapInputs<float, int, double>> inputsf_i32_d = {
  {0.000001f, 1024 * 1024, 1234ULL, 5.9}};
typedef MapTest<float, int, double> MapTestF_i32_D;
TEST_P(MapTestF_i32_D, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapTests, MapTestF_i32_D, ::testing::ValuesIn(inputsf_i32_d));

const std::vector<MapInputs<double, int>> inputsd_i32 = {{0.00000001, 1024 * 1024, 1234ULL, 7.5}};
typedef MapTest<double, int> MapTestD_i32;
TEST_P(MapTestD_i32, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapTests, MapTestD_i32, ::testing::ValuesIn(inputsd_i32));

const std::vector<MapInputs<double, size_t>> inputsd_i64 = {
  {0.00000001, 1024 * 1024, 1234ULL, 5.2}};
typedef MapTest<double, size_t> MapTestD_i64;
TEST_P(MapTestD_i64, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapTests, MapTestD_i64, ::testing::ValuesIn(inputsd_i64));

}  // namespace linalg
}  // namespace raft
