/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include "unary_op.cuh"
#include <gtest/gtest.h>
#include <raft/linalg/eltwise.cuh>
#include <raft/linalg/map.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename InType, typename IdxType, typename OutType>
void mapLaunch(OutType* out,
               const InType* in1,
               const InType* in2,
               const InType* in3,
               InType scalar,
               IdxType len,
               hipStream_t stream)
{
  raft::device_resources handle{stream};
  auto out_view = raft::make_device_vector_view(out, len);
  auto in1_view = raft::make_device_vector_view(in1, len);
  map(
    handle,
    in1_view,
    out_view,
    [=] __device__(InType a, InType b, InType c) { return a + b + c + scalar; },
    in2,
    in3);
}

template <typename InType, typename IdxType = int, typename OutType = InType>
struct MapInputs {
  InType tolerance;
  IdxType len;
  unsigned long long int seed;
  InType scalar;
};

template <typename InType, typename IdxType, typename OutType = InType>
void create_ref(OutType* out_ref,
                const InType* in1,
                const InType* in2,
                const InType* in3,
                InType scalar,
                IdxType len,
                hipStream_t stream)
{
  rmm::device_uvector<InType> tmp(len, stream);
  eltwiseAdd(tmp.data(), in1, in2, len, stream);
  eltwiseAdd(out_ref, tmp.data(), in3, len, stream);
  scalarAdd(out_ref, out_ref, (OutType)scalar, len, stream);
  RAFT_CUDA_TRY(hipStreamSynchronize(stream));
}

template <typename InType, typename IdxType, typename OutType = InType>
class MapTest : public ::testing::TestWithParam<MapInputs<InType, IdxType, OutType>> {
 public:
  MapTest()
    : params(::testing::TestWithParam<MapInputs<InType, IdxType, OutType>>::GetParam()),
      stream(handle.get_stream()),
      in1(params.len, stream),
      in2(params.len, stream),
      in3(params.len, stream),
      out_ref(params.len, stream),
      out(params.len, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);

    IdxType len = params.len;
    uniform(handle, r, in1.data(), len, InType(-1.0), InType(1.0));
    uniform(handle, r, in2.data(), len, InType(-1.0), InType(1.0));
    uniform(handle, r, in3.data(), len, InType(-1.0), InType(1.0));

    create_ref(out_ref.data(), in1.data(), in2.data(), in3.data(), params.scalar, len, stream);
    mapLaunch(out.data(), in1.data(), in2.data(), in3.data(), params.scalar, len, stream);
    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }

 protected:
  raft::device_resources handle;
  hipStream_t stream;

  MapInputs<InType, IdxType, OutType> params;
  rmm::device_uvector<InType> in1, in2, in3;
  rmm::device_uvector<OutType> out_ref, out;
};

template <typename OutType, typename IdxType>
class MapOffsetTest : public ::testing::TestWithParam<MapInputs<OutType, IdxType, OutType>> {
 public:
  MapOffsetTest()
    : params(::testing::TestWithParam<MapInputs<OutType, IdxType, OutType>>::GetParam()),
      stream(handle.get_stream()),
      out_ref(params.len, stream),
      out(params.len, stream)
  {
  }

 protected:
  void SetUp() override
  {
    IdxType len    = params.len;
    OutType scalar = params.scalar;
    naiveScale(out_ref.data(), (OutType*)nullptr, scalar, len, stream);

    auto out_view = raft::make_device_vector_view(out.data(), len);
    map_offset(handle,
               out_view,
               raft::compose_op(raft::cast_op<OutType>(), raft::mul_const_op<OutType>(scalar)));
    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }

 protected:
  raft::device_resources handle;
  hipStream_t stream;

  MapInputs<OutType, IdxType, OutType> params;
  rmm::device_uvector<OutType> out_ref, out;
};

#define MAP_TEST(test_type, test_name, inputs)                       \
  typedef RAFT_DEPAREN(test_type) test_name;                         \
  TEST_P(test_name, Result)                                          \
  {                                                                  \
    ASSERT_TRUE(devArrMatch(this->out_ref.data(),                    \
                            this->out.data(),                        \
                            this->params.len,                        \
                            CompareApprox(this->params.tolerance))); \
  }                                                                  \
  INSTANTIATE_TEST_SUITE_P(MapTests, test_name, ::testing::ValuesIn(inputs))

const std::vector<MapInputs<float, int>> inputsf_i32 = {{0.000001f, 1024 * 1024, 1234ULL, 3.2}};
MAP_TEST((MapTest<float, int>), MapTestF_i32, inputsf_i32);
MAP_TEST((MapOffsetTest<float, int>), MapOffsetTestF_i32, inputsf_i32);

const std::vector<MapInputs<float, size_t>> inputsf_i64 = {{0.000001f, 1024 * 1024, 1234ULL, 9.4}};
MAP_TEST((MapTest<float, size_t>), MapTestF_i64, inputsf_i64);
MAP_TEST((MapOffsetTest<float, size_t>), MapOffsetTestF_i64, inputsf_i64);

const std::vector<MapInputs<float, int, double>> inputsf_i32_d = {
  {0.000001f, 1024 * 1024, 1234ULL, 5.9}};
MAP_TEST((MapTest<float, int, double>), MapTestF_i32_D, inputsf_i32_d);

const std::vector<MapInputs<double, int>> inputsd_i32 = {{0.00000001, 1024 * 1024, 1234ULL, 7.5}};
MAP_TEST((MapTest<double, int>), MapTestD_i32, inputsd_i32);
MAP_TEST((MapOffsetTest<double, int>), MapOffsetTestD_i32, inputsd_i32);

const std::vector<MapInputs<double, size_t>> inputsd_i64 = {
  {0.00000001, 1024 * 1024, 1234ULL, 5.2}};
MAP_TEST((MapTest<double, size_t>), MapTestD_i64, inputsd_i64);
MAP_TEST((MapOffsetTest<double, size_t>), MapOffsetTestD_i64, inputsd_i64);

}  // namespace linalg
}  // namespace raft
