#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <limits>
#include <raft/linalg/map_then_reduce.cuh>
#include <raft/random/rng.cuh>
#include "../test_utils.h"

namespace raft {
namespace linalg {

template <typename InType, typename OutType, typename MapOp>
__global__ void naiveMapReduceKernel(OutType *out, const InType *in, size_t len,
                                     MapOp map) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    raft::myAtomicAdd(out, (OutType)map(in[idx]));
  }
}

template <typename InType, typename OutType, typename MapOp>
void naiveMapReduce(OutType *out, const InType *in, size_t len, MapOp map,
                    hipStream_t stream) {
  static const int TPB = 64;
  int nblks = raft::ceildiv(len, (size_t)TPB);
  naiveMapReduceKernel<InType, OutType, MapOp>
    <<<nblks, TPB, 0, stream>>>(out, in, len, map);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct MapReduceInputs {
  T tolerance;
  size_t len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const MapReduceInputs<T> &dims) {
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename InType, typename OutType>
void mapReduceLaunch(OutType *out_ref, OutType *out, const InType *in,
                     size_t len, hipStream_t stream) {
  auto op = [] __device__(InType in) { return in; };
  naiveMapReduce(out_ref, in, len, op, stream);
  mapThenSumReduce(out, len, op, 0, in);
}

template <typename InType, typename OutType>
class MapReduceTest : public ::testing::TestWithParam<MapReduceInputs<InType>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MapReduceInputs<InType>>::GetParam();
    raft::random::Rng r(params.seed);
    auto len = params.len;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in, len, InType(-1.0), InType(1.0), stream);
    mapReduceLaunch(out_ref, out, in, len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

 protected:
  MapReduceInputs<InType> params;
  InType *in;
  OutType *out_ref, *out;
};

const std::vector<MapReduceInputs<float>> inputsf = {
  {0.001f, 1024 * 1024, 1234ULL}};
typedef MapReduceTest<float, float> MapReduceTestFF;
TEST_P(MapReduceTestFF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapReduceTests, MapReduceTestFF,
                         ::testing::ValuesIn(inputsf));

typedef MapReduceTest<float, double> MapReduceTestFD;
TEST_P(MapReduceTestFD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapReduceTests, MapReduceTestFD,
                         ::testing::ValuesIn(inputsf));

const std::vector<MapReduceInputs<double>> inputsd = {
  {0.000001, 1024 * 1024, 1234ULL}};
typedef MapReduceTest<double, double> MapReduceTestDD;
TEST_P(MapReduceTestDD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapReduceTests, MapReduceTestDD,
                         ::testing::ValuesIn(inputsd));

template <typename T>
class MapGenericReduceTest : public ::testing::Test {
  using InType = typename T::first_type;
  using OutType = typename T::second_type;

 protected:
  MapGenericReduceTest()
    : allocator(handle.get_device_allocator()),
      input(allocator, handle.get_stream(), n),
      output(allocator, handle.get_stream(), 1) {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);
    initInput(input.data(), input.size(), stream);
  }

  void TearDown() override { CUDA_CHECK(hipStreamDestroy(stream)); }

 public:
  void initInput(InType *input, int n, hipStream_t stream) {
    raft::random::Rng r(137);
    r.uniform(input, n, InType(2), InType(3), stream);
    InType val = 1;
    raft::update_device(input + 42, &val, 1, stream);
    val = 5;
    raft::update_device(input + 337, &val, 1, stream);
  }

  void testMin() {
    auto op = [] __device__(InType in) { return in; };
    const OutType neutral = std::numeric_limits<InType>::max();
    mapThenReduce(output.data(), input.size(), neutral, op, hipcub::Min(), stream,
                  input.data());
    EXPECT_TRUE(raft::devArrMatch(OutType(1), output.data(), 1,
                                  raft::Compare<OutType>()));
  }
  void testMax() {
    auto op = [] __device__(InType in) { return in; };
    const OutType neutral = std::numeric_limits<InType>::min();
    mapThenReduce(output.data(), input.size(), neutral, op, hipcub::Max(), stream,
                  input.data());
    EXPECT_TRUE(raft::devArrMatch(OutType(5), output.data(), 1,
                                  raft::Compare<OutType>()));
  }

 protected:
  int n = 1237;
  raft::handle_t handle;
  hipStream_t stream;
  std::shared_ptr<raft::mr::device::allocator> allocator;
  raft::mr::device::buffer<InType> input;
  raft::mr::device::buffer<OutType> output;
};

using IoTypePair =
  ::testing::Types<std::pair<float, float>, std::pair<float, double>,
                   std::pair<double, double>>;

TYPED_TEST_CASE(MapGenericReduceTest, IoTypePair);
TYPED_TEST(MapGenericReduceTest, min) { this->testMin(); }
TYPED_TEST(MapGenericReduceTest, max) { this->testMax(); }
}  // end namespace linalg
}  // end namespace raft
