#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <limits>
#include <raft/linalg/map_then_reduce.cuh>
#include <raft/random/rng.cuh>
#include "../test_utils.h"

namespace raft {
namespace linalg {

template <typename Type, typename MapOp>
__global__ void naiveMapReduceKernel(Type *out, const Type *in, size_t len,
                                     MapOp map) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < len) {
    raft::myAtomicAdd(out, map(in[idx]));
  }
}

template <typename Type, typename MapOp>
void naiveMapReduce(Type *out, const Type *in, size_t len, MapOp map,
                    hipStream_t stream) {
  static const int TPB = 64;
  int nblks = raft::ceildiv(len, (size_t)TPB);
  naiveMapReduceKernel<Type, MapOp>
    <<<nblks, TPB, 0, stream>>>(out, in, len, map);
  CUDA_CHECK(hipPeekAtLastError());
}

template <typename T>
struct MapReduceInputs {
  T tolerance;
  size_t len;
  unsigned long long int seed;
};

template <typename T>
::std::ostream &operator<<(::std::ostream &os, const MapReduceInputs<T> &dims) {
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T>
void mapReduceLaunch(T *out_ref, T *out, const T *in, size_t len,
                     hipStream_t stream) {
  auto op = [] __device__(T in) { return in; };
  naiveMapReduce(out_ref, in, len, op, stream);
  mapThenSumReduce(out, len, op, 0, in);
}

template <typename T>
class MapReduceTest : public ::testing::TestWithParam<MapReduceInputs<T>> {
 protected:
  void SetUp() override {
    params = ::testing::TestWithParam<MapReduceInputs<T>>::GetParam();
    raft::random::Rng r(params.seed);
    auto len = params.len;

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));
    allocate(in, len);
    allocate(out_ref, len);
    allocate(out, len);
    r.uniform(in, len, T(-1.0), T(1.0), stream);
    mapReduceLaunch(out_ref, out, in, len, stream);
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  void TearDown() override {
    CUDA_CHECK(hipFree(in));
    CUDA_CHECK(hipFree(out_ref));
    CUDA_CHECK(hipFree(out));
  }

 protected:
  MapReduceInputs<T> params;
  T *in, *out_ref, *out;
};

const std::vector<MapReduceInputs<float>> inputsf = {
  {0.001f, 1024 * 1024, 1234ULL}};
typedef MapReduceTest<float> MapReduceTestF;
TEST_P(MapReduceTestF, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapReduceTests, MapReduceTestF,
                         ::testing::ValuesIn(inputsf));

const std::vector<MapReduceInputs<double>> inputsd = {
  {0.000001, 1024 * 1024, 1234ULL}};
typedef MapReduceTest<double> MapReduceTestD;
TEST_P(MapReduceTestD, Result) {
  ASSERT_TRUE(devArrMatch(out_ref, out, params.len,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MapReduceTests, MapReduceTestD,
                         ::testing::ValuesIn(inputsd));

template <typename math_t>
class MapGenericReduceTest : public ::testing::Test {
 protected:
  MapGenericReduceTest()
    : allocator(handle.get_device_allocator()),
      input(allocator, handle.get_stream(), n),
      output(allocator, handle.get_stream(), 1) {
    CUDA_CHECK(hipStreamCreate(&stream));
    handle.set_stream(stream);
    initInput(input.data(), input.size(), stream);
  }

  void TearDown() override { CUDA_CHECK(hipStreamDestroy(stream)); }

 public:
  void initInput(math_t *input, int n, hipStream_t stream) {
    raft::random::Rng r(137);
    r.uniform(input, n, math_t(2), math_t(3), stream);
    math_t val = 1;
    raft::update_device(input + 42, &val, 1, stream);
    val = 5;
    raft::update_device(input + 337, &val, 1, stream);
  }

  void testMin() {
    auto op = [] __device__(math_t in) { return in; };
    const math_t neutral = std::numeric_limits<math_t>::max();
    mapThenReduce(output.data(), input.size(), neutral, op, hipcub::Min(), stream,
                  input.data());
    EXPECT_TRUE(
      raft::devArrMatch(math_t(1), output.data(), 1, raft::Compare<math_t>()));
  }
  void testMax() {
    auto op = [] __device__(math_t in) { return in; };
    const math_t neutral = std::numeric_limits<math_t>::min();
    mapThenReduce(output.data(), input.size(), neutral, op, hipcub::Max(), stream,
                  input.data());
    EXPECT_TRUE(
      raft::devArrMatch(math_t(5), output.data(), 1, raft::Compare<math_t>()));
  }

 protected:
  int n = 1237;
  raft::handle_t handle;
  hipStream_t stream;
  std::shared_ptr<raft::mr::device::allocator> allocator;
  raft::mr::device::buffer<math_t> input;
  raft::mr::device::buffer<math_t> output;
};

typedef ::testing::Types<float, double> FloatTypes;

TYPED_TEST_CASE(MapGenericReduceTest, FloatTypes);
TYPED_TEST(MapGenericReduceTest, min) { this->testMin(); }
TYPED_TEST(MapGenericReduceTest, max) { this->testMax(); }
}  // end namespace linalg
}  // end namespace raft
