/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "matrix_vector_op.cuh"
#include <gtest/gtest.h>
#include <raft/linalg/matrix_vector_op.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename T, typename IdxType = int>
struct MatVecOpInputs {
  T tolerance;
  IdxType rows, cols;
  bool rowMajor, bcastAlongRows, useTwoVectors;
  unsigned long long int seed;
};

template <typename T, typename IdxType>
::std::ostream& operator<<(::std::ostream& os, const MatVecOpInputs<T, IdxType>& dims)
{
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T, typename IdxType>
void matrixVectorOpLaunch(const raft::handle_t& handle,
                          T* out,
                          const T* in,
                          const T* vec1,
                          const T* vec2,
                          IdxType D,
                          IdxType N,
                          bool rowMajor,
                          bool bcastAlongRows,
                          bool useTwoVectors)
{
  auto out_row_major = raft::make_device_matrix_view<T, IdxType, raft::row_major>(out, N, D);
  auto in_row_major  = raft::make_device_matrix_view<const T, IdxType, raft::row_major>(in, N, D);

  auto out_col_major = raft::make_device_matrix_view<T, IdxType, raft::col_major>(out, N, D);
  auto in_col_major  = raft::make_device_matrix_view<const T, IdxType, raft::col_major>(in, N, D);

  auto apply     = bcastAlongRows ? Apply::ALONG_ROWS : Apply::ALONG_COLUMNS;
  auto len       = bcastAlongRows ? D : N;
  auto vec1_view = raft::make_device_vector_view<const T, IdxType>(vec1, len);
  auto vec2_view = raft::make_device_vector_view<const T, IdxType>(vec2, len);

  if (useTwoVectors) {
    if (rowMajor) {
      matrix_vector_op(handle,
                       in_row_major,
                       vec1_view,
                       vec2_view,
                       out_row_major,
                       apply,
                       [] __device__(T a, T b, T c) { return a + b + c; });
    } else {
      matrix_vector_op(handle,
                       in_col_major,
                       vec1_view,
                       vec2_view,
                       out_col_major,

                       apply,
                       [] __device__(T a, T b, T c) { return a + b + c; });
    }
  } else {
    if (rowMajor) {
      matrix_vector_op(
        handle, in_row_major, vec1_view, out_row_major, apply, [] __device__(T a, T b) {
          return a + b;
        });
    } else {
      matrix_vector_op(
        handle, in_col_major, vec1_view, out_col_major, apply, [] __device__(T a, T b) {
          return a + b;
        });
    }
  }
}

template <typename T, typename IdxType>
class MatVecOpTest : public ::testing::TestWithParam<MatVecOpInputs<T, IdxType>> {
 public:
  MatVecOpTest()
    : params(::testing::TestWithParam<MatVecOpInputs<T, IdxType>>::GetParam()),
      stream(handle.get_stream()),
      in(params.rows * params.cols, stream),
      out_ref(params.rows * params.cols, stream),
      out(params.rows * params.cols, stream),
      vec1(params.bcastAlongRows ? params.cols : params.rows, stream),
      vec2(params.bcastAlongRows ? params.cols : params.rows, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);
    IdxType N = params.rows, D = params.cols;
    IdxType len    = N * D;
    IdxType vecLen = params.bcastAlongRows ? D : N;
    uniform(handle, r, in.data(), len, (T)-1.0, (T)1.0);
    uniform(handle, r, vec1.data(), vecLen, (T)-1.0, (T)1.0);
    uniform(handle, r, vec2.data(), vecLen, (T)-1.0, (T)1.0);
    if (params.useTwoVectors) {
      naiveMatVec(out_ref.data(),
                  in.data(),
                  vec1.data(),
                  vec2.data(),
                  D,
                  N,
                  params.rowMajor,
                  params.bcastAlongRows,
                  (T)1.0,
                  stream);
    } else {
      naiveMatVec(out_ref.data(),
                  in.data(),
                  vec1.data(),
                  D,
                  N,
                  params.rowMajor,
                  params.bcastAlongRows,
                  (T)1.0,
                  stream);
    }
    matrixVectorOpLaunch(handle,
                         out.data(),
                         in.data(),
                         vec1.data(),
                         vec2.data(),
                         D,
                         N,
                         params.rowMajor,
                         params.bcastAlongRows,
                         params.useTwoVectors);
    handle.sync_stream();
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  MatVecOpInputs<T, IdxType> params;
  rmm::device_uvector<T> in, out, out_ref, vec1, vec2;
};

const std::vector<MatVecOpInputs<float, int>> inputsf_i32 = {
  {0.00001f, 1024, 32, true, true, false, 1234ULL},
  {0.00001f, 1024, 64, true, true, false, 1234ULL},
  {0.00001f, 1024, 32, true, false, false, 1234ULL},
  {0.00001f, 1024, 64, true, false, false, 1234ULL},
  {0.00001f, 1024, 32, false, true, false, 1234ULL},
  {0.00001f, 1024, 64, false, true, false, 1234ULL},
  {0.00001f, 1024, 32, false, false, false, 1234ULL},
  {0.00001f, 1024, 64, false, false, false, 1234ULL},

  {0.00001f, 1024, 32, true, true, true, 1234ULL},
  {0.00001f, 1024, 64, true, true, true, 1234ULL},
  {0.00001f, 1024, 32, true, false, true, 1234ULL},
  {0.00001f, 1024, 64, true, false, true, 1234ULL},
  {0.00001f, 1024, 32, false, true, true, 1234ULL},
  {0.00001f, 1024, 64, false, true, true, 1234ULL},
  {0.00001f, 1024, 32, false, false, true, 1234ULL},
  {0.00001f, 1024, 64, false, false, true, 1234ULL}};
typedef MatVecOpTest<float, int> MatVecOpTestF_i32;
TEST_P(MatVecOpTestF_i32, Result)
{
  ASSERT_TRUE(devArrMatch(
    out_ref.data(), out.data(), params.rows * params.cols, CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MatVecOpTests, MatVecOpTestF_i32, ::testing::ValuesIn(inputsf_i32));

const std::vector<MatVecOpInputs<float, size_t>> inputsf_i64 = {
  {0.00001f, 2500, 250, false, false, false, 1234ULL},
  {0.00001f, 2500, 250, false, false, true, 1234ULL}};
typedef MatVecOpTest<float, size_t> MatVecOpTestF_i64;
TEST_P(MatVecOpTestF_i64, Result)
{
  ASSERT_TRUE(devArrMatch(
    out_ref.data(), out.data(), params.rows * params.cols, CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MatVecOpTests, MatVecOpTestF_i64, ::testing::ValuesIn(inputsf_i64));

const std::vector<MatVecOpInputs<double, int>> inputsd_i32 = {
  {0.0000001, 1024, 32, true, true, false, 1234ULL},
  {0.0000001, 1024, 64, true, true, false, 1234ULL},
  {0.0000001, 1024, 32, true, false, false, 1234ULL},
  {0.0000001, 1024, 64, true, false, false, 1234ULL},
  {0.0000001, 1024, 32, false, true, false, 1234ULL},
  {0.0000001, 1024, 64, false, true, false, 1234ULL},
  {0.0000001, 1024, 32, false, false, false, 1234ULL},
  {0.0000001, 1024, 64, false, false, false, 1234ULL},

  {0.0000001, 1024, 32, true, true, true, 1234ULL},
  {0.0000001, 1024, 64, true, true, true, 1234ULL},
  {0.0000001, 1024, 32, true, false, true, 1234ULL},
  {0.0000001, 1024, 64, true, false, true, 1234ULL},
  {0.0000001, 1024, 32, false, true, true, 1234ULL},
  {0.0000001, 1024, 64, false, true, true, 1234ULL},
  {0.0000001, 1024, 32, false, false, true, 1234ULL},
  {0.0000001, 1024, 64, false, false, true, 1234ULL}};
typedef MatVecOpTest<double, int> MatVecOpTestD_i32;
TEST_P(MatVecOpTestD_i32, Result)
{
  ASSERT_TRUE(devArrMatch(out_ref.data(),
                          out.data(),
                          params.rows * params.cols,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MatVecOpTests, MatVecOpTestD_i32, ::testing::ValuesIn(inputsd_i32));

const std::vector<MatVecOpInputs<double, size_t>> inputsd_i64 = {
  {0.0000001, 2500, 250, false, false, false, 1234ULL},
  {0.0000001, 2500, 250, false, false, true, 1234ULL}};
typedef MatVecOpTest<double, size_t> MatVecOpTestD_i64;
TEST_P(MatVecOpTestD_i64, Result)
{
  ASSERT_TRUE(devArrMatch(out_ref.data(),
                          out.data(),
                          params.rows * params.cols,
                          CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(MatVecOpTests, MatVecOpTestD_i64, ::testing::ValuesIn(inputsd_i64));

}  // end namespace linalg
}  // end namespace raft
