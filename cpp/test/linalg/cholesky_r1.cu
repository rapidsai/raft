#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resource/cusolver_dn_handle.hpp>
#include <raft/core/resources.hpp>
#include <raft/linalg/cholesky_r1_update.cuh>
#include <raft/linalg/detail/cusolver_wrappers.hpp>
#include <raft/util/cudart_utils.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include "../test_utils.cuh"
#include <sstream>
#include <vector>
namespace raft {
namespace linalg {

template <typename math_t>
class CholeskyR1Test : public ::testing::Test {
 protected:
  CholeskyR1Test()
    : G(n_rows * n_rows, resource::get_cuda_stream(handle)),
      L(n_rows * n_rows, resource::get_cuda_stream(handle)),
      L_exp(n_rows * n_rows, resource::get_cuda_stream(handle)),
      devInfo(resource::get_cuda_stream(handle)),
      workspace(0, resource::get_cuda_stream(handle))
  {
    raft::update_device(G.data(), G_host, n_rows * n_rows, resource::get_cuda_stream(handle));

    // Allocate workspace
    solver_handle = resource::get_cusolver_dn_handle(handle);
    // TODO: Call from public API when ready
    RAFT_CUSOLVER_TRY(raft::linalg::detail::cusolverDnpotrf_bufferSize(
      solver_handle, HIPBLAS_FILL_MODE_LOWER, n_rows, L.data(), n_rows, &Lwork));
    int n_bytes = 0;
    // Initializing in HIPBLAS_FILL_MODE_LOWER, because that has larger workspace
    // requirements.
    raft::linalg::choleskyRank1Update(handle,
                                      L.data(),
                                      n_rows,
                                      n_rows,
                                      nullptr,
                                      &n_bytes,
                                      HIPBLAS_FILL_MODE_LOWER,
                                      resource::get_cuda_stream(handle));
    Lwork = std::max(Lwork * sizeof(math_t), (size_t)n_bytes);
    workspace.resize(Lwork, resource::get_cuda_stream(handle));
  }

  void testR1Update()
  {
    int n = n_rows * n_rows;
    std::vector<hipblasFillMode_t> fillmode{HIPBLAS_FILL_MODE_LOWER, HIPBLAS_FILL_MODE_UPPER};
    for (auto uplo : fillmode) {
      raft::copy(L.data(), G.data(), n, resource::get_cuda_stream(handle));
      for (int rank = 1; rank <= n_rows; rank++) {
        std::stringstream ss;
        ss << "Rank " << rank << ((uplo == HIPBLAS_FILL_MODE_LOWER) ? ", lower" : ", upper");
        SCOPED_TRACE(ss.str());

        // Expected solution using Cholesky factorization from scratch
        raft::copy(L_exp.data(), G.data(), n, resource::get_cuda_stream(handle));
        // TODO: Call from public API when ready
        RAFT_CUSOLVER_TRY(raft::linalg::detail::cusolverDnpotrf(solver_handle,
                                                                uplo,
                                                                rank,
                                                                L_exp.data(),
                                                                n_rows,
                                                                (math_t*)workspace.data(),
                                                                Lwork,
                                                                devInfo.data(),
                                                                resource::get_cuda_stream(handle)));

        // Incremental Cholesky factorization using rank one updates.
        raft::linalg::choleskyRank1Update(handle,
                                          L.data(),
                                          rank,
                                          n_rows,
                                          workspace.data(),
                                          &Lwork,
                                          uplo,
                                          resource::get_cuda_stream(handle));

        ASSERT_TRUE(raft::devArrMatch(L_exp.data(),
                                      L.data(),
                                      n_rows * rank,
                                      raft::CompareApprox<math_t>(3e-3),
                                      resource::get_cuda_stream(handle)));
      }
    }
  }

  void testR1Error()
  {
    raft::update_device(G.data(), G2_host, 4, resource::get_cuda_stream(handle));
    std::vector<hipblasFillMode_t> fillmode{HIPBLAS_FILL_MODE_LOWER, HIPBLAS_FILL_MODE_UPPER};
    for (auto uplo : fillmode) {
      raft::copy(L.data(), G.data(), 4, resource::get_cuda_stream(handle));
      ASSERT_NO_THROW(raft::linalg::choleskyRank1Update(
        handle, L.data(), 1, 2, workspace.data(), &Lwork, uplo, resource::get_cuda_stream(handle)));
      ASSERT_THROW(raft::linalg::choleskyRank1Update(handle,
                                                     L.data(),
                                                     2,
                                                     2,
                                                     workspace.data(),
                                                     &Lwork,
                                                     uplo,
                                                     resource::get_cuda_stream(handle)),
                   raft::exception);

      math_t eps = std::numeric_limits<math_t>::epsilon();
      ASSERT_NO_THROW(raft::linalg::choleskyRank1Update(handle,
                                                        L.data(),
                                                        2,
                                                        2,
                                                        workspace.data(),
                                                        &Lwork,
                                                        uplo,
                                                        resource::get_cuda_stream(handle),
                                                        eps));
    }
  }

  raft::resources handle;
  hipStream_t stream;

  hipsolverHandle_t solver_handle;

  int n_rows = 4;
  int Lwork;
  math_t G_host[16] =  // clang-format off
     {107.,  1393.,  1141.,  91.,
      1393., 21132., 15689., 9539.,
      1141., 15689., 13103., 2889.,
      91.,   9539.,  2889.,  23649.};
                       // clang-format on

  math_t G2_host[4] = {3, 4, 2, 1};

  rmm::device_scalar<int> devInfo;
  rmm::device_uvector<math_t> G;
  rmm::device_uvector<math_t> L_exp;
  rmm::device_uvector<math_t> L;
  rmm::device_uvector<char> workspace;
};

typedef ::testing::Types<float, double> FloatTypes;

TYPED_TEST_CASE(CholeskyR1Test, FloatTypes);

TYPED_TEST(CholeskyR1Test, update) { this->testR1Update(); }
TYPED_TEST(CholeskyR1Test, throwError) { this->testR1Error(); }

};  // namespace linalg
};  // namespace raft