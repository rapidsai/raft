#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <raft/linalg/axpy.cuh>

#include "../test_utils.h"
#include <gtest/gtest.h>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>

namespace raft {
namespace linalg {

// Reference axpy implementation.
template <typename T>
__global__ void naiveAxpy(const int n, const T alpha, const T* x, T* y, int incx, int incy)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) { y[idx * incy] += alpha * x[idx * incx]; }
}

template <typename ElementType,
          typename IndexType    = std::uint32_t,
          typename LayoutPolicy = layout_stride>
auto make_strided_device_vector_view(ElementType* ptr, IndexType n, IndexType stride)
{
  vector_extent<IndexType> exts{n};
  std::array<IndexType, 1> strides{stride};
  auto layout = typename LayoutPolicy::mapping<vector_extent<IndexType>>{exts, strides};
  return device_vector_view<ElementType, IndexType, LayoutPolicy>{ptr, layout};
}

template <typename InType, typename IndexType = int, typename OutType = InType>
struct AxpyInputs {
  OutType tolerance;
  IndexType len;
  InType alpha;
  IndexType incx;
  IndexType incy;
  unsigned long long int seed;
};

template <typename T>
class AxpyTest : public ::testing::TestWithParam<AxpyInputs<T>> {
 protected:
  raft::handle_t handle;
  AxpyInputs<T> params;
  rmm::device_uvector<T> refy;
  rmm::device_uvector<T> y;

 public:
  AxpyTest()
    : testing::TestWithParam<AxpyInputs<T>>(),
      refy(0, handle.get_stream()),
      y(0, handle.get_stream())
  {
    handle.sync_stream();
  }

 protected:
  void SetUp() override
  {
    params = ::testing::TestWithParam<AxpyInputs<T>>::GetParam();

    hipStream_t stream = handle.get_stream();

    raft::random::RngState r(params.seed);

    int x_len = params.len * params.incx;
    int y_len = params.len * params.incy;
    rmm::device_uvector<T> x(x_len, stream);
    y.resize(y_len, stream);
    refy.resize(y_len, stream);

    uniform(handle, r, x.data(), x_len, T(-1.0), T(1.0));
    uniform(handle, r, y.data(), y_len, T(-1.0), T(1.0));

    // Take a copy of the random generated values in y for the naive reference implementation
    // this is necessary since axpy uses y for both input and output
    raft::copy(refy.data(), y.data(), y_len, stream);

    int threads = 64;
    int blocks  = raft::ceildiv<int>(params.len, threads);

    naiveAxpy<T><<<blocks, threads, 0, stream>>>(
      params.len, params.alpha, x.data(), refy.data(), params.incx, params.incy);

    if ((params.incx > 1) && (params.incy > 1)) {
      axpy(handle,
           make_host_scalar_view<const T>(&params.alpha),
           make_strided_device_vector_view<const T>(x.data(), params.len, params.incx),
           make_strided_device_vector_view<T>(y.data(), params.len, params.incy));
    } else if (params.incx > 1) {
      axpy(handle,
           make_host_scalar_view<const T>(&params.alpha),
           make_strided_device_vector_view<const T>(x.data(), params.len, params.incx),
           make_device_vector_view<T>(y.data(), params.len));
    } else if (params.incy > 1) {
      axpy(handle,
           make_host_scalar_view<const T>(&params.alpha),
           make_device_vector_view<const T>(x.data(), params.len),
           make_strided_device_vector_view<T>(y.data(), params.len, params.incy));
    } else {
      axpy(handle,
           make_host_scalar_view<const T>(&params.alpha),
           make_device_vector_view<const T>(x.data(), params.len),
           make_device_vector_view<T>(y.data(), params.len));
    }

    handle.sync_stream();
  }

  void TearDown() override {}
};

const std::vector<AxpyInputs<float>> inputsf = {
  {0.000001f, 1024 * 1024, 2.f, 1, 1, 1234ULL},
  {0.000001f, 16 * 1024 * 1024, 128.f, 1, 1, 1234ULL},
  {0.000001f, 98689, 4.f, 1, 1, 1234ULL},
  {0.000001f, 4 * 1024 * 1024, -1, 1, 1, 1234ULL},
  {0.000001f, 1024 * 1024, 6, 4, 1, 1234ULL},
  {0.000001f, 1024 * 1024, 7, 1, 3, 1234ULL},
  {0.000001f, 1024 * 1024, 8, 4, 3, 1234ULL},
};

const std::vector<AxpyInputs<double>> inputsd = {
  {0.000001f, 1024 * 1024, 2.f, 1, 1, 1234ULL},
  {0.000001f, 16 * 1024 * 1024, 128.f, 1, 1, 1234ULL},
  {0.000001f, 98689, 4.f, 1, 1, 1234ULL},
  {0.000001f, 4 * 1024 * 1024, -1, 1, 1, 1234ULL},
  {0.000001f, 1024 * 1024, 6, 4, 1, 1234ULL},
  {0.000001f, 1024 * 1024, 7, 1, 3, 1234ULL},
  {0.000001f, 1024 * 1024, 8, 4, 3, 1234ULL},
};

typedef AxpyTest<float> AxpyTestF;
TEST_P(AxpyTestF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(
    refy.data(), y.data(), params.len * params.incy, raft::CompareApprox<float>(params.tolerance)));
}

typedef AxpyTest<double> AxpyTestD;
TEST_P(AxpyTestD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(refy.data(),
                                y.data(),
                                params.len * params.incy,
                                raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_SUITE_P(AxpyTests, AxpyTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(AxpyTests, AxpyTestD, ::testing::ValuesIn(inputsd));

}  // end namespace linalg
}  // end namespace raft
