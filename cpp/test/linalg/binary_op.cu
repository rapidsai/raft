/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <raft/cudart_utils.h>
#include <raft/linalg/binary_op.cuh>
#include <raft/random/rng.hpp>
#include <rmm/device_uvector.hpp>
#include "../test_utils.h"
#include "binary_op.cuh"

namespace raft {
namespace linalg {

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename InType, typename IdxType, typename OutType>
void binaryOpLaunch(
  OutType* out, const InType* in1, const InType* in2, IdxType len, hipStream_t stream)
{
  binaryOp(
    out, in1, in2, len, [] __device__(InType a, InType b) { return a + b; }, stream);
}

template <typename InType, typename IdxType, typename OutType = InType>
class BinaryOpTest : public ::testing::TestWithParam<BinaryOpInputs<InType, IdxType, OutType>> {
 public:
  BinaryOpTest()
    : params(::testing::TestWithParam<BinaryOpInputs<InType, IdxType, OutType>>::GetParam()),
      stream(handle.get_stream()),
      in1(params.len, stream),
      in2(params.len, stream),
      out_ref(params.len, stream),
      out(params.len, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::Rng r(params.seed);
    IdxType len = params.len;
    r.uniform(in1.data(), len, InType(-1.0), InType(1.0), stream);
    r.uniform(in2.data(), len, InType(-1.0), InType(1.0), stream);
    naiveAdd(out_ref.data(), in1.data(), in2.data(), len);
    binaryOpLaunch(out.data(), in1.data(), in2.data(), len, stream);
    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  BinaryOpInputs<InType, IdxType, OutType> params;
  rmm::device_uvector<InType> in1;
  rmm::device_uvector<InType> in2;
  rmm::device_uvector<OutType> out_ref;
  rmm::device_uvector<OutType> out;
};

const std::vector<BinaryOpInputs<float, int>> inputsf_i32 = {{0.000001f, 1024 * 1024, 1234ULL}};
typedef BinaryOpTest<float, int> BinaryOpTestF_i32;
TEST_P(BinaryOpTestF_i32, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(BinaryOpTests, BinaryOpTestF_i32, ::testing::ValuesIn(inputsf_i32));

const std::vector<BinaryOpInputs<float, size_t>> inputsf_i64 = {{0.000001f, 1024 * 1024, 1234ULL}};
typedef BinaryOpTest<float, size_t> BinaryOpTestF_i64;
TEST_P(BinaryOpTestF_i64, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(BinaryOpTests, BinaryOpTestF_i64, ::testing::ValuesIn(inputsf_i64));

const std::vector<BinaryOpInputs<float, int, double>> inputsf_i32_d = {
  {0.000001f, 1024 * 1024, 1234ULL}};
typedef BinaryOpTest<float, int, double> BinaryOpTestF_i32_D;
TEST_P(BinaryOpTestF_i32_D, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(BinaryOpTests, BinaryOpTestF_i32_D, ::testing::ValuesIn(inputsf_i32_d));

const std::vector<BinaryOpInputs<double, int>> inputsd_i32 = {{0.00000001, 1024 * 1024, 1234ULL}};
typedef BinaryOpTest<double, int> BinaryOpTestD_i32;
TEST_P(BinaryOpTestD_i32, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(BinaryOpTests, BinaryOpTestD_i32, ::testing::ValuesIn(inputsd_i32));

const std::vector<BinaryOpInputs<double, size_t>> inputsd_i64 = {
  {0.00000001, 1024 * 1024, 1234ULL}};
typedef BinaryOpTest<double, size_t> BinaryOpTestD_i64;
TEST_P(BinaryOpTestD_i64, Result)
{
  ASSERT_TRUE(
    devArrMatch(out_ref.data(), out.data(), params.len, CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_SUITE_P(BinaryOpTests, BinaryOpTestD_i64, ::testing::ValuesIn(inputsd_i64));

template <typename math_t>
class BinaryOpAlignment : public ::testing::Test {
 protected:
  BinaryOpAlignment()
  {
    RAFT_CUDA_TRY(hipStreamCreate(&stream));
    handle.set_stream(stream);
  }
  void TearDown() override { RAFT_CUDA_TRY(hipStreamDestroy(stream)); }

 public:
  void Misaligned()
  {
    // Test to trigger cudaErrorMisalignedAddress if veclen is incorrectly
    // chosen.
    int n = 1024;
    rmm::device_uvector<math_t> x(n, stream);
    rmm::device_uvector<math_t> y(n, stream);
    rmm::device_uvector<math_t> z(n, stream);
    RAFT_CUDA_TRY(hipMemsetAsync(x.data(), 0, n * sizeof(math_t), stream));
    RAFT_CUDA_TRY(hipMemsetAsync(y.data(), 0, n * sizeof(math_t), stream));
    raft::linalg::binaryOp(
      z.data() + 9,
      x.data() + 137,
      y.data() + 19,
      256,
      [] __device__(math_t x, math_t y) { return x + y; },
      stream);
  }

  raft::handle_t handle;
  hipStream_t stream;
};
typedef ::testing::Types<float, double> FloatTypes;
TYPED_TEST_CASE(BinaryOpAlignment, FloatTypes);
TYPED_TEST(BinaryOpAlignment, Misaligned) { this->Misaligned(); }
}  // namespace linalg
}  // namespace raft
