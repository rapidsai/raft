/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include "add.cuh"
#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/linalg/add.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename InT, typename OutT = InT>
class AddTest : public ::testing::TestWithParam<AddInputs<InT, OutT>> {
 public:
  AddTest()
    : params(::testing::TestWithParam<AddInputs<InT, OutT>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      in1(params.len, stream),
      in2(params.len, stream),
      out_ref(params.len, stream),
      out(params.len, stream)
  {
  }

 protected:
  void SetUp() override
  {
    params = ::testing::TestWithParam<AddInputs<InT, OutT>>::GetParam();
    raft::random::RngState r{params.seed};
    int len = params.len;
    uniform(handle, r, in1.data(), len, InT(-1.0), InT(1.0));
    uniform(handle, r, in2.data(), len, InT(-1.0), InT(1.0));
    naiveAddElem<InT, OutT>(out_ref.data(), in1.data(), in2.data(), len, stream);

    auto out_view = raft::make_device_vector_view(out.data(), out.size());
    auto in1_view = raft::make_device_vector_view<const InT>(in1.data(), in1.size());
    auto in2_view = raft::make_device_vector_view<const InT>(in2.data(), in2.size());

    add(handle, in1_view, in2_view, out_view);
    resource::sync_stream(handle, stream);
  }

  void compare()
  {
    ASSERT_TRUE(raft::devArrMatch(
      out_ref.data(), out.data(), params.len, raft::CompareApprox<OutT>(params.tolerance), stream));
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  AddInputs<InT, OutT> params;
  rmm::device_uvector<InT> in1;
  rmm::device_uvector<InT> in2;
  rmm::device_uvector<OutT> out_ref;
  rmm::device_uvector<OutT> out;
};

const std::vector<AddInputs<float>> inputsf = {
  {0.000001f, 1024 * 1024, 1234ULL},
  {0.000001f, 1024 * 1024 + 2, 1234ULL},
  {0.000001f, 1024 * 1024 + 1, 1234ULL},
};
typedef AddTest<float> AddTestF;
TEST_P(AddTestF, Result) { compare(); }
INSTANTIATE_TEST_SUITE_P(AddTests, AddTestF, ::testing::ValuesIn(inputsf));

const std::vector<AddInputs<double>> inputsd = {
  {0.00000001, 1024 * 1024, 1234ULL},
  {0.00000001, 1024 * 1024 + 2, 1234ULL},
  {0.00000001, 1024 * 1024 + 1, 1234ULL},
};
typedef AddTest<double> AddTestD;
TEST_P(AddTestD, Result) { compare(); }
INSTANTIATE_TEST_SUITE_P(AddTests, AddTestD, ::testing::ValuesIn(inputsd));

const std::vector<AddInputs<float, double>> inputsfd = {
  {0.00000001, 1024 * 1024, 1234ULL},
  {0.00000001, 1024 * 1024 + 2, 1234ULL},
  {0.00000001, 1024 * 1024 + 1, 1234ULL},
};
typedef AddTest<float, double> AddTestFD;
TEST_P(AddTestFD, Result) { compare(); }
INSTANTIATE_TEST_SUITE_P(AddTests, AddTestFD, ::testing::ValuesIn(inputsfd));

}  // end namespace linalg
}  // end namespace raft
