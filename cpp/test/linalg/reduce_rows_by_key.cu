/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <gtest/gtest.h>
#include <iostream>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/linalg/reduce_rows_by_key.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename Type>
RAFT_KERNEL naiveReduceRowsByKeyKernel(const Type* d_A,
                                       int lda,
                                       uint32_t* d_keys,
                                       const Type* d_weight,
                                       char* d_char_keys,
                                       int nrows,
                                       int ncols,
                                       int nkeys,
                                       Type* d_sums)
{
  int c = threadIdx.x + blockIdx.x * blockDim.x;
  if (c >= ncols) return;
  int this_key = threadIdx.y + blockIdx.y * blockDim.y;

  Type sum = 0.0;
  for (int r = 0; r < nrows; r++) {
    if (this_key != d_keys[r]) continue;
    Type wt = 1;
    if (d_weight) wt = d_weight[r];
    sum += d_A[lda * r + c] * wt;
  }
  d_sums[this_key * ncols + c] = sum;
}
template <typename Type>
void naiveReduceRowsByKey(const Type* d_A,
                          int lda,
                          uint32_t* d_keys,
                          const Type* d_weight,
                          char* d_char_keys,
                          int nrows,
                          int ncols,
                          int nkeys,
                          Type* d_sums,
                          hipStream_t stream)
{
  hipMemset(d_sums, 0, sizeof(Type) * nkeys * ncols);

  naiveReduceRowsByKeyKernel<<<dim3((ncols + 31) / 32, nkeys), dim3(32, 1), 0, stream>>>(
    d_A, lda, d_keys, d_weight, d_char_keys, nrows, ncols, nkeys, d_sums);
}

template <typename T>
struct ReduceRowsInputs {
  T tolerance;
  int nobs;
  uint32_t cols;
  uint32_t nkeys;
  unsigned long long int seed;
  bool weighted;
  T max_weight;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const ReduceRowsInputs<T>& dims)
{
  return os;
}

template <typename T>
class ReduceRowTest : public ::testing::TestWithParam<ReduceRowsInputs<T>> {
 public:
  ReduceRowTest()
    : params(::testing::TestWithParam<ReduceRowsInputs<T>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      in(params.nobs * params.cols, stream),
      out(params.nkeys * params.cols, stream),
      out_ref(params.nkeys * params.cols, stream),
      keys(params.nobs, stream),
      scratch_buf(params.nobs, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);
    raft::random::RngState r_int(params.seed);

    uint32_t nobs  = params.nobs;
    uint32_t cols  = params.cols;
    uint32_t nkeys = params.nkeys;
    uniform(handle, r, in.data(), nobs * cols, T(0.0), T(2.0 / nobs));
    uniformInt(handle, r_int, keys.data(), nobs, (uint32_t)0, nkeys);

    rmm::device_uvector<T> weight(0, stream);
    if (params.weighted) {
      weight.resize(nobs, stream);
      raft::random::RngState r(params.seed);
      uniform(handle, r, weight.data(), nobs, T(1), params.max_weight);
    }

    naiveReduceRowsByKey(in.data(),
                         cols,
                         keys.data(),
                         params.weighted ? weight.data() : nullptr,
                         scratch_buf.data(),
                         nobs,
                         cols,
                         nkeys,
                         out_ref.data(),
                         stream);
    auto input_view = raft::make_device_matrix_view<const T>(
      in.data(), params.cols, static_cast<uint32_t>(params.nobs));
    auto output_view = raft::make_device_matrix_view(out.data(), params.cols, params.nkeys);
    auto keys_view   = raft::make_device_vector_view<const uint32_t>(
      keys.data(), static_cast<uint32_t>(params.nobs));
    auto scratch_buf_view =
      raft::make_device_vector_view(scratch_buf.data(), static_cast<uint32_t>(params.nobs));
    std::optional<raft::device_vector_view<const T>> weights_view;
    if (params.weighted) {
      weights_view.emplace(weight.data(), static_cast<uint32_t>(params.nobs));
    }

    reduce_rows_by_key(
      handle, input_view, keys_view, output_view, params.nkeys, scratch_buf_view, weights_view);
    resource::sync_stream(handle, stream);
  }

 protected:
  ReduceRowsInputs<T> params;
  raft::resources handle;
  hipStream_t stream = 0;

  int device_count = 0;
  rmm::device_uvector<T> in, out, out_ref;
  rmm::device_uvector<uint32_t> keys;
  rmm::device_uvector<char> scratch_buf;
};

// ReduceRowTestF
// 128 Obs, 32 cols, 6 clusters
const std::vector<ReduceRowsInputs<float>> inputsf2 = {{0.000001f, 128, 32, 6, 1234ULL, false},
                                                       {0.000001f, 128, 32, 6, 1234ULL, true, 1.0},
                                                       {0.000001f, 128, 32, 6, 1234ULL, true, 2.0}};
typedef ReduceRowTest<float> ReduceRowTestF;
TEST_P(ReduceRowTestF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(out_ref.data(),
                                out.data(),
                                params.cols * params.nkeys,
                                raft::CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestF, ::testing::ValuesIn(inputsf2));

// ReduceRowTestD
// 128 Obs, 32 cols, 6 clusters, double precision
const std::vector<ReduceRowsInputs<double>> inputsd2 = {
  {0.00000001, 128, 32, 6, 1234ULL, false},
  {0.00000001, 128, 32, 6, 1234ULL, true, 2.0},
  {0.00000001, 128, 32, 6, 1234ULL, true, 8.0}};
typedef ReduceRowTest<double> ReduceRowTestD;
TEST_P(ReduceRowTestD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(out_ref.data(),
                                out.data(),
                                params.cols * params.nkeys,
                                raft::CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests, ReduceRowTestD, ::testing::ValuesIn(inputsd2));

// ReduceRowTestSmallnKey
// 128 Obs, 32 cols, 3 clusters
const std::vector<ReduceRowsInputs<float>> inputsf_small_nkey = {
  {0.000001f, 128, 32, 3, 1234ULL, false},
  {0.000001f, 128, 32, 3, 1234ULL, true, 5.0},
  {0.000001f, 128, 32, 3, 1234ULL, true, 8.0}};
typedef ReduceRowTest<float> ReduceRowTestSmallnKey;
TEST_P(ReduceRowTestSmallnKey, Result)
{
  ASSERT_TRUE(raft::devArrMatch(out_ref.data(),
                                out.data(),
                                params.cols * params.nkeys,
                                raft::CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests,
                        ReduceRowTestSmallnKey,
                        ::testing::ValuesIn(inputsf_small_nkey));

// ReduceRowTestBigSpace
// 512 Obs, 1024 cols, 32 clusters, double precision
const std::vector<ReduceRowsInputs<double>> inputsd_big_space = {
  {0.00000001, 512, 1024, 40, 1234ULL, false},
  {0.00000001, 512, 1024, 40, 1234ULL, true, 4.0},
  {0.00000001, 512, 1024, 40, 1234ULL, true, 16.0}};
typedef ReduceRowTest<double> ReduceRowTestBigSpace;
TEST_P(ReduceRowTestBigSpace, Result)
{
  ASSERT_TRUE(raft::devArrMatch(out_ref.data(),
                                out.data(),
                                params.cols * params.nkeys,
                                raft::CompareApprox<double>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests,
                        ReduceRowTestBigSpace,
                        ::testing::ValuesIn(inputsd_big_space));

// ReduceRowTestManyObs
// 100000 Obs, 37 cols, 32 clusters
const std::vector<ReduceRowsInputs<float>> inputsf_many_obs = {
  {0.00001f, 100000, 37, 32, 1234ULL, false},
  {0.00001f, 100000, 37, 32, 1234ULL, true, 4.0},
  {0.00001f, 100000, 37, 32, 1234ULL, true, 16.0}};
typedef ReduceRowTest<float> ReduceRowTestManyObs;
TEST_P(ReduceRowTestManyObs, Result)
{
  ASSERT_TRUE(raft::devArrMatch(out_ref.data(),
                                out.data(),
                                params.cols * params.nkeys,
                                raft::CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests,
                        ReduceRowTestManyObs,
                        ::testing::ValuesIn(inputsf_many_obs));

// ReduceRowTestManyClusters
// 100000 Obs, 37 cols, 2048 clusters
const std::vector<ReduceRowsInputs<float>> inputsf_many_cluster = {
  {0.00001f, 100000, 37, 2048, 1234ULL, false},
  {0.00001f, 100000, 37, 2048, 1234ULL, true, 32.0},
  {0.00001f, 100000, 37, 2048, 1234ULL, true, 16.0}};
typedef ReduceRowTest<float> ReduceRowTestManyClusters;
TEST_P(ReduceRowTestManyClusters, Result)
{
  ASSERT_TRUE(raft::devArrMatch(out_ref.data(),
                                out.data(),
                                params.cols * params.nkeys,
                                raft::CompareApprox<float>(params.tolerance)));
}
INSTANTIATE_TEST_CASE_P(ReduceRowTests,
                        ReduceRowTestManyClusters,
                        ::testing::ValuesIn(inputsf_many_cluster));

}  // end namespace linalg
}  // end namespace raft
