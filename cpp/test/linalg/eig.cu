/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"
#include <gtest/gtest.h>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/linalg/eig.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename T>
struct EigInputs {
  T tolerance;
  int len;
  int n_row;
  int n_col;
  unsigned long long int seed;
  int n;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const EigInputs<T>& dims)
{
  return os;
}

template <typename T>
class EigTest : public ::testing::TestWithParam<EigInputs<T>> {
 public:
  EigTest()
    : params(::testing::TestWithParam<EigInputs<T>>::GetParam()),
      stream(resource::get_cuda_stream(handle)),
      cov_matrix(params.len, stream),
      eig_vectors(params.len, stream),
      eig_vectors_jacobi(params.len, stream),
      eig_vectors_ref(params.len, stream),
      eig_vals(params.n_col, stream),
      eig_vals_jacobi(params.n_col, stream),
      eig_vals_ref(params.n_col, stream),
      cov_matrix_large(params.n * params.n, stream),
      eig_vectors_large(params.n * params.n, stream),
      eig_vectors_jacobi_large(params.n * params.n, stream),
      eig_vals_large(params.n, stream),
      eig_vals_jacobi_large(params.n, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);
    int len = params.len;

    T cov_matrix_h[] = {
      1.0, 0.9, 0.81, 0.729, 0.9, 1.0, 0.9, 0.81, 0.81, 0.9, 1.0, 0.9, 0.729, 0.81, 0.9, 1.0};
    ASSERT(len == 16, "This test only works with 4x4 matrices!");
    raft::update_device(cov_matrix.data(), cov_matrix_h, len, stream);

    T eig_vectors_ref_h[] = {0.2790,
                             -0.6498,
                             0.6498,
                             -0.2789,
                             -0.5123,
                             0.4874,
                             0.4874,
                             -0.5123,
                             0.6498,
                             0.2789,
                             -0.2789,
                             -0.6498,
                             0.4874,
                             0.5123,
                             0.5123,
                             0.4874};
    T eig_vals_ref_h[]    = {0.0614, 0.1024, 0.3096, 3.5266};

    raft::update_device(eig_vectors_ref.data(), eig_vectors_ref_h, len, stream);
    raft::update_device(eig_vals_ref.data(), eig_vals_ref_h, params.n_col, stream);

    auto cov_matrix_view = raft::make_device_matrix_view<const T, std::uint32_t, raft::col_major>(
      cov_matrix.data(), params.n_row, params.n_col);
    auto eig_vectors_view = raft::make_device_matrix_view<T, std::uint32_t, raft::col_major>(
      eig_vectors.data(), params.n_row, params.n_col);
    auto eig_vals_view =
      raft::make_device_vector_view<T, std::uint32_t>(eig_vals.data(), params.n_row);

    auto eig_vectors_jacobi_view = raft::make_device_matrix_view<T, std::uint32_t, raft::col_major>(
      eig_vectors_jacobi.data(), params.n_row, params.n_col);
    auto eig_vals_jacobi_view =
      raft::make_device_vector_view<T, std::uint32_t>(eig_vals_jacobi.data(), params.n_row);

    eig_dc(handle, cov_matrix_view, eig_vectors_view, eig_vals_view);

    T tol      = 1.e-7;
    int sweeps = 15;
    eig_jacobi(handle, cov_matrix_view, eig_vectors_jacobi_view, eig_vals_jacobi_view, tol, sweeps);

    // test code for comparing two methods
    len = params.n * params.n;

    uniform(handle, r, cov_matrix_large.data(), len, T(-1.0), T(1.0));

    auto cov_matrix_large_view =
      raft::make_device_matrix_view<const T, std::uint32_t, raft::col_major>(
        cov_matrix_large.data(), params.n, params.n);
    auto eig_vectors_large_view = raft::make_device_matrix_view<T, std::uint32_t, raft::col_major>(
      eig_vectors_large.data(), params.n, params.n);
    auto eig_vals_large_view =
      raft::make_device_vector_view<T, std::uint32_t>(eig_vals_large.data(), params.n);

    auto eig_vectors_jacobi_large_view =
      raft::make_device_matrix_view<T, std::uint32_t, raft::col_major>(
        eig_vectors_jacobi_large.data(), params.n, params.n);
    auto eig_vals_jacobi_large_view =
      raft::make_device_vector_view<T, std::uint32_t>(eig_vals_jacobi_large.data(), params.n);

    eig_dc(handle, cov_matrix_large_view, eig_vectors_large_view, eig_vals_large_view);
    eig_jacobi(handle,
               cov_matrix_large_view,
               eig_vectors_jacobi_large_view,
               eig_vals_jacobi_large_view,
               tol,
               sweeps);
    resource::sync_stream(handle, stream);
  }

 protected:
  raft::resources handle;
  hipStream_t stream;

  EigInputs<T> params;

  rmm::device_uvector<T> cov_matrix, eig_vectors, eig_vectors_jacobi, eig_vectors_ref, eig_vals,
    eig_vals_jacobi, eig_vals_ref;

  rmm::device_uvector<T> cov_matrix_large, eig_vectors_large, eig_vectors_jacobi_large,
    eig_vals_large, eig_vals_jacobi_large;
};

const std::vector<EigInputs<float>> inputsf2 = {{0.001f, 4 * 4, 4, 4, 1234ULL, 256}};

const std::vector<EigInputs<double>> inputsd2 = {{0.001, 4 * 4, 4, 4, 1234ULL, 256}};

typedef EigTest<float> EigTestValF;
TEST_P(EigTestValF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vals_ref.data(),
                                eig_vals.data(),
                                params.n_col,
                                raft::CompareApproxAbs<float>(params.tolerance),
                                stream));
}

typedef EigTest<double> EigTestValD;
TEST_P(EigTestValD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vals_ref.data(),
                                eig_vals.data(),
                                params.n_col,
                                raft::CompareApproxAbs<double>(params.tolerance),
                                stream));
}

typedef EigTest<float> EigTestVecF;
TEST_P(EigTestVecF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vectors_ref.data(),
                                eig_vectors.data(),
                                params.len,
                                raft::CompareApproxAbs<float>(params.tolerance),
                                stream));
}

typedef EigTest<double> EigTestVecD;
TEST_P(EigTestVecD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vectors_ref.data(),
                                eig_vectors.data(),
                                params.len,
                                raft::CompareApproxAbs<double>(params.tolerance),
                                stream));
}

typedef EigTest<float> EigTestValJacobiF;
TEST_P(EigTestValJacobiF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vals_ref.data(),
                                eig_vals_jacobi.data(),
                                params.n_col,
                                raft::CompareApproxAbs<float>(params.tolerance),
                                stream));
}

typedef EigTest<double> EigTestValJacobiD;
TEST_P(EigTestValJacobiD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vals_ref.data(),
                                eig_vals_jacobi.data(),
                                params.n_col,
                                raft::CompareApproxAbs<double>(params.tolerance),
                                stream));
}

typedef EigTest<float> EigTestVecJacobiF;
TEST_P(EigTestVecJacobiF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vectors_ref.data(),
                                eig_vectors_jacobi.data(),
                                params.len,
                                raft::CompareApproxAbs<float>(params.tolerance),
                                stream));
}

typedef EigTest<double> EigTestVecJacobiD;
TEST_P(EigTestVecJacobiD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vectors_ref.data(),
                                eig_vectors_jacobi.data(),
                                params.len,
                                raft::CompareApproxAbs<double>(params.tolerance),
                                stream));
}

typedef EigTest<float> EigTestVecCompareF;
TEST_P(EigTestVecCompareF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vectors_large.data(),
                                eig_vectors_jacobi_large.data(),
                                (params.n * params.n),
                                raft::CompareApproxAbs<float>(params.tolerance),
                                stream));
}

typedef EigTest<double> EigTestVecCompareD;
TEST_P(EigTestVecCompareD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(eig_vectors_large.data(),
                                eig_vectors_jacobi_large.data(),
                                (params.n * params.n),
                                raft::CompareApproxAbs<double>(params.tolerance),
                                stream));
}

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestValF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestValD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestVecF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestVecD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestValJacobiF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestValJacobiD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestVecJacobiF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestVecJacobiD, ::testing::ValuesIn(inputsd2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestVecCompareF, ::testing::ValuesIn(inputsf2));

INSTANTIATE_TEST_SUITE_P(EigTests, EigTestVecCompareD, ::testing::ValuesIn(inputsd2));

}  // namespace linalg
}  // namespace raft
