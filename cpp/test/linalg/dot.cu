#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <raft/linalg/dot.cuh>

#include "../test_utils.h"
#include <gtest/gtest.h>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>
#include <rmm/device_scalar.hpp>

namespace raft {
namespace linalg {

// Reference dot implementation.
template <typename T>
__global__ void naiveDot(const int n, const T* x, int incx, const T* y, int incy, T* out)
{
  T sum = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    sum += x[i * incx] * y[i * incy];
  }
  atomicAdd(out, sum);
}

template <typename InType, typename IndexType = int, typename OutType = InType>
struct DotInputs {
  OutType tolerance;
  IndexType len;
  IndexType incx;
  IndexType incy;
  unsigned long long int seed;
};

template <typename T>
class DotTest : public ::testing::TestWithParam<DotInputs<T>> {
 protected:
  raft::handle_t handle;
  DotInputs<T> params;
  rmm::device_scalar<T> output;
  rmm::device_scalar<T> refoutput;

 public:
  DotTest()
    : testing::TestWithParam<DotInputs<T>>(),
      output(0, handle.get_stream()),
      refoutput(0, handle.get_stream())
  {
    handle.sync_stream();
  }

 protected:
  void SetUp() override
  {
    params = ::testing::TestWithParam<DotInputs<T>>::GetParam();

    hipStream_t stream = handle.get_stream();

    raft::random::RngState r(params.seed);

    int x_len = params.len * params.incx;
    int y_len = params.len * params.incy;

    rmm::device_uvector<T> x(x_len, stream);
    rmm::device_uvector<T> y(y_len, stream);
    uniform(handle, r, x.data(), x_len, T(-1.0), T(1.0));
    uniform(handle, r, y.data(), y_len, T(-1.0), T(1.0));

    naiveDot<<<256, 256, 0, stream>>>(
      params.len, x.data(), params.incx, y.data(), params.incy, refoutput.data());

    auto out_view = make_device_scalar_view<T, int>(output.data());

    if ((params.incx > 1) && (params.incy > 1)) {
      dot(handle,
          make_strided_device_vector_view<const T>(x.data(), params.len, params.incx),
          make_strided_device_vector_view<const T>(y.data(), params.len, params.incy),
          out_view);
    } else if (params.incx > 1) {
      dot(handle,
          make_strided_device_vector_view<const T>(x.data(), params.len, params.incx),
          make_device_vector_view<const T>(y.data(), params.len),
          out_view);
    } else if (params.incy > 1) {
      dot(handle,
          make_device_vector_view<const T>(x.data(), params.len),
          make_strided_device_vector_view<const T>(y.data(), params.len, params.incy),
          out_view);
    } else {
      dot(handle,
          make_device_vector_view<const T>(x.data(), params.len),
          make_device_vector_view<const T>(y.data(), params.len),
          out_view);
    }
    handle.sync_stream();
  }

  void TearDown() override {}
};

const std::vector<DotInputs<float>> inputsf = {
  {0.0001f, 1024 * 1024, 1, 1, 1234ULL},
  {0.0001f, 16 * 1024 * 1024, 1, 1, 1234ULL},
  {0.0001f, 98689, 1, 1, 1234ULL},
  {0.0001f, 4 * 1024 * 1024, 1, 1, 1234ULL},
  {0.0001f, 1024 * 1024, 4, 1, 1234ULL},
  {0.0001f, 1024 * 1024, 1, 3, 1234ULL},
  {0.0001f, 1024 * 1024, 4, 3, 1234ULL},
};

const std::vector<DotInputs<double>> inputsd = {
  {0.000001f, 1024 * 1024, 1, 1, 1234ULL},
  {0.000001f, 16 * 1024 * 1024, 1, 1, 1234ULL},
  {0.000001f, 98689, 1, 1, 1234ULL},
  {0.000001f, 4 * 1024 * 1024, 1, 1, 1234ULL},
  {0.000001f, 1024 * 1024, 4, 1, 1234ULL},
  {0.000001f, 1024 * 1024, 1, 3, 1234ULL},
  {0.000001f, 1024 * 1024, 4, 3, 1234ULL},
};

typedef DotTest<float> DotTestF;
TEST_P(DotTestF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(
    refoutput.data(), output.data(), 1, raft::CompareApprox<float>(params.tolerance)));
}

typedef DotTest<double> DotTestD;
TEST_P(DotTestD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(
    refoutput.data(), output.data(), 1, raft::CompareApprox<double>(params.tolerance)));
}

INSTANTIATE_TEST_SUITE_P(DotTests, DotTestF, ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_SUITE_P(DotTests, DotTestD, ::testing::ValuesIn(inputsd));

}  // end namespace linalg
}  // end namespace raft
