/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "reduce.cuh"
#include <gtest/gtest.h>
#include <raft/linalg/coalesced_reduction.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft {
namespace linalg {

template <typename T>
struct coalescedReductionInputs {
  T tolerance;
  int rows, cols;
  unsigned long long int seed;
};

template <typename T>
::std::ostream& operator<<(::std::ostream& os, const coalescedReductionInputs<T>& dims)
{
  return os;
}

// Or else, we get the following compilation error
// for an extended __device__ lambda cannot have private or protected access
// within its class
template <typename T>
void coalescedReductionLaunch(
  T* dots, const T* data, int cols, int rows, hipStream_t stream, bool inplace = false)
{
  coalescedReduction(
    dots, data, cols, rows, (T)0, stream, inplace, [] __device__(T in, int i) { return in * in; });
}

template <typename T>
class coalescedReductionTest : public ::testing::TestWithParam<coalescedReductionInputs<T>> {
 public:
  coalescedReductionTest()
    : params(::testing::TestWithParam<coalescedReductionInputs<T>>::GetParam()),
      stream(handle.get_stream()),
      data(params.rows * params.cols, stream),
      dots_exp(params.rows * params.cols, stream),
      dots_act(params.rows * params.cols, stream)
  {
  }

 protected:
  void SetUp() override
  {
    raft::random::RngState r(params.seed);
    int rows = params.rows, cols = params.cols;
    int len = rows * cols;
    uniform(handle, r, data.data(), len, T(-1.0), T(1.0));
    naiveCoalescedReduction(dots_exp.data(), data.data(), cols, rows, stream);

    // Perform reduction with default inplace = false first
    coalescedReductionLaunch(dots_act.data(), data.data(), cols, rows, stream);
    // Add to result with inplace = true next
    coalescedReductionLaunch(dots_act.data(), data.data(), cols, rows, stream, true);

    handle.sync_stream(stream);
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  coalescedReductionInputs<T> params;
  rmm::device_uvector<T> data;
  rmm::device_uvector<T> dots_exp;
  rmm::device_uvector<T> dots_act;
};

const std::vector<coalescedReductionInputs<float>> inputsf = {{0.000002f, 1024, 32, 1234ULL},
                                                              {0.000002f, 1024, 64, 1234ULL},
                                                              {0.000002f, 1024, 128, 1234ULL},
                                                              {0.000002f, 1024, 256, 1234ULL}};

const std::vector<coalescedReductionInputs<double>> inputsd = {{0.000000001, 1024, 32, 1234ULL},
                                                               {0.000000001, 1024, 64, 1234ULL},
                                                               {0.000000001, 1024, 128, 1234ULL},
                                                               {0.000000001, 1024, 256, 1234ULL}};

typedef coalescedReductionTest<float> coalescedReductionTestF;
TEST_P(coalescedReductionTestF, Result)
{
  ASSERT_TRUE(raft::devArrMatch(dots_exp.data(),
                                dots_act.data(),
                                params.rows,
                                raft::CompareApprox<float>(params.tolerance),
                                stream));
}

typedef coalescedReductionTest<double> coalescedReductionTestD;
TEST_P(coalescedReductionTestD, Result)
{
  ASSERT_TRUE(raft::devArrMatch(dots_exp.data(),
                                dots_act.data(),
                                params.rows,
                                raft::CompareApprox<double>(params.tolerance),
                                stream));
}

INSTANTIATE_TEST_CASE_P(coalescedReductionTests,
                        coalescedReductionTestF,
                        ::testing::ValuesIn(inputsf));

INSTANTIATE_TEST_CASE_P(coalescedReductionTests,
                        coalescedReductionTestD,
                        ::testing::ValuesIn(inputsd));

}  // end namespace linalg
}  // end namespace raft
