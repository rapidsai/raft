#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "spatial_data.h"
#include <raft/cudart_utils.h>
#include <raft/distance/distance_type.hpp>
#include <raft/random/make_blobs.cuh>
#include <raft/spatial/knn/ball_cover.cuh>
#include <raft/spatial/knn/detail/knn_brute_force_faiss.cuh>
#if defined RAFT_NN_COMPILED
#include <raft/spatial/knn/specializations.cuh>
#endif

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/count.h>
#include <thrust/fill.h>
#include <thrust/transform.h>

#include <cstdint>
#include <gtest/gtest.h>
#include <iostream>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {

using namespace std;

template <typename value_idx, typename value_t>
__global__ void count_discrepancies_kernel(value_idx* actual_idx,
                                           value_idx* expected_idx,
                                           value_t* actual,
                                           value_t* expected,
                                           uint32_t m,
                                           uint32_t n,
                                           uint32_t* out,
                                           float thres = 1e-3)
{
  uint32_t row = blockDim.x * blockIdx.x + threadIdx.x;

  int n_diffs = 0;
  if (row < m) {
    for (uint32_t i = 0; i < n; i++) {
      value_t d    = actual[row * n + i] - expected[row * n + i];
      bool matches = (fabsf(d) <= thres) || (actual_idx[row * n + i] == expected_idx[row * n + i] &&
                                             actual_idx[row * n + i] == row);

      if (!matches) {
        printf(
          "row=%ud, n=%ud, actual_dist=%f, actual_ind=%ld, expected_dist=%f, expected_ind=%ld\n",
          row,
          i,
          actual[row * n + i],
          actual_idx[row * n + i],
          expected[row * n + i],
          expected_idx[row * n + i]);
      }
      n_diffs += !matches;
      out[row] = n_diffs;
    }
  }
}

struct is_nonzero {
  __host__ __device__ bool operator()(uint32_t& i) { return i > 0; }
};

template <typename value_idx, typename value_t>
uint32_t count_discrepancies(value_idx* actual_idx,
                             value_idx* expected_idx,
                             value_t* actual,
                             value_t* expected,
                             uint32_t m,
                             uint32_t n,
                             uint32_t* out,
                             hipStream_t stream)
{
  uint32_t tpb = 256;
  count_discrepancies_kernel<<<raft::ceildiv(m, tpb), tpb, 0, stream>>>(
    actual_idx, expected_idx, actual, expected, m, n, out);

  auto exec_policy = rmm::exec_policy(stream);

  uint32_t result = thrust::count_if(exec_policy, out, out + m, is_nonzero());
  return result;
}

template <typename value_t>
void compute_bfknn(const raft::handle_t& handle,
                   const value_t* X1,
                   const value_t* X2,
                   uint32_t n_rows,
                   uint32_t n_query_rows,
                   uint32_t d,
                   uint32_t k,
                   const raft::distance::DistanceType metric,
                   value_t* dists,
                   int64_t* inds)
{
  std::vector<value_t*> input_vec = {const_cast<value_t*>(X1)};
  std::vector<uint32_t> sizes_vec = {n_rows};

  std::vector<int64_t>* translations = nullptr;

  raft::spatial::knn::detail::brute_force_knn_impl<uint32_t, int64_t>(handle,
                                                                      input_vec,
                                                                      sizes_vec,
                                                                      d,
                                                                      const_cast<value_t*>(X2),
                                                                      n_query_rows,
                                                                      inds,
                                                                      dists,
                                                                      k,
                                                                      true,
                                                                      true,
                                                                      translations,
                                                                      metric);
}

struct ToRadians {
  __device__ __host__ float operator()(float a) { return a * (HIP_PI_F / 180.0); }
};

struct BallCoverInputs {
  uint32_t k;
  uint32_t n_rows;
  uint32_t n_cols;
  float weight;
  uint32_t n_query;
  raft::distance::DistanceType metric;
};

template <typename value_idx, typename value_t>
class BallCoverKNNQueryTest : public ::testing::TestWithParam<BallCoverInputs> {
 protected:
  void basicTest()
  {
    params = ::testing::TestWithParam<BallCoverInputs>::GetParam();
    raft::handle_t handle;

    uint32_t k         = params.k;
    uint32_t n_centers = 25;
    float weight       = params.weight;
    auto metric        = params.metric;

    rmm::device_uvector<value_t> X(params.n_rows * params.n_cols, handle.get_stream());
    rmm::device_uvector<uint32_t> Y(params.n_rows, handle.get_stream());

    // Make sure the train and query sets are completely disjoint
    rmm::device_uvector<value_t> X2(params.n_query * params.n_cols, handle.get_stream());
    rmm::device_uvector<uint32_t> Y2(params.n_query, handle.get_stream());

    raft::random::make_blobs(
      X.data(), Y.data(), params.n_rows, params.n_cols, n_centers, handle.get_stream());

    raft::random::make_blobs(
      X2.data(), Y2.data(), params.n_query, params.n_cols, n_centers, handle.get_stream());

    rmm::device_uvector<value_idx> d_ref_I(params.n_query * k, handle.get_stream());
    rmm::device_uvector<value_t> d_ref_D(params.n_query * k, handle.get_stream());

    if (metric == raft::distance::DistanceType::Haversine) {
      thrust::transform(
        handle.get_thrust_policy(), X.data(), X.data() + X.size(), X.data(), ToRadians());
      thrust::transform(
        handle.get_thrust_policy(), X2.data(), X2.data() + X2.size(), X2.data(), ToRadians());
    }

    compute_bfknn(handle,
                  X.data(),
                  X2.data(),
                  params.n_rows,
                  params.n_query,
                  params.n_cols,
                  k,
                  metric,
                  d_ref_D.data(),
                  d_ref_I.data());

    handle.sync_stream();

    // Allocate predicted arrays
    rmm::device_uvector<value_idx> d_pred_I(params.n_query * k, handle.get_stream());
    rmm::device_uvector<value_t> d_pred_D(params.n_query * k, handle.get_stream());

    BallCoverIndex<value_idx, value_t> index(
      handle, X.data(), params.n_rows, params.n_cols, metric);

    raft::spatial::knn::rbc_build_index(handle, index);
    raft::spatial::knn::rbc_knn_query(
      handle, index, k, X2.data(), params.n_query, d_pred_I.data(), d_pred_D.data(), true, weight);

    handle.sync_stream();
    // What we really want are for the distances to match exactly. The
    // indices may or may not match exactly, depending upon the ordering which
    // can be nondeterministic.

    rmm::device_uvector<uint32_t> discrepancies(params.n_query, handle.get_stream());
    thrust::fill(handle.get_thrust_policy(),
                 discrepancies.data(),
                 discrepancies.data() + discrepancies.size(),
                 0);
    //
    int res = count_discrepancies(d_ref_I.data(),
                                  d_pred_I.data(),
                                  d_ref_D.data(),
                                  d_pred_D.data(),
                                  params.n_query,
                                  k,
                                  discrepancies.data(),
                                  handle.get_stream());

    ASSERT_TRUE(res == 0);
  }

  void SetUp() override {}

  void TearDown() override {}

 protected:
  uint32_t d = 2;
  BallCoverInputs params;
};

template <typename value_idx, typename value_t>
class BallCoverAllKNNTest : public ::testing::TestWithParam<BallCoverInputs> {
 protected:
  void basicTest()
  {
    params = ::testing::TestWithParam<BallCoverInputs>::GetParam();
    raft::handle_t handle;

    uint32_t k         = params.k;
    uint32_t n_centers = 25;
    float weight       = params.weight;
    auto metric        = params.metric;

    rmm::device_uvector<value_t> X(params.n_rows * params.n_cols, handle.get_stream());
    rmm::device_uvector<uint32_t> Y(params.n_rows, handle.get_stream());

    raft::random::make_blobs(
      X.data(), Y.data(), params.n_rows, params.n_cols, n_centers, handle.get_stream());

    rmm::device_uvector<value_idx> d_ref_I(params.n_rows * k, handle.get_stream());
    rmm::device_uvector<value_t> d_ref_D(params.n_rows * k, handle.get_stream());

    if (metric == raft::distance::DistanceType::Haversine) {
      thrust::transform(
        handle.get_thrust_policy(), X.data(), X.data() + X.size(), X.data(), ToRadians());
    }

    compute_bfknn(handle,
                  X.data(),
                  X.data(),
                  params.n_rows,
                  params.n_rows,
                  params.n_cols,
                  k,
                  metric,
                  d_ref_D.data(),
                  d_ref_I.data());

    handle.sync_stream();

    // Allocate predicted arrays
    rmm::device_uvector<value_idx> d_pred_I(params.n_rows * k, handle.get_stream());
    rmm::device_uvector<value_t> d_pred_D(params.n_rows * k, handle.get_stream());

    BallCoverIndex<value_idx, value_t> index(
      handle, X.data(), params.n_rows, params.n_cols, metric);

    raft::spatial::knn::rbc_all_knn_query(
      handle, index, k, d_pred_I.data(), d_pred_D.data(), true, weight);

    handle.sync_stream();
    // What we really want are for the distances to match exactly. The
    // indices may or may not match exactly, depending upon the ordering which
    // can be nondeterministic.

    rmm::device_uvector<uint32_t> discrepancies(params.n_rows, handle.get_stream());
    thrust::fill(handle.get_thrust_policy(),
                 discrepancies.data(),
                 discrepancies.data() + discrepancies.size(),
                 0);
    //
    uint32_t res = count_discrepancies(d_ref_I.data(),
                                       d_pred_I.data(),
                                       d_ref_D.data(),
                                       d_pred_D.data(),
                                       params.n_rows,
                                       k,
                                       discrepancies.data(),
                                       handle.get_stream());

    // TODO: There seem to be discrepancies here only when
    // the entire test suite is executed.
    // Ref: https://github.com/rapidsai/raft/issues/
    // 1-5 mismatches in 8000 samples is 0.0125% - 0.0625%
    ASSERT_TRUE(res <= 5);
  }

  void SetUp() override {}

  void TearDown() override {}

 protected:
  BallCoverInputs params;
};

typedef BallCoverAllKNNTest<int64_t, float> BallCoverAllKNNTestF;
typedef BallCoverKNNQueryTest<int64_t, float> BallCoverKNNQueryTestF;

const std::vector<BallCoverInputs> ballcover_inputs = {
  {11, 5000, 2, 1.0, 10000, raft::distance::DistanceType::Haversine},
  {25, 10000, 2, 1.0, 5000, raft::distance::DistanceType::Haversine},
  {2, 10000, 2, 1.0, 5000, raft::distance::DistanceType::L2SqrtUnexpanded},
  {2, 5000, 2, 1.0, 10000, raft::distance::DistanceType::Haversine},
  {11, 10000, 2, 1.0, 5000, raft::distance::DistanceType::L2SqrtUnexpanded},
  {25, 5000, 2, 1.0, 10000, raft::distance::DistanceType::L2SqrtUnexpanded},
  {5, 8000, 3, 1.0, 10000, raft::distance::DistanceType::L2SqrtUnexpanded},
  {11, 6000, 3, 1.0, 10000, raft::distance::DistanceType::L2SqrtUnexpanded},
  {25, 10000, 3, 1.0, 5000, raft::distance::DistanceType::L2SqrtUnexpanded}};

INSTANTIATE_TEST_CASE_P(BallCoverAllKNNTest,
                        BallCoverAllKNNTestF,
                        ::testing::ValuesIn(ballcover_inputs));
INSTANTIATE_TEST_CASE_P(BallCoverKNNQueryTest,
                        BallCoverKNNQueryTestF,
                        ::testing::ValuesIn(ballcover_inputs));

TEST_P(BallCoverAllKNNTestF, Fit) { basicTest(); }
TEST_P(BallCoverKNNQueryTestF, Fit) { basicTest(); }

}  // namespace knn
}  // namespace spatial
}  // namespace raft
