#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "spatial_data.h"
#include <raft/cudart_utils.h>
#include <raft/linalg/distance_type.h>
#include <raft/spatial/knn/ball_cover.hpp>
#include <raft/spatial/knn/detail/knn_brute_force_faiss.cuh>
#if defined RAFT_NN_COMPILED
#include <raft/spatial/knn/specializations.hpp>
#endif

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <thrust/transform.h>

#include <cstdint>
#include <gtest/gtest.h>
#include <iostream>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {

using namespace std;

template <typename value_idx, typename value_t>
__global__ void count_discrepancies_kernel(value_idx* actual_idx,
                                           value_idx* expected_idx,
                                           value_t* actual,
                                           value_t* expected,
                                           uint32_t m,
                                           uint32_t n,
                                           uint32_t* out,
                                           float thres = 1e-3)
{
  uint32_t row = blockDim.x * blockIdx.x + threadIdx.x;

  int n_diffs = 0;
  if (row < m) {
    for (uint32_t i = 0; i < n; i++) {
      value_t d    = actual[row * n + i] - expected[row * n + i];
      bool matches = (fabsf(d) <= thres) || (actual_idx[row * n + i] == expected_idx[row * n + i] &&
                                             actual_idx[row * n + i] == row);
      //      if (!matches) {
      //                  printf("row=%d, actual_idx=%ld, actual=%f, expected_id=%ld,
      //                  expected=%f\n",
      //                         row, actual_idx[row*n+i], actual[row*n+i], expected_idx[row*n+i],
      //                         expected[row*n+i]);
      //      }

      n_diffs += !matches;
      out[row] = n_diffs;
    }
  }
}

struct is_nonzero {
  __host__ __device__ bool operator()(uint32_t& i) { return i > 0; }
};

template <typename value_idx, typename value_t>
uint32_t count_discrepancies(value_idx* actual_idx,
                             value_idx* expected_idx,
                             value_t* actual,
                             value_t* expected,
                             uint32_t m,
                             uint32_t n,
                             uint32_t* out,
                             hipStream_t stream)
{
  uint32_t tpb = 256;
  count_discrepancies_kernel<<<raft::ceildiv(m, tpb), tpb, 0, stream>>>(
    actual_idx, expected_idx, actual, expected, m, n, out);

  auto exec_policy = rmm::exec_policy(stream);

  uint32_t result = thrust::count_if(exec_policy, out, out + m, is_nonzero());
  return result;
}

template <typename value_t>
void compute_bfknn(const raft::handle_t& handle,
                   const value_t* X1,
                   const value_t* X2,
                   uint32_t n,
                   uint32_t d,
                   uint32_t k,
                   const raft::distance::DistanceType metric,
                   value_t* dists,
                   int64_t* inds)
{
  std::vector<value_t*> input_vec = {const_cast<value_t*>(X1)};
  std::vector<uint32_t> sizes_vec = {n};

  std::vector<int64_t>* translations = nullptr;

  raft::spatial::knn::detail::brute_force_knn_impl<uint32_t, int64_t>(handle,
                                                                      input_vec,
                                                                      sizes_vec,
                                                                      d,
                                                                      const_cast<value_t*>(X2),
                                                                      n,
                                                                      inds,
                                                                      dists,
                                                                      k,
                                                                      true,
                                                                      true,
                                                                      translations,
                                                                      metric);
}

struct ToRadians {
  __device__ __host__ float operator()(float a) { return a * (HIP_PI_F / 180.0); }
};

struct BallCoverInputs {
  uint32_t k;
  float weight;
  raft::distance::DistanceType metric;
};

template <typename value_idx, typename value_t>
class BallCoverKNNQueryTest : public ::testing::TestWithParam<BallCoverInputs> {
 protected:
  void basicTest()
  {
    params = ::testing::TestWithParam<BallCoverInputs>::GetParam();
    raft::handle_t handle;

    uint32_t k   = params.k;
    float weight = params.weight;
    auto metric  = params.metric;

    std::vector<value_t> h_train_inputs = spatial_data;

    uint32_t n = h_train_inputs.size() / d;

    rmm::device_uvector<value_idx> d_ref_I(n * k, handle.get_stream());
    rmm::device_uvector<value_t> d_ref_D(n * k, handle.get_stream());

    // Allocate input
    rmm::device_uvector<value_t> d_train_inputs(n * d, handle.get_stream());
    raft::update_device(d_train_inputs.data(), h_train_inputs.data(), n * d, handle.get_stream());

    if (metric == raft::distance::DistanceType::Haversine) {
      thrust::transform(handle.get_thrust_policy(),
                        d_train_inputs.data(),
                        d_train_inputs.data() + d_train_inputs.size(),
                        d_train_inputs.data(),
                        ToRadians());
    }

    compute_bfknn(handle,
                  d_train_inputs.data(),
                  d_train_inputs.data(),
                  n,
                  d,
                  k,
                  metric,
                  d_ref_D.data(),
                  d_ref_I.data());

    RAFT_CUDA_TRY(hipStreamSynchronize(handle.get_stream()));

    // Allocate predicted arrays
    rmm::device_uvector<value_idx> d_pred_I(n * k, handle.get_stream());
    rmm::device_uvector<value_t> d_pred_D(n * k, handle.get_stream());

    BallCoverIndex<value_idx, value_t> index(handle, d_train_inputs.data(), n, d, metric);

    raft::spatial::knn::rbc_build_index(handle, index);
    raft::spatial::knn::rbc_knn_query(
      handle, index, k, d_train_inputs.data(), n, d_pred_I.data(), d_pred_D.data(), true, weight);

    RAFT_CUDA_TRY(hipStreamSynchronize(handle.get_stream()));
    // What we really want are for the distances to match exactly. The
    // indices may or may not match exactly, depending upon the ordering which
    // can be nondeterministic.

    rmm::device_uvector<uint32_t> discrepancies(n, handle.get_stream());
    thrust::fill(handle.get_thrust_policy(),
                 discrepancies.data(),
                 discrepancies.data() + discrepancies.size(),
                 0);
    //
    int res = count_discrepancies(d_ref_I.data(),
                                  d_pred_I.data(),
                                  d_ref_D.data(),
                                  d_pred_D.data(),
                                  n,
                                  k,
                                  discrepancies.data(),
                                  handle.get_stream());

    ASSERT_TRUE(res == 0);
  }

  void SetUp() override {}

  void TearDown() override {}

 protected:
  uint32_t d = 2;
  BallCoverInputs params;
};

template <typename value_idx, typename value_t>
class BallCoverAllKNNTest : public ::testing::TestWithParam<BallCoverInputs> {
 protected:
  void basicTest()
  {
    params = ::testing::TestWithParam<BallCoverInputs>::GetParam();
    raft::handle_t handle;

    uint32_t k   = params.k;
    float weight = params.weight;
    auto metric  = params.metric;

    std::vector<value_t> h_train_inputs = spatial_data;

    uint32_t n = h_train_inputs.size() / d;

    rmm::device_uvector<value_idx> d_ref_I(n * k, handle.get_stream());
    rmm::device_uvector<value_t> d_ref_D(n * k, handle.get_stream());

    // Allocate input
    rmm::device_uvector<value_t> d_train_inputs(n * d, handle.get_stream());
    raft::update_device(d_train_inputs.data(), h_train_inputs.data(), n * d, handle.get_stream());

    if (metric == raft::distance::DistanceType::Haversine) {
      thrust::transform(handle.get_thrust_policy(),
                        d_train_inputs.data(),
                        d_train_inputs.data() + d_train_inputs.size(),
                        d_train_inputs.data(),
                        ToRadians());
    }

    std::vector<int64_t>* translations = nullptr;

    std::vector<float*> input_vec   = {d_train_inputs.data()};
    std::vector<uint32_t> sizes_vec = {n};

    raft::spatial::knn::detail::brute_force_knn_impl<uint32_t, int64_t>(handle,
                                                                        input_vec,
                                                                        sizes_vec,
                                                                        d,
                                                                        d_train_inputs.data(),
                                                                        n,
                                                                        d_ref_I.data(),
                                                                        d_ref_D.data(),
                                                                        k,
                                                                        true,
                                                                        true,
                                                                        translations,
                                                                        metric);

    RAFT_CUDA_TRY(hipStreamSynchronize(handle.get_stream()));

    // Allocate predicted arrays
    rmm::device_uvector<value_idx> d_pred_I(n * k, handle.get_stream());
    rmm::device_uvector<value_t> d_pred_D(n * k, handle.get_stream());

    BallCoverIndex<value_idx, value_t> index(handle, d_train_inputs.data(), n, d, metric);

    raft::spatial::knn::rbc_all_knn_query(
      handle, index, k, d_pred_I.data(), d_pred_D.data(), true, weight);

    RAFT_CUDA_TRY(hipStreamSynchronize(handle.get_stream()));
    // What we really want are for the distances to match exactly. The
    // indices may or may not match exactly, depending upon the ordering which
    // can be nondeterministic.

    rmm::device_uvector<uint32_t> discrepancies(n, handle.get_stream());
    thrust::fill(handle.get_thrust_policy(),
                 discrepancies.data(),
                 discrepancies.data() + discrepancies.size(),
                 0);
    //
    uint32_t res = count_discrepancies(d_ref_I.data(),
                                       d_pred_I.data(),
                                       d_ref_D.data(),
                                       d_pred_D.data(),
                                       n,
                                       k,
                                       discrepancies.data(),
                                       handle.get_stream());
    ASSERT_TRUE(res == 0);
  }

  void SetUp() override {}

  void TearDown() override {}

 protected:
  uint32_t d = 2;
  BallCoverInputs params;
};

typedef BallCoverAllKNNTest<int64_t, float> BallCoverAllKNNTestF;
typedef BallCoverKNNQueryTest<int64_t, float> BallCoverKNNQueryTestF;

const std::vector<BallCoverInputs> ballcover_inputs = {
  {2, 1.0, raft::distance::DistanceType::Haversine},
  {4, 1.0, raft::distance::DistanceType::Haversine},
  {7, 1.0, raft::distance::DistanceType::Haversine},
  {2, 1.0, raft::distance::DistanceType::L2SqrtUnexpanded},
  {4, 1.0, raft::distance::DistanceType::L2SqrtUnexpanded},
  {7, 1.0, raft::distance::DistanceType::L2SqrtUnexpanded},
};

INSTANTIATE_TEST_CASE_P(BallCoverAllKNNTest,
                        BallCoverAllKNNTestF,
                        ::testing::ValuesIn(ballcover_inputs));
INSTANTIATE_TEST_CASE_P(BallCoverKNNQueryTest,
                        BallCoverKNNQueryTestF,
                        ::testing::ValuesIn(ballcover_inputs));

TEST_P(BallCoverAllKNNTestF, Fit) { basicTest(); }
TEST_P(BallCoverKNNQueryTestF, Fit) { basicTest(); }

}  // namespace knn
}  // namespace spatial
}  // namespace raft
