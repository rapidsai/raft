#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "./ann_base_kernel.cuh"

#include <raft/core/logger.hpp>
#include <raft/distance/distance_type.hpp>
#include <raft/random/rng.cuh>
#include <raft/sparse/detail/utils.h>
#include <raft/spatial/knn/ivf_pq.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/mr/device/managed_memory_resource.hpp>

#include <gtest/gtest.h>

#include <cstddef>
#include <iostream>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {
struct IvfPqInputs {
  int num_queries;
  int num_db_vecs;
  int dim;
  int k;
  int nprobe;
  int nlist;
  raft::distance::DistanceType metric;
};

template <typename IdxT, typename DistT, typename compareDist>
struct idx_dist_pair {
  IdxT idx;
  DistT dist;
  compareDist eq_compare;
  bool operator==(const idx_dist_pair<IdxT, DistT, compareDist>& a) const
  {
    if (idx == a.idx) return true;
    if (eq_compare(dist, a.dist)) return true;
    return false;
  }
  idx_dist_pair(IdxT x, DistT y, compareDist op) : idx(x), dist(y), eq_compare(op) {}
};

template <typename T, typename DistT>
auto eval_knn(const std::vector<T>& expected_idx,
              const std::vector<T>& actual_idx,
              const std::vector<DistT>& expected_dist,
              const std::vector<DistT>& actual_dist,
              size_t rows,
              size_t cols,
              const DistT eps,
              double min_recall) -> testing::AssertionResult
{
  size_t match_count = 0;
  size_t total_count = static_cast<size_t>(rows) * static_cast<size_t>(cols);
  for (size_t i = 0; i < rows; ++i) {
    for (size_t k = 0; k < cols; ++k) {
      size_t idx_k  = i * cols + k;  // row major assumption!
      auto act_idx  = actual_idx[idx_k];
      auto act_dist = actual_dist[idx_k];
      for (size_t j = 0; j < cols; ++j) {
        size_t idx    = i * cols + j;  // row major assumption!
        auto exp_idx  = expected_idx[idx];
        auto exp_dist = expected_dist[idx];
        idx_dist_pair exp_kvp(exp_idx, exp_dist, raft::CompareApprox<DistT>(eps));
        idx_dist_pair act_kvp(act_idx, act_dist, raft::CompareApprox<DistT>(eps));
        if (exp_kvp == act_kvp) {
          match_count++;
          break;
        }
      }
    }
  }
  RAFT_LOG_INFO("Recall = %zu/%zu", match_count, total_count);
  double actual_recall = static_cast<double>(match_count) / static_cast<double>(total_count);
  if (actual_recall < min_recall - eps) {
    if (actual_recall < min_recall * min_recall - eps) {
      RAFT_LOG_ERROR("Recall is much lower than the minimum (%f < %f)", actual_recall, min_recall);
    } else {
      RAFT_LOG_WARN("Recall is suspiciously too low (%f < %f)", actual_recall, min_recall);
    }
    if (match_count == 0 || actual_recall < min_recall * std::min(min_recall, 0.5) - eps) {
      return testing::AssertionFailure()
             << "actual recall (" << actual_recall
             << ") is much smaller than the minimum expected recall (" << min_recall << ").";
    }
  }
  return testing::AssertionSuccess();
}

#define CUANN_CHECK(ret) RAFT_EXPECTS(ret == ivf_pq::CUANN_STATUS_SUCCESS, "cuann failure: %d", ret)

template <typename T, typename DataT>
class IvfPqTest : public ::testing::TestWithParam<IvfPqInputs> {
 public:
  IvfPqTest()
    : stream_(handle_.get_stream()),
      ps(::testing::TestWithParam<IvfPqInputs>::GetParam()),
      database(0, stream_, &managed_memory),
      search_queries(0, stream_)
  {
  }

 protected:
  void testIvfPq()
  {
    size_t queries_size = ps.num_queries * ps.k;
    std::vector<int64_t> indices_ivf_pq(queries_size);
    std::vector<int64_t> indices_naive(queries_size);
    std::vector<T> distances_ivf_pq(queries_size);
    std::vector<T> distances_naive(queries_size);

    {
      rmm::device_uvector<T> distances_naive_dev(queries_size, stream_);
      rmm::device_uvector<int64_t> indices_naive_dev(queries_size, stream_);
      using acc_t = typename detail::utils::config<DataT>::value_t;
      naiveBfKnn<DataT, acc_t>(distances_naive_dev.data(),
                               indices_naive_dev.data(),
                               search_queries.data(),
                               database.data(),
                               ps.num_queries,
                               ps.num_db_vecs,
                               ps.dim,
                               ps.k,
                               ps.metric,
                               2.0f,
                               stream_);
      update_host(distances_naive.data(), distances_naive_dev.data(), queries_size, stream_);
      update_host(indices_naive.data(), indices_naive_dev.data(), queries_size, stream_);
      handle_.sync_stream(stream_);
    }

    {
      // unless something is really wrong with clustering, this could serve as a lower bound on
      // recall
      double min_recall = static_cast<double>(ps.nprobe) / static_cast<double>(ps.nlist);

      rmm::device_uvector<T> distances_ivf_pq_dev(queries_size, stream_);
      rmm::device_uvector<uint64_t> indices_ivf_pq_dev(queries_size, stream_);

      {
        std::unique_ptr<ivf_pq::cuannContext, std::function<void(ivf_pq::cuannHandle_t)>>
          cuann_handle{[]() {
                         ivf_pq::cuannHandle_t h;
                         CUANN_CHECK(ivf_pq::cuannCreate(&h));
                         return h;
                       }(),
                       [](ivf_pq::cuannHandle_t h) { ivf_pq::cuannDestroy(h); }};
        std::unique_ptr<ivf_pq::cuannIvfPqDescriptor,
                        std::function<void(ivf_pq::cuannIvfPqDescriptor_t)>>
          cuann_desc{
            []() {
              ivf_pq::cuannIvfPqDescriptor_t d;
              CUANN_CHECK(ivf_pq::cuannIvfPqCreateDescriptor(&d));
              return d;
            }(),
            [](ivf_pq::cuannIvfPqDescriptor_t d) { ivf_pq::cuannIvfPqDestroyDescriptor(d); }};

        CUANN_CHECK(ivf_pq::cuannSetDevice(cuann_handle.get(), handle_.get_device()));
        CUANN_CHECK(ivf_pq::cuannSetStream(cuann_handle.get(), handle_.get_stream()));

        // Number of kmeans clusters.
        //
        // The number of vectors per cluster, or 'numDataset' / 'numClusters',
        // should be approximately 1,000 to 10,000.
        uint32_t n_clusters = ps.nlist;
        // Important parameters of the index to create.
        //
        // 'bitPq' is the bit length of the vector element after compression by PQ.
        // 'dimPq' is the dimensionality of the vector after compression by PQ.
        //
        // 'bitPq' is 4, 5, 6, 7, or 8. The smaller the 'bitPq', the smaller the
        // index size and the better the search performance, but the lower the recall.
        //
        // Similarly, a smaller 'dimPq' results in a smaller index size and better
        // search performance, but lower recall. If 'bitPq' is 8, 'dimPq' can be set
        // to any number, but multiple of 8 are desirable for good performance.
        // If 'bitPq' is not 8, 'dimPq' must be basically multiple of 8. For good
        // performance, multiple 32 is desirable.
        //
        uint32_t bitPq = 8;
        uint32_t dimPq = ps.dim;
        if (dimPq >= 128) {
          dimPq = raft::alignDown<uint32_t>(dimPq / 2, 32);
        } else if (dimPq >= 32) {
          dimPq = raft::alignDown<uint32_t>(dimPq, 32);
        } else if (dimPq >= 8) {
          dimPq = raft::alignDown<uint32_t>(dimPq, 8);
        }
        // If true, dataset and query vectors are rotated by random rotation matrix
        // created at indexing time.
        //
        bool randomRotation = ps.dim < 1024;  // disable for large-dimensional data (CPU intensive)
        // Number of iterations for kmeans training.
        uint32_t numIterations = 25;
        // metric
        ivf_pq::cuannSimilarity_t similarity =
          ps.metric == raft::distance::DistanceType::InnerProduct ? ivf_pq::CUANN_SIMILARITY_INNER
                                                                  : ivf_pq::CUANN_SIMILARITY_L2;
        // Specify whether PQ codebooks are created per subspace or per cluster.
        ivf_pq::cuannPqCenter_t typePqCenter = ivf_pq::CUANN_PQ_CENTER_PER_SUBSPACE;
        CUANN_CHECK(ivf_pq::cuannIvfPqSetIndexParameters(
          cuann_desc.get(),
          n_clusters,               /* Number of clusters */
          uint32_t(ps.num_db_vecs), /* Number of dataset entries */
          uint32_t(ps.dim),         /* Dimension of each entry */
          dimPq,                    /* Dimension of each entry after product quantization */
          bitPq,                    /* Bit length of PQ */
          similarity,
          typePqCenter));

        // Allocate memory for index
        size_t ivf_pq_index_size;
        CUANN_CHECK(ivf_pq::cuannIvfPqGetIndexSize(cuann_desc.get(), &ivf_pq_index_size));
        rmm::device_buffer ivf_pq_index_buf_managed(ivf_pq_index_size, stream_, &managed_memory);

        // Build index
        hipDataType dtype;
        if constexpr (std::is_same_v<DataT, uint8_t>) {
          dtype = HIP_R_8U;
        } else if constexpr (std::is_same_v<DataT, int8_t>) {
          dtype = HIP_R_8I;
        } else if constexpr (std::is_same_v<DataT, float>) {
          dtype = HIP_R_32F;
        }
        CUANN_CHECK(ivf_pq::cuannIvfPqBuildIndex(
          cuann_handle.get(),
          cuann_desc.get(),
          database.data(),  // dataset
          database.data(),  // ?kmeans? trainset
          dtype,
          uint32_t(ps.num_db_vecs),  // size of the trainset (I guess for kmeans)
          numIterations,
          randomRotation,
          true,                            // hierarchialClustering: always true in raft
          ivf_pq_index_buf_managed.data()  // memory allocated for the index
          ));
        handle_.sync_stream(stream_);

        // set search parameters
        CUANN_CHECK(ivf_pq::cuannIvfPqSetSearchParameters(cuann_desc.get(), ps.nprobe, ps.k));
        // Data type of LUT to be created dynamically at search time.
        //
        // The use of low-precision types reduces the amount of shared memory
        // required at search time, so fast shared memory kernels can be used even
        // for datasets with large dimansionality. Note that the recall is slightly
        // degraded when low-precision type is selected.
        //
        hipDataType smemLutDtype = HIP_R_32F;
        // smemLutDtype = HIP_R_16F;
        // smemLutDtype = HIP_R_8U;
        // Storage data type for distance/similarity computed at search time.
        //
        // If the performance limiter at search time is device memory access,
        // selecting FP16 will improve performance slightly.
        //
        hipDataType internalDistanceDtype = HIP_R_32F;
        // internalDistanceDtype = HIP_R_16F;

        // Thread block size of the distance calculation kernel at search time.
        //
        // If 0, the thread block size is determined automatically.
        //
        uint32_t preferredThreadBlockSize = 0;  // 0, 256, 512, or 1024
        CUANN_CHECK(ivf_pq::cuannIvfPqSetSearchTuningParameters(
          cuann_desc.get(), internalDistanceDtype, smemLutDtype, preferredThreadBlockSize));
        // Maximum number of query vectors to search.
        uint32_t maxQueries = 1000000;
        // Maximum number of query vectors to search at the same time.
        uint32_t batchSize = maxQueries;
        // Maximum device memory size that may be used as workspace at search time.
        // maxSearchWorkspaceSize = 0;  // default
        size_t maxSearchWorkspaceSize = (size_t)2 * 1024 * 1024 * 1024;  // 2 GiB

        // Allocate memory for index
        size_t ivf_pq_search_workspace_size;
        CUANN_CHECK(ivf_pq::cuannIvfPqSearch_bufferSize(cuann_handle.get(),
                                                        cuann_desc.get(),
                                                        ivf_pq_index_buf_managed.data(),
                                                        batchSize,
                                                        maxSearchWorkspaceSize,
                                                        &ivf_pq_search_workspace_size));
        rmm::device_buffer ivf_pq_search_ws_buf(ivf_pq_search_workspace_size, stream_);

        // finally, search!
        CUANN_CHECK(cuannIvfPqSearch(cuann_handle.get(),
                                     cuann_desc.get(),
                                     ivf_pq_index_buf_managed.data(),
                                     search_queries.data(),
                                     dtype,
                                     ps.num_queries,
                                     indices_ivf_pq_dev.data(),
                                     distances_ivf_pq_dev.data(),
                                     ivf_pq_search_ws_buf.data()));
        handle_.sync_stream(stream_);

        update_host(distances_ivf_pq.data(), distances_ivf_pq_dev.data(), queries_size, stream_);
        update_host(indices_ivf_pq.data(),
                    reinterpret_cast<int64_t*>(indices_ivf_pq_dev.data()),
                    queries_size,
                    stream_);
        handle_.sync_stream(stream_);
      }
      handle_.sync_stream(stream_);
      ASSERT_TRUE(eval_knn(indices_naive,
                           indices_ivf_pq,
                           distances_naive,
                           distances_ivf_pq,
                           ps.num_queries,
                           ps.k,
                           float(0.001),
                           min_recall));
    }
  }

  void SetUp() override
  {
    database.resize(ps.num_db_vecs * ps.dim, stream_);
    search_queries.resize(ps.num_queries * ps.dim, stream_);

    raft::random::Rng r(1234ULL);
    if constexpr (std::is_same<DataT, float>{}) {
      r.uniform(database.data(), ps.num_db_vecs * ps.dim, DataT(0.1), DataT(2.0), stream_);
      r.uniform(search_queries.data(), ps.num_queries * ps.dim, DataT(0.1), DataT(2.0), stream_);
    } else {
      r.uniformInt(database.data(), ps.num_db_vecs * ps.dim, DataT(1), DataT(20), stream_);
      r.uniformInt(search_queries.data(), ps.num_queries * ps.dim, DataT(1), DataT(20), stream_);
    }
    handle_.sync_stream(stream_);
  }

  void TearDown() override
  {
    handle_.sync_stream(stream_);
    database.resize(0, stream_);
    search_queries.resize(0, stream_);
  }

 private:
  raft::handle_t handle_;
  rmm::cuda_stream_view stream_;
  rmm::mr::managed_memory_resource managed_memory;
  IvfPqInputs ps;
  rmm::device_uvector<DataT> database;
  rmm::device_uvector<DataT> search_queries;
};

const std::vector<IvfPqInputs> inputs = {
  // test various dims (aligned and not aligned to vector sizes)
  {1000, 10000, 1, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 3, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 4, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 5, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 8, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},

  // test dims that do not fit into kernel shared memory limits
  {1000, 10000, 2048, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2049, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2050, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2051, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2052, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2053, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2056, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},

  // various random combinations
  {1000, 10000, 16, 10, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, 50, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, 70, 1024, raft::distance::DistanceType::L2Expanded},
  {100, 10000, 16, 10, 20, 512, raft::distance::DistanceType::L2Expanded},
  {20, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},
  {10000, 131072, 8, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},

  {1000, 10000, 16, 10, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 16, 10, 50, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 16, 10, 70, 1024, raft::distance::DistanceType::InnerProduct},
  {100, 10000, 16, 10, 20, 512, raft::distance::DistanceType::InnerProduct},
  {20, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::InnerProduct},
  {10000, 131072, 8, 10, 50, 1024, raft::distance::DistanceType::InnerProduct},

  {1000, 10000, 4096, 20, 50, 1024, raft::distance::DistanceType::InnerProduct},

  // test splitting the big query batches  (> max gridDim.y) into smaller batches
  {100000, 1024, 32, 10, 64, 64, raft::distance::DistanceType::InnerProduct},
  {98306, 1024, 32, 10, 64, 64, raft::distance::DistanceType::InnerProduct},

  // test radix_sort for getting the cluster selection
  {1000,
   10000,
   16,
   10,
   raft::spatial::knn::detail::topk::kMaxCapacity * 2,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::distance::DistanceType::L2Expanded},
  {1000,
   10000,
   16,
   10,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::distance::DistanceType::InnerProduct}};

typedef IvfPqTest<float, float> IvfPqTestF;
TEST_P(IvfPqTestF, IvfPq) { this->testIvfPq(); }

INSTANTIATE_TEST_CASE_P(IvfPqTest, IvfPqTestF, ::testing::ValuesIn(inputs));

typedef IvfPqTest<float, uint8_t> IvfPqTestF_uint8;
TEST_P(IvfPqTestF_uint8, IvfPq) { this->testIvfPq(); }

INSTANTIATE_TEST_CASE_P(IvfPqTest, IvfPqTestF_uint8, ::testing::ValuesIn(inputs));

typedef IvfPqTest<float, int8_t> IvfPqTestF_int8;
TEST_P(IvfPqTestF_int8, IvfPq) { this->testIvfPq(); }

INSTANTIATE_TEST_CASE_P(IvfPqTest, IvfPqTestF_int8, ::testing::ValuesIn(inputs));

}  // namespace knn
}  // namespace spatial
}  // namespace raft
