#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "./ann_base_kernel.cuh"

#include <raft/core/logger.hpp>
#include <raft/distance/distance_type.hpp>
#include <raft/random/rng.cuh>
#include <raft/spatial/knn/ann.cuh>
#include <raft/spatial/knn/ivf_flat.cuh>
#include <raft/spatial/knn/knn.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>

#include <gtest/gtest.h>

#include <thrust/sequence.h>

#include <cstddef>
#include <iostream>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {
struct AnnIvfFlatInputs {
  int num_queries;
  int num_db_vecs;
  int dim;
  int k;
  int nprobe;
  int nlist;
  raft::distance::DistanceType metric;
};

template <typename IdxT, typename DistT, typename compareDist>
struct idx_dist_pair {
  IdxT idx;
  DistT dist;
  compareDist eq_compare;
  bool operator==(const idx_dist_pair<IdxT, DistT, compareDist>& a) const
  {
    if (idx == a.idx) return true;
    if (eq_compare(dist, a.dist)) return true;
    return false;
  }
  idx_dist_pair(IdxT x, DistT y, compareDist op) : idx(x), dist(y), eq_compare(op) {}
};

template <typename T, typename DistT>
auto eval_knn(const std::vector<T>& expected_idx,
              const std::vector<T>& actual_idx,
              const std::vector<DistT>& expected_dist,
              const std::vector<DistT>& actual_dist,
              size_t rows,
              size_t cols,
              const DistT eps,
              double min_recall) -> testing::AssertionResult
{
  size_t match_count = 0;
  size_t total_count = static_cast<size_t>(rows) * static_cast<size_t>(cols);
  for (size_t i = 0; i < rows; ++i) {
    for (size_t k = 0; k < cols; ++k) {
      size_t idx_k  = i * cols + k;  // row major assumption!
      auto act_idx  = actual_idx[idx_k];
      auto act_dist = actual_dist[idx_k];
      for (size_t j = 0; j < cols; ++j) {
        size_t idx    = i * cols + j;  // row major assumption!
        auto exp_idx  = expected_idx[idx];
        auto exp_dist = expected_dist[idx];
        idx_dist_pair exp_kvp(exp_idx, exp_dist, raft::CompareApprox<DistT>(eps));
        idx_dist_pair act_kvp(act_idx, act_dist, raft::CompareApprox<DistT>(eps));
        if (exp_kvp == act_kvp) {
          match_count++;
          break;
        }
      }
    }
  }
  RAFT_LOG_INFO("Recall = %zu/%zu", match_count, total_count);
  double actual_recall = static_cast<double>(match_count) / static_cast<double>(total_count);
  if (actual_recall < min_recall - eps) {
    if (actual_recall < min_recall * min_recall - eps) {
      RAFT_LOG_ERROR("Recall is much lower than the minimum (%f < %f)", actual_recall, min_recall);
    } else {
      RAFT_LOG_WARN("Recall is suspiciously too low (%f < %f)", actual_recall, min_recall);
    }
    if (match_count == 0 || actual_recall < min_recall * std::min(min_recall, 0.5) - eps) {
      return testing::AssertionFailure()
             << "actual recall (" << actual_recall
             << ") is much smaller than the minimum expected recall (" << min_recall << ").";
    }
  }
  return testing::AssertionSuccess();
}

template <typename T, typename DataT>
class AnnIVFFlatTest : public ::testing::TestWithParam<AnnIvfFlatInputs> {
 public:
  AnnIVFFlatTest()
    : stream_(handle_.get_stream()),
      ps(::testing::TestWithParam<AnnIvfFlatInputs>::GetParam()),
      database(0, stream_),
      search_queries(0, stream_)
  {
  }

 protected:
  void testIVFFlat()
  {
    size_t queries_size = ps.num_queries * ps.k;
    std::vector<int64_t> indices_ivfflat(queries_size);
    std::vector<int64_t> indices_naive(queries_size);
    std::vector<T> distances_ivfflat(queries_size);
    std::vector<T> distances_naive(queries_size);

    {
      rmm::device_uvector<T> distances_naive_dev(queries_size, stream_);
      rmm::device_uvector<int64_t> indices_naive_dev(queries_size, stream_);
      using acc_t = typename detail::utils::config<DataT>::value_t;
      naiveBfKnn<DataT, acc_t>(distances_naive_dev.data(),
                               indices_naive_dev.data(),
                               search_queries.data(),
                               database.data(),
                               ps.num_queries,
                               ps.num_db_vecs,
                               ps.dim,
                               ps.k,
                               ps.metric,
                               2.0f,
                               stream_);
      update_host(distances_naive.data(), distances_naive_dev.data(), queries_size, stream_);
      update_host(indices_naive.data(), indices_naive_dev.data(), queries_size, stream_);
      handle_.sync_stream(stream_);
    }

    {
      // unless something is really wrong with clustering, this could serve as a lower bound on
      // recall
      double min_recall = static_cast<double>(ps.nprobe) / static_cast<double>(ps.nlist);

      rmm::device_uvector<T> distances_ivfflat_dev(queries_size, stream_);
      rmm::device_uvector<int64_t> indices_ivfflat_dev(queries_size, stream_);

      {
        // legacy interface
        raft::spatial::knn::IVFFlatParam ivfParams;
        ivfParams.nprobe = ps.nprobe;
        ivfParams.nlist  = ps.nlist;
        raft::spatial::knn::knnIndex index;
        index.index   = nullptr;
        index.gpu_res = nullptr;

        approx_knn_build_index(handle_,
                               &index,
                               dynamic_cast<raft::spatial::knn::knnIndexParam*>(&ivfParams),
                               ps.metric,
                               0,
                               database.data(),
                               ps.num_db_vecs,
                               ps.dim);
        handle_.sync_stream(stream_);
        approx_knn_search(handle_,
                          distances_ivfflat_dev.data(),
                          indices_ivfflat_dev.data(),
                          &index,
                          ps.k,
                          search_queries.data(),
                          ps.num_queries);

        update_host(distances_ivfflat.data(), distances_ivfflat_dev.data(), queries_size, stream_);
        update_host(indices_ivfflat.data(), indices_ivfflat_dev.data(), queries_size, stream_);
        handle_.sync_stream(stream_);
      }

      ASSERT_TRUE(eval_knn(indices_naive,
                           indices_ivfflat,
                           distances_naive,
                           distances_ivfflat,
                           ps.num_queries,
                           ps.k,
                           float(0.001),
                           min_recall));
      {
        // new interface
        raft::spatial::knn::ivf_flat::index_params index_params;
        raft::spatial::knn::ivf_flat::search_params search_params;
        index_params.n_lists   = ps.nlist;
        index_params.metric    = ps.metric;
        search_params.n_probes = ps.nprobe;

        index_params.add_data_on_build        = false;
        index_params.kmeans_trainset_fraction = 0.5;
        auto index =
          ivf_flat::build(handle_, index_params, database.data(), int64_t(ps.num_db_vecs), ps.dim);

        rmm::device_uvector<int64_t> vector_indices(ps.num_db_vecs, stream_);
        thrust::sequence(handle_.get_thrust_policy(),
                         thrust::device_pointer_cast(vector_indices.data()),
                         thrust::device_pointer_cast(vector_indices.data() + ps.num_db_vecs));
        handle_.sync_stream(stream_);

        int64_t half_of_data = ps.num_db_vecs / 2;

        auto index_2 =
          ivf_flat::extend<DataT, int64_t>(handle_, index, database.data(), nullptr, half_of_data);

        ivf_flat::extend<DataT, int64_t>(handle_,
                                         &index_2,
                                         database.data() + half_of_data * ps.dim,
                                         vector_indices.data() + half_of_data,
                                         int64_t(ps.num_db_vecs) - half_of_data);

        ivf_flat::search(handle_,
                         search_params,
                         index_2,
                         search_queries.data(),
                         ps.num_queries,
                         ps.k,
                         indices_ivfflat_dev.data(),
                         distances_ivfflat_dev.data());

        update_host(distances_ivfflat.data(), distances_ivfflat_dev.data(), queries_size, stream_);
        update_host(indices_ivfflat.data(), indices_ivfflat_dev.data(), queries_size, stream_);
        handle_.sync_stream(stream_);
      }
      ASSERT_TRUE(eval_knn(indices_naive,
                           indices_ivfflat,
                           distances_naive,
                           distances_ivfflat,
                           ps.num_queries,
                           ps.k,
                           float(0.001),
                           min_recall));
    }
  }

  void SetUp() override
  {
    database.resize(ps.num_db_vecs * ps.dim, stream_);
    search_queries.resize(ps.num_queries * ps.dim, stream_);

    raft::random::Rng r(1234ULL);
    if constexpr (std::is_same<DataT, float>{}) {
      r.uniform(database.data(), ps.num_db_vecs * ps.dim, DataT(0.1), DataT(2.0), stream_);
      r.uniform(search_queries.data(), ps.num_queries * ps.dim, DataT(0.1), DataT(2.0), stream_);
    } else {
      r.uniformInt(database.data(), ps.num_db_vecs * ps.dim, DataT(1), DataT(20), stream_);
      r.uniformInt(search_queries.data(), ps.num_queries * ps.dim, DataT(1), DataT(20), stream_);
    }
    handle_.sync_stream(stream_);
  }

  void TearDown() override
  {
    handle_.sync_stream(stream_);
    database.resize(0, stream_);
    search_queries.resize(0, stream_);
  }

 private:
  raft::handle_t handle_;
  rmm::cuda_stream_view stream_;
  AnnIvfFlatInputs ps;
  rmm::device_uvector<DataT> database;
  rmm::device_uvector<DataT> search_queries;
};

const std::vector<AnnIvfFlatInputs> inputs = {
  // test various dims (aligned and not aligned to vector sizes)
  {1000, 10000, 1, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 3, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 4, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 5, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 8, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},

  // test dims that do not fit into kernel shared memory limits
  {1000, 10000, 2048, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2049, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2050, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2051, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2052, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2053, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2056, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},

  // various random combinations
  {1000, 10000, 16, 10, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, 50, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, 70, 1024, raft::distance::DistanceType::L2Expanded},
  {100, 10000, 16, 10, 20, 512, raft::distance::DistanceType::L2Expanded},
  {20, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},
  {10000, 131072, 8, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},

  {1000, 10000, 16, 10, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 16, 10, 50, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 16, 10, 70, 1024, raft::distance::DistanceType::InnerProduct},
  {100, 10000, 16, 10, 20, 512, raft::distance::DistanceType::InnerProduct},
  {20, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::InnerProduct},
  {10000, 131072, 8, 10, 50, 1024, raft::distance::DistanceType::InnerProduct},

  {1000, 10000, 4096, 20, 50, 1024, raft::distance::DistanceType::InnerProduct},

  // test splitting the big query batches  (> max gridDim.y) into smaller batches
  {100000, 1024, 32, 10, 64, 64, raft::distance::DistanceType::InnerProduct},
  {98306, 1024, 32, 10, 64, 64, raft::distance::DistanceType::InnerProduct},

  // test radix_sort for getting the cluster selection
  {1000,
   10000,
   16,
   10,
   raft::spatial::knn::detail::topk::kMaxCapacity * 2,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::distance::DistanceType::L2Expanded},
  {1000,
   10000,
   16,
   10,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::distance::DistanceType::InnerProduct}};

typedef AnnIVFFlatTest<float, float> AnnIVFFlatTestF;
TEST_P(AnnIVFFlatTestF, AnnIVFFlat) { this->testIVFFlat(); }

INSTANTIATE_TEST_CASE_P(AnnIVFFlatTest, AnnIVFFlatTestF, ::testing::ValuesIn(inputs));

typedef AnnIVFFlatTest<float, uint8_t> AnnIVFFlatTestF_uint8;
TEST_P(AnnIVFFlatTestF_uint8, AnnIVFFlat) { this->testIVFFlat(); }

INSTANTIATE_TEST_CASE_P(AnnIVFFlatTest, AnnIVFFlatTestF_uint8, ::testing::ValuesIn(inputs));

typedef AnnIVFFlatTest<float, int8_t> AnnIVFFlatTestF_int8;
TEST_P(AnnIVFFlatTestF_int8, AnnIVFFlat) { this->testIVFFlat(); }

INSTANTIATE_TEST_CASE_P(AnnIVFFlatTest, AnnIVFFlatTestF_int8, ::testing::ValuesIn(inputs));

}  // namespace knn
}  // namespace spatial
}  // namespace raft
