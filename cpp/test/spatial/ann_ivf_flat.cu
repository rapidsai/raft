#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.h"
#include "ann_utils.cuh"

#include <raft/core/logger.hpp>
#include <raft/distance/distance_types.hpp>
#include <raft/random/rng.cuh>
#include <raft/spatial/knn/ann.cuh>
#include <raft/spatial/knn/ivf_flat.cuh>
#include <raft/spatial/knn/knn.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>

#include <gtest/gtest.h>

#include <thrust/sequence.h>

#include <cstddef>
#include <iostream>
#include <vector>

namespace raft {
namespace spatial {
namespace knn {
struct AnnIvfFlatInputs {
  int num_queries;
  int num_db_vecs;
  int dim;
  int k;
  int nprobe;
  int nlist;
  raft::distance::DistanceType metric;
};

template <typename T, typename DataT>
class AnnIVFFlatTest : public ::testing::TestWithParam<AnnIvfFlatInputs> {
 public:
  AnnIVFFlatTest()
    : stream_(handle_.get_stream()),
      ps(::testing::TestWithParam<AnnIvfFlatInputs>::GetParam()),
      database(0, stream_),
      search_queries(0, stream_)
  {
  }

 protected:
  void testIVFFlat()
  {
    size_t queries_size = ps.num_queries * ps.k;
    std::vector<int64_t> indices_ivfflat(queries_size);
    std::vector<int64_t> indices_naive(queries_size);
    std::vector<T> distances_ivfflat(queries_size);
    std::vector<T> distances_naive(queries_size);

    {
      rmm::device_uvector<T> distances_naive_dev(queries_size, stream_);
      rmm::device_uvector<int64_t> indices_naive_dev(queries_size, stream_);
      naiveBfKnn<T, DataT, int64_t>(distances_naive_dev.data(),
                                    indices_naive_dev.data(),
                                    search_queries.data(),
                                    database.data(),
                                    ps.num_queries,
                                    ps.num_db_vecs,
                                    ps.dim,
                                    ps.k,
                                    ps.metric,
                                    stream_);
      update_host(distances_naive.data(), distances_naive_dev.data(), queries_size, stream_);
      update_host(indices_naive.data(), indices_naive_dev.data(), queries_size, stream_);
      handle_.sync_stream(stream_);
    }

    {
      // unless something is really wrong with clustering, this could serve as a lower bound on
      // recall
      double min_recall = static_cast<double>(ps.nprobe) / static_cast<double>(ps.nlist);

      rmm::device_uvector<T> distances_ivfflat_dev(queries_size, stream_);
      rmm::device_uvector<int64_t> indices_ivfflat_dev(queries_size, stream_);

      {
        // legacy interface
        raft::spatial::knn::IVFFlatParam ivfParams;
        ivfParams.nprobe = ps.nprobe;
        ivfParams.nlist  = ps.nlist;
        raft::spatial::knn::knnIndex index;
        index.index   = nullptr;
        index.gpu_res = nullptr;

        approx_knn_build_index(handle_,
                               &index,
                               dynamic_cast<raft::spatial::knn::knnIndexParam*>(&ivfParams),
                               ps.metric,
                               0,
                               database.data(),
                               ps.num_db_vecs,
                               ps.dim);
        handle_.sync_stream(stream_);
        approx_knn_search(handle_,
                          distances_ivfflat_dev.data(),
                          indices_ivfflat_dev.data(),
                          &index,
                          ps.k,
                          search_queries.data(),
                          ps.num_queries);

        update_host(distances_ivfflat.data(), distances_ivfflat_dev.data(), queries_size, stream_);
        update_host(indices_ivfflat.data(), indices_ivfflat_dev.data(), queries_size, stream_);
        handle_.sync_stream(stream_);
      }

      ASSERT_TRUE(eval_neighbours(indices_naive,
                                  indices_ivfflat,
                                  distances_naive,
                                  distances_ivfflat,
                                  ps.num_queries,
                                  ps.k,
                                  0.001,
                                  min_recall));
      {
        // new interface
        raft::spatial::knn::ivf_flat::index_params index_params;
        raft::spatial::knn::ivf_flat::search_params search_params;
        index_params.n_lists   = ps.nlist;
        index_params.metric    = ps.metric;
        search_params.n_probes = ps.nprobe;

        index_params.add_data_on_build        = false;
        index_params.kmeans_trainset_fraction = 0.5;
        auto index =
          ivf_flat::build(handle_, index_params, database.data(), int64_t(ps.num_db_vecs), ps.dim);

        rmm::device_uvector<int64_t> vector_indices(ps.num_db_vecs, stream_);
        thrust::sequence(handle_.get_thrust_policy(),
                         thrust::device_pointer_cast(vector_indices.data()),
                         thrust::device_pointer_cast(vector_indices.data() + ps.num_db_vecs));
        handle_.sync_stream(stream_);

        int64_t half_of_data = ps.num_db_vecs / 2;

        auto index_2 =
          ivf_flat::extend<DataT, int64_t>(handle_, index, database.data(), nullptr, half_of_data);

        ivf_flat::extend<DataT, int64_t>(handle_,
                                         &index_2,
                                         database.data() + half_of_data * ps.dim,
                                         vector_indices.data() + half_of_data,
                                         int64_t(ps.num_db_vecs) - half_of_data);

        ivf_flat::search(handle_,
                         search_params,
                         index_2,
                         search_queries.data(),
                         ps.num_queries,
                         ps.k,
                         indices_ivfflat_dev.data(),
                         distances_ivfflat_dev.data());

        update_host(distances_ivfflat.data(), distances_ivfflat_dev.data(), queries_size, stream_);
        update_host(indices_ivfflat.data(), indices_ivfflat_dev.data(), queries_size, stream_);
        handle_.sync_stream(stream_);
      }
      ASSERT_TRUE(eval_neighbours(indices_naive,
                                  indices_ivfflat,
                                  distances_naive,
                                  distances_ivfflat,
                                  ps.num_queries,
                                  ps.k,
                                  0.001,
                                  min_recall));
    }
  }

  void SetUp() override
  {
    database.resize(ps.num_db_vecs * ps.dim, stream_);
    search_queries.resize(ps.num_queries * ps.dim, stream_);

    raft::random::Rng r(1234ULL);
    if constexpr (std::is_same<DataT, float>{}) {
      r.uniform(database.data(), ps.num_db_vecs * ps.dim, DataT(0.1), DataT(2.0), stream_);
      r.uniform(search_queries.data(), ps.num_queries * ps.dim, DataT(0.1), DataT(2.0), stream_);
    } else {
      r.uniformInt(database.data(), ps.num_db_vecs * ps.dim, DataT(1), DataT(20), stream_);
      r.uniformInt(search_queries.data(), ps.num_queries * ps.dim, DataT(1), DataT(20), stream_);
    }
    handle_.sync_stream(stream_);
  }

  void TearDown() override
  {
    handle_.sync_stream(stream_);
    database.resize(0, stream_);
    search_queries.resize(0, stream_);
  }

 private:
  raft::handle_t handle_;
  rmm::cuda_stream_view stream_;
  AnnIvfFlatInputs ps;
  rmm::device_uvector<DataT> database;
  rmm::device_uvector<DataT> search_queries;
};

const std::vector<AnnIvfFlatInputs> inputs = {
  // test various dims (aligned and not aligned to vector sizes)
  {1000, 10000, 1, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 3, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 4, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 5, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 8, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},

  // test dims that do not fit into kernel shared memory limits
  {1000, 10000, 2048, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2049, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2050, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2051, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2052, 16, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 2053, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 2056, 16, 40, 1024, raft::distance::DistanceType::L2Expanded},

  // various random combinations
  {1000, 10000, 16, 10, 40, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, 50, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 10000, 16, 10, 70, 1024, raft::distance::DistanceType::L2Expanded},
  {100, 10000, 16, 10, 20, 512, raft::distance::DistanceType::L2Expanded},
  {20, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},
  {1000, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},
  {10000, 131072, 8, 10, 20, 1024, raft::distance::DistanceType::L2Expanded},

  {1000, 10000, 16, 10, 40, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 16, 10, 50, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 10000, 16, 10, 70, 1024, raft::distance::DistanceType::InnerProduct},
  {100, 10000, 16, 10, 20, 512, raft::distance::DistanceType::InnerProduct},
  {20, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::InnerProduct},
  {1000, 100000, 16, 10, 20, 1024, raft::distance::DistanceType::InnerProduct},
  {10000, 131072, 8, 10, 50, 1024, raft::distance::DistanceType::InnerProduct},

  {1000, 10000, 4096, 20, 50, 1024, raft::distance::DistanceType::InnerProduct},

  // test splitting the big query batches  (> max gridDim.y) into smaller batches
  {100000, 1024, 32, 10, 64, 64, raft::distance::DistanceType::InnerProduct},
  {98306, 1024, 32, 10, 64, 64, raft::distance::DistanceType::InnerProduct},

  // test radix_sort for getting the cluster selection
  {1000,
   10000,
   16,
   10,
   raft::spatial::knn::detail::topk::kMaxCapacity * 2,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::distance::DistanceType::L2Expanded},
  {1000,
   10000,
   16,
   10,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::spatial::knn::detail::topk::kMaxCapacity * 4,
   raft::distance::DistanceType::InnerProduct}};

typedef AnnIVFFlatTest<float, float> AnnIVFFlatTestF;
TEST_P(AnnIVFFlatTestF, AnnIVFFlat) { this->testIVFFlat(); }

INSTANTIATE_TEST_CASE_P(AnnIVFFlatTest, AnnIVFFlatTestF, ::testing::ValuesIn(inputs));

typedef AnnIVFFlatTest<float, uint8_t> AnnIVFFlatTestF_uint8;
TEST_P(AnnIVFFlatTestF_uint8, AnnIVFFlat) { this->testIVFFlat(); }

INSTANTIATE_TEST_CASE_P(AnnIVFFlatTest, AnnIVFFlatTestF_uint8, ::testing::ValuesIn(inputs));

typedef AnnIVFFlatTest<float, int8_t> AnnIVFFlatTestF_int8;
TEST_P(AnnIVFFlatTestF_int8, AnnIVFFlat) { this->testIVFFlat(); }

INSTANTIATE_TEST_CASE_P(AnnIVFFlatTest, AnnIVFFlatTestF_int8, ::testing::ValuesIn(inputs));

}  // namespace knn
}  // namespace spatial
}  // namespace raft
