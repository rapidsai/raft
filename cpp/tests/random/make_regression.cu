/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../test_utils.cuh"

#include <raft/core/operators.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/linalg/gemm.cuh>
#include <raft/linalg/map_reduce.cuh>
#include <raft/linalg/subtract.cuh>
#include <raft/linalg/transpose.cuh>
#include <raft/random/make_regression.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/cudart_utils.hpp>

#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <gtest/gtest.h>

namespace raft::random {

template <typename T>
struct MakeRegressionInputs {
  T tolerance;
  int n_samples, n_features, n_informative, n_targets, effective_rank;
  T bias;
  bool shuffle;
  raft::random::GeneratorType gtype;
  uint64_t seed;
};

template <typename T>
class MakeRegressionTest : public ::testing::TestWithParam<MakeRegressionInputs<T>> {
 protected:
  void SetUp() override
  {
    // Noise must be zero to compare the actual and expected values
    T noise = (T)0.0, tail_strength = (T)0.5;

    rmm::device_uvector<T> data(params.n_samples * params.n_features, stream);
    rmm::device_uvector<T> values_cm(params.n_samples * params.n_targets, stream);
    rmm::device_uvector<T> coef(params.n_features * params.n_targets, stream);

    // Create the regression problem
    make_regression(handle,
                    data.data(),
                    values_ret.data(),
                    params.n_samples,
                    params.n_features,
                    params.n_informative,
                    stream,
                    coef.data(),
                    params.n_targets,
                    params.bias,
                    params.effective_rank,
                    tail_strength,
                    noise,
                    params.shuffle,
                    params.seed,
                    params.gtype);

    // FIXME (mfh 2022/09/07) This test passes even if it doesn't call
    // make_regression.  Please see
    // https://github.com/rapidsai/raft/issues/814.

    // Calculate the values from the data and coefficients (column-major)
    T alpha = (T)1.0, beta = (T)0.0;
    raft::linalg::gemm(handle,
                       true,
                       true,
                       params.n_samples,
                       params.n_targets,
                       params.n_features,
                       &alpha,
                       data.data(),
                       params.n_features,
                       coef.data(),
                       params.n_targets,
                       &beta,
                       values_cm.data(),
                       params.n_samples,
                       stream);

    // Transpose the values to row-major
    raft::linalg::transpose(
      handle, values_cm.data(), values_prod.data(), params.n_samples, params.n_targets, stream);

    // Add the bias
    raft::linalg::addScalar(values_prod.data(),
                            values_prod.data(),
                            params.bias,
                            params.n_samples * params.n_targets,
                            stream);

    // Count the number of zeroes in the coefficients
    rmm::device_scalar<int> zc_device(stream);
    raft::linalg::mapReduce(zc_device.data(),
                            coef.size(),
                            0,
                            raft::compose_op{raft::cast_op<int>{}, raft::equal_const_op<T>{0}},
                            raft::add_op{},
                            stream,
                            coef.data());
    zero_count = zc_device.value(stream);
  }

 protected:
  MakeRegressionInputs<T> params{::testing::TestWithParam<MakeRegressionInputs<T>>::GetParam()};
  raft::resources handle;
  rmm::cuda_stream_view stream{resource::get_cuda_stream(handle)};
  rmm::device_uvector<T> values_ret{size_t(params.n_samples) * size_t(params.n_targets), stream};
  rmm::device_uvector<T> values_prod{size_t(params.n_samples) * size_t(params.n_targets), stream};

  int zero_count;
};

typedef MakeRegressionTest<float> MakeRegressionTestF;
const std::vector<MakeRegressionInputs<float>> inputsf_t = {
  {0.01f, 256, 32, 16, 1, -1, 0.f, true, raft::random::GenPC, 1234ULL},
  {0.01f, 1000, 100, 47, 4, 65, 4.2f, true, raft::random::GenPC, 1234ULL},
  {0.01f, 20000, 500, 450, 13, -1, -3.f, false, raft::random::GenPC, 1234ULL},
  {0.01f, 256, 32, 16, 1, -1, 0.f, true, raft::random::GenPhilox, 1234ULL},
  {0.01f, 1000, 100, 47, 4, 65, 4.2f, true, raft::random::GenPhilox, 1234ULL},
  {0.01f, 20000, 500, 450, 13, -1, -3.f, false, raft::random::GenPhilox, 1234ULL}};

TEST_P(MakeRegressionTestF, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<float>(params.tolerance),
                          stream));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionTests, MakeRegressionTestF, ::testing::ValuesIn(inputsf_t));

typedef MakeRegressionTest<double> MakeRegressionTestD;
const std::vector<MakeRegressionInputs<double>> inputsd_t = {
  {0.01, 256, 32, 16, 1, -1, 0.0, true, raft::random::GenPC, 1234ULL},
  {0.01, 1000, 100, 47, 4, 65, 4.2, true, raft::random::GenPC, 1234ULL},
  {0.01, 20000, 500, 450, 13, -1, -3.0, false, raft::random::GenPC, 1234ULL},
  {0.01, 256, 32, 16, 1, -1, 0.0, true, raft::random::GenPhilox, 1234ULL},
  {0.01, 1000, 100, 47, 4, 65, 4.2, true, raft::random::GenPhilox, 1234ULL},
  {0.01, 20000, 500, 450, 13, -1, -3.0, false, raft::random::GenPhilox, 1234ULL}};

TEST_P(MakeRegressionTestD, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<double>(params.tolerance),
                          stream));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionTests, MakeRegressionTestD, ::testing::ValuesIn(inputsd_t));

template <typename T>
class MakeRegressionMdspanTest : public ::testing::TestWithParam<MakeRegressionInputs<T>> {
 public:
  MakeRegressionMdspanTest() = default;

 protected:
  void SetUp() override
  {
    // Noise must be zero to compare the actual and expected values
    T noise = (T)0.0, tail_strength = (T)0.5;

    rmm::device_uvector<T> data(params.n_samples * params.n_features, stream);
    rmm::device_uvector<T> values_cm(params.n_samples * params.n_targets, stream);
    rmm::device_uvector<T> coef(params.n_features * params.n_targets, stream);

    using index_type = typename rmm::device_uvector<T>::index_type;
    using matrix_view =
      raft::device_matrix_view<T, raft::matrix_extent<index_type>, raft::row_major>;
    matrix_view out_mat(data.data(), params.n_samples, params.n_features);
    matrix_view values_mat(values_ret.data(), params.n_samples, params.n_targets);
    matrix_view coef_mat(coef.data(), params.n_features, params.n_targets);

    // Create the regression problem
    make_regression(handle,
                    out_mat,
                    values_mat,
                    params.n_informative,
                    coef_mat,
                    params.bias,
                    params.effective_rank,
                    tail_strength,
                    noise,
                    params.shuffle,
                    params.seed,
                    params.gtype);

    // FIXME (mfh 2022/09/07) This test passes even if it doesn't call
    // make_regression.  Please see
    // https://github.com/rapidsai/raft/issues/814.

    // Calculate the values from the data and coefficients (column-major)
    T alpha{};
    T beta{};
    raft::linalg::gemm(handle,
                       true,
                       true,
                       params.n_samples,
                       params.n_targets,
                       params.n_features,
                       &alpha,
                       data.data(),
                       params.n_features,
                       coef.data(),
                       params.n_targets,
                       &beta,
                       values_cm.data(),
                       params.n_samples,
                       stream);

    // Transpose the values to row-major
    raft::linalg::transpose(
      handle, values_cm.data(), values_prod.data(), params.n_samples, params.n_targets, stream);

    // Add the bias
    raft::linalg::addScalar(values_prod.data(),
                            values_prod.data(),
                            params.bias,
                            params.n_samples * params.n_targets,
                            stream);

    // Count the number of zeroes in the coefficients
    rmm::device_scalar<int> zc_device(stream);
    raft::linalg::mapReduce(zc_device.data(),
                            coef.size(),
                            0,
                            raft::compose_op{raft::cast_op<int>{}, raft::equal_const_op<T>{0}},
                            raft::add_op{},
                            stream,
                            coef.data());
    zero_count = zc_device.value(stream);
  }

 private:
  MakeRegressionInputs<T> params{::testing::TestWithParam<MakeRegressionInputs<T>>::GetParam()};
  raft::resources handle;
  rmm::cuda_stream_view stream{resource::get_cuda_stream(handle)};
  rmm::device_uvector<T> values_ret{size_t(params.n_samples) * size_t(params.n_targets), stream};
  rmm::device_uvector<T> values_prod{size_t(params.n_samples) * size_t(params.n_targets), stream};

  int zero_count = -1;
};

using MakeRegressionMdspanTestF = MakeRegressionTest<float>;

TEST_P(MakeRegressionMdspanTestF, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<float>(params.tolerance),
                          resource::get_cuda_stream(handle)));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionMdspanTests,
                        MakeRegressionMdspanTestF,
                        ::testing::ValuesIn(inputsf_t));

using MakeRegressionMdspanTestD = MakeRegressionTest<double>;

TEST_P(MakeRegressionMdspanTestD, Result)
{
  ASSERT_TRUE(match(params.n_targets * (params.n_features - params.n_informative),
                    zero_count,
                    raft::Compare<int>()));
  ASSERT_TRUE(devArrMatch(values_ret.data(),
                          values_prod.data(),
                          params.n_samples,
                          params.n_targets,
                          raft::CompareApprox<double>(params.tolerance),
                          resource::get_cuda_stream(handle)));
}
INSTANTIATE_TEST_CASE_P(MakeRegressionMdspanTests,
                        MakeRegressionMdspanTestD,
                        ::testing::ValuesIn(inputsd_t));

}  // end namespace raft::random
