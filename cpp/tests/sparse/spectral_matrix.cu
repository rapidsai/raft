/*
 * SPDX-FileCopyrightText: Copyright (c) 2020-2024, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resource/device_id.hpp>
#include <raft/core/resources.hpp>
#include <raft/sparse/linalg/laplacian.cuh>
#include <raft/spectral/matrix_wrappers.hpp>
#include <raft/util/cudart_utils.hpp>

#include <gtest/gtest.h>

#include <iostream>
#include <memory>

namespace raft {
namespace spectral {
namespace matrix {
namespace {
template <typename index_type, typename value_type>
struct csr_view_t {
  index_type* offsets;
  index_type* indices;
  value_type* edge_data;
  index_type number_of_vertices;
  index_type number_of_edges;
};
}  // namespace
TEST(Raft, SpectralMatrices)
{
  using index_type = int;
  using value_type = double;
  using nnz_type   = uint64_t;

  raft::resources h;
  ASSERT_EQ(0, raft::resource::get_device_id(h));

  csr_view_t<index_type, value_type> csr_v{nullptr, nullptr, nullptr, 0, 0};

  int const sz = 10;
  vector_t<index_type> d_v{h, sz};

  index_type* ro{nullptr};
  index_type* ci{nullptr};
  value_type* vs{nullptr};
  nnz_type nnz     = 0;
  index_type nrows = 0;
  sparse_matrix_t<index_type, value_type, nnz_type> sm1{h, ro, ci, vs, nrows, nnz};
  sparse_matrix_t<index_type, value_type, nnz_type> sm2{h, csr_v};
  ASSERT_EQ(nullptr, sm1.row_offsets_);
  ASSERT_EQ(nullptr, sm2.row_offsets_);

  auto stream = resource::get_cuda_stream(h);

  auto cnstr_lm1 = [&h, ro, ci, vs, nrows, nnz](void) {
    laplacian_matrix_t<index_type, value_type, nnz_type> lm1{h, ro, ci, vs, nrows, nnz};
  };
  EXPECT_ANY_THROW(cnstr_lm1());  // because of nullptr ptr args

  auto cnstr_lm2 = [&h, &sm2](void) {
    laplacian_matrix_t<index_type, value_type, nnz_type> lm2{h, sm2};
  };
  EXPECT_ANY_THROW(cnstr_lm2());  // because of nullptr ptr args

  auto cnstr_mm1 = [&h, ro, ci, vs, nrows, nnz](void) {
    modularity_matrix_t<index_type, value_type, nnz_type> mm1{h, ro, ci, vs, nrows, nnz};
  };
  EXPECT_ANY_THROW(cnstr_mm1());  // because of nullptr ptr args

  auto cnstr_mm2 = [&h, &sm2](void) {
    modularity_matrix_t<index_type, value_type, nnz_type> mm2{h, sm2};
  };
  EXPECT_ANY_THROW(cnstr_mm2());  // because of nullptr ptr args
}

}  // namespace matrix
}  // namespace spectral
}  // namespace raft
