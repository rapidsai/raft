#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/core/device_csr_matrix.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/sparse/linalg/laplacian.cuh>
#include <raft/util/cudart_utils.hpp>

#include <gtest/gtest.h>

#include <vector>

namespace raft::sparse::linalg {

TEST(Raft, ComputeGraphLaplacian)
{
  // The following adjacency matrix will be used to allow for manual
  // verification of results:
  // [[0 1 1 1]
  //  [1 0 0 1]
  //  [1 0 0 0]
  //  [1 1 0 0]]

  auto data    = std::vector<float>{1, 1, 1, 1, 1, 1, 1, 1};
  auto indices = std::vector<int>{1, 2, 3, 0, 3, 0, 0, 1};
  auto indptr  = std::vector<int>{0, 3, 5, 6, 8};

  auto res = raft::resources{};
  auto adjacency_matrix =
    make_device_csr_matrix<float>(res, int(indptr.size() - 1), int(indptr.size() - 1), data.size());
  auto adjacency_structure = adjacency_matrix.structure_view();
  raft::copy(adjacency_matrix.get_elements().data(),
             &(data[0]),
             data.size(),
             raft::resource::get_cuda_stream(res));
  raft::copy(adjacency_structure.get_indices().data(),
             &(indices[0]),
             indices.size(),
             raft::resource::get_cuda_stream(res));
  raft::copy(adjacency_structure.get_indptr().data(),
             &(indptr[0]),
             indptr.size(),
             raft::resource::get_cuda_stream(res));
  auto laplacian           = compute_graph_laplacian(res, adjacency_matrix.view());
  auto laplacian_structure = laplacian.structure_view();
  auto laplacian_data      = std::vector<float>(laplacian_structure.get_nnz());
  auto laplacian_indices   = std::vector<int>(laplacian_structure.get_nnz());
  auto laplacian_indptr    = std::vector<int>(laplacian_structure.get_n_rows() + 1);
  raft::copy(&(laplacian_data[0]),
             laplacian.get_elements().data(),
             laplacian_structure.get_nnz(),
             raft::resource::get_cuda_stream(res));
  raft::copy(&(laplacian_indices[0]),
             laplacian_structure.get_indices().data(),
             laplacian_structure.get_nnz(),
             raft::resource::get_cuda_stream(res));
  raft::copy(&(laplacian_indptr[0]),
             laplacian_structure.get_indptr().data(),
             laplacian_structure.get_n_rows() + 1,
             raft::resource::get_cuda_stream(res));
  auto expected_data    = std::vector<float>{3, -1, -1, -1, -1, 2, -1, -1, 1, -1, -1, 2};
  auto expected_indices = std::vector<int>{0, 1, 2, 3, 0, 1, 3, 0, 2, 0, 1, 3};
  auto expected_indptr  = std::vector<int>{0, 4, 7, 9, 12};
  raft::resource::sync_stream(res);

  EXPECT_EQ(expected_data, laplacian_data);
  EXPECT_EQ(expected_indices, laplacian_indices);
  EXPECT_EQ(expected_indptr, laplacian_indptr);
}

TEST(Raft, ComputeGraphLaplacianNormalized)
{
  // Using the same adjacency matrix as in the ComputeGraphLaplacian test:
  // [[0 1 1 1]
  //  [1 0 0 1]
  //  [1 0 0 0]
  //  [1 1 0 0]]

  auto data    = std::vector<float>{1, 1, 1, 1, 1, 1, 1, 1};
  auto indices = std::vector<int>{1, 2, 3, 0, 3, 0, 0, 1};
  auto indptr  = std::vector<int>{0, 3, 5, 6, 8};

  auto res              = raft::resources{};
  auto adjacency_matrix = make_device_csr_matrix<float, int, int, int>(
    res, (indptr.size() - 1), (indptr.size() - 1), data.size());
  auto adjacency_structure = adjacency_matrix.structure_view();
  raft::copy(adjacency_matrix.get_elements().data(),
             &(data[0]),
             data.size(),
             raft::resource::get_cuda_stream(res));
  raft::copy(adjacency_structure.get_indices().data(),
             &(indices[0]),
             indices.size(),
             raft::resource::get_cuda_stream(res));
  raft::copy(adjacency_structure.get_indptr().data(),
             &(indptr[0]),
             indptr.size(),
             raft::resource::get_cuda_stream(res));

  // Create diagonal output vector
  auto diagonal_out = raft::make_device_vector<float, int>(res, adjacency_structure.get_n_rows());

  // Compute normalized Laplacian
  auto normalized_laplacian =
    laplacian_normalized(res, adjacency_matrix.view(), diagonal_out.view());
  auto normalized_laplacian_structure = normalized_laplacian.structure_view();

  // Copy results back to host
  auto normalized_laplacian_data    = std::vector<float>(normalized_laplacian_structure.get_nnz());
  auto normalized_laplacian_indices = std::vector<int>(normalized_laplacian_structure.get_nnz());
  auto normalized_laplacian_indptr =
    std::vector<int>(normalized_laplacian_structure.get_n_rows() + 1);
  auto diagonal_data = std::vector<float>(adjacency_structure.get_n_rows());

  raft::copy(&(normalized_laplacian_data[0]),
             normalized_laplacian.get_elements().data(),
             normalized_laplacian_structure.get_nnz(),
             raft::resource::get_cuda_stream(res));
  raft::copy(&(normalized_laplacian_indices[0]),
             normalized_laplacian_structure.get_indices().data(),
             normalized_laplacian_structure.get_nnz(),
             raft::resource::get_cuda_stream(res));
  raft::copy(&(normalized_laplacian_indptr[0]),
             normalized_laplacian_structure.get_indptr().data(),
             normalized_laplacian_structure.get_n_rows() + 1,
             raft::resource::get_cuda_stream(res));
  raft::copy(&(diagonal_data[0]),
             diagonal_out.data_handle(),
             diagonal_out.size(),
             raft::resource::get_cuda_stream(res));
  raft::resource::sync_stream(res);

  // For the given adjacency matrix, the diagonal degree matrix D has values [3, 2, 1, 2]
  // The square root of these values is [√3, √2, 1, √2]
  // The normalized Laplacian should have values close to:
  // [1, -1/√(3*2), -1/√(3*1), -1/√(3*2),
  //  -1/√(2*3), 1, -1/√(2*2),
  //  -1/√(1*3), 1,
  //  -1/√(2*3), -1/√(2*2), 1]

  // Expected diagonal values (sqrt of diagonal degree matrix)
  auto expected_diagonal =
    std::vector<float>{std::sqrt(3.0f), std::sqrt(2.0f), 1.0f, std::sqrt(2.0f)};

  // Expected normalized Laplacian values
  auto expected_data = std::vector<float>{1.0f,
                                          -1.0f / std::sqrt(3.0f * 2.0f),
                                          -1.0f / std::sqrt(3.0f * 1.0f),
                                          -1.0f / std::sqrt(3.0f * 2.0f),
                                          -1.0f / std::sqrt(2.0f * 3.0f),
                                          1.0f,
                                          -1.0f / std::sqrt(2.0f * 2.0f),
                                          -1.0f / std::sqrt(1.0f * 3.0f),
                                          1.0f,
                                          -1.0f / std::sqrt(2.0f * 3.0f),
                                          -1.0f / std::sqrt(2.0f * 2.0f),
                                          1.0f};

  // Same indices and indptr as non-normalized Laplacian
  auto expected_indices = std::vector<int>{0, 1, 2, 3, 0, 1, 3, 0, 2, 0, 1, 3};
  auto expected_indptr  = std::vector<int>{0, 4, 7, 9, 12};

  // Compare results with expected values with a small tolerance for floating point differences
  const float tol = 1e-6f;
  ASSERT_EQ(expected_diagonal.size(), diagonal_data.size());
  for (size_t i = 0; i < expected_diagonal.size(); ++i) {
    EXPECT_NEAR(expected_diagonal[i], diagonal_data[i], tol);
  }

  ASSERT_EQ(expected_data.size(), normalized_laplacian_data.size());
  for (size_t i = 0; i < expected_data.size(); ++i) {
    EXPECT_NEAR(expected_data[i], normalized_laplacian_data[i], tol);
  }

  EXPECT_EQ(expected_indices, normalized_laplacian_indices);
  EXPECT_EQ(expected_indptr, normalized_laplacian_indptr);
}

}  // namespace raft::sparse::linalg
