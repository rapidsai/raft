/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common.cuh"

#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/neighbors/ivf_pq.cuh>
#include <raft/neighbors/refine.cuh>

#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include <cstdint>

void ivf_pq_build_search(raft::device_resources const& dev_resources,
                         raft::device_matrix_view<const float, int64_t> dataset,
                         raft::device_matrix_view<const float, int64_t> queries)
{
  using namespace raft::neighbors;  // NOLINT

  ivf_pq::index_params index_params;
  index_params.n_lists                  = 1024;
  index_params.kmeans_trainset_fraction = 0.1;
  index_params.metric                   = raft::distance::DistanceType::L2Expanded;
  index_params.pq_bits                  = 8;
  index_params.pq_dim                   = 2;

  std::cout << "Building IVF-PQ index" << std::endl;
  auto index = ivf_pq::build(dev_resources, index_params, dataset);

  std::cout << "Number of clusters " << index.n_lists() << ", number of vectors added to index "
            << index.size() << std::endl;

  // Set search parameters.
  ivf_pq::search_params search_params;
  search_params.n_probes = 50;
  // Set the internal search precision to 16-bit floats;
  // usually, this improves the performance at a slight cost to the recall.
  search_params.internal_distance_dtype = HIP_R_16F;
  search_params.lut_dtype               = HIP_R_16F;

  // Create output arrays.
  int64_t topk      = 10;
  int64_t n_queries = queries.extent(0);
  auto neighbors    = raft::make_device_matrix<int64_t>(dev_resources, n_queries, topk);
  auto distances    = raft::make_device_matrix<float>(dev_resources, n_queries, topk);

  // Search K nearest neighbors for each of the queries.
  ivf_pq::search<float, int64_t>(
    dev_resources, search_params, index, queries, neighbors.view(), distances.view());

  // Re-ranking operation: refine the initial search results by computing exact distances
  int64_t topk_refined = 7;
  auto neighbors_refined =
    raft::make_device_matrix<int64_t>(dev_resources, n_queries, topk_refined);
  auto distances_refined = raft::make_device_matrix<float>(dev_resources, n_queries, topk_refined);

  // Note, refinement requires the original dataset and the queries.
  // Don't forget to specify the same distance metric as used by the index.
  raft::neighbors::refine(dev_resources,
                          dataset,
                          queries,
                          raft::make_const_mdspan(neighbors.view()),
                          neighbors_refined.view(),
                          distances_refined.view(),
                          index.metric());

  // Show both the original and the refined results
  std::cout << std::endl << "Original results:" << std::endl;
  print_results(dev_resources, neighbors.view(), distances.view());
  std::cout << std::endl << "Refined results:" << std::endl;
  print_results(dev_resources, neighbors_refined.view(), distances_refined.view());
}

int main()
{
  raft::device_resources dev_resources;

  // Set pool memory resource with 1 GiB initial pool size. All allocations use the same pool.
  rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> pool_mr(
    rmm::mr::get_current_device_resource(), 1024 * 1024 * 1024ull);
  rmm::mr::set_current_device_resource(&pool_mr);

  // Alternatively, one could define a pool allocator for temporary arrays (used within RAFT
  // algorithms). In that case only the internal arrays would use the pool, any other allocation
  // uses the default RMM memory resource. Here is how to change the workspace memory resource to
  // a pool with 2 GiB upper limit.
  // raft::resource::set_workspace_to_pool_resource(dev_resources, 2 * 1024 * 1024 * 1024ull);

  // Create input arrays.
  int64_t n_samples = 10000;
  int64_t n_dim     = 3;
  int64_t n_queries = 10;
  auto dataset      = raft::make_device_matrix<float, int64_t>(dev_resources, n_samples, n_dim);
  auto queries      = raft::make_device_matrix<float, int64_t>(dev_resources, n_queries, n_dim);
  generate_dataset(dev_resources, dataset.view(), queries.view());

  // Simple build and search example.
  ivf_pq_build_search(dev_resources,
                      raft::make_const_mdspan(dataset.view()),
                      raft::make_const_mdspan(queries.view()));
}
