/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/core/operators.hpp>                            // raft::identity_op
#include <raft/distance/detail/distance_ops/all_ops.cuh>      // ops::*
#include <raft/distance/detail/pairwise_matrix/dispatch.cuh>  // pairwise_matrix_instantiation_point
#include <raft/distance/detail/pairwise_matrix/dispatch_sm60.cuh>
#include <raft/distance/detail/pairwise_matrix/dispatch_sm80.cuh>

namespace raft::distance::detail {

template void pairwise_matrix_instantiation_point<ops::l2_exp_distance_op<float, float, int>,
                                                  int,
                                                  float,
                                                  float,
                                                  decltype(raft::identity_op())>(
  ops::l2_exp_distance_op<float, float, int>,
  pairwise_matrix_params<int, float, float, decltype(raft::identity_op())>,
  hipStream_t);

}  // namespace raft::distance::detail
