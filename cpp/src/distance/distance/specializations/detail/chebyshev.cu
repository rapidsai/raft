/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/distance/detail/distance.cuh>

namespace raft {
namespace distance {
namespace detail {
template void distance<raft::distance::DistanceType::Linf, float, float, float, int>(
  const float* x,
  const float* y,
  float* dist,
  int m,
  int n,
  int k,
  void* workspace,
  std::size_t worksize,
  hipStream_t stream,
  bool isRowMajor,
  float metric_arg);

template void distance<raft::distance::DistanceType::Linf, double, double, double, int>(
  const double* x,
  const double* y,
  double* dist,
  int m,
  int n,
  int k,
  void* workspace,
  std::size_t worksize,
  hipStream_t stream,
  bool isRowMajor,
  double metric_arg);

template void distance<raft::distance::DistanceType::Linf, float, float, float, std::uint32_t>(
  const float* x,
  const float* y,
  float* dist,
  std::uint32_t m,
  std::uint32_t n,
  std::uint32_t k,
  void* workspace,
  std::size_t worksize,
  hipStream_t stream,
  bool isRowMajor,
  float metric_arg);

}  // namespace detail
}  // namespace distance
}  // namespace raft
