/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cluster_cost.cuh"
#include <raft/core/resources.hpp>
#include <raft/distance/distance_types.hpp>

namespace raft::runtime::cluster::kmeans {

void cluster_cost(raft::resources const& handle,
                  const double* X,
                  int n_samples,
                  int n_features,
                  int n_clusters,
                  const double* centroids,
                  double* cost)
{
  cluster_cost<double, int>(handle, X, n_samples, n_features, n_clusters, centroids, cost);
}
}  // namespace raft::runtime::cluster::kmeans
