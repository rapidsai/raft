/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/cluster/kmeans.cuh>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>

namespace raft::runtime::cluster::kmeans {

void init_plus_plus(raft::resources const& handle,
                    const raft::cluster::kmeans::KMeansParams& params,
                    raft::device_matrix_view<const float, int> X,
                    raft::device_matrix_view<float, int> centroids)
{
  rmm::device_uvector<char> workspace(0, resource::get_cuda_stream(handle));
  raft::cluster::kmeans::init_plus_plus<float, int>(handle, params, X, centroids, workspace);
}
}  // namespace raft::runtime::cluster::kmeans
