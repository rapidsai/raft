#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "fused_distance_min_arg.hpp"

#include <raft/core/device_mdarray.hpp>
#include <raft/core/kvp.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resource/thrust_policy.hpp>
#include <raft/core/resources.hpp>
#include <raft/distance/distance_types.hpp>
#include <raft/distance/fused_l2_nn.cuh>
#include <raft/linalg/norm.cuh>

#include <thrust/for_each.h>
#include <thrust/tuple.h>

namespace raft::runtime::distance {

[[deprecated("use fused_distance_nn_min_arg instead")]] void fused_l2_nn_min_arg(
  raft::resources const& handle,
  int* min,
  const float* x,
  const float* y,
  int m,
  int n,
  int k,
  bool sqrt)
{
  compute_fused_l2_nn_min_arg<float, int>(handle, min, x, y, m, n, k, sqrt);
}

[[deprecated("use fused_distance_nn_min_arg instead")]] void fused_l2_nn_min_arg(
  raft::resources const& handle,
  int* min,
  const double* x,
  const double* y,
  int m,
  int n,
  int k,
  bool sqrt)
{
  compute_fused_l2_nn_min_arg<double, int>(handle, min, x, y, m, n, k, sqrt);
}

}  // end namespace raft::runtime::distance
