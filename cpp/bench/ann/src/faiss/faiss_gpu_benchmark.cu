/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <memory>
#include <stdexcept>
#include <string>
#include <type_traits>
#include <utility>

#include "../common/ann_types.hpp"
#undef WARP_SIZE
#include "faiss_gpu_wrapper.h"
#define JSON_DIAGNOSTICS 1
#include <nlohmann/json.hpp>

namespace raft::bench::ann {

template <typename T>
void parse_base_build_param(const nlohmann::json& conf,
                            typename raft::bench::ann::FaissGpu<T>::BuildParam& param)
{
  param.nlist = conf.at("nlist");
  if (conf.contains("ratio")) { param.ratio = conf.at("ratio"); }
}

template <typename T>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::FaissGpuIVFFlat<T>::BuildParam& param)
{
  parse_base_build_param<T>(conf, param);
}

template <typename T>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::FaissGpuIVFPQ<T>::BuildParam& param)
{
  parse_base_build_param<T>(conf, param);
  param.M = conf.at("M");
  if (conf.contains("usePrecomputed")) {
    param.usePrecomputed = conf.at("usePrecomputed");
  } else {
    param.usePrecomputed = false;
  }
  if (conf.contains("useFloat16")) {
    param.useFloat16 = conf.at("useFloat16");
  } else {
    param.useFloat16 = false;
  }
}

template <typename T>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::FaissGpuIVFSQ<T>::BuildParam& param)
{
  parse_base_build_param<T>(conf, param);
  param.quantizer_type = conf.at("quantizer_type");
}

template <typename T>
void parse_search_param(const nlohmann::json& conf,
                        typename raft::bench::ann::FaissGpu<T>::SearchParam& param)
{
  param.nprobe = conf.at("nprobe");
  if (conf.contains("refine_ratio")) { param.refine_ratio = conf.at("refine_ratio"); }
}

template <typename T, template <typename> class Algo>
std::unique_ptr<raft::bench::ann::ANN<T>> make_algo(raft::bench::ann::Metric metric,
                                                    int dim,
                                                    const nlohmann::json& conf)
{
  typename Algo<T>::BuildParam param;
  parse_build_param<T>(conf, param);
  return std::make_unique<Algo<T>>(metric, dim, param);
}

template <typename T, template <typename> class Algo>
std::unique_ptr<raft::bench::ann::ANN<T>> make_algo(raft::bench::ann::Metric metric,
                                                    int dim,
                                                    const nlohmann::json& conf,
                                                    const std::vector<int>& dev_list)
{
  typename Algo<T>::BuildParam param;
  parse_build_param<T>(conf, param);

  (void)dev_list;
  return std::make_unique<Algo<T>>(metric, dim, param);
}

template <typename T>
std::unique_ptr<raft::bench::ann::ANN<T>> create_algo(const std::string& algo,
                                                      const std::string& distance,
                                                      int dim,
                                                      const nlohmann::json& conf,
                                                      const std::vector<int>& dev_list)
{
  // stop compiler warning; not all algorithms support multi-GPU so it may not be used
  (void)dev_list;

  std::unique_ptr<raft::bench::ann::ANN<T>> ann;

  if constexpr (std::is_same_v<T, float>) {
    raft::bench::ann::Metric metric = parse_metric(distance);
    if (algo == "faiss_gpu_ivf_flat") {
      ann = make_algo<T, raft::bench::ann::FaissGpuIVFFlat>(metric, dim, conf, dev_list);
    } else if (algo == "faiss_gpu_ivf_pq") {
      ann = make_algo<T, raft::bench::ann::FaissGpuIVFPQ>(metric, dim, conf);
    } else if (algo == "faiss_gpu_ivf_sq") {
      ann = make_algo<T, raft::bench::ann::FaissGpuIVFSQ>(metric, dim, conf);
    } else if (algo == "faiss_gpu_flat") {
      ann = std::make_unique<raft::bench::ann::FaissGpuFlat<T>>(metric, dim);
    }
  }

  if constexpr (std::is_same_v<T, uint8_t>) {}

  if (!ann) { throw std::runtime_error("invalid algo: '" + algo + "'"); }

  return ann;
}

template <typename T>
std::unique_ptr<typename raft::bench::ann::ANN<T>::AnnSearchParam> create_search_param(
  const std::string& algo, const nlohmann::json& conf)
{
  if (algo == "faiss_gpu_ivf_flat" || algo == "faiss_gpu_ivf_pq" || algo == "faiss_gpu_ivf_sq") {
    auto param = std::make_unique<typename raft::bench::ann::FaissGpu<T>::SearchParam>();
    parse_search_param<T>(conf, *param);
    return param;
  } else if (algo == "faiss_gpu_flat") {
    auto param = std::make_unique<typename raft::bench::ann::FaissGpu<T>::SearchParam>();
    return param;
  }
  // else
  throw std::runtime_error("invalid algo: '" + algo + "'");
}

}  // namespace raft::bench::ann

REGISTER_ALGO_INSTANCE(float);
REGISTER_ALGO_INSTANCE(std::int8_t);
REGISTER_ALGO_INSTANCE(std::uint8_t);

#ifdef ANN_BENCH_BUILD_MAIN
#include "../common/benchmark.hpp"
int main(int argc, char** argv) { return raft::bench::ann::run_main(argc, argv); }
#endif
