/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <memory>
#include <stdexcept>
#include <string>
#include <type_traits>
#include <utility>

#ifdef RAFT_COMPILED
#include <raft/neighbors/specializations.cuh>
#endif

#include "../common/ann_types.hpp"
#include "../common/benchmark_util.hpp"
#undef WARP_SIZE
#ifdef RAFT_ANN_BENCH_USE_RAFT_BFKNN
#include "raft_wrapper.h"
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
#include "raft_ivf_flat_wrapper.h"
extern template class raft::bench::ann::RaftIvfFlatGpu<float, int64_t>;
extern template class raft::bench::ann::RaftIvfFlatGpu<uint8_t, int64_t>;
extern template class raft::bench::ann::RaftIvfFlatGpu<int8_t, int64_t>;
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
#include "raft_ivf_pq_wrapper.h"
extern template class raft::bench::ann::RaftIvfPQ<float, int64_t>;
extern template class raft::bench::ann::RaftIvfPQ<uint8_t, int64_t>;
extern template class raft::bench::ann::RaftIvfPQ<int8_t, int64_t>;
#endif
#define JSON_DIAGNOSTICS 1
#include <nlohmann/json.hpp>

namespace raft::bench::ann {

#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
template <typename T, typename IdxT>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::RaftIvfFlatGpu<T, IdxT>::BuildParam& param)
{
  param.n_lists = conf.at("nlist");
  if (conf.contains("niter")) { param.kmeans_n_iters = conf.at("niter"); }
  if (conf.contains("ratio")) {
    param.kmeans_trainset_fraction = 1.0 / (double)conf.at("ratio");
    std::cout << "kmeans_trainset_fraction " << param.kmeans_trainset_fraction;
  }
}

template <typename T, typename IdxT>
void parse_search_param(const nlohmann::json& conf,
                        typename raft::bench::ann::RaftIvfFlatGpu<T, IdxT>::SearchParam& param)
{
  param.ivf_flat_params.n_probes = conf.at("nprobe");
}
#endif

#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
template <typename T, typename IdxT>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::RaftIvfPQ<T, IdxT>::BuildParam& param)
{
  param.n_lists = conf.at("nlist");
  if (conf.contains("niter")) { param.kmeans_n_iters = conf.at("niter"); }
  if (conf.contains("ratio")) { param.kmeans_trainset_fraction = 1.0 / (double)conf.at("ratio"); }
  if (conf.contains("pq_bits")) { param.pq_bits = conf.at("pq_bits"); }
  if (conf.contains("pq_dim")) { param.pq_dim = conf.at("pq_dim"); }
}

template <typename T, typename IdxT>
void parse_search_param(const nlohmann::json& conf,
                        typename raft::bench::ann::RaftIvfPQ<T, IdxT>::SearchParam& param)
{
  param.pq_param.n_probes = conf.at("numProbes");
  if (conf.contains("internalDistanceDtype")) {
    std::string type = conf.at("internalDistanceDtype");
    if (type == "float") {
      param.pq_param.internal_distance_dtype = HIP_R_32F;
    } else if (type == "half") {
      param.pq_param.internal_distance_dtype = HIP_R_16F;
    } else {
      throw std::runtime_error("internalDistanceDtype: '" + type +
                               "', should be either 'float' or 'half'");
    }
  } else {
    // set half as default type
    param.pq_param.internal_distance_dtype = HIP_R_16F;
  }

  if (conf.contains("smemLutDtype")) {
    std::string type = conf.at("smemLutDtype");
    if (type == "float") {
      param.pq_param.lut_dtype = HIP_R_32F;
    } else if (type == "half") {
      param.pq_param.lut_dtype = HIP_R_16F;
    } else if (type == "fp8") {
      param.pq_param.lut_dtype = HIP_R_8U;
    } else {
      throw std::runtime_error("smemLutDtype: '" + type +
                               "', should be either 'float', 'half' or 'fp8'");
    }
  } else {
    // set half as default
    param.pq_param.lut_dtype = HIP_R_16F;
  }
}
#endif

template <typename T, template <typename> class Algo>
std::unique_ptr<raft::bench::ann::ANN<T>> make_algo(raft::bench::ann::Metric metric,
                                                    int dim,
                                                    const nlohmann::json& conf)
{
  typename Algo<T>::BuildParam param;
  parse_build_param<T>(conf, param);
  return std::make_unique<Algo<T>>(metric, dim, param);
}

template <typename T, template <typename> class Algo>
std::unique_ptr<raft::bench::ann::ANN<T>> make_algo(raft::bench::ann::Metric metric,
                                                    int dim,
                                                    const nlohmann::json& conf,
                                                    const std::vector<int>& dev_list)
{
  typename Algo<T>::BuildParam param;
  parse_build_param<T>(conf, param);

  (void)dev_list;
  return std::make_unique<Algo<T>>(metric, dim, param);
}

template <typename T>
std::unique_ptr<raft::bench::ann::ANN<T>> create_algo(const std::string& algo,
                                                      const std::string& distance,
                                                      int dim,
                                                      float refine_ratio,
                                                      const nlohmann::json& conf,
                                                      const std::vector<int>& dev_list)
{
  // stop compiler warning; not all algorithms support multi-GPU so it may not be used
  (void)dev_list;

  raft::bench::ann::Metric metric = parse_metric(distance);
  std::unique_ptr<raft::bench::ann::ANN<T>> ann;

  if constexpr (std::is_same_v<T, float>) {
#ifdef RAFT_ANN_BENCH_USE_RAFT_BFKNN
    if (algo == "raft_bfknn") { ann = std::make_unique<raft::bench::ann::RaftGpu<T>>(metric, dim); }
#endif
  }

  if constexpr (std::is_same_v<T, uint8_t>) {}

#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
  if (algo == "raft_ivf_flat") {
    typename raft::bench::ann::RaftIvfFlatGpu<T, int64_t>::BuildParam param;
    parse_build_param<T, int64_t>(conf, param);
    ann = std::make_unique<raft::bench::ann::RaftIvfFlatGpu<T, int64_t>>(metric, dim, param);
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
  if (algo == "raft_ivf_pq") {
    typename raft::bench::ann::RaftIvfPQ<T, int64_t>::BuildParam param;
    parse_build_param<T, int64_t>(conf, param);
    ann =
      std::make_unique<raft::bench::ann::RaftIvfPQ<T, int64_t>>(metric, dim, param, refine_ratio);
  }
#endif
  if (!ann) { throw std::runtime_error("invalid algo: '" + algo + "'"); }

  if (refine_ratio > 1.0) {}
  return ann;
}

template <typename T>
std::unique_ptr<typename raft::bench::ann::ANN<T>::AnnSearchParam> create_search_param(
  const std::string& algo, const nlohmann::json& conf)
{
#ifdef RAFT_ANN_BENCH_USE_RAFT_BFKNN
  if (algo == "raft_bfknn") {
    auto param = std::make_unique<typename raft::bench::ann::ANN<T>::AnnSearchParam>();
    return param;
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
  if (algo == "raft_ivf_flat") {
    auto param =
      std::make_unique<typename raft::bench::ann::RaftIvfFlatGpu<T, int64_t>::SearchParam>();
    parse_search_param<T, int64_t>(conf, *param);
    return param;
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
  if (algo == "raft_ivf_pq") {
    auto param = std::make_unique<typename raft::bench::ann::RaftIvfPQ<T, int64_t>::SearchParam>();
    parse_search_param<T, int64_t>(conf, *param);
    return param;
  }
#endif
  // else
  throw std::runtime_error("invalid algo: '" + algo + "'");
}

};  // namespace raft::bench::ann

#include "../common/benchmark.hpp"

int main(int argc, char** argv) { return raft::bench::ann::run_main(argc, argv); }