/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../common/ann_types.hpp"

#include <algorithm>
#include <cmath>
#include <memory>
#include <stdexcept>
#include <string>
#include <type_traits>
#include <utility>

#undef WARP_SIZE
#ifdef RAFT_ANN_BENCH_USE_RAFT_BFKNN
#include "raft_wrapper.h"
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
#include "raft_ivf_flat_wrapper.h"
extern template class raft::bench::ann::RaftIvfFlatGpu<float, int64_t>;
extern template class raft::bench::ann::RaftIvfFlatGpu<uint8_t, int64_t>;
extern template class raft::bench::ann::RaftIvfFlatGpu<int8_t, int64_t>;
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
#include "raft_ivf_pq_wrapper.h"
extern template class raft::bench::ann::RaftIvfPQ<float, int64_t>;
extern template class raft::bench::ann::RaftIvfPQ<uint8_t, int64_t>;
extern template class raft::bench::ann::RaftIvfPQ<int8_t, int64_t>;
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_CAGRA
#include "raft_cagra_wrapper.h"
extern template class raft::bench::ann::RaftCagra<float, uint32_t>;
extern template class raft::bench::ann::RaftCagra<uint8_t, uint32_t>;
extern template class raft::bench::ann::RaftCagra<int8_t, uint32_t>;
#endif
#define JSON_DIAGNOSTICS 1
#include <nlohmann/json.hpp>

namespace raft::bench::ann {

#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
template <typename T, typename IdxT>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::RaftIvfFlatGpu<T, IdxT>::BuildParam& param)
{
  param.n_lists = conf.at("nlist");
  if (conf.contains("niter")) { param.kmeans_n_iters = conf.at("niter"); }
  if (conf.contains("ratio")) {
    param.kmeans_trainset_fraction = 1.0 / (double)conf.at("ratio");
    std::cout << "kmeans_trainset_fraction " << param.kmeans_trainset_fraction;
  }
}

template <typename T, typename IdxT>
void parse_search_param(const nlohmann::json& conf,
                        typename raft::bench::ann::RaftIvfFlatGpu<T, IdxT>::SearchParam& param)
{
  param.ivf_flat_params.n_probes = conf.at("nprobe");
}
#endif

#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
template <typename T, typename IdxT>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::RaftIvfPQ<T, IdxT>::BuildParam& param)
{
  param.n_lists = conf.at("nlist");
  if (conf.contains("niter")) { param.kmeans_n_iters = conf.at("niter"); }
  if (conf.contains("ratio")) { param.kmeans_trainset_fraction = 1.0 / (double)conf.at("ratio"); }
  if (conf.contains("pq_bits")) { param.pq_bits = conf.at("pq_bits"); }
  if (conf.contains("pq_dim")) { param.pq_dim = conf.at("pq_dim"); }
}

template <typename T, typename IdxT>
void parse_search_param(const nlohmann::json& conf,
                        typename raft::bench::ann::RaftIvfPQ<T, IdxT>::SearchParam& param)
{
  param.pq_param.n_probes = conf.at("nprobe");
  if (conf.contains("internalDistanceDtype")) {
    std::string type = conf.at("internalDistanceDtype");
    if (type == "float") {
      param.pq_param.internal_distance_dtype = HIP_R_32F;
    } else if (type == "half") {
      param.pq_param.internal_distance_dtype = HIP_R_16F;
    } else {
      throw std::runtime_error("internalDistanceDtype: '" + type +
                               "', should be either 'float' or 'half'");
    }
  } else {
    // set half as default type
    param.pq_param.internal_distance_dtype = HIP_R_16F;
  }

  if (conf.contains("smemLutDtype")) {
    std::string type = conf.at("smemLutDtype");
    if (type == "float") {
      param.pq_param.lut_dtype = HIP_R_32F;
    } else if (type == "half") {
      param.pq_param.lut_dtype = HIP_R_16F;
    } else if (type == "fp8") {
      param.pq_param.lut_dtype = HIP_R_8U;
    } else {
      throw std::runtime_error("smemLutDtype: '" + type +
                               "', should be either 'float', 'half' or 'fp8'");
    }
  } else {
    // set half as default
    param.pq_param.lut_dtype = HIP_R_16F;
  }
  if (conf.contains("refine_ratio")) {
    param.refine_ratio = conf.at("refine_ratio");
    if (param.refine_ratio < 1.0f) { throw std::runtime_error("refine_ratio should be >= 1.0"); }
  }
}
#endif

#ifdef RAFT_ANN_BENCH_USE_RAFT_CAGRA
template <typename T, typename IdxT>
void parse_build_param(const nlohmann::json& conf,
                       typename raft::bench::ann::RaftCagra<T, IdxT>::BuildParam& param)
{
  if (conf.contains("graph_degree")) {
    param.graph_degree              = conf.at("graph_degree");
    param.intermediate_graph_degree = param.graph_degree * 2;
  }
  if (conf.contains("intermediate_graph_degree")) {
    param.intermediate_graph_degree = conf.at("intermediate_graph_degree");
  }
}

template <typename T, typename IdxT>
void parse_search_param(const nlohmann::json& conf,
                        typename raft::bench::ann::RaftCagra<T, IdxT>::SearchParam& param)
{
  if (conf.contains("itopk")) { param.p.itopk_size = conf.at("itopk"); }
  if (conf.contains("search_width")) { param.p.search_width = conf.at("search_width"); }
  if (conf.contains("max_iterations")) { param.p.max_iterations = conf.at("max_iterations"); }
  if (conf.contains("algo")) {
    if (conf.at("algo") == "single_cta") {
      param.p.algo = raft::neighbors::experimental::cagra::search_algo::SINGLE_CTA;
    } else if (conf.at("algo") == "multi_cta") {
      param.p.algo = raft::neighbors::experimental::cagra::search_algo::MULTI_CTA;
    } else if (conf.at("algo") == "multi_kernel") {
      param.p.algo = raft::neighbors::experimental::cagra::search_algo::MULTI_KERNEL;
    } else if (conf.at("algo") == "auto") {
      param.p.algo = raft::neighbors::experimental::cagra::search_algo::AUTO;
    } else {
      std::string tmp = conf.at("algo");
      THROW("Invalid value for algo: %s", tmp.c_str());
    }
  }
}
#endif

template <typename T>
std::unique_ptr<raft::bench::ann::ANN<T>> create_algo(const std::string& algo,
                                                      const std::string& distance,
                                                      int dim,
                                                      const nlohmann::json& conf,
                                                      const std::vector<int>& dev_list)
{
  // stop compiler warning; not all algorithms support multi-GPU so it may not be used
  (void)dev_list;

  raft::bench::ann::Metric metric = parse_metric(distance);
  std::unique_ptr<raft::bench::ann::ANN<T>> ann;

  if constexpr (std::is_same_v<T, float>) {
#ifdef RAFT_ANN_BENCH_USE_RAFT_BFKNN
    if (algo == "raft_bfknn") { ann = std::make_unique<raft::bench::ann::RaftGpu<T>>(metric, dim); }
#endif
  }

  if constexpr (std::is_same_v<T, uint8_t>) {}

#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
  if (algo == "raft_ivf_flat") {
    typename raft::bench::ann::RaftIvfFlatGpu<T, int64_t>::BuildParam param;
    parse_build_param<T, int64_t>(conf, param);
    ann = std::make_unique<raft::bench::ann::RaftIvfFlatGpu<T, int64_t>>(metric, dim, param);
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
  if (algo == "raft_ivf_pq") {
    typename raft::bench::ann::RaftIvfPQ<T, int64_t>::BuildParam param;
    parse_build_param<T, int64_t>(conf, param);
    ann = std::make_unique<raft::bench::ann::RaftIvfPQ<T, int64_t>>(metric, dim, param);
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_CAGRA
  if (algo == "raft_cagra") {
    typename raft::bench::ann::RaftCagra<T, uint32_t>::BuildParam param;
    parse_build_param<T, uint32_t>(conf, param);
    ann = std::make_unique<raft::bench::ann::RaftCagra<T, uint32_t>>(metric, dim, param);
  }
#endif
  if (!ann) { throw std::runtime_error("invalid algo: '" + algo + "'"); }

  return ann;
}

template <typename T>
std::unique_ptr<typename raft::bench::ann::ANN<T>::AnnSearchParam> create_search_param(
  const std::string& algo, const nlohmann::json& conf)
{
#ifdef RAFT_ANN_BENCH_USE_RAFT_BFKNN
  if (algo == "raft_bfknn") {
    auto param = std::make_unique<typename raft::bench::ann::ANN<T>::AnnSearchParam>();
    return param;
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_FLAT
  if (algo == "raft_ivf_flat") {
    auto param =
      std::make_unique<typename raft::bench::ann::RaftIvfFlatGpu<T, int64_t>::SearchParam>();
    parse_search_param<T, int64_t>(conf, *param);
    return param;
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_IVF_PQ
  if (algo == "raft_ivf_pq") {
    auto param = std::make_unique<typename raft::bench::ann::RaftIvfPQ<T, int64_t>::SearchParam>();
    parse_search_param<T, int64_t>(conf, *param);
    return param;
  }
#endif
#ifdef RAFT_ANN_BENCH_USE_RAFT_CAGRA
  if (algo == "raft_cagra") {
    auto param = std::make_unique<typename raft::bench::ann::RaftCagra<T, uint32_t>::SearchParam>();
    parse_search_param<T, uint32_t>(conf, *param);
    return param;
  }
#endif
  // else
  throw std::runtime_error("invalid algo: '" + algo + "'");
}

};  // namespace raft::bench::ann

REGISTER_ALGO_INSTANCE(float);
REGISTER_ALGO_INSTANCE(std::int8_t);
REGISTER_ALGO_INSTANCE(std::uint8_t);

#ifdef ANN_BENCH_BUILD_MAIN
#include "../common/benchmark.hpp"
int main(int argc, char** argv) { return raft::bench::ann::run_main(argc, argv); }
#endif
