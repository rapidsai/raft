#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
#include <stdio.h>

#include <common/benchmark.hpp>
#include <raft/sparse/convert/csr.cuh>
#include <rmm/device_uvector.hpp>

namespace raft::bench::sparse {

struct bench_param {
  size_t num_cols;
  size_t num_rows;
  size_t divisor;
};

template <typename index_t>
__global__ void init_adj(bool* adj, index_t num_rows, index_t num_cols, int divisor)
{
  index_t r = blockDim.y * blockIdx.y + threadIdx.y;
  index_t c = blockDim.x * blockIdx.x + threadIdx.x;

  if (r < num_rows && c < num_cols) { adj[r * num_cols + c] = c % divisor == 0; }
}

template <typename index_t>
struct bench_base : public fixture {
  bench_base(const bench_param& p)
    : params(p),
      handle(stream),
      adj(p.num_rows * p.num_cols, stream),
      row_ind(p.num_rows, stream),
      row_ind_host(p.num_rows),
      row_counters(p.num_rows, stream),
      col_ind(p.num_rows * p.num_cols,
              stream)  // This is over-dimensioned because nnz is unknown at this point
  {
    dim3 block(32, 32);
    dim3 grid(p.num_cols / 32 + 1, p.num_rows / 32 + 1);
    init_adj<index_t><<<grid, block, 0, stream>>>(adj.data(), p.num_rows, p.num_cols, p.divisor);

    std::vector<index_t> row_ind_host(p.num_rows);
    for (size_t i = 0; i < row_ind_host.size(); ++i) {
      size_t nnz_per_row = raft::ceildiv(p.num_cols, p.divisor);
      row_ind_host[i]    = nnz_per_row * i;
    }
    raft::update_device(row_ind.data(), row_ind_host.data(), row_ind.size(), stream);
  }

  void run_benchmark(::benchmark::State& state) override
  {
    loop_on_state(state, [this]() {
      raft::sparse::convert::dense_bool_to_unsorted_csr<index_t>(handle,
                                                                 adj.data(),
                                                                 row_ind.data(),
                                                                 params.num_rows,
                                                                 params.num_cols,
                                                                 row_counters.data(),
                                                                 col_ind.data());
    });

    // Estimate bandwidth:
    index_t num_entries = params.num_rows * params.num_cols;
    index_t bytes_read  = num_entries * sizeof(bool);
    index_t bytes_write = num_entries / params.divisor * sizeof(index_t);

    state.counters["BW"]      = benchmark::Counter(bytes_read + bytes_write,
                                              benchmark::Counter::kIsIterationInvariantRate,
                                              benchmark::Counter::OneK::kIs1024);
    state.counters["BW read"] = benchmark::Counter(
      bytes_read, benchmark::Counter::kIsIterationInvariantRate, benchmark::Counter::OneK::kIs1024);
    state.counters["BW write"] = benchmark::Counter(bytes_write,
                                                    benchmark::Counter::kIsIterationInvariantRate,
                                                    benchmark::Counter::OneK::kIs1024);

    state.counters["Fraction nz"] = benchmark::Counter(100.0 / ((double)params.divisor));
    state.counters["Columns"]     = benchmark::Counter(params.num_cols);
    state.counters["Rows"]        = benchmark::Counter(params.num_rows);
  }

 protected:
  raft::handle_t handle;
  bench_param params;
  rmm::device_uvector<bool> adj;
  rmm::device_uvector<index_t> row_ind;
  std::vector<index_t> row_ind_host;
  rmm::device_uvector<index_t> row_counters;
  rmm::device_uvector<index_t> col_ind;
};  // struct bench_base

const int64_t num_cols = 1 << 30;

const std::vector<bench_param> bench_params = {
  {num_cols, 1, 8},
  {num_cols >> 3, 1 << 3, 8},
  {num_cols >> 6, 1 << 6, 8},

  {num_cols, 1, 64},
  {num_cols >> 3, 1 << 3, 64},
  {num_cols >> 6, 1 << 6, 64},

  {num_cols, 1, 2048},
  {num_cols >> 3, 1 << 3, 2048},
  {num_cols >> 6, 1 << 6, 2048},
};

RAFT_BENCH_REGISTER(bench_base<int64_t>, "", bench_params);
// RAFT_BENCH_REGISTER(bench_base<int>, "", bench_params);

}  // namespace raft::bench::sparse
