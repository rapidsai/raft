/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/benchmark.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/matrix/gather.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/itertools.hpp>

#include <rmm/device_uvector.hpp>

namespace raft::bench::matrix {

template <typename IdxT>
struct GatherParams {
  IdxT rows, cols, map_length;
};

template <typename IdxT>
inline auto operator<<(std::ostream& os, const GatherParams<IdxT>& p) -> std::ostream&
{
  os << p.rows << "#" << p.cols << "#" << p.map_length;
  return os;
}

template <typename T, typename MapT, typename IdxT, bool Conditional = false>
struct Gather : public fixture {
  Gather(const GatherParams<IdxT>& p)
    : params(p), matrix(this->handle), map(this->handle), out(this->handle), stencil(this->handle)
  {
  }

  void allocate_data(const ::benchmark::State& state) override
  {
    matrix  = raft::make_device_matrix<T, IdxT>(handle, params.rows, params.cols);
    map     = raft::make_device_vector<MapT, IdxT>(handle, params.map_length);
    out     = raft::make_device_matrix<T, IdxT>(handle, params.map_length, params.cols);
    stencil = raft::make_device_vector<T, IdxT>(handle, Conditional ? params.map_length : IdxT(0));

    raft::random::RngState rng{1234};
    raft::random::uniform(
      handle, rng, matrix.data_handle(), params.rows * params.cols, T(-1), T(1));
    raft::random::uniformInt(
      handle, rng, map.data_handle(), params.map_length, (MapT)0, (MapT)params.rows);
    if constexpr (Conditional) {
      raft::random::uniform(handle, rng, stencil.data_handle(), params.map_length, T(-1), T(1));
    }
    resource::sync_stream(handle, stream);
  }

  void run_benchmark(::benchmark::State& state) override
  {
    std::ostringstream label_stream;
    label_stream << params;
    state.SetLabel(label_stream.str());

    loop_on_state(state, [this]() {
      auto matrix_const_view = raft::make_const_mdspan(matrix.view());
      auto map_const_view    = raft::make_const_mdspan(map.view());
      if constexpr (Conditional) {
        auto stencil_const_view = raft::make_const_mdspan(stencil.view());
        auto pred_op            = raft::plug_const_op(T(0.0), raft::greater_op());
        raft::matrix::gather_if(
          handle, matrix_const_view, out.view(), map_const_view, stencil_const_view, pred_op);
      } else {
        raft::matrix::gather(handle, matrix_const_view, map_const_view, out.view());
      }
    });
  }

 private:
  GatherParams<IdxT> params;
  raft::device_matrix<T, IdxT> matrix, out;
  raft::device_vector<T, IdxT> stencil;
  raft::device_vector<MapT, IdxT> map;
};  // struct Gather

template <typename T, typename MapT, typename IdxT>
using GatherIf = Gather<T, MapT, IdxT, true>;

const std::vector<GatherParams<int64_t>> gather_inputs_i64 =
  raft::util::itertools::product<GatherParams<int64_t>>(
    {1000000}, {10, 20, 50, 100, 200, 500}, {1000, 10000, 100000, 1000000});

RAFT_BENCH_REGISTER((Gather<float, uint32_t, int64_t>), "", gather_inputs_i64);
RAFT_BENCH_REGISTER((Gather<double, uint32_t, int64_t>), "", gather_inputs_i64);
RAFT_BENCH_REGISTER((GatherIf<float, uint32_t, int64_t>), "", gather_inputs_i64);
RAFT_BENCH_REGISTER((GatherIf<double, uint32_t, int64_t>), "", gather_inputs_i64);
}  // namespace raft::bench::matrix
