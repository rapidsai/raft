/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <common/benchmark.hpp>

#include <raft/core/device_csr_matrix.hpp>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_mdspan.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/matrix/copy.cuh>
#include <raft/matrix/select_k.cuh>
#include <raft/random/make_blobs.cuh>
#include <raft/random/rng_state.hpp>
#include <raft/sparse/convert/csr.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/itertools.hpp>

#include <rmm/device_uvector.hpp>

#include <random>
#include <sstream>
#include <unordered_set>
#include <vector>

namespace raft::bench::sparse {

template <typename index_t>
struct bench_param {
  index_t n_rows;
  index_t n_cols;
  index_t top_k;
  float sparsity;
  bool select_min         = true;
  bool customized_indices = false;
};

template <typename index_t>
inline auto operator<<(std::ostream& os, const bench_param<index_t>& params) -> std::ostream&
{
  os << " rows*cols=" << params.n_rows << "*" << params.n_cols << "\ttop_k=" << params.top_k
     << "\tsparsity=" << params.sparsity;
  return os;
}

template <typename value_t, typename index_t>
struct SelectKCsrTest : public fixture {
  SelectKCsrTest(const bench_param<index_t>& p)
    : fixture(true),
      params(p),
      handle(stream),
      values_d(0, stream),
      indptr_d(0, stream),
      indices_d(0, stream),
      customized_indices_d(0, stream),
      dst_values_d(0, stream),
      dst_indices_d(0, stream)
  {
    std::vector<bool> dense_values_h(params.n_rows * params.n_cols, false);
    nnz = create_sparse_matrix(params.n_rows, params.n_cols, params.sparsity, dense_values_h);

    std::vector<index_t> indices_h(nnz);
    std::vector<index_t> customized_indices_h(nnz);
    std::vector<index_t> indptr_h(params.n_rows + 1);

    convert_to_csr(dense_values_h, params.n_rows, params.n_cols, indices_h, indptr_h);

    std::vector<value_t> dst_values_h(params.n_rows * params.top_k, static_cast<value_t>(2.0f));
    std::vector<index_t> dst_indices_h(params.n_rows * params.top_k,
                                       static_cast<index_t>(params.n_rows * params.n_cols * 100));

    dst_values_d.resize(params.n_rows * params.top_k, stream);
    dst_indices_d.resize(params.n_rows * params.top_k, stream);
    values_d.resize(nnz, stream);

    if (nnz) {
      auto blobs_values = raft::make_device_matrix<value_t, index_t>(handle, 1, nnz);
      auto labels       = raft::make_device_vector<index_t, index_t>(handle, 1);

      raft::random::make_blobs<value_t, index_t>(blobs_values.data_handle(),
                                                 labels.data_handle(),
                                                 1,
                                                 nnz,
                                                 1,
                                                 stream,
                                                 false,
                                                 nullptr,
                                                 nullptr,
                                                 value_t(1.0),
                                                 false,
                                                 value_t(-10.0f),
                                                 value_t(10.0f),
                                                 uint64_t(2024));
      raft::copy(values_d.data(), blobs_values.data_handle(), nnz, stream);
      resource::sync_stream(handle);
    }

    indices_d.resize(nnz, stream);
    indptr_d.resize(params.n_rows + 1, stream);

    update_device(indices_d.data(), indices_h.data(), indices_h.size(), stream);
    update_device(indptr_d.data(), indptr_h.data(), indptr_h.size(), stream);

    if (params.customized_indices) {
      customized_indices_d.resize(nnz, stream);
      update_device(customized_indices_d.data(),
                    customized_indices_h.data(),
                    customized_indices_h.size(),
                    stream);
    }
  }

  index_t create_sparse_matrix(index_t m, index_t n, value_t sparsity, std::vector<bool>& matrix)
  {
    index_t total_elements = static_cast<index_t>(m * n);
    index_t num_ones       = static_cast<index_t>((total_elements * 1.0f) * sparsity);
    index_t res            = num_ones;

    for (index_t i = 0; i < total_elements; ++i) {
      matrix[i] = false;
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis_idx(0, total_elements - 1);

    while (num_ones > 0) {
      size_t index = dis_idx(gen);
      if (matrix[index] == false) {
        matrix[index] = true;
        num_ones--;
      }
    }
    return res;
  }

  void convert_to_csr(std::vector<bool>& matrix,
                      index_t rows,
                      index_t cols,
                      std::vector<index_t>& indices,
                      std::vector<index_t>& indptr)
  {
    index_t offset_indptr   = 0;
    index_t offset_values   = 0;
    indptr[offset_indptr++] = 0;

    for (index_t i = 0; i < rows; ++i) {
      for (index_t j = 0; j < cols; ++j) {
        if (matrix[i * cols + j]) {
          indices[offset_values] = static_cast<index_t>(j);
          offset_values++;
        }
      }
      indptr[offset_indptr++] = static_cast<index_t>(offset_values);
    }
  }

  template <typename data_t>
  std::optional<data_t> get_opt_var(data_t x)
  {
    if (params.customized_indices) {
      return x;
    } else {
      return std::nullopt;
    }
  }

  void run_benchmark(::benchmark::State& state) override
  {
    std::ostringstream label_stream;
    label_stream << params;
    state.SetLabel(label_stream.str());

    auto in_val_structure = raft::make_device_compressed_structure_view<index_t, index_t, index_t>(
      indptr_d.data(),
      indices_d.data(),
      params.n_rows,
      params.n_cols,
      static_cast<index_t>(indices_d.size()));

    auto in_val =
      raft::make_device_csr_matrix_view<const value_t>(values_d.data(), in_val_structure);

    std::optional<raft::device_vector_view<const index_t, index_t>> in_idx;

    in_idx = get_opt_var(
      raft::make_device_vector_view<const index_t, index_t>(customized_indices_d.data(), nnz));

    auto out_val = raft::make_device_matrix_view<value_t, index_t, raft::row_major>(
      dst_values_d.data(), params.n_rows, params.top_k);
    auto out_idx = raft::make_device_matrix_view<index_t, index_t, raft::row_major>(
      dst_indices_d.data(), params.n_rows, params.top_k);

    raft::matrix::select_k(handle, in_val, in_idx, out_val, out_idx, params.select_min);
    resource::sync_stream(handle);
    loop_on_state(state, [this, &in_val, &in_idx, &out_val, &out_idx]() {
      raft::matrix::select_k(handle, in_val, in_idx, out_val, out_idx, params.select_min);
      resource::sync_stream(handle);
    });
  }

 protected:
  const raft::device_resources handle;

  bench_param<index_t> params;
  index_t nnz;

  rmm::device_uvector<value_t> values_d;
  rmm::device_uvector<index_t> indptr_d;
  rmm::device_uvector<index_t> indices_d;
  rmm::device_uvector<index_t> customized_indices_d;

  rmm::device_uvector<value_t> dst_values_d;
  rmm::device_uvector<index_t> dst_indices_d;
};  // struct SelectKCsrTest

template <typename index_t>
const std::vector<bench_param<index_t>> getInputs()
{
  std::vector<bench_param<index_t>> param_vec;
  struct TestParams {
    index_t m;
    index_t n;
    index_t k;
    float sparsity;
  };

  const std::vector<TestParams> params_group =
    raft::util::itertools::product<TestParams>({index_t(10), index_t(1024)},
                                               {index_t(1024 * 10), index_t(1024 * 1024)},
                                               {index_t(128), index_t(100), index_t(2048)},
                                               {0.1f, 0.2f, 0.5f});

  param_vec.reserve(params_group.size());
  for (TestParams params : params_group) {
    param_vec.push_back(bench_param<index_t>({params.m, params.n, params.k, params.sparsity}));
  }
  return param_vec;
}

RAFT_BENCH_REGISTER((SelectKCsrTest<float, int>), "", getInputs<int>());

}  // namespace raft::bench::sparse
