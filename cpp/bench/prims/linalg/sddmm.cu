/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <common/benchmark.hpp>
#include <hipsparse.h>
#include <raft/core/device_resources.hpp>
#include <raft/core/resource/cublas_handle.hpp>
#include <raft/distance/distance.cuh>
#include <raft/distance/distance_types.hpp>
#include <raft/random/rng.cuh>
#include <raft/sparse/linalg/sddmm.hpp>
#include <raft/util/itertools.hpp>

#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>

#include <random>
#include <sstream>
#include <vector>

namespace raft::bench::linalg {

template <typename ValueType>
struct SDDMMBenchParams {
  size_t m;
  size_t k;
  size_t n;
  float sparsity;
  bool transpose_a;
  bool transpose_b;
  ValueType alpha = 1.0;
  ValueType beta  = 0.0;
};

enum Alg { SDDMM, Inner };

template <typename ValueType>
inline auto operator<<(std::ostream& os, const SDDMMBenchParams<ValueType>& params) -> std::ostream&
{
  os << " m*k*n=" << params.m << "*" << params.k << "*" << params.n
     << "\tsparsity=" << params.sparsity << "\ttrans_a=" << (params.transpose_a ? "T" : "F")
     << " trans_b=" << (params.transpose_b ? "T" : "F");
  return os;
}

template <typename ValueType,
          typename LayoutPolicyA = row_major,
          typename LayoutPolicyB = col_major,
          const int SDDMMorInner = Alg::SDDMM,
          typename IndexType     = int64_t>
struct SDDMMBench : public fixture {
  SDDMMBench(const SDDMMBenchParams<ValueType>& p)
    : fixture(true),
      params(p),
      handle(stream),
      a_data_d(0, stream),
      b_data_d(0, stream),
      c_indptr_d(0, stream),
      c_indices_d(0, stream),
      c_data_d(0, stream),
      c_dense_data_d(0, stream)
  {
    a_data_d.resize(params.m * params.k, stream);
    b_data_d.resize(params.k * params.n, stream);

    raft::random::RngState rng(2024ULL);
    raft::random::uniform(
      handle, rng, a_data_d.data(), params.m * params.k, ValueType(-1.0), ValueType(1.0));
    raft::random::uniform(
      handle, rng, b_data_d.data(), params.k * params.n, ValueType(-1.0), ValueType(1.0));

    std::vector<bool> c_dense_data_h(params.m * params.n);

    c_true_nnz = create_sparse_matrix(c_dense_data_h);
    std::vector<ValueType> values(c_true_nnz);
    std::vector<IndexType> indices(c_true_nnz);
    std::vector<IndexType> indptr(params.m + 1);

    c_data_d.resize(c_true_nnz, stream);
    c_indptr_d.resize(params.m + 1, stream);
    c_indices_d.resize(c_true_nnz, stream);

    if (SDDMMorInner == Alg::Inner) { c_dense_data_d.resize(params.m * params.n, stream); }

    convert_to_csr(c_dense_data_h, params.m, params.n, values, indices, indptr);
    RAFT_EXPECTS(c_true_nnz == c_indices_d.size(),
                 "Something wrong. The c_true_nnz != c_indices_d.size()!");

    update_device(c_data_d.data(), values.data(), c_true_nnz, stream);
    update_device(c_indices_d.data(), indices.data(), c_true_nnz, stream);
    update_device(c_indptr_d.data(), indptr.data(), params.m + 1, stream);
  }

  void convert_to_csr(std::vector<bool>& matrix,
                      IndexType rows,
                      IndexType cols,
                      std::vector<ValueType>& values,
                      std::vector<IndexType>& indices,
                      std::vector<IndexType>& indptr)
  {
    IndexType offset_indptr = 0;
    IndexType offset_values = 0;
    indptr[offset_indptr++] = 0;

    for (IndexType i = 0; i < rows; ++i) {
      for (IndexType j = 0; j < cols; ++j) {
        if (matrix[i * cols + j]) {
          values[offset_values]  = static_cast<ValueType>(1.0);
          indices[offset_values] = static_cast<IndexType>(j);
          offset_values++;
        }
      }
      indptr[offset_indptr++] = static_cast<IndexType>(offset_values);
    }
  }

  size_t create_sparse_matrix(std::vector<bool>& matrix)
  {
    size_t total_elements = static_cast<size_t>(params.m * params.n);
    size_t num_ones       = static_cast<size_t>((total_elements * 1.0f) * params.sparsity);
    size_t res            = num_ones;

    for (size_t i = 0; i < total_elements; ++i) {
      matrix[i] = false;
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, total_elements - 1);

    while (num_ones > 0) {
      size_t index = dis(gen);

      if (matrix[index] == false) {
        matrix[index] = true;
        num_ones--;
      }
    }
    return res;
  }

  ~SDDMMBench() {}

  void run_benchmark(::benchmark::State& state) override
  {
    std::ostringstream label_stream;
    label_stream << params;
    state.SetLabel(label_stream.str());

    auto a = raft::make_device_matrix_view<const ValueType, IndexType, LayoutPolicyA>(
      a_data_d.data(),
      (!params.transpose_a ? params.m : params.k),
      (!params.transpose_a ? params.k : params.m));

    auto b = raft::make_device_matrix_view<const ValueType, IndexType, LayoutPolicyB>(
      b_data_d.data(),
      (!params.transpose_b ? params.k : params.n),
      (!params.transpose_b ? params.n : params.k));

    auto c_structure = raft::make_device_compressed_structure_view<int64_t, int64_t, int64_t>(
      c_indptr_d.data(),
      c_indices_d.data(),
      params.m,
      params.n,
      static_cast<IndexType>(c_indices_d.size()));

    auto c = raft::make_device_csr_matrix_view<ValueType>(c_data_d.data(), c_structure);
    raft::resource::get_cusparse_handle(handle);

    resource::sync_stream(handle);

    auto op_a = params.transpose_a ? raft::linalg::Operation::TRANSPOSE
                                   : raft::linalg::Operation::NON_TRANSPOSE;
    auto op_b = params.transpose_b ? raft::linalg::Operation::TRANSPOSE
                                   : raft::linalg::Operation::NON_TRANSPOSE;

    raft::sparse::linalg::sddmm(handle,
                                a,
                                b,
                                c,
                                op_a,
                                op_b,
                                raft::make_host_scalar_view<ValueType>(&params.alpha),
                                raft::make_host_scalar_view<ValueType>(&params.beta));
    resource::sync_stream(handle);

    loop_on_state(state, [this, &a, &b, &c, &op_a, &op_b]() {
      if (SDDMMorInner == Alg::SDDMM) {
        raft::sparse::linalg::sddmm(handle,
                                    a,
                                    b,
                                    c,
                                    op_a,
                                    op_b,
                                    raft::make_host_scalar_view<ValueType>(&params.alpha),
                                    raft::make_host_scalar_view<ValueType>(&params.beta));
        resource::sync_stream(handle);
      } else {
        raft::distance::pairwise_distance(handle,
                                          a_data_d.data(),
                                          b_data_d.data(),
                                          c_dense_data_d.data(),
                                          static_cast<int>(params.m),
                                          static_cast<int>(params.n),
                                          static_cast<int>(params.k),
                                          raft::distance::DistanceType::InnerProduct,
                                          std::is_same_v<LayoutPolicyA, row_major>);
        resource::sync_stream(handle);
      }
    });
  }

 private:
  const raft::device_resources handle;
  SDDMMBenchParams<ValueType> params;

  rmm::device_uvector<ValueType> a_data_d;
  rmm::device_uvector<ValueType> b_data_d;
  rmm::device_uvector<ValueType> c_dense_data_d;

  size_t c_true_nnz = 0;
  rmm::device_uvector<IndexType> c_indptr_d;
  rmm::device_uvector<IndexType> c_indices_d;
  rmm::device_uvector<ValueType> c_data_d;
};

template <typename ValueType>
static std::vector<SDDMMBenchParams<ValueType>> getInputs()
{
  std::vector<SDDMMBenchParams<ValueType>> param_vec;
  struct TestParams {
    bool transpose_a;
    bool transpose_b;
    size_t m;
    size_t k;
    size_t n;
    float sparsity;
  };

  const std::vector<TestParams> params_group =
    raft::util::itertools::product<TestParams>({false, true},
                                               {false, true},
                                               {size_t(10), size_t(1024)},
                                               {size_t(128), size_t(1024)},
                                               {size_t(1024 * 1024)},
                                               {0.01f, 0.1f, 0.2f, 0.5f});

  param_vec.reserve(params_group.size());
  for (TestParams params : params_group) {
    param_vec.push_back(SDDMMBenchParams<ValueType>(
      {params.m, params.k, params.n, params.sparsity, params.transpose_a, params.transpose_b}));
  }
  return param_vec;
}

RAFT_BENCH_REGISTER((SDDMMBench<float, row_major, col_major, Alg::SDDMM>), "", getInputs<float>());
RAFT_BENCH_REGISTER((SDDMMBench<float, col_major, row_major, Alg::SDDMM>), "", getInputs<float>());
RAFT_BENCH_REGISTER((SDDMMBench<float, row_major, row_major, Alg::SDDMM>), "", getInputs<float>());
RAFT_BENCH_REGISTER((SDDMMBench<float, col_major, col_major, Alg::SDDMM>), "", getInputs<float>());

RAFT_BENCH_REGISTER((SDDMMBench<float, row_major, col_major, Alg::Inner>), "", getInputs<float>());

}  // namespace raft::bench::linalg
