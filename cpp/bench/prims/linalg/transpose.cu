/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/benchmark.hpp>

#include <raft/core/resource/hip_stream.hpp>
#include <raft/linalg/matrix_vector_op.cuh>
#include <raft/linalg/transpose.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/itertools.hpp>

#include <rmm/device_uvector.hpp>

namespace raft::bench::linalg {

template <typename IdxT>
struct transpose_input {
  IdxT rows, cols;
};

template <typename IdxT>
inline auto operator<<(std::ostream& os, const transpose_input<IdxT>& p) -> std::ostream&
{
  os << p.rows << "#" << p.cols;
  return os;
}

template <typename T, typename IdxT, typename Layout>
struct TransposeBench : public fixture {
  TransposeBench(const transpose_input<IdxT>& p)
    : params(p), in(p.rows * p.cols, stream), out(p.rows * p.cols, stream)
  {
    raft::random::RngState rng{1234};
    raft::random::uniform(handle, rng, in.data(), p.rows * p.cols, (T)-10.0, (T)10.0);
  }

  void run_benchmark(::benchmark::State& state) override
  {
    std::ostringstream label_stream;
    label_stream << params;
    state.SetLabel(label_stream.str());

    loop_on_state(state, [this]() {
      auto input_view =
        raft::make_device_matrix_view<T, IdxT, Layout>(in.data(), params.rows, params.cols);
      auto output_view = raft::make_device_vector_view<T, IdxT, Layout>(out.data(), params.rows);
      raft::linalg::transpose(handle,
                              input_view.data_handle(),
                              output_view.data_handle(),
                              params.rows,
                              params.cols,
                              handle.get_stream());
    });
  }

 private:
  transpose_input<IdxT> params;
  rmm::device_uvector<T> in, out;
};  // struct TransposeBench

const std::vector<transpose_input<int>> transpose_inputs_i32 =
  raft::util::itertools::product<transpose_input<int>>({10, 128, 256, 512, 1024},
                                                       {10000, 100000, 1000000});

RAFT_BENCH_REGISTER((TransposeBench<float, int, raft::row_major>), "", transpose_inputs_i32);
RAFT_BENCH_REGISTER((TransposeBench<half, int, raft::row_major>), "", transpose_inputs_i32);

RAFT_BENCH_REGISTER((TransposeBench<float, int, raft::col_major>), "", transpose_inputs_i32);
RAFT_BENCH_REGISTER((TransposeBench<half, int, raft::col_major>), "", transpose_inputs_i32);

}  // namespace raft::bench::linalg
