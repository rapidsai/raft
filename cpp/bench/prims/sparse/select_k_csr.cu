/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <common/benchmark.hpp>

#include <raft/core/device_csr_matrix.hpp>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_mdspan.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/core/resources.hpp>
#include <raft/matrix/copy.cuh>
#include <raft/random/make_blobs.cuh>
#include <raft/random/rng_state.hpp>
#include <raft/sparse/convert/csr.cuh>
#include <raft/sparse/matrix/select_k.cuh>
#include <raft/util/cuda_utils.cuh>
#include <raft/util/itertools.hpp>

#include <rmm/device_uvector.hpp>

#include <random>
#include <sstream>
#include <unordered_set>
#include <vector>

namespace raft::bench::sparse {

template <typename index_t>
struct bench_param {
  index_t n_rows;
  index_t n_cols;
  index_t top_k;
  float sparsity;
  bool select_min         = true;
  bool customized_indices = false;
};

template <typename index_t>
inline auto operator<<(std::ostream& os, const bench_param<index_t>& params) -> std::ostream&
{
  os << params.n_rows << "#" << params.n_cols << "#" << params.top_k << "#" << params.sparsity;
  return os;
}

template <typename value_t, typename index_t>
struct SelectKCsrTest : public fixture {
  SelectKCsrTest(const bench_param<index_t>& p)
    : fixture(true),
      params(p),
      handle(stream),
      values_d(0, stream),
      indptr_d(0, stream),
      indices_d(0, stream),
      customized_indices_d(0, stream),
      dst_values_d(0, stream),
      dst_indices_d(0, stream)
  {
    std::vector<bool> dense_values_h(params.n_rows * params.n_cols);
    nnz = create_sparse_matrix(params.n_rows, params.n_cols, params.sparsity, dense_values_h);

    std::vector<index_t> indices_h(nnz);
    std::vector<index_t> customized_indices_h(nnz);
    std::vector<index_t> indptr_h(params.n_rows + 1);

    convert_to_csr(dense_values_h, params.n_rows, params.n_cols, indices_h, indptr_h);

    std::vector<value_t> dst_values_h(params.n_rows * params.top_k, static_cast<value_t>(2.0f));
    std::vector<index_t> dst_indices_h(params.n_rows * params.top_k,
                                       static_cast<index_t>(params.n_rows * params.n_cols * 100));

    dst_values_d.resize(params.n_rows * params.top_k, stream);
    dst_indices_d.resize(params.n_rows * params.top_k, stream);
    values_d.resize(nnz, stream);

    if (nnz) {
      auto blobs_values = raft::make_device_matrix<value_t, index_t>(handle, 1, nnz);
      auto labels       = raft::make_device_vector<index_t, index_t>(handle, 1);

      raft::random::make_blobs<value_t, index_t>(blobs_values.data_handle(),
                                                 labels.data_handle(),
                                                 1,
                                                 nnz,
                                                 1,
                                                 stream,
                                                 false,
                                                 nullptr,
                                                 nullptr,
                                                 value_t(1.0),
                                                 false,
                                                 value_t(-10.0f),
                                                 value_t(10.0f),
                                                 uint64_t(2024));
      raft::copy(values_d.data(), blobs_values.data_handle(), nnz, stream);
      resource::sync_stream(handle);
    }

    indices_d.resize(nnz, stream);
    indptr_d.resize(params.n_rows + 1, stream);

    update_device(indices_d.data(), indices_h.data(), indices_h.size(), stream);
    update_device(indptr_d.data(), indptr_h.data(), indptr_h.size(), stream);

    if (params.customized_indices) {
      customized_indices_d.resize(nnz, stream);
      update_device(customized_indices_d.data(),
                    customized_indices_h.data(),
                    customized_indices_h.size(),
                    stream);
    }
  }

  index_t create_sparse_matrix(index_t m, index_t n, value_t sparsity, std::vector<bool>& matrix)
  {
    index_t total_elements = static_cast<index_t>(m * n);
    index_t num_ones       = static_cast<index_t>((total_elements * 1.0f) * sparsity);
    index_t res            = num_ones;

    for (index_t i = 0; i < total_elements; ++i) {
      matrix[i] = false;
    }

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis_idx(0, total_elements - 1);

    while (num_ones > 0) {
      size_t index = dis_idx(gen);
      if (matrix[index] == false) {
        matrix[index] = true;
        num_ones--;
      }
    }
    return res;
  }

  void convert_to_csr(std::vector<bool>& matrix,
                      index_t rows,
                      index_t cols,
                      std::vector<index_t>& indices,
                      std::vector<index_t>& indptr)
  {
    index_t offset_indptr   = 0;
    index_t offset_values   = 0;
    indptr[offset_indptr++] = 0;

    for (index_t i = 0; i < rows; ++i) {
      for (index_t j = 0; j < cols; ++j) {
        if (matrix[i * cols + j]) {
          indices[offset_values] = static_cast<index_t>(j);
          offset_values++;
        }
      }
      indptr[offset_indptr++] = static_cast<index_t>(offset_values);
    }
  }

  template <typename data_t>
  std::optional<data_t> get_opt_var(data_t x)
  {
    if (params.customized_indices) {
      return x;
    } else {
      return std::nullopt;
    }
  }

  void run_benchmark(::benchmark::State& state) override
  {
    std::ostringstream label_stream;
    label_stream << params;
    state.SetLabel(label_stream.str());

    auto in_val_structure = raft::make_device_compressed_structure_view<index_t, index_t, index_t>(
      indptr_d.data(),
      indices_d.data(),
      params.n_rows,
      params.n_cols,
      static_cast<index_t>(indices_d.size()));

    auto in_val =
      raft::make_device_csr_matrix_view<const value_t>(values_d.data(), in_val_structure);

    std::optional<raft::device_vector_view<const index_t, index_t>> in_idx;

    in_idx = get_opt_var(
      raft::make_device_vector_view<const index_t, index_t>(customized_indices_d.data(), nnz));

    auto out_val = raft::make_device_matrix_view<value_t, index_t, raft::row_major>(
      dst_values_d.data(), params.n_rows, params.top_k);
    auto out_idx = raft::make_device_matrix_view<index_t, index_t, raft::row_major>(
      dst_indices_d.data(), params.n_rows, params.top_k);

    raft::sparse::matrix::select_k(handle, in_val, in_idx, out_val, out_idx, params.select_min);
    resource::sync_stream(handle);
    loop_on_state(state, [this, &in_val, &in_idx, &out_val, &out_idx]() {
      raft::sparse::matrix::select_k(
        handle, in_val, in_idx, out_val, out_idx, params.select_min, false);
      resource::sync_stream(handle);
    });
  }

 protected:
  const raft::device_resources handle;

  bench_param<index_t> params;
  index_t nnz;

  rmm::device_uvector<value_t> values_d;
  rmm::device_uvector<index_t> indptr_d;
  rmm::device_uvector<index_t> indices_d;
  rmm::device_uvector<index_t> customized_indices_d;

  rmm::device_uvector<value_t> dst_values_d;
  rmm::device_uvector<index_t> dst_indices_d;
};  // struct SelectKCsrTest

template <typename index_t>
const std::vector<bench_param<index_t>> getInputs()
{
  std::vector<bench_param<index_t>> param_vec;
  struct TestParams {
    index_t m;
    index_t n;
    index_t k;
  };

  const std::vector<TestParams> params_group{
    {20000, 500, 1},    {20000, 500, 2},    {20000, 500, 4},   {20000, 500, 8},
    {20000, 500, 16},   {20000, 500, 32},   {20000, 500, 64},  {20000, 500, 128},
    {20000, 500, 256},

    {1000, 10000, 1},   {1000, 10000, 2},   {1000, 10000, 4},  {1000, 10000, 8},
    {1000, 10000, 16},  {1000, 10000, 32},  {1000, 10000, 64}, {1000, 10000, 128},
    {1000, 10000, 256},

    {100, 100000, 1},   {100, 100000, 2},   {100, 100000, 4},  {100, 100000, 8},
    {100, 100000, 16},  {100, 100000, 32},  {100, 100000, 64}, {100, 100000, 128},
    {100, 100000, 256},

    {10, 1000000, 1},   {10, 1000000, 2},   {10, 1000000, 4},  {10, 1000000, 8},
    {10, 1000000, 16},  {10, 1000000, 32},  {10, 1000000, 64}, {10, 1000000, 128},
    {10, 1000000, 256},

    {10, 1000000, 1},   {10, 1000000, 2},   {10, 1000000, 4},  {10, 1000000, 8},
    {10, 1000000, 16},  {10, 1000000, 32},  {10, 1000000, 64}, {10, 1000000, 128},
    {10, 1000000, 256},

    {10, 1000000, 1},   {10, 1000000, 16},  {10, 1000000, 64}, {10, 1000000, 128},
    {10, 1000000, 256},

    {10, 1000000, 1},   {10, 1000000, 16},  {10, 1000000, 64}, {10, 1000000, 128},
    {10, 1000000, 256}, {1000, 10000, 1},   {1000, 10000, 16}, {1000, 10000, 64},
    {1000, 10000, 128}, {1000, 10000, 256},

    {10, 1000000, 1},   {10, 1000000, 16},  {10, 1000000, 64}, {10, 1000000, 128},
    {10, 1000000, 256}, {1000, 10000, 1},   {1000, 10000, 16}, {1000, 10000, 64},
    {1000, 10000, 128}, {1000, 10000, 256}};

  param_vec.reserve(params_group.size());
  for (TestParams params : params_group) {
    param_vec.push_back(bench_param<index_t>({params.m, params.n, params.k, 0.1}));
  }
  for (TestParams params : params_group) {
    param_vec.push_back(bench_param<index_t>({params.m, params.n, params.k, 0.2}));
  }
  for (TestParams params : params_group) {
    param_vec.push_back(bench_param<index_t>({params.m, params.n, params.k, 0.5}));
  }
  return param_vec;
}

RAFT_BENCH_REGISTER((SelectKCsrTest<float, uint32_t>), "", getInputs<uint32_t>());

}  // namespace raft::bench::sparse
