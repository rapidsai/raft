#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/benchmark.hpp>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/distance/fused_l2_nn.cuh>
#include <raft/linalg/norm.cuh>
#include <raft/util/cudart_utils.hpp>
#include <rmm/device_uvector.hpp>

namespace raft::bench::distance {

struct fusedl2nn_inputs {
  int64_t m, n, k;
};  // struct fusedl2nn_inputs

inline auto operator<<(std::ostream& os, const fusedl2nn_inputs& p) -> std::ostream&
{
  os << p.m << "#" << p.n << "#" << p.k;
  return os;
}

template <typename DataT, typename IdxT, typename OutT>
struct fusedl2nn : public fixture {
  fusedl2nn(const fusedl2nn_inputs& p)
    : params(p),
      workspace(this->handle),
      x(this->handle),
      y(this->handle),
      x_norm(this->handle),
      y_norm(this->handle),
      out(this->handle)
  {
  }

  void allocate_data(const ::benchmark::State& state) override
  {
    x      = raft::make_device_matrix<DataT, IdxT>(handle, params.m, params.k);
    y      = raft::make_device_matrix<DataT, IdxT>(handle, params.n, params.k);
    x_norm = raft::make_device_vector<DataT, IdxT>(handle, params.m);
    y_norm = raft::make_device_vector<DataT, IdxT>(handle, params.n);
    out    = raft::make_device_vector<OutT, IdxT>(handle, params.m);

    raft::random::RngState rng{1234};
    raft::random::uniform(
      handle, rng, x.data_handle(), params.m * params.k, (DataT)-1.0, (DataT)1.0);
    raft::random::uniform(
      handle, rng, y.data_handle(), params.n * params.k, (DataT)-1.0, (DataT)1.0);

    // Pre-compute norms
    raft::linalg::rowNorm(x_norm.data_handle(),
                          x.data_handle(),
                          params.k,
                          params.m,
                          raft::linalg::L2Norm,
                          true,
                          stream);
    raft::linalg::rowNorm(y_norm.data_handle(),
                          y.data_handle(),
                          params.k,
                          params.n,
                          raft::linalg::L2Norm,
                          true,
                          stream);
    resource::sync_stream(handle, stream);
  }

  void allocate_temp_buffers(const ::benchmark::State& state) override
  {
    workspace = raft::make_device_vector<char, IdxT>(handle, params.m * sizeof(IdxT));
  }

  void run_benchmark(::benchmark::State& state) override
  {
    std::ostringstream label_stream;
    label_stream << params;
    state.SetLabel(label_stream.str());

    loop_on_state(state, [this]() {
      raft::distance::fusedL2NNMinReduce<DataT, OutT, IdxT>(out.data_handle(),
                                                            x.data_handle(),
                                                            y.data_handle(),
                                                            x_norm.data_handle(),
                                                            y_norm.data_handle(),
                                                            static_cast<IdxT>(params.m),
                                                            static_cast<IdxT>(params.n),
                                                            static_cast<IdxT>(params.k),
                                                            (void*)workspace.data_handle(),
                                                            false,
                                                            true,
                                                            stream);
    });

    int64_t num_flops = 2 * params.m * params.n * params.k;

    int64_t read_elts  = params.n * params.k + params.m * params.k;
    int64_t write_elts = params.m;

    state.counters["FLOP/s"] = benchmark::Counter(
      num_flops, benchmark::Counter::kIsIterationInvariantRate, benchmark::Counter::OneK::kIs1000);

    state.counters["BW Wr"] = benchmark::Counter(write_elts * sizeof(OutT),
                                                 benchmark::Counter::kIsIterationInvariantRate,
                                                 benchmark::Counter::OneK::kIs1000);
    state.counters["BW Rd"] = benchmark::Counter(read_elts * sizeof(DataT),
                                                 benchmark::Counter::kIsIterationInvariantRate,
                                                 benchmark::Counter::OneK::kIs1000);
  }

 private:
  fusedl2nn_inputs params;
  raft::device_matrix<DataT, IdxT> x, y;
  raft::device_vector<DataT, IdxT> x_norm, y_norm;
  raft::device_vector<OutT, IdxT> out;
  raft::device_vector<char, IdxT> workspace;
};  // struct fusedl2nn

template <typename IdxT>
std::vector<fusedl2nn_inputs> getFusedL2NNInputs()
{
  std::vector<fusedl2nn_inputs> inputs;
  std::vector<int64_t> m_list = {100000, 1000000};
  if constexpr (sizeof(IdxT) == 8) { m_list.push_back(10000000); }
  std::vector<int64_t> n_list = {100, 1000, 10000};
  std::vector<int64_t> k_list = {64, 128, 256};
  for (auto m : m_list) {
    for (auto n : n_list) {
      for (auto k : k_list) {
        inputs.push_back({m, n, k});
      }
    }
  }
  return inputs;
}

#define FUSEDL2NN_BENCH(DataT, IdxT, OutT) \
  RAFT_BENCH_REGISTER((fusedl2nn<DataT, IdxT, RAFT_DEPAREN(OutT)>), "", getFusedL2NNInputs<IdxT>())

FUSEDL2NN_BENCH(float, int, float);
FUSEDL2NN_BENCH(double, int, double);
FUSEDL2NN_BENCH(float, int, (raft::KeyValuePair<int, float>));
FUSEDL2NN_BENCH(double, int, (raft::KeyValuePair<int, double>));
FUSEDL2NN_BENCH(float, int64_t, float);
FUSEDL2NN_BENCH(double, int64_t, double);
FUSEDL2NN_BENCH(float, int64_t, (raft::KeyValuePair<int64_t, float>));
FUSEDL2NN_BENCH(double, int64_t, (raft::KeyValuePair<int64_t, double>));

}  // namespace raft::bench::distance
