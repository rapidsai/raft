#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdint>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>

#include <common/benchmark.hpp>
#include <limits>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_mdspan.hpp>
#include <raft/core/handle.hpp>
#include <raft/distance/masked_nn.cuh>
#include <raft/linalg/norm.cuh>
#include <raft/random/rng.cuh>
#include <raft/util/cudart_utils.hpp>

namespace raft::bench::distance::masked_nn {

// Introduce various sparsity patterns
enum AdjacencyPattern {
  checkerboard    = 0,
  checkerboard_4  = 1,
  checkerboard_64 = 2,
  all_true        = 3,
  all_false       = 4
};

struct Params {
  int m, n, k, num_groups;
  AdjacencyPattern pattern;
};  // struct Params

RAFT_KERNEL init_adj(AdjacencyPattern pattern,
                     int n,
                     raft::device_matrix_view<bool, int, raft::layout_c_contiguous> adj,
                     raft::device_vector_view<int, int, raft::layout_c_contiguous> group_idxs)
{
  int m          = adj.extent(0);
  int num_groups = adj.extent(1);

  for (int idx_m = blockIdx.y * blockDim.y + threadIdx.y; idx_m < m;
       idx_m += blockDim.y * gridDim.y) {
    for (int idx_g = blockIdx.x * blockDim.x + threadIdx.x; idx_g < num_groups;
         idx_g += blockDim.x * gridDim.x) {
      switch (pattern) {
        case checkerboard: adj(idx_m, idx_g) = (idx_m + idx_g) % 2; break;
        case checkerboard_4: adj(idx_m, idx_g) = (idx_m / 4 + idx_g) % 2; break;
        case checkerboard_64: adj(idx_m, idx_g) = (idx_m / 64 + idx_g) % 2; break;
        case all_true: adj(idx_m, idx_g) = true; break;
        case all_false: adj(idx_m, idx_g) = false; break;
        default: assert(false && "unknown pattern");
      }
    }
  }
  // Each group is of size n / num_groups.
  //
  // - group_idxs[j] indicates the start of group j + 1 (i.e. is the inclusive
  // scan of the group lengths)
  //
  // - The first group always starts at index zero, so we do not store it.
  //
  // - The group_idxs[num_groups - 1] should always equal n.

  if (blockIdx.y == 0 && threadIdx.y == 0) {
    const int g_stride = blockDim.x * gridDim.x;
    for (int idx_g = blockIdx.x * blockDim.x + threadIdx.x; idx_g < num_groups; idx_g += g_stride) {
      group_idxs(idx_g) = (idx_g + 1) * (n / num_groups);
    }
    group_idxs(num_groups - 1) = n;
  }
}

template <typename T>
struct masked_l2_nn : public fixture {
  using DataT      = T;
  using IdxT       = int;
  using OutT       = raft::KeyValuePair<IdxT, DataT>;
  using RedOpT     = raft::distance::MinAndDistanceReduceOp<int, DataT>;
  using PairRedOpT = raft::distance::KVPMinReduce<int, DataT>;
  using ParamT     = raft::distance::masked_l2_nn_params<RedOpT, PairRedOpT>;

  // Parameters
  Params params;
  // Data
  raft::device_vector<OutT, IdxT> out;
  raft::device_matrix<T, IdxT> x, y;
  raft::device_vector<DataT, IdxT> xn, yn;
  raft::device_matrix<bool, IdxT> adj;
  raft::device_vector<IdxT, IdxT> group_idxs;

  masked_l2_nn(const Params& p)
    : params(p),
      out{raft::make_device_vector<OutT, IdxT>(handle, p.m)},
      x{raft::make_device_matrix<DataT, IdxT>(handle, p.m, p.k)},
      y{raft::make_device_matrix<DataT, IdxT>(handle, p.n, p.k)},
      xn{raft::make_device_vector<DataT, IdxT>(handle, p.m)},
      yn{raft::make_device_vector<DataT, IdxT>(handle, p.n)},
      adj{raft::make_device_matrix<bool, IdxT>(handle, p.m, p.num_groups)},
      group_idxs{raft::make_device_vector<IdxT, IdxT>(handle, p.num_groups)}
  {
    raft::random::RngState r(123456ULL);

    uniform(handle, r, x.data_handle(), p.m * p.k, T(-1.0), T(1.0));
    uniform(handle, r, y.data_handle(), p.n * p.k, T(-1.0), T(1.0));
    raft::linalg::rowNorm(
      xn.data_handle(), x.data_handle(), p.k, p.m, raft::linalg::L2Norm, true, stream);
    raft::linalg::rowNorm(
      yn.data_handle(), y.data_handle(), p.k, p.n, raft::linalg::L2Norm, true, stream);
    raft::distance::initialize<T, raft::KeyValuePair<int, T>, int>(
      handle, out.data_handle(), p.m, std::numeric_limits<T>::max(), RedOpT{});

    dim3 block(32, 32);
    dim3 grid(10, 10);
    init_adj<<<grid, block, 0, stream>>>(p.pattern, p.n, adj.view(), group_idxs.view());
    RAFT_CUDA_TRY(hipGetLastError());
  }

  void run_benchmark(::benchmark::State& state) override
  {
    bool init_out = true;
    bool sqrt     = false;
    ParamT masked_l2_params{RedOpT{}, PairRedOpT{}, sqrt, init_out};

    loop_on_state(state, [this, masked_l2_params]() {
      // It is sufficient to only benchmark the L2-squared metric
      raft::distance::masked_l2_nn<DataT, OutT, IdxT>(handle,
                                                      masked_l2_params,
                                                      x.view(),
                                                      y.view(),
                                                      xn.view(),
                                                      yn.view(),
                                                      adj.view(),
                                                      group_idxs.view(),
                                                      out.view());
    });

    // Virtual flop count if no skipping had occurred.
    size_t virtual_flops = size_t(2) * size_t(params.m) * size_t(params.n) * size_t(params.k);

    int64_t read_elts  = params.n * params.k + params.m * params.k;
    int64_t write_elts = params.m;

    // Virtual min flops is the number of flops that would have been executed if
    // the algorithm had actually skipped each computation that it could have
    // skipped.
    size_t virtual_min_flops = 0;
    switch (params.pattern) {
      case checkerboard:
      case checkerboard_4:
      case checkerboard_64: virtual_min_flops = virtual_flops / 2; break;
      case all_true: virtual_min_flops = virtual_flops; break;
      case all_false: virtual_min_flops = 0; break;
      default: assert(false && "unknown pattern");
    }

    // VFLOP/s is the "virtual" flop count that would have executed if there was
    // no adjacency pattern. This is useful for comparing to fusedL2NN
    state.counters["VFLOP/s"] = benchmark::Counter(virtual_flops,
                                                   benchmark::Counter::kIsIterationInvariantRate,
                                                   benchmark::Counter::OneK::kIs1000);
    // Virtual min flops is the number of flops that would have been executed if
    // the algorithm had actually skipped each computation that it could have
    // skipped.
    state.counters["VminFLOP/s"] = benchmark::Counter(virtual_min_flops,
                                                      benchmark::Counter::kIsIterationInvariantRate,
                                                      benchmark::Counter::OneK::kIs1000);

    state.counters["BW Wr"] = benchmark::Counter(write_elts * sizeof(OutT),
                                                 benchmark::Counter::kIsIterationInvariantRate,
                                                 benchmark::Counter::OneK::kIs1000);
    state.counters["BW Rd"] = benchmark::Counter(read_elts * sizeof(DataT),
                                                 benchmark::Counter::kIsIterationInvariantRate,
                                                 benchmark::Counter::OneK::kIs1000);

    state.counters["m"]          = benchmark::Counter(params.m);
    state.counters["n"]          = benchmark::Counter(params.n);
    state.counters["k"]          = benchmark::Counter(params.k);
    state.counters["num_groups"] = benchmark::Counter(params.num_groups);
    state.counters["group size"] = benchmark::Counter(params.n / params.num_groups);
    state.counters["Pat"]        = benchmark::Counter(static_cast<int>(params.pattern));

    state.counters["SM count"] = raft::getMultiProcessorCount();
  }
};

const std::vector<Params> masked_l2_nn_input_vecs = {
  // Very fat matrices...
  {32, 16384, 16384, 32, AdjacencyPattern::checkerboard},
  {64, 16384, 16384, 32, AdjacencyPattern::checkerboard},
  {128, 16384, 16384, 32, AdjacencyPattern::checkerboard},
  {256, 16384, 16384, 32, AdjacencyPattern::checkerboard},
  {512, 16384, 16384, 32, AdjacencyPattern::checkerboard},
  {1024, 16384, 16384, 32, AdjacencyPattern::checkerboard},
  {16384, 32, 16384, 32, AdjacencyPattern::checkerboard},
  {16384, 64, 16384, 32, AdjacencyPattern::checkerboard},
  {16384, 128, 16384, 32, AdjacencyPattern::checkerboard},
  {16384, 256, 16384, 32, AdjacencyPattern::checkerboard},
  {16384, 512, 16384, 32, AdjacencyPattern::checkerboard},
  {16384, 1024, 16384, 32, AdjacencyPattern::checkerboard},

  // Representative matrices...
  {16384, 16384, 32, 32, AdjacencyPattern::checkerboard},
  {16384, 16384, 64, 32, AdjacencyPattern::checkerboard},
  {16384, 16384, 128, 32, AdjacencyPattern::checkerboard},
  {16384, 16384, 256, 32, AdjacencyPattern::checkerboard},
  {16384, 16384, 512, 32, AdjacencyPattern::checkerboard},
  {16384, 16384, 1024, 32, AdjacencyPattern::checkerboard},
  {16384, 16384, 16384, 32, AdjacencyPattern::checkerboard},

  {16384, 16384, 32, 32, AdjacencyPattern::checkerboard_4},
  {16384, 16384, 64, 32, AdjacencyPattern::checkerboard_4},
  {16384, 16384, 128, 32, AdjacencyPattern::checkerboard_4},
  {16384, 16384, 256, 32, AdjacencyPattern::checkerboard_4},
  {16384, 16384, 512, 32, AdjacencyPattern::checkerboard_4},
  {16384, 16384, 1024, 32, AdjacencyPattern::checkerboard_4},
  {16384, 16384, 16384, 32, AdjacencyPattern::checkerboard_4},

  {16384, 16384, 32, 32, AdjacencyPattern::checkerboard_64},
  {16384, 16384, 64, 32, AdjacencyPattern::checkerboard_64},
  {16384, 16384, 128, 32, AdjacencyPattern::checkerboard_64},
  {16384, 16384, 256, 32, AdjacencyPattern::checkerboard_64},
  {16384, 16384, 512, 32, AdjacencyPattern::checkerboard_64},
  {16384, 16384, 1024, 32, AdjacencyPattern::checkerboard_64},
  {16384, 16384, 16384, 32, AdjacencyPattern::checkerboard_64},

  {16384, 16384, 32, 32, AdjacencyPattern::all_true},
  {16384, 16384, 64, 32, AdjacencyPattern::all_true},
  {16384, 16384, 128, 32, AdjacencyPattern::all_true},
  {16384, 16384, 256, 32, AdjacencyPattern::all_true},
  {16384, 16384, 512, 32, AdjacencyPattern::all_true},
  {16384, 16384, 1024, 32, AdjacencyPattern::all_true},
  {16384, 16384, 16384, 32, AdjacencyPattern::all_true},

  {16384, 16384, 32, 32, AdjacencyPattern::all_false},
  {16384, 16384, 64, 32, AdjacencyPattern::all_false},
  {16384, 16384, 128, 32, AdjacencyPattern::all_false},
  {16384, 16384, 256, 32, AdjacencyPattern::all_false},
  {16384, 16384, 512, 32, AdjacencyPattern::all_false},
  {16384, 16384, 1024, 32, AdjacencyPattern::all_false},
  {16384, 16384, 16384, 32, AdjacencyPattern::all_false},
};

RAFT_BENCH_REGISTER(masked_l2_nn<float>, "", masked_l2_nn_input_vecs);
// We don't benchmark double to keep compile times in check when not using the
// distance library.

}  // namespace raft::bench::distance::masked_nn
