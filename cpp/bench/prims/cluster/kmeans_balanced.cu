#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <common/benchmark.hpp>
#include <raft/cluster/kmeans_balanced.cuh>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/random/rng.cuh>

namespace raft::bench::cluster {

struct KMeansBalancedBenchParams {
  DatasetParams data;
  uint32_t n_lists;
  raft::cluster::kmeans_balanced_params kb_params;
};

template <typename T, typename IndexT = int>
struct KMeansBalanced : public fixture {
  KMeansBalanced(const KMeansBalancedBenchParams& p) : params(p), X(handle), centroids(handle) {}

  void run_benchmark(::benchmark::State& state) override
  {
    this->loop_on_state(state, [this]() {
      raft::device_matrix_view<const T, IndexT> X_view   = this->X.view();
      raft::device_matrix_view<T, IndexT> centroids_view = this->centroids.view();
      raft::cluster::kmeans_balanced::fit(
        this->handle, this->params.kb_params, X_view, centroids_view);
    });
  }

  void allocate_data(const ::benchmark::State& state) override
  {
    X = raft::make_device_matrix<T, IndexT>(handle, params.data.rows, params.data.cols);

    raft::random::RngState rng{1234};
    constexpr T kRangeMax = std::is_integral_v<T> ? std::numeric_limits<T>::max() : T(1);
    constexpr T kRangeMin = std::is_integral_v<T> ? std::numeric_limits<T>::min() : T(-1);
    if constexpr (std::is_integral_v<T>) {
      raft::random::uniformInt(
        handle, rng, X.data_handle(), params.data.rows * params.data.cols, kRangeMin, kRangeMax);
    } else {
      raft::random::uniform(
        handle, rng, X.data_handle(), params.data.rows * params.data.cols, kRangeMin, kRangeMax);
    }
    resource::sync_stream(handle, stream);
  }

  void allocate_temp_buffers(const ::benchmark::State& state) override
  {
    centroids =
      raft::make_device_matrix<float, IndexT>(this->handle, params.n_lists, params.data.cols);
  }

 private:
  KMeansBalancedBenchParams params;
  raft::device_matrix<T, IndexT> X;
  raft::device_matrix<float, IndexT> centroids;
};  // struct KMeansBalanced

std::vector<KMeansBalancedBenchParams> getKMeansBalancedInputs()
{
  std::vector<KMeansBalancedBenchParams> out;
  KMeansBalancedBenchParams p;
  p.data.row_major                          = true;
  p.kb_params.n_iters                       = 20;
  p.kb_params.metric                        = raft::distance::DistanceType::L2Expanded;
  std::vector<std::pair<int, int>> row_cols = {
    {100000, 128}, {1000000, 128}, {10000000, 128},
    // The following dataset sizes are too large for most GPUs.
    // {100000000, 128},
  };
  for (auto& rc : row_cols) {
    p.data.rows = rc.first;
    p.data.cols = rc.second;
    for (auto n_lists : std::vector<int>({1000, 10000, 100000})) {
      p.n_lists = n_lists;
      out.push_back(p);
    }
  }
  return out;
}

// Note: the datasets sizes are too large for 32-bit index types.
RAFT_BENCH_REGISTER((KMeansBalanced<float, int64_t>), "", getKMeansBalancedInputs());

}  // namespace raft::bench::cluster
